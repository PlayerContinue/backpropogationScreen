#include "hip/hip_runtime.h"
#include "modifyNetwork.cuh"

modifyNetwork::modifyNetwork(){
	this->buildNetwork();
}

double* modifyNetwork::getLayerOutputUse(){
	return this->getLayerOutput();
}

double modifyNetwork::getLayerForValue(int i){
	return this->getValueForLayer(i);
}


void modifyNetwork::buildNetwork(){
		CSettings settings = CSettings();
		modifyNetwork temp1 = modifyNetwork();
		settings.d_alpha = 0;
		settings.d_beta = .1;
		vector<int> vec = vector<int>();
		vec.push_back(2);
		vec.push_back(3);
		vec.push_back(3);
		vec.push_back(3);
		this->testNetwork = CGraphicsNetwork(vec, &settings);
	}



double* modifyNetwork::getLayerOutput(){
		double* temp = new double(2);
		temp[0] = 1;
		temp[1] = 1;
		double* output1 = new double[3];
		//Modify the layer values
		output1 = this->testNetwork.getRootMeanSquareErrorForAllLayer(temp);
		return output1;
	}

	//Check if the subtract function works correctly
double modifyNetwork::getValueForLayer(int layer){
	thrust::host_vector<double> temp = this->testNetwork.v_layers[layer].getOutput();
		double k = 0;
		for (int i = 0; i < temp.size(); i++){
			for (int j = i; j < temp.size(); j++){
				k += (temp[i] - temp[j]) * (temp[i] - temp[j]);
			}
		}
		return sqrt(k);
	}


