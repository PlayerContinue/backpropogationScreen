#include "hip/hip_runtime.h"


//---------------------------------------------------------------------------------------
//Author:David Greenberg
//Desc: Initizializing algorithm. Contains the main function and only the main function
//----------------------------------------------------------------------------------------

//#define PROBLEMS 5
//#define FIRST_TEST
#define PAUSE
#pragma once
#include <vector>
#include <stdlib.h>
#include <bitset>
#include <math.h>
#include <iostream>
#include <fstream>
#include <string>
#include <csignal>
#include "CSettings.h"
#include "ReccurentLoops.cuh"
#include "CGraphicNetwork.cuh"

#ifdef WINDOWS_COMPUTER
#include <wincon.h>
#endif

using namespace std;


static volatile bool pause;


//PROTOTYPES
int writeToFile(CGraphicsNetwork &network, CSettings settings);
void testOutput2(double** value, CGraphicsNetwork &test, int size);
bool addToNetwork(CGraphicsNetwork &test, CSettings settings, SCheckpoint& checkpoint, double** testSet, double mean_square_error);
int getDataFromFile(string fileName, int start, int end, int numberResults, double** input, int type_of_input);
void testOutput3(double* value, CGraphicsNetwork &test, int size);

void printVectorOutput(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}
	std::cout << value << endl;
	if (value <= 255 & value > 0){
		std::cout << (char)value << endl;
	}

}

int printVectorOutputChar(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}

	if (value <= 255 & value > 0){
		std::cout << (char)value;
	}
	else{
		std::cout << value;
	}

	return value;

}

void printVectorOutput2(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}

	std::cout << value;
}

void printArray(double* arrayA, int size){
	for (int i = 0; i < size; i++){
		std::cout << (char)arrayA[i];
		std::cout << endl;
	}
}

void trainNetwork(double* value_1, double* value_2, double* value_3, double* value_4, CGraphicsNetwork &test, int rounds){
	//Train the network on simple test values
	for (int i = 0; i < rounds; i++){
		if (i % 2 == 0){
			test.backprop(value_1, value_2);
		}
		else{
			test.backprop(value_3, value_4);
		}
	}

}

void trainNetwork2(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, int rounds){

	int count = 0;
	//Train the network on simple test values
	for (int i = start; i < rounds; i++){
		test.backprop(value[count], results[count]);
		count++;
		if (count >= end){
			count = start;
		}
	}

}

//Returns true if the current mean_square_error is less than the lowest_mean_square
template <typename T>
bool checkThreshold(T mean_square, T lowest_mean_square, T threshold){
	if (mean_square < threshold && threshold >= 0){
		return false;
	}
	else if (mean_square <= lowest_mean_square){
		return true;
	}
	else{
		return false;
	}
}

int numberFullSame(CGraphicsNetwork test, double** in, int size){
	vector<double> output;
	vector<double> output2;
	int count_success = 0;
	for (int i = 0; i < size; i++){
		test.feedForward(in[i]);
		if (i != 0){
			output2 = output;
		}
		output = test.getOutput();
		if (i != 0){
			for (unsigned int j = 0; j < output.size(); j++){
				if (output[j] != output2[j]){
					count_success++;
					break;
				}
			}
		}
	}

	return (size - count_success);
}

//******************************
//Save, Load, Create Checkpoints
//******************************

//Create a checkpoint containing all the current values of the system
void createCheckpoint(CGraphicsNetwork test, SCheckpoint& checkpoint, CSettings settings){
	int checkpoint_number = writeToFile(test, settings);
	checkpoint.s_network_file_name = "networks/" + settings.s_network_name + std::to_string(checkpoint_number) + ".txt";
	//Write the checkpoint to a file
	std::ofstream outputfile;
	outputfile.precision(30);
	outputfile.open("checkpoints/" + settings.s_network_name + "_checkpoint_" + std::to_string(checkpoint_number) + ".txt", ios::trunc);
	if (outputfile.is_open()){
		//Output the network
		outputfile << checkpoint << flush;
		outputfile.close();
	}
	else{
		std::cout << "Unable to write checkpoint to file." << endl;
		std::cout << "continue?";
	}

}

void LoadCheckpointFromFile(SCheckpoint& checkpoint, string s_file_name){
	std::ifstream inputfile;

	inputfile.open(s_file_name, ios_base::beg);
	if (inputfile.is_open()){
		inputfile >> checkpoint;
		inputfile.close();
	}
	else{
		std::cout << "Unable to read checkpoint from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}
	}
}

//Create a new checkpoint object from the created settings
void createNewCheckpoint(SCheckpoint& checkpoint, CSettings settings, double** in){
	checkpoint.i_number_of_loops_checkpoint = 0;
	checkpoint.i_number_of_loops = 0;
	checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
	checkpoint.d_mean_square_error = (double)INT_MAX;
	checkpoint.d_lowest_mean_square_error = (double)INT_MAX;
	checkpoint.d_previous_mean_square_error = 0;
	checkpoint.d_row_distance_threshold = settings.d_row_distance_threshold;
	checkpoint.d_neuron_distance_threshold = settings.d_neuron_distance_threshold;
	checkpoint.i_current_position_in_output_file = 0;
	checkpoint.i_current_position_in_output_file = 0;
	std::cout << "Getting Layer Or Row Number " << endl;
	checkpoint.get_layer_or_row(in, settings.i_number_of_training, settings.i_output);
}

//*********************************
//Signal Handlers
//*********************************

void signal_handler(int signal)
{
	if (signal == SIGINT){
		pause = true;
	}
	std::signal(SIGINT, signal_handler);
}

//*********************************
//User Signal
//*********************************
#ifdef PAUSE
void userSignal(double** results, double** value, SCheckpoint checkpoint, double** testSetIn, double** testSetOut, CGraphicsNetwork test, int testLength, CSettings settings){




	char userin = 'l';
	double* tempDistanceHolderForPause;
	double tempDistanceHolderForPauseSingle;
	std::cout << "1) Print output " << endl;
	std::cout << "2) Print Output On Current Input " << endl;
	std::cout << "3) Run same check " << endl;
	std::cout << "4) Get mean square " << endl;
	std::cout << "5) Save Checkpoint " << endl;
	std::cout << "6) Check MSE on current input " << endl;
	std::cout << "7) Get distance check " << endl;
	std::cout << "8) Continue " << endl;
	std::cout << "0) Exit " << endl;

	std::cout << " loop " << checkpoint.i_number_of_loops_checkpoint << endl;
	std::cout.precision(30);

	do{

		if (userin != 'l'){
			std::cout << "Anything else? ";
		}
		std::cin.sync();
		userin = std::cin.get();
		switch (userin){
		case '1':
			//Test the output
			std::cout << "Getting Output " << endl;
			testOutput2(testSetIn, test, testLength);
			break;
		case '2':
			std::cout << "Getting Single Output " << endl;
			test.feedForward(value[0]);
			testOutput3(value[checkpoint.i_number_of_loops], test, test.getI_input());
			break;
		case '3':
			std::cout << "Getting Number Same " << endl;
			std::cout << numberFullSame(test, testSetIn, settings.i_number_of_training) << endl;
			break;
		case '4':
			std::cout << "Getting Mean Square " << endl;
			std::cout << test.getMeanSquareError(testSetIn, testSetOut, testLength) << endl;
			break;
		case '5':
			std::cout << "Creating Checkpoint " << endl;
			createCheckpoint(test, checkpoint, settings);
			std::cout << "Checkpoint created " << endl;
			break;
		case '6':
			std::cout << "Getting MSE " << endl;
			std::cout << test.getSingleMeanSquareError(value[checkpoint.i_number_of_loops - 1], results[checkpoint.i_number_of_loops - 1], testLength) << endl;
			break;
		case '7':
			std::cout << "Getting Distance Measure " << endl;
			tempDistanceHolderForPause = test.getRootMeanSquareErrorForAllLayer(value[checkpoint.i_number_of_loops]);
			tempDistanceHolderForPauseSingle = 0;
			for (int i = 0; i < test.getNumLayers() - 1; i++){
				std::cout << i << ") " << tempDistanceHolderForPause[i] << endl;
				tempDistanceHolderForPauseSingle += tempDistanceHolderForPause[i];
			}
			tempDistanceHolderForPauseSingle /= (test.getNumLayers() - 1);
			std::cout << tempDistanceHolderForPauseSingle << endl;
			break;
		case '0':
			std::wcout << "Would you like to create a checkpoint?";
			std::cin.sync();
			if (std::cin.get() == 'y'){
				std::cout << "Creating Checkpoint " << endl;
				createCheckpoint(test, checkpoint, settings);
				std::cout << "Checkpoint created " << endl;
			}
			exit(0);
			break;
		default:
			break;
		}
	} while (userin != '9' && userin != 'n');

	pause = false;
	std::cout << "finished" << endl;



}
#endif

//******************************
//Training the Network
//******************************

//Train the current network
//Check if the limit has been reached as the stopping point
//Use mean square error to check distance
void trainNetworkDelta(double* value[], double* results[], CGraphicsNetwork &test, int input_start, int output_start, double* testSetIn[], double* testSetOut[], int testLength, CSettings settings, SCheckpoint& checkpoint){

	int new_end = settings.i_number_of_training;

	do{

		//Pause the program and perform one of the following options
#ifdef PAUSE
		if (pause){
			userSignal(results, value, checkpoint, testSetIn, testSetOut, test, testLength, settings);
		}
#endif

		//Run the backpropogation algorithm to both get the current output and train the network
		test.backprop(value[checkpoint.i_number_of_loops], results[checkpoint.i_number_of_loops]);

		//Create a checkpoint when X number of iterations have occured, where X is defined by the user
		if (checkpoint.i_number_of_loops_checkpoint % settings.i_loops == 0){
			createCheckpoint(test, checkpoint, settings);
		}
		checkpoint.i_number_of_loops++;
		checkpoint.i_number_of_loops_checkpoint++;

		//Grab more training data when the current list of training data has been exhausted

		try{
			if (checkpoint.i_number_of_loops >= settings.i_number_of_training){
				checkpoint.i_number_of_loops = 0;

				if (new_end < settings.i_number_of_training){
					//TODO Add asking for a new file or to reuse file
					createCheckpoint(test, checkpoint, settings);
					exit(0);
				}

				if (settings.b_trainingFromFile){

					//Store training set
					checkpoint.i_current_position_in_input_file = getDataFromFile(settings.s_trainingSet, checkpoint.i_current_position_in_input_file, settings.i_number_of_training, settings.i_input, value, settings.i_trainingSetType);

				}

				if (settings.b_testingFromFile){
					checkpoint.i_current_position_in_output_file = getDataFromFile(settings.s_outputTrainingFile, checkpoint.i_current_position_in_output_file, settings.i_number_of_training, settings.i_output, results, settings.i_outputTrainingSetType);
				}

				new_end = (checkpoint.i_current_position_in_input_file < checkpoint.i_current_position_in_output_file ? checkpoint.i_current_position_in_input_file : checkpoint.i_current_position_in_output_file);

			}
		}
		catch (exception e){
			createCheckpoint(test, checkpoint, settings);
			exit(0);
		}
		//Get the mean_square_error when the number of loops reaches a user defined values
		if (checkpoint.i_number_of_loops_checkpoint%settings.i_number_before_growth_potential == 0){

			//Set the current mean_square_error as the previous error
			checkpoint.d_previous_mean_square_error = checkpoint.d_mean_square_error;

			//Retrieve the new mean_square_error
			checkpoint.d_mean_square_error = test.getMeanSquareError(testSetIn, testSetOut, testLength);

			//Set the current lowest
			if (checkpoint.d_mean_square_error < checkpoint.d_lowest_mean_square_error && checkpoint.d_mean_square_error != checkpoint.d_previous_mean_square_error){
				checkpoint.d_lowest_mean_square_error = checkpoint.d_mean_square_error;

				//Reset value since error was lowered
				checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
			}
			else{
				checkpoint.i_times_lowest_mean_square_error_to_large--;

				//Keep track of the number of times the mean square error has been equal
				if (checkpoint.d_previous_mean_square_error == checkpoint.d_mean_square_error){
					checkpoint.i_equal_square_errors++;
				}
				//If the below is true, something close to the limit has been reached, the network needs to change size 
				if (checkpoint.i_times_lowest_mean_square_error_to_large <= 0){
					//Add new nodes to the network
					if (addToNetwork(test, settings, checkpoint, testSetIn, checkpoint.d_mean_square_error)){
						//Reset the number allowed
						//Since new ones may have been
						checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures + 10;
					}
					else{
						//Reset the number of times before a growth is attempted
						checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
					}

					//Reset the number of times the mean_square_error has been equal
					checkpoint.i_equal_square_errors = 0;

				}
			}

		}

		//Loop until the error is smaller than the threshold
	} while (settings.d_threshold < checkpoint.d_mean_square_error);
	createCheckpoint(test, checkpoint, settings);
}

//******************************
//Modifying the Network
//******************************

//Returns true if a neuron was added
bool addToNetwork(CGraphicsNetwork &test, CSettings settings, SCheckpoint& checkpoint, double** testSet, double mean_square_error){


	double* differencePerLayer = test.getRootMeanSquareErrorForAllLayer(testSet[checkpoint.i_number_of_loops]);

	//Get the average difference per layer between the outputs in the layers
	double averageDifferencePerLayer = 0;

	for (int i = 0; i < test.getNumLayers() - 1; i++){
		averageDifferencePerLayer += differencePerLayer[i];
	}

	averageDifferencePerLayer /= (test.getNumLayers() - 1);


	//double success = test.getSuccessRate() - test.getPreviousSuccessRate();
	//double mean_square_error_dif = mean_square_error - settings.d_threshold;
#ifdef FULL_SUCCESS
	double full_success = test.getFullSuccessRate() - test.getFullPreviousSuccessRate();
#endif
	//Get the number of test sets returning the exact same values
	//int numberSame = numberFullSame(test, testSet, settings.i_number_of_training);

	//Add a new layer if the success is too low and the threshold has not been reached
	//A layer should be added if the mean square error remains constant as the current layer has been fully trained to give a particular output
	//Therefore a function should be added to deal with that particular output
	if (averageDifferencePerLayer > checkpoint.d_neuron_or_layer_threshold){ //&& numberSame < settings.i_number_allowed_same){
		test.addLayer(-1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) * 5);
		//Increment the size of the need mean distance to get a new layer
		//And decrease the size of the needed mean distance to get a new neuron
		/*if (checkpoint.d_neuron_distance_threshold > 0){
			checkpoint.d_row_distance_threshold += settings.d_row_distance_threshold * .1;
			checkpoint.d_neuron_distance_threshold -= settings.d_neuron_distance_threshold * .1;
			}*/
		test.resetNetwork();
		return true;
	}
	else if (averageDifferencePerLayer < checkpoint.d_neuron_or_layer_threshold){
		if (test.getNumLayers() == 2){
			//Since neurons cannot be added, if only the input/output layer exists, and new neuron is chosen, than a new layer is added instead
			test.addLayer(-1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) * 5);
		}
		else{
			//Double the number of nodes in every non input/output row 
			//Keeps size of each row equivalent
			//Good for initial growth
			for (int i = 1; i < test.getNumLayers() - 1; i++){
				if (differencePerLayer[i - 1] < checkpoint.d_neuron_or_layer_threshold){
					test.addNeuronToLayer(i, i, 10);
				}
			}
			//Increment the size of the need mean distance to get a new neuron
			//And decrease the size of the needed mean distance to get a new row

			/*if (checkpoint.d_row_distance_threshold > 0){
				checkpoint.d_row_distance_threshold -= settings.d_row_distance_threshold * .1;
				checkpoint.d_neuron_distance_threshold += settings.d_neuron_distance_threshold * .1;
				}*/
		}
		test.resetNetwork();
		return true;
	}
	//If both fail, but a large gap still exists between the threshold and the set distances, decrease them both.
	//This is to allow the system to grow further when needed.
	//However, only occur when the network is fully trained
	/*else if (checkpoint.i_equal_square_errors >= settings.i_number_allowed_failures){
		checkpoint.d_row_distance_threshold -= settings.d_row_distance_threshold * .1;
		checkpoint.d_neuron_distance_threshold -= settings.d_neuron_distance_threshold * .1;
		}*/
	return false;


}

//******************************
//Testing Output
//******************************

void testOutput(double* value_1, double* value_3, CGraphicsNetwork &test){
	vector<double> temp2;
	std::cout << "input";
	std::cout << endl;
	printArray(value_1, 2);
	test.feedForward(value_1);
	temp2 = test.getOutput();
	std::cout << "output";
	std::cout << endl;
	printVectorOutput(temp2);
	std::cout << endl;


	test.feedForward(value_3);
	temp2 = test.getOutput();

	std::cout << "input";
	std::cout << endl;
	printArray(value_3, 2);

	temp2 = test.getOutput();
	std::cout << "output";
	std::cout << endl;
	printVectorOutput(temp2);
	std::cout << endl;
}

void testOutput2(double** value, CGraphicsNetwork &test, int size){
	vector<double> temp2;

	for (int i = 0; i < size; i++){
		std::cout << "input";
		std::cout << endl;

		printArray(value[i], 6);

		test.feedForward(value[i]);

		temp2 = test.getOutput();

		std::cout << "output";
		std::cout << endl;
		printVectorOutput(temp2);
		std::cout << endl;
	}
}

void testOutput3(double* value, CGraphicsNetwork &test, int size){
	vector<double> temp2;


	std::cout << "input";
	std::cout << endl;

	printArray(value, size);

	test.feedForward(value);

	temp2 = test.getOutput();

	std::cout << "output";
	std::cout << endl;
	printVectorOutput(temp2);
	std::cout << endl;

}

//******************************
//Writing And Loading From Files
//******************************

//Output the network to a file
int writeToFile(CGraphicsNetwork &network, CSettings settings){
	static int file_number = 0;
	file_number++;
	std::ofstream outputfile;
	outputfile.open("networks/" + settings.s_network_name + std::to_string(file_number) + ".txt", ios::trunc);
	if (outputfile.is_open()){
		//Output the network
		outputfile << network << flush;
		outputfile.close();
	}
	else{
		std::cout << "Unable to write checkpoint to file." << endl;
		std::cout << "continue?";
	}

	return file_number;

}

bool loadFromFile(CGraphicsNetwork& network, string fileName){
	std::ifstream inputfile;

	inputfile.open(fileName, ios_base::beg);
	if (inputfile.is_open()){
		inputfile >> network;
		inputfile.close();
		return true;
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}
		return false;
	}
}

CSettings loadSettings(string fileName){
	std::ifstream inputfile;
	inputfile.open(fileName, ios_base::beg);
	CSettings settings;
	if (inputfile.is_open()){

		inputfile >> settings;
		inputfile.close();
		return settings;
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}

		return settings;
	}
}


#ifdef WINDOWS_COMPUTER
BOOL WINAPI ConsoleHandlerRoutine(DWORD dwCtrlType)
{
	if (dwCtrlType == CTRL_CLOSE_EVENT)
	{
		return TRUE;
	}

	return FALSE;
}


void initialize(){
	if (SetConsoleCtrlHandler(ConsoleHandlerRoutine, TRUE) == false){
		printf("Unable to attach Handler");
		std::cout << "continue? ";
		if (cin.get() == 'n'){
			exit(0);
		}
	}
}

#endif

//Retrieve the data from a file
//start: Where to start gathering characters from in the file
//numberOfRounds: The number of character sets which are needed to be retrieve
//numberResults: How many characters should be retrieved for a single round
//input : the storage container for the input
//type_of_input : The type of input/output i.e. is char, double, ect
int getDataFromFile(string fileName, int start, int numberOfRounds, int numberResults, double** input, int type_of_input){
	std::ifstream inputfile;
	char individual_delimiter = (char)29;
	char group_delimiter = (char)30;
	char current_char = '1';
	string current_string = "";
	inputfile.open(fileName);
	if (inputfile.is_open()){
		inputfile.seekg(start);
		int letterPosition = 0;
		for (int i = 0; i < (numberOfRounds); i++){
			//Reset Everything
			input[i] = new double[numberResults];
			current_char = '1';
			current_string = "";
			letterPosition = 0;
			//While not at the end of a group, retrieve the current dataset
			while (current_char != group_delimiter){
				if (inputfile.eof()){
					return i;
				}
				//Get the current char
				current_char = inputfile.get();

				if (current_char == individual_delimiter){//Reached the end of the current set


					if (type_of_input == 0){
						//The type is double, convert to double
						input[i][letterPosition] = stod(current_string);

					}
					else if (type_of_input == 1){
						//Is a char, get the first and only letter and make a double
						input[i][letterPosition] = (double)current_string.at(0);

					}
					else if (type_of_input == 2){
						//Is a string, work on conversion later
					}

					letterPosition++;
					current_string = "";
				}
				else{//String has not ended and is still correct
					current_string += current_char;
				}

			}

		}

		//Get current location in file
		int currentPosition = inputfile.tellg();
		inputfile.close();
		return currentPosition;
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){
			exit(0);
		}
		return numberOfRounds;
	}
}

//*********************************
//Running The Network
//*********************************

void recursiveTestInput(CGraphicsNetwork network){
	char userchoice;
	string userstartentry;
	double* input = new double[network.getI_input()];
	//Ask User for input
	std::cout << "Please enter " << network.getI_input() << " inputs seperated by spaces " << endl;
	cin.sync();
	std::getline(std::cin, userstartentry);
	for (int i = 0, k = 0; i < network.getI_input(); i++, k++){
		if (userstartentry.at(k) != ' '){
			input[i] = (double)userstartentry.at(k);
		}
		else{
			k++;
			input[i] = (double)userstartentry.at(k);
		}
	}


	while (true){
		if (pause){
			std::cout << endl << "Would you like to quit? ";
			cin.sync();
			userchoice = cin.get();
			if (userchoice == 'y'){
				exit(0);
			}
			else{
				pause = false;
				//Attach the signal handler
				//std::signal(SIGINT, signal_handler);
				continue;
			}
		}
		network.feedForward(input);
		//Move all in the array to the left by one
		for (int i = 0; i < network.getI_input() - 1; i++){
			input[i] = input[i + 1];
		}

		input[network.getI_input() - 1] = printVectorOutputChar(network.getOutput());


	}
}



void initializeFeedForwardNetwork(int argc, char** argv, CSettings settings){
	CGraphicsNetwork test;
	std::cout << "Would you like to train?";
	cin.sync();
	char in = cin.get();
	bool loadCheckFromFile = false;
	int PROBLEMS = std::stoi(argv[2]);
	SCheckpoint checkpoint = SCheckpoint();
	if (settings.b_loadFromCheckpoint){
		std::cout << "loading checkpoint " << endl;
		//Load the checkpoint from a file
		LoadCheckpointFromFile(checkpoint, settings.s_checkpoint_file);
		//Load the information from the checkpoint
		std::cout << "loading network " << endl;
		test = CGraphicsNetwork(&settings);
		loadFromFile(test, checkpoint.s_network_file_name);
		test.setSettings(&settings);
		loadCheckFromFile = true;
	}
	else if (settings.b_loadNetworkFromFile){//Load only the network from file
		std::cout << "loading network " << endl;
		test = CGraphicsNetwork(&settings);
		loadFromFile(test, settings.s_loadNetworkFile);
		test.setSettings(&settings);
	}
	else{//Start with a brand new network
		std::cout << "creating new network " << endl;
		vector<int> temp = vector<int>();
		temp.push_back(settings.i_input);
		temp.push_back(settings.i_output);
		vector<double> temp2 = vector<double>(settings.i_output);

		test = CGraphicsNetwork(temp, &settings);
	}
	if (in == 'y'){
		double **value;
		double **results;
		double **testIn;
		double **testOut;
		int start_Input;
		int start_output;
		value = new double*[settings.i_number_of_training];
		results = new double*[settings.i_number_of_training];
		testIn = new double*[settings.i_number_of_training];
		testOut = new double*[settings.i_number_of_training];

		if (settings.b_trainingFromFile){
			//Store training set
			std::cout << "loading training set " << endl;
			start_Input = getDataFromFile(settings.s_trainingSet, checkpoint.i_current_position_in_input_file, settings.i_number_of_training, settings.i_input, value, settings.i_trainingSetType);
			start_output = getDataFromFile(settings.s_outputTrainingFile, checkpoint.i_current_position_in_output_file, settings.i_number_of_training, settings.i_output, results, settings.i_outputTrainingSetType);

			if (!loadCheckFromFile){//Create a new checkpoint
				createNewCheckpoint(checkpoint, settings, results);
			}

			if (settings.b_testingFromFile){
				std::cout << "loading testing data " << endl;
				//Store the data to test the neural network
				getDataFromFile(settings.s_testSet, 0, settings.i_number_of_training, settings.i_input, testIn, settings.i_trainingSetType);
				getDataFromFile(settings.s_outputTestSet, 0, settings.i_number_of_training, settings.i_output, testOut, settings.i_outputTrainingSetType);
			}
			else{
				std::cout << "loading testing data " << endl;
				//If no test file is given, use some from the training set
				//Store the testing set
				getDataFromFile(settings.s_trainingSet, 0, settings.i_number_of_training, settings.i_input, testIn, settings.i_trainingSetType);
				getDataFromFile(settings.s_outputTrainingFile, 0, settings.i_number_of_training, settings.i_output, testOut, settings.i_outputTrainingSetType);

			}
		}
		std::cout << "training start " << endl;
		trainNetworkDelta(value, results, test, start_Input, start_output, testIn, testOut, settings.i_number_of_training, settings, checkpoint);

		for (int i = 0; i < settings.i_number_of_training; i++){
			delete value[i];
			delete results[i];
		}

		//Clean up memory
		delete value;
		delete results;
	}
	else{
		std::cout << "Starting output Loops" << endl;
		recursiveTestInput(test);
	}
}

//
//

void initializeRecurrentNetwork(int argc, char** argv, CSettings settings){
	ReccurentLoops RLoops = ReccurentLoops(settings,ReccurentLoops::LongTermShortTerm);
	
	double* temp = new double[settings.i_input];
#ifdef _DEBUG
	for (int i = 0; i < settings.i_input; i++){
		temp[i] = i;
	}
	//RLoops.runNetwork(temp);
	RLoops.testTraining();

#else
	RLoops.startTraining();
#endif
	
}

void initialize_loops(int argc, char** argv){

	CSettings settings;
	if (argc > 1){
		std::cout << "loading settings " << endl;
		settings = loadSettings(argv[1]);
	}
	else{
		string settingsLocation;
		std::cout << "Where are the settings? " << endl;
		std::getline(std::cin, settingsLocation);
		settings = loadSettings(settingsLocation);
	}
	std::cout << "1) Recurrent Neural Network" << endl;
	std::cout << "2) Feedforward Neural Network" << endl;
	switch (cin.get()){
	case '1':
		initializeRecurrentNetwork(argc, argv, settings);
		exit(0);
		break;
	case '2':
		pause = false;
		//Attach the signal handler
		std::signal(SIGINT, signal_handler);
		initializeFeedForwardNetwork(argc, argv, settings);
		exit(0);
		break;


	}


}



//*********************************
//Main Function
//*********************************

int main(int argc, char** argv){
#ifdef WINDOWS_COMPUTER
	initialize();
#endif


	pause = false;

	std::cout << "Starting Program... " << endl;


	initialize_loops(argc, argv);




	return 0;
}

