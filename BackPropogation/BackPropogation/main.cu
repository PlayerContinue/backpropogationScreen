#include "hip/hip_runtime.h"


//---------------------------------------------------------------------------------------
//Author:David Greenberg
//Desc: Initizializing algorithm. Contains the main function and only the main function
//----------------------------------------------------------------------------------------

//#define PROBLEMS 5
#pragma once
#include <vector>
#include <stdlib.h>
#include <bitset>
#include <math.h>
#include <iostream>
#include <fstream>
#include <string>
#include "CSettings.h"
#include "CGraphicNetwork.cuh"
#ifdef WINDOWS_COMPUTER
#include <wincon.h>
#endif

using namespace std;

//PROTOTYPES
bool writeToFile(CGraphicsNetwork &network, CSettings settings);
void testOutput2(double** value, CGraphicsNetwork &test, int size);
bool addToNetwork(CGraphicsNetwork &test, CSettings settings, double mean_square_error);
void getDataFromFile(string fileName, int start, int end, int numberResults, double** input);

void printVectorOutput(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}
	cout << (char)value;
}

void printVectorOutput2(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}
	cout << value;
}

void printArray(double* arrayA, int size){
	for (int i = 0; i < size; i++){
		cout << arrayA[i];
		cout << endl;
	}
}

void trainNetwork(double* value_1, double* value_2, double* value_3, double* value_4, CGraphicsNetwork &test, int rounds){
	//Train the network on simple test values
	for (int i = 0; i < rounds; i++){
		if (i % 2 == 0){
			test.backprop(value_1, value_2);
		}
		else{
			test.backprop(value_3, value_4);
		}
	}

}

void trainNetwork2(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, int rounds){

	int count = 0;
	//Train the network on simple test values
	for (int i = start; i < rounds; i++){
		test.backprop(value[count], results[count]);
		count++;
		if (count >= end){
			count = start;
		}
	}

}

//Returns true if the current mean_square_error is less than the lowest_mean_square
template <typename T>
bool checkThreshold(T mean_square, T lowest_mean_square, T threshold){
	if (mean_square < threshold && threshold >= 0){
		return false;
	}
	else if (mean_square <= lowest_mean_square){
		return true;
	}
	else{
		return false;
	}
}

//Train the current network
//Check if the limit has been reached as the stopping point
//Use mean square error to check distance
void trainNetworkDelta(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, double* testSetIn[], double* testSetOut[], int testLength, CSettings settings){
	int training_position = start;
	double mean_square_error = 0;
	int count_loops = 0;
	int times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
	double lowest_mean_square_error = (double)INT_MAX;
	do{

		test.backprop(value[training_position], results[training_position]);


		if (count_loops % settings.i_loops == 0){
			writeToFile(test, settings);
			//Test the output
			testOutput2(testSetIn, test, testLength);
			cout << " loop " << count_loops << endl;
		}
		training_position++;
		count_loops++;

		if (training_position >= end){
			training_position = start;
		}

		if (count_loops%settings.i_number_before_growth_potential == 0){

			mean_square_error = test.getMeanSquareError(testSetIn, testSetOut, testLength);
			//Set the current lowest
			if (mean_square_error < lowest_mean_square_error){
				lowest_mean_square_error = mean_square_error;
				//Reset value since error was lowered
				times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
			}
			else{
				times_lowest_mean_square_error_to_large--;
				//If the below is true, something close to the limit has been reached, the network needs to change size 
				if (times_lowest_mean_square_error_to_large == 0){
					//Add new nodes to the network
					if (addToNetwork(test, settings, mean_square_error)){
						//Reset the number allowed
						//Since new ones may have been
						times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures + RandInt(10, 50);
					}
					else{
						//Reset the number of times before a growth is attempted
						times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
					}
				}
			}
		}

		//Loop until the error is smaller than the threshold
	} while (settings.d_threshold < mean_square_error && training_position == start || training_position != start);
}

//Returns true if a neuron was added
bool addToNetwork(CGraphicsNetwork &test, CSettings settings, double mean_square_error){

	//Get delta in success
	double success = test.getSuccessRate() - test.getPreviousSuccessRate();
	double averagedistance = abs(test.getPreviousAverageDistance() - test.getAverageDistance());
	double delta = abs(test.getAverageDelta());
	double mean_square_error_dif = mean_square_error - settings.d_threshold;
#ifdef FULL_SUCCESS
	double full_success = test.getFullSuccessRate() - test.getFullPreviousSuccessRate();
#endif
	//Add a new layer if the success is too low and the threshold has not been reached
	if (success <= settings.d_neuron_success_threshold && mean_square_error_dif > 0 && mean_square_error_dif > settings.d_neuron_distance_threshold &&  mean_square_error_dif < settings.d_row_distance_threshold){
		if (test.getNumLayers() == 2){
			test.addLayer(test.getNumLayers() + 1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) / 5);
		}
		else{
			test.addNeuronToLayer(5, test.getNumLayers() - 2, 2);
		}
		test.resetNetwork();
		return true;
	}
	else if (success <= settings.d_row_success_threshold && mean_square_error_dif > 0 && mean_square_error_dif >= settings.d_row_distance_threshold){
		test.addLayer(test.getNumLayers() + 1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) / 5);
		test.resetNetwork();
		return true;
	}
	return false;


}

void testOutput(double* value_1, double* value_3, CGraphicsNetwork &test){
	vector<double> temp2;
	cout << "input";
	cout << endl;
	printArray(value_1, 2);
	test.feedForward(value_1);
	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;


	test.feedForward(value_3);
	temp2 = test.getOutput();

	cout << "input";
	cout << endl;
	printArray(value_3, 2);

	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;
}

void testOutput2(double** value, CGraphicsNetwork &test, int size){
	vector<double> temp2;

	for (int i = 0; i < size; i++){
		cout << "input";
		cout << endl;
		printArray(value[i], 2);
		test.feedForward(value[i]);
		temp2 = test.getOutput();
		cout << "output";
		cout << endl;
		printVectorOutput(temp2);
		cout << endl;
	}
}

//Output the network to a file
bool writeToFile(CGraphicsNetwork &network, CSettings settings){
	static int file_number = 0;
	file_number++;
	std::ofstream outputfile;
	outputfile.open("networks/" + settings.s_network_name + std::to_string(file_number) + ".txt", ios::trunc);
	if (outputfile.is_open()){
		//Output the network
		outputfile << network << flush;
		outputfile.close();
		return true;
	}
	else{
		cout << "Unable to write checkpoint to file." << endl;
		cout << "continue?";
		return false;
	}


}

bool loadFromFile(CGraphicsNetwork& network, string fileName){
	std::ifstream inputfile;

	inputfile.open(fileName, ios_base::beg);
	if (inputfile.is_open()){
		inputfile >> network;
		inputfile.close();
		return true;
	}
	else{
		cout << "Unable to read from file." << endl;
		cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}
		return false;
	}
}

CSettings loadSettings(string fileName){
	std::ifstream inputfile;
	inputfile.open(fileName, ios_base::beg);
	CSettings settings;
	if (inputfile.is_open()){

		inputfile >> settings;
		inputfile.close();
		return settings;
	}
	else{
		cout << "Unable to read from file." << endl;
		cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}

		return settings;
	}
}


#ifdef WINDOWS_COMPUTER
BOOL WINAPI ConsoleHandlerRoutine(DWORD dwCtrlType)
{
	if (dwCtrlType == CTRL_CLOSE_EVENT)
	{
		return TRUE;
	}

	return FALSE;
}


void initialize(){
	if (SetConsoleCtrlHandler(ConsoleHandlerRoutine, TRUE) == false){
		printf("Unable to attach Handler");
		cout << "continue? ";
		if (cin.get() == 'n'){
			exit(0);
		}
	}
}

#endif

void getDataFromFile(string fileName, int start, int end, int numberResults, double** input){
	std::ifstream inputfile;

	inputfile.open(fileName);
	if (inputfile.is_open()){
		inputfile.seekg(start);
		int k = -1;
		int letterPosition = 0;
		for (int i = 0; i < end - start; i++){
			if (i%numberResults == 0){
				{
					k++;
					input[k] = new double[numberResults];
					letterPosition = 0;
				}

				input[k][letterPosition] = (int)inputfile.get();
				letterPosition++;
			}


		}
		inputfile.close();
	}
	else{
		cout << "Unable to read from file." << endl;
		cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}
	}
}


int main(int argc, char** argv){
#ifdef WINDOWS_COMPUTER
	initialize();
#endif
	CGraphicsNetwork test;
	CSettings settings;
	if (argc > 1){
		settings = loadSettings(argv[1]);
	}

	int PROBLEMS = std::stoi(argv[2]);

	if (settings.b_loadNetworkFromFile){
		test = CGraphicsNetwork();
		loadFromFile(test, settings.s_loadNetworkFile);
	}
	else{

		vector<int> temp = vector<int>();
		temp.push_back(settings.i_input);
		temp.push_back(settings.i_output);
		vector<double> temp2 = vector<double>(settings.i_output);

		test = CGraphicsNetwork(temp, settings.d_beta, settings.d_alpha);
	}
	double **value;
	double **results;
	double **testIn;
	double **testOut;
	if (settings.b_trainingFromFile && settings.b_trainingFromFile){
		int zero;
		int number2;
		value = new double*[PROBLEMS];
		results = new double*[PROBLEMS];
		for (int i = 0; i < PROBLEMS; i++){
			value[i] = new double[2];
			double number = (double)RandInt(0, PROBLEMS) + 1;
			value[i][0] = number;
			value[i][1] = number + 1;
			//number = (double)(1 / (number + number + 1));
			results[i] = new double[32];
			number2 = number + number + 1;
			zero = 1;
			for (int j = 31; j >= 0; j--){
				results[i][j] = (double)(((int)(number2 & zero)) != 0 ? .7 : .1);
				//Shift left by one
				zero = zero << 1;
			}
		}

		double **testIn = new double*[PROBLEMS];
		double **testOut = new double*[PROBLEMS];

		for (int i = 0; i < 100; i++){
			testIn[i] = new double[2];
			double number = (double)RandInt(0, PROBLEMS) + 1;
			testIn[i][0] = number;
			testIn[i][1] = number + 1;
			testOut[i] = new double[32];
			number2 = number + number + 1;
			zero = 1;
			for (int k = 31; k >= 0; k--){
				testOut[i][k] = (double)(((int)(number2 & zero)) != 0 ? .7 : .1);
				//Shift left by one
				zero = zero << 1;
			}
		}
		//trainNetwork2(value, results, test, 0, PROBLEMS, settings.i_loops);
		trainNetworkDelta(value, results, test, 0, PROBLEMS, testIn, testOut, 100, settings);

	}
	else{
		value = new double*[settings.i_number_of_training];
		results = new double*[settings.i_number_of_training];
		getDataFromFile(settings.s_trainingSet, 0, settings.i_number_of_training, settings.i_input, value);
		//trainNetwork2(value, results, test, 0, PROBLEMS, settings.i_loops);
		trainNetworkDelta(value, results, test, 0, settings.i_number_of_training, testIn, testOut, 100, settings);
	}






	testOutput2(value, test, PROBLEMS);

	for (int i = 0; i < PROBLEMS; i++){
		delete value[i];
		delete results[i];
	}

	//Clean up memory
	delete value;
	delete results;

	return 0;
}

