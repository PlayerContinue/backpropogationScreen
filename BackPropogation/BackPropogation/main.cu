#include "hip/hip_runtime.h"


//---------------------------------------------------------------------------------------
//Author:David Greenberg
//Desc: Initizializing algorithm. Contains the main function and only the main function
//----------------------------------------------------------------------------------------

//#define PROBLEMS 5
#pragma once
#include <vector>
#include <stdlib.h>
#include <bitset>
#include <math.h>
#include <iostream>
#include "util.h"
#include "CGraphicNetwork.cuh"
using namespace std;


void printVectorOutput(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){
		
		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}
	cout << value;
}

void printArray(double* arrayA, int size){
	for (int i = 0; i < size; i++){
		cout << arrayA[i];
		cout << endl;
	}
}

void trainNetwork(double* value_1, double* value_2, double* value_3, double* value_4, CGraphicsNetwork &test, int rounds){
	//Train the network on simple test values
	for (int i = 0; i < rounds; i++){
		if (i % 2 == 0){
			test.backprop(value_1, value_2);
		}
		else{
			test.backprop(value_3, value_4);
		}
	}

}

void trainNetwork2(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, int rounds){

	int count = 0;
	//Train the network on simple test values
	for (int i = start; i < rounds; i++){
		test.backprop(value[count], results[count]);
		count++;
		if (count >= end){
			count = start;
		}
	}

}

void addToNetwork(CGraphicsNetwork &test){

	//Get delta in success
	double success = test.getSuccessRate() - test.getPreviousSuccessRate();
	double averagedistance = abs(test.getAverageDistance() - test.getPreviousAverageDistance());
	double delta = abs(test.getAverageDelta());
	double distanceMeasure = .01;
	//Add a new layer if the success is too low and either the there are no hidden layers or the previous layer has too many nodes
	if (success < .1 && averagedistance < distanceMeasure * .0000001 && (test.getNumLayers() == 2 || test.getSuccessRate() < .3 && test.getPreviousSuccessRate() < .3)){
		test.addLayer(-1, test.getNumNeuronsInLayer(test.getNumLayers()-1)/5);
	}else if (success < .2 && averagedistance < distanceMeasure * .0003 && test.getSuccessRate() < .8){
		if (test.getNumLayers() == 2){
			test.addLayer(-1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) / 5);
		}
		else{
			test.addNeuronToLayer(1, test.getNumLayers() - 2, 2);
		}
	}
	test.resetNetwork();

}

void testOutput(double* value_1, double* value_3, CGraphicsNetwork &test){
	vector<double> temp2;
	cout << "input";
	cout << endl;
	printArray(value_1, 2);
	test.feedForward(value_1);
	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;


	test.feedForward(value_3);
	temp2 = test.getOutput();

	cout << "input";
	cout << endl;
	printArray(value_3, 2);

	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;
}

void testOutput2(double** value, CGraphicsNetwork &test, int size){
	vector<double> temp2;

	for (int i = 0; i < size; i++){
		cout << "input";
		cout << endl;
		printArray(value[i], 2);
		test.feedForward(value[i]);
		temp2 = test.getOutput();
		cout << "output";
		cout << endl;
		printVectorOutput(temp2);
		cout << endl;
	}
}

int main(int argc, char** argv){
	int PROBLEMS = (((int)argv[1][0])-48);
	vector<int> temp = vector<int>();
	temp.push_back(2);
	temp.push_back(32);
	vector<double> temp2 = vector<double>(32);

	CGraphicsNetwork test = CGraphicsNetwork(temp, 1, 2);
	int zero;
	int number2;
	double **value = new double*[PROBLEMS];
	double **results = new double*[PROBLEMS];
	for (int i = 0; i < PROBLEMS; i++){
		value[i] = new double[2];
		double number = (double)i + 1;
		value[i][0] = number;
		value[i][1] = number + 1;
		//number = (double)(1 / (number + number + 1));
		results[i] = new double[32];
		number2 = number + number + 1;
		zero = 1;
		for (int j = 31; j >= 0; j--){
			results[i][j] = (double)(((int)(number2 & zero)) != 0 ? .7 : .1);
			//Shift left by one
			zero = zero << 1;
		}

	}

	for (int i = 0; i < 500; i++){
		trainNetwork2(value, results, test, 0, PROBLEMS, 1000);

		if (i % 2 == 0 || true){
			//Test the output
			testOutput2(value, test, PROBLEMS);
		}

		//Add new nodes to the network
		addToNetwork(test);

	}

	testOutput2(value, test, PROBLEMS);

	for (int i = 0; i < PROBLEMS; i++){
		delete value[i];
		delete results[i];
	}

	//Clean up memory
	delete value;
	delete results;

	return 0;
}

