#include "hip/hip_runtime.h"


//---------------------------------------------------------------------------------------
//Author:David Greenberg
//Desc: Initizializing algorithm. Contains the main function and only the main function
//----------------------------------------------------------------------------------------

#define PROBLEMS 1000
#pragma once
#include <vector>
#include <stdlib.h>
#include <bitset>
#include <math.h>
#include <iostream>
#include "util.h"
#include "CGraphicNetwork.cuh"
using namespace std;


void printVectorOutput(vector<double> vectorA){
	for (int i = 0; i < vectorA.size(); i++){
		cout << 1 / vectorA.at(i);
		cout << endl;
		cout << vectorA.at(i);
		cout << endl;
	}
}

void printArray(double* arrayA, int size){
	for (int i = 0; i < size; i++){
		cout << arrayA[i];
		cout << endl;
	}
}

void trainNetwork(double* value_1, double* value_2, double* value_3, double* value_4, CGraphicsNetwork &test, int rounds){
	//Train the network on simple test values
	for (int i = 0; i < rounds; i++){
		if (i % 2 == 0){
			test.backprop(value_1, value_2);
		}
		else{
			test.backprop(value_3, value_4);
		}
	}

}

void trainNetwork2(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, int rounds){

	int count = 0;
	//Train the network on simple test values
	for (int i = start; i < rounds; i++){
		test.backprop(value[count], results[count]);
		count++;
		if (count >= end){
			count = start;
		}
	}

}

void addToNetwork(CGraphicsNetwork &test){

	//Get delta in success
	double success = test.getSuccessRate() - test.getPreviousSuccessRate();
	double averagedistance = abs(test.getAverageDistance());
		//Add a new layer if the success is too low and either the there are no hidden layers or the previous layer has too many nodes
	if (success < .1 && averagedistance > .002 && (test.getNumLayers() == 2 || test.getSuccessRate() < .3 && test.getPreviousSuccessRate() < .3 && RandBool())){
			test.addLayer(-1, 1);
		}

	if (success < .2 && averagedistance > .001 && test.getSuccessRate() < .8){
			test.addNeuronToLayer(test.getNumLayers() - 2);
		}
	test.resetNetwork();

}

void testOutput(double* value_1, double* value_3, CGraphicsNetwork &test){
	vector<double> temp2;
	cout << "input";
	cout << endl;
	printArray(value_1, 2);
	test.feedForward(value_1);
	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;


	test.feedForward(value_3);
	temp2 = test.getOutput();

	cout << "input";
	cout << endl;
	printArray(value_3, 2);

	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;
}

void testOutput2(double** value, CGraphicsNetwork &test, int size){
	vector<double> temp2;

	for (int i = 0; i < size; i++){
		cout << "input";
		cout << endl;
		printArray(value[i], 2);
		test.feedForward(value[i]);
		temp2 = test.getOutput();
		cout << "output";
		cout << endl;
		printVectorOutput(temp2);
		cout << endl;
	}
}

int main(int argc, char* argv){
	vector<int> temp = vector<int>();
	temp.push_back(2);
	temp.push_back(sizeof(double));
	CGraphicsNetwork test = CGraphicsNetwork(temp, 1, 2);
	double zero =(double) ~0;
	double **value = new double*[PROBLEMS];
	double **results = new double*[PROBLEMS];
	for (int i = 0; i < PROBLEMS; i++){
		value[i] = new double[2];
		double number = (double)i + 1;
		value[i][0] = number;
		value[i][1] = number + 1;
		number = (double)(1 / (number + number + 1));
		results[i] = new double[sizeof(double)];
		for (int j = 0; j < sizeof(double); j++){
			results[i][j] = (double)((number & zero) ? 0 : 1)
		}
		
	}


	for (int i = 0; i < 500; i++){
		trainNetwork2(value, results, test, 0, PROBLEMS, 1000);

		if (i % 2 == 0){
			//Test the output
			testOutput2(value, test, PROBLEMS);
		}

		//Add new nodes to the network
		addToNetwork(test);

	}

	testOutput2(value, test, PROBLEMS);

	for (int i = 0; i < PROBLEMS; i++){
		delete value[i];
		delete results[i];
	}

	//Clean up memory
	delete value;
	delete results;

	return 0;
}

