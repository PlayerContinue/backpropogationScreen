#include "hip/hip_runtime.h"


//---------------------------------------------------------------------------------------
//Author:David Greenberg
//Desc: Initizializing algorithm. Contains the main function and only the main function
//----------------------------------------------------------------------------------------

//#define PROBLEMS 5
#define FIRST_TEST
#define PAUSE
#pragma once
#include <vector>
#include <stdlib.h>
#include <bitset>
#include <math.h>
#include <iostream>
#include <fstream>
#include <string>
#include <csignal>
#include "CSettings.h"
#include "CGraphicNetwork.cuh"
#ifdef WINDOWS_COMPUTER
#include <wincon.h>
#endif

using namespace std;


static volatile bool pause;


//PROTOTYPES
int writeToFile(CGraphicsNetwork &network, CSettings settings);
void testOutput2(double** value, CGraphicsNetwork &test, int size);
bool addToNetwork(CGraphicsNetwork &test, CSettings settings, SCheckpoint& checkpoint, double** testSet, double mean_square_error);
int getDataFromFile(string fileName, int start, int end, int numberResults, double** input);

void printVectorOutput(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}
	std::cout << value << endl;
	if (value <= 255 & value > 0){
		std::cout << (char)value << endl;
	}

}

int printVectorOutputChar(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}

	if (value <= 255 & value > 0){
		std::cout << (char)value;
	}
	else{
		std::cout << value;
	}

	return value;

}

void printVectorOutput2(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}
	std::cout << value;
}

void printArray(double* arrayA, int size){
	for (int i = 0; i < size; i++){
		std::cout << (char)arrayA[i];
		std::cout << endl;
	}
}

void trainNetwork(double* value_1, double* value_2, double* value_3, double* value_4, CGraphicsNetwork &test, int rounds){
	//Train the network on simple test values
	for (int i = 0; i < rounds; i++){
		if (i % 2 == 0){
			test.backprop(value_1, value_2);
		}
		else{
			test.backprop(value_3, value_4);
		}
	}

}

void trainNetwork2(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, int rounds){

	int count = 0;
	//Train the network on simple test values
	for (int i = start; i < rounds; i++){
		test.backprop(value[count], results[count]);
		count++;
		if (count >= end){
			count = start;
		}
	}

}

//Returns true if the current mean_square_error is less than the lowest_mean_square
template <typename T>
bool checkThreshold(T mean_square, T lowest_mean_square, T threshold){
	if (mean_square < threshold && threshold >= 0){
		return false;
	}
	else if (mean_square <= lowest_mean_square){
		return true;
	}
	else{
		return false;
	}
}

int numberFullSame(CGraphicsNetwork test, double** in, int size){
	vector<double> output;
	vector<double> output2;
	int count_success = 0;
	for (int i = 0; i < size; i++){
		test.feedForward(in[i]);
		if (i != 0){
			output2 = output;
		}
		output = test.getOutput();
		if (i != 0){
			for (int j = 0; j < output.size(); j++){
				if (output[j] != output2[j]){
					count_success++;
					break;
				}
			}
		}
	}

	return (size - count_success);
}

//******************************
//Save, Load, Create Checkpoints
//******************************

//Create a checkpoint containing all the current values of the system
void createCheckpoint(CGraphicsNetwork test, SCheckpoint& checkpoint, CSettings settings){
	int checkpoint_number = writeToFile(test, settings);
	checkpoint.s_network_file_name = "networks/" + settings.s_network_name + std::to_string(checkpoint_number) + ".txt";
	//Write the checkpoint to a file
	std::ofstream outputfile;
	outputfile.precision(30);
	outputfile.open("checkpoints/" + settings.s_network_name + "_checkpoint_" + std::to_string(checkpoint_number) + ".txt", ios::trunc);
	if (outputfile.is_open()){
		//Output the network
		outputfile << checkpoint << flush;
		outputfile.close();
	}
	else{
		std::cout << "Unable to write checkpoint to file." << endl;
		std::cout << "continue?";
	}

}

void LoadCheckpointFromFile(SCheckpoint& checkpoint, string s_file_name){
	std::ifstream inputfile;

	inputfile.open(s_file_name, ios_base::beg);
	if (inputfile.is_open()){
		inputfile >> checkpoint;
		inputfile.close();
	}
	else{
		std::cout << "Unable to read checkpoint from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}
	}
}

//Create a new checkpoint object from the created settings
void createNewCheckpoint(SCheckpoint& checkpoint, CSettings settings){
	checkpoint.i_number_of_loops_checkpoint = 0;
	checkpoint.i_number_of_loops = 0;
	checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
	checkpoint.d_mean_square_error = (double)INT_MAX;
	checkpoint.d_lowest_mean_square_error = (double)INT_MAX;
	checkpoint.d_previous_mean_square_error = 0;
	checkpoint.d_row_distance_threshold = settings.d_row_distance_threshold;
	checkpoint.d_neuron_distance_threshold = settings.d_neuron_distance_threshold;
}

//*********************************
//Signal Handlers
//*********************************

void signal_handler(int signal)
{
	if (signal == SIGINT){
		pause = true;
	}
	std::signal(SIGINT, signal_handler);
}

//******************************
//Training the Network
//******************************

//Train the current network
//Check if the limit has been reached as the stopping point
//Use mean square error to check distance
void trainNetworkDelta(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, double* testSetIn[], double* testSetOut[], int testLength, CSettings settings, SCheckpoint& checkpoint){

	int number_of_rounds_returned[2];
	int new_end = end;
	do{

		//Pause the program and perform one of the following options
#ifdef PAUSE
		char userin = 'l';
		if (pause){
			std::cout << "1) print output " << endl;;
			std::cout << "2) run same check " << endl;
			std::cout << "3) get mean square " << endl;
			std::cout << "4) Save Checkpoint " << endl;
			std::cout << "5) check MSE on current input " << endl;
			std::cout << "6) continue " << endl;
			std::cout << "7) exit " << endl;

			std::cout << " loop " << checkpoint.i_number_of_loops_checkpoint << endl;
			std::cout.precision(30);

			do{
				
				if (userin != 'l'){
					std::cout << "Anything else? ";
				}
				cin.sync();
				userin = cin.get();
				switch (userin){
				case '1':
					//Test the output
					std::cout << "Getting Output " << endl;
					testOutput2(testSetIn, test, testLength);
					break;
				case '2':
					std::cout << "Getting Number Same " << endl;
					std::cout << numberFullSame(test, testSetIn, settings.i_number_of_training) << endl;
					break;
				case '3':
					std::cout << "Getting Mean Square " << endl;
					std::cout << test.getMeanSquareError(testSetIn, testSetOut, testLength) << endl;
					break;
				case '4':
					std::cout << "Creating Checkpoint " << endl;
					createCheckpoint(test, checkpoint, settings);
					std::cout << "Checkpoint created " << endl;
					break;

				case '5':
					std::cout << "Getting MSE " << endl;
					std::cout << test.getSingleMeanSquareError(value[checkpoint.i_number_of_loops-1], results[checkpoint.i_number_of_loops-1], testLength) << endl;
					break;
				case '6':
					break;
				case '7':
					std::wcout << "Would you like to create a checkpoint?";
					cin.sync();
					if (cin.get() == 'y'){
						std::cout << "Creating Checkpoint " << endl;
						createCheckpoint(test, checkpoint, settings);
						std::cout << "Checkpoint created " << endl;
					}
					exit(0);
					break;
				}
			} while (userin != '6' && userin != 'n');

			pause = false;
			std::cout << "finished" << endl;
		}
#endif

		test.backprop(value[checkpoint.i_number_of_loops], results[checkpoint.i_number_of_loops]);


		if (checkpoint.i_number_of_loops_checkpoint % settings.i_loops == 0){
			createCheckpoint(test, checkpoint, settings);
		}
		checkpoint.i_number_of_loops++;
		checkpoint.i_number_of_loops_checkpoint++;

		if (checkpoint.i_number_of_loops >= new_end){
			checkpoint.i_number_of_loops = start;

			if (new_end < end){
				//TODO Add asking for a new file or to reuse file
				exit(0);
			}

			if (settings.b_trainingFromFile){

				//Store training set
				number_of_rounds_returned[0] = getDataFromFile(settings.s_trainingSet, checkpoint.i_number_of_loops_checkpoint*settings.i_input, settings.i_number_of_training, settings.i_input, value);

			}

			if (settings.b_testingFromFile){
				number_of_rounds_returned[1] = getDataFromFile(settings.s_outputTrainingFile, checkpoint.i_number_of_loops_checkpoint*settings.i_output, settings.i_number_of_training, settings.i_output, results);
			}

			new_end = (number_of_rounds_returned[0] < number_of_rounds_returned[1] ? number_of_rounds_returned[0] : number_of_rounds_returned[1]);

		}
		//Get the mean_square_error when the number of loops reaches a user defined values
		if (checkpoint.i_number_of_loops_checkpoint%settings.i_number_before_growth_potential == 0){

			//Set the current mean_square_error as the previous error
			checkpoint.d_previous_mean_square_error = checkpoint.d_mean_square_error;

			//Retrieve the new mean_square_error
			checkpoint.d_mean_square_error = test.getMeanSquareError(testSetIn, testSetOut, testLength);

			//Set the current lowest
			if (checkpoint.d_mean_square_error < checkpoint.d_lowest_mean_square_error && checkpoint.d_mean_square_error != checkpoint.d_previous_mean_square_error){
				checkpoint.d_lowest_mean_square_error = checkpoint.d_mean_square_error;

				//Reset value since error was lowered
				checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
			}
			else{
				checkpoint.i_times_lowest_mean_square_error_to_large--;

				//Keep track of the number of times the mean square error has been equal
				if (checkpoint.d_previous_mean_square_error == checkpoint.d_mean_square_error){
					checkpoint.i_equal_square_errors++;
				}
				//If the below is true, something close to the limit has been reached, the network needs to change size 
				if (checkpoint.i_times_lowest_mean_square_error_to_large <= 0){
					//Add new nodes to the network
					if (addToNetwork(test, settings, checkpoint, testSetIn, checkpoint.d_mean_square_error)){
						//Reset the number allowed
						//Since new ones may have been
						checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures + 10;
					}
					else{
						//Reset the number of times before a growth is attempted
						checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
					}

					//Reset the number of times the mean_square_error has been equal
					checkpoint.i_equal_square_errors = 0;

				}
			}

		}

		//Loop until the error is smaller than the threshold
	} while (settings.d_threshold < checkpoint.d_mean_square_error && checkpoint.i_number_of_loops == start || checkpoint.i_number_of_loops != start);
}

//******************************
//Modifying the Network
//******************************

//Returns true if a neuron was added
bool addToNetwork(CGraphicsNetwork &test, CSettings settings, SCheckpoint& checkpoint, double** testSet, double mean_square_error){

	//Get delta in success
	double success = test.getSuccessRate() - test.getPreviousSuccessRate();
	double mean_square_error_dif = mean_square_error - settings.d_threshold;
#ifdef FULL_SUCCESS
	double full_success = test.getFullSuccessRate() - test.getFullPreviousSuccessRate();
#endif
	//Get the number of test sets returning the exact same values
	//int numberSame = numberFullSame(test, testSet, settings.i_number_of_training);

	//Add a new layer if the success is too low and the threshold has not been reached
	//A layer should be added if the mean square error remains constant as the current layer has been fully trained to give a particular output
	//Therefore a function should be added to deal with that particular output
	if (success <= settings.d_row_success_threshold && mean_square_error_dif > 0 && mean_square_error_dif >= checkpoint.d_row_distance_threshold){ //&& numberSame < settings.i_number_allowed_same){
		test.addLayer(-1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) * 5);
		//Increment the size of the need mean distance to get a new layer
		//And decrease the size of the needed mean distance to get a new neuron
		if (checkpoint.d_neuron_distance_threshold > 0){
			checkpoint.d_row_distance_threshold += settings.d_row_distance_threshold * .1;
			checkpoint.d_neuron_distance_threshold -= settings.d_neuron_distance_threshold * .1;
		}
		test.resetNetwork();
		return true;
	}
	else if (success <= settings.d_neuron_success_threshold && mean_square_error_dif > 0 && mean_square_error_dif >= checkpoint.d_neuron_distance_threshold){
		if (test.getNumLayers() == 2){
			test.addLayer(-1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) * 5);
		}
		else{
			//Double the number of nodes in every non input/output row 
			//Keeps size of each row equivalent
			//Good for initial growth
			for (int i = 1; i < test.getNumLayers() - 1; i++){
				test.addNeuronToLayer(i, i, test.getNumNeuronsInLayer(i));
			}
			//Increment the size of the need mean distance to get a new neuron
			//And decrease the size of the needed mean distance to get a new row
			if (checkpoint.d_row_distance_threshold > 0){
				checkpoint.d_row_distance_threshold -= settings.d_row_distance_threshold * .1;
				checkpoint.d_neuron_distance_threshold += settings.d_neuron_distance_threshold * .1;
			}
		}
		test.resetNetwork();
		return true;
	}
	//If both fail, but a large gap still exists between the threshold and the set distances, decrease them both.
	//This is to allow the system to grow further when needed.
	//However, only occur when the network is fully trained
	else if (checkpoint.i_equal_square_errors >= settings.i_number_allowed_failures){
		checkpoint.d_row_distance_threshold -= settings.d_row_distance_threshold * .1;
		checkpoint.d_neuron_distance_threshold -= settings.d_neuron_distance_threshold * .1;
	}
	return false;


}

//******************************
//Testing Output
//******************************

void testOutput(double* value_1, double* value_3, CGraphicsNetwork &test){
	vector<double> temp2;
	std::cout << "input";
	std::cout << endl;
	printArray(value_1, 2);
	test.feedForward(value_1);
	temp2 = test.getOutput();
	std::cout << "output";
	std::cout << endl;
	printVectorOutput(temp2);
	std::cout << endl;


	test.feedForward(value_3);
	temp2 = test.getOutput();

	std::cout << "input";
	std::cout << endl;
	printArray(value_3, 2);

	temp2 = test.getOutput();
	std::cout << "output";
	std::cout << endl;
	printVectorOutput(temp2);
	std::cout << endl;
}

void testOutput2(double** value, CGraphicsNetwork &test, int size){
	vector<double> temp2;

	for (int i = 0; i < size; i++){
		std::cout << "input";
		std::cout << endl;

		printArray(value[i], 6);

		test.feedForward(value[i]);

		temp2 = test.getOutput();

		std::cout << "output";
		std::cout << endl;
		printVectorOutput(temp2);
		std::cout << endl;
	}
}

//******************************
//Writing And Loading From Files
//******************************

//Output the network to a file
int writeToFile(CGraphicsNetwork &network, CSettings settings){
	static int file_number = 0;
	file_number++;
	std::ofstream outputfile;
	outputfile.open("networks/" + settings.s_network_name + std::to_string(file_number) + ".txt", ios::trunc);
	if (outputfile.is_open()){
		//Output the network
		outputfile << network << flush;
		outputfile.close();
	}
	else{
		std::cout << "Unable to write checkpoint to file." << endl;
		std::cout << "continue?";
	}

	return file_number;

}

bool loadFromFile(CGraphicsNetwork& network, string fileName){
	std::ifstream inputfile;

	inputfile.open(fileName, ios_base::beg);
	if (inputfile.is_open()){
		inputfile >> network;
		inputfile.close();
		return true;
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}
		return false;
	}
}

CSettings loadSettings(string fileName){
	std::ifstream inputfile;
	inputfile.open(fileName, ios_base::beg);
	CSettings settings;
	if (inputfile.is_open()){

		inputfile >> settings;
		inputfile.close();
		return settings;
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}

		return settings;
	}
}


#ifdef WINDOWS_COMPUTER
BOOL WINAPI ConsoleHandlerRoutine(DWORD dwCtrlType)
{
	if (dwCtrlType == CTRL_CLOSE_EVENT)
	{
		return TRUE;
	}

	return FALSE;
}


void initialize(){
	if (SetConsoleCtrlHandler(ConsoleHandlerRoutine, TRUE) == false){
		printf("Unable to attach Handler");
		std::cout << "continue? ";
		if (cin.get() == 'n'){
			exit(0);
		}
	}
}

#endif

//Retrieve the data from a file
//start: Where to start gathering characters from in the file
//numberOfRounds: The number of character sets which are needed to be retrieve
//numberResults: How many characters should be retrieved for a single round
//input : the storage container for the input
int getDataFromFile(string fileName, int start, int numberOfRounds, int numberResults, double** input){
	std::ifstream inputfile;

	inputfile.open(fileName);
	if (inputfile.is_open()){
		inputfile.seekg(start);
		int k = -1;
		int letterPosition = 0;
		for (int i = 0; i < (numberOfRounds)*numberResults; i++){
			if (i%numberResults == 0){
				k++;
				input[k] = new double[numberResults];
				letterPosition = 0;
			}

			input[k][letterPosition] = (int)inputfile.get();
#ifdef FIRST_TEST 
			if (input[k][letterPosition] == 48.0){
				input[k][letterPosition] = .1;
			}
			else if (input[k][letterPosition] == 49.0){
				input[k][letterPosition] = .9;
			}
#endif
			if (inputfile.eof()){
				return k;
			}

			letterPosition++;



		}
		inputfile.close();
		return numberOfRounds;
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){
			exit(0);
		}
		return numberOfRounds;
	}
}

void recursiveTestInput(CGraphicsNetwork network){
	char userchoice;
	string userstartentry;
	double* input = new double[network.getI_input()];
	//Ask User for input
	std::cout << "Please enter " << network.getI_input() << " inputs seperated by spaces " << endl;
	cin.sync();
	std::getline(std::cin,userstartentry);
	for (int i = 0, k=0; i < network.getI_input(); i++,k++){
		if (userstartentry.at(k) != ' '){
			input[i] = (double)userstartentry.at(k);
		}
		else{
			k++;
			input[i] = (double)userstartentry.at(k);
		}
	}

	
	while (true){
		if (pause){
			std::cout << endl << "Would you like to quit? ";
			cin.sync();
			userchoice = cin.get();
			if (userchoice == 'y'){
				exit(0);
			}
			else{
				pause = false;
				//Attach the signal handler
				//std::signal(SIGINT, signal_handler);
				continue;
			}
		}
		network.feedForward(input);
		//Move all in the array to the left by one
		for (int i = 0; i < network.getI_input() - 1; i++){
			input[i] = input[i + 1];
		}

		input[network.getI_input() - 1] = printVectorOutputChar(network.getOutput());


	}
}

//
//

void initialize_loops(int argc, char** argv){
	CGraphicsNetwork test;
	CSettings settings;
	if (argc > 1){
		std::cout << "loading settings " << endl;
		settings = loadSettings(argv[1]);
	}

	std::cout << "Would you like to train?";
	cin.sync();
	char in = cin.get();

	int PROBLEMS = std::stoi(argv[2]);
	SCheckpoint checkpoint = SCheckpoint();
	if (settings.b_loadFromCheckpoint){
		std::cout << "loading checkpoint " << endl;
		//Load the checkpoint from a file
		LoadCheckpointFromFile(checkpoint, settings.s_checkpoint_file);
		//Load the information from the checkpoint
		std::cout << "loading network " << endl;
		test = CGraphicsNetwork(&settings);
		loadFromFile(test, checkpoint.s_network_file_name);
		test.setSettings(&settings);
	}
	else if (settings.b_loadNetworkFromFile){//Load only the network from file
		std::cout << "loading network " << endl;
		test = CGraphicsNetwork(&settings);
		createNewCheckpoint(checkpoint, settings);
		loadFromFile(test, settings.s_loadNetworkFile);
		test.setSettings(&settings);
	}
	else{//Start with a brand new network
		std::cout << "creating new network " << endl;
		vector<int> temp = vector<int>();
		createNewCheckpoint(checkpoint, settings);
		temp.push_back(settings.i_input);
		temp.push_back(settings.i_output);
		vector<double> temp2 = vector<double>(settings.i_output);

		test = CGraphicsNetwork(temp, &settings);
	}
	if (in == 'y'){
		double **value;
		double **results;
		double **testIn;
		double **testOut;

		value = new double*[settings.i_number_of_training];
		results = new double*[settings.i_number_of_training];
		testIn = new double*[settings.i_number_of_training];
		testOut = new double*[settings.i_number_of_training];

		if (settings.b_trainingFromFile){
			//Store training set
			std::cout << "loading training set " << endl;
			getDataFromFile(settings.s_trainingSet, checkpoint.i_number_of_loops_checkpoint, settings.i_number_of_training, settings.i_input, value);
			getDataFromFile(settings.s_outputTrainingFile, checkpoint.i_number_of_loops_checkpoint, settings.i_number_of_training, settings.i_output, results);

			if (settings.b_testingFromFile){
				std::cout << "loading testing data " << endl;
				//Store the data to test the neural network
				getDataFromFile(settings.s_testSet, 0, settings.i_number_of_training, settings.i_input, testIn);
				getDataFromFile(settings.s_outputTestSet, 0, settings.i_number_of_training, settings.i_output, testOut);
			}
			else{
				std::cout << "loading testing data " << endl;
				//If no test file is given, use some from the training set
				//Store the testing set
				getDataFromFile(settings.s_trainingSet, 0, settings.i_number_of_training, settings.i_input, testIn);
				getDataFromFile(settings.s_outputTrainingFile, 0, settings.i_number_of_training, settings.i_output, testOut);

			}
		}
		std::cout << "training start " << endl;
		trainNetworkDelta(value, results, test, 0, settings.i_number_of_training, testIn, testOut, settings.i_number_of_training, settings, checkpoint);

		for (int i = 0; i < settings.i_number_of_training; i++){
			delete value[i];
			delete results[i];
		}

		//Clean up memory
		delete value;
		delete results;
	}
	else{
		std::cout << "Starting output Loops" << endl;
		recursiveTestInput(test);
	}
}



//*********************************
//Main Function
//*********************************

int main(int argc, char** argv){
#ifdef WINDOWS_COMPUTER
	initialize();
#endif

	pause = false;

	std::cout << "Starting Program... " << endl;
	//Attach the signal handler
	std::signal(SIGINT, signal_handler);

	initialize_loops(argc, argv);

	return 0;
	}

