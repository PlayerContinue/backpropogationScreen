#include "hip/hip_runtime.h"


//---------------------------------------------------------------------------------------
//Author:David Greenberg
//Desc: Initizializing algorithm. Contains the main function and only the main function
//----------------------------------------------------------------------------------------

//#define PROBLEMS 5
#define FIRST_TEST
#pragma once
#include <vector>
#include <stdlib.h>
#include <bitset>
#include <math.h>
#include <iostream>
#include <fstream>
#include <string>
#include "CSettings.h"
#include "CGraphicNetwork.cuh"
#ifdef WINDOWS_COMPUTER
#include <wincon.h>
#endif

using namespace std;

//PROTOTYPES
int writeToFile(CGraphicsNetwork &network, CSettings settings);
void testOutput2(double** value, CGraphicsNetwork &test, int size);
bool addToNetwork(CGraphicsNetwork &test, CSettings settings, SCheckpoint& checkpoint, double** testSet, double mean_square_error);
void getDataFromFile(string fileName, int start, int end, int numberResults, double** input);

void printVectorOutput(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}
	cout << value << endl;
	if (value <= 255 & value > 0){
		cout << (char)value << endl;
	}
	
}

void printVectorOutput2(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}
	cout << value;
}

void printArray(double* arrayA, int size){
	for (int i = 0; i < size; i++){
		cout << (char)arrayA[i];
		cout << endl;
	}
}

void trainNetwork(double* value_1, double* value_2, double* value_3, double* value_4, CGraphicsNetwork &test, int rounds){
	//Train the network on simple test values
	for (int i = 0; i < rounds; i++){
		if (i % 2 == 0){
			test.backprop(value_1, value_2);
		}
		else{
			test.backprop(value_3, value_4);
		}
	}

}

void trainNetwork2(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, int rounds){

	int count = 0;
	//Train the network on simple test values
	for (int i = start; i < rounds; i++){
		test.backprop(value[count], results[count]);
		count++;
		if (count >= end){
			count = start;
		}
	}

}

//Returns true if the current mean_square_error is less than the lowest_mean_square
template <typename T>
bool checkThreshold(T mean_square, T lowest_mean_square, T threshold){
	if (mean_square < threshold && threshold >= 0){
		return false;
	}
	else if (mean_square <= lowest_mean_square){
		return true;
	}
	else{
		return false;
	}
}

int numberFullSame(CGraphicsNetwork test, double** in, int size){
	vector<double> output;
	vector<double> output2;
	int count_success = 0;
	for (int i = 0; i < size; i++){
		test.feedForward(in[i]);
		if (i != 0){
			output2 = output;
		}
		output = test.getOutput();
		if (i != 0){
			for (int j = 0; j < output.size(); j++){
				if (output[j] != output2[j]){
					count_success++;
					break;
				}
			}
		}
	}

	return (size-count_success);
}

//******************************
//Save, Load, Create Checkpoints
//******************************

//Create a checkpoint containing all the current values of the system
void createCheckpoint(CGraphicsNetwork test,SCheckpoint& checkpoint,CSettings settings){
	int checkpoint_number = writeToFile(test, settings);
	checkpoint.s_network_file_name = "networks/" + settings.s_network_name + std::to_string(checkpoint_number) + ".txt";
	//Write the checkpoint to a file
	std::ofstream outputfile;
	outputfile.precision(30);
	outputfile.open("checkpoints/" + settings.s_network_name + "_checkpoint_" + std::to_string(checkpoint_number) + ".txt", ios::trunc);
	if (outputfile.is_open()){
		//Output the network
		outputfile << checkpoint << flush;
		outputfile.close();
	}
	else{
		cout << "Unable to write checkpoint to file." << endl;
		cout << "continue?";
	}

}

void LoadCheckpointFromFile(SCheckpoint& checkpoint,string s_file_name){
	std::ifstream inputfile;

	inputfile.open(s_file_name, ios_base::beg);
	if (inputfile.is_open()){
		inputfile >> checkpoint;
		inputfile.close();
	}
	else{
		cout << "Unable to read checkpoint from file." << endl;
		cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}
	}
}

//Create a new checkpoint object from the created settings
void createNewCheckpoint(SCheckpoint& checkpoint, CSettings settings){
	checkpoint.i_number_of_loops_checkpoint = 0;
	checkpoint.i_number_of_loops = 0;
	checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
	checkpoint.d_mean_square_error = (double)INT_MAX;
	checkpoint.d_lowest_mean_square_error = (double)INT_MAX;
	checkpoint.d_previous_mean_square_error = 0;
	checkpoint.d_row_distance_threshold = settings.d_row_distance_threshold;
	checkpoint.d_neuron_distance_threshold = settings.d_neuron_distance_threshold;
}

//******************************
//Training the Network
//******************************

//Train the current network
//Check if the limit has been reached as the stopping point
//Use mean square error to check distance
void trainNetworkDelta(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, double* testSetIn[], double* testSetOut[], int testLength, CSettings settings, SCheckpoint checkpoint){
	do{

		test.backprop(value[checkpoint.i_number_of_loops], results[checkpoint.i_number_of_loops]);


		if (checkpoint.i_number_of_loops_checkpoint % settings.i_loops == 0){
			createCheckpoint(test,checkpoint, settings);
			//Test the output
			testOutput2(testSetIn, test, testLength);
			cout << " loop " << checkpoint.i_number_of_loops_checkpoint << endl;
		}
		checkpoint.i_number_of_loops++;
		checkpoint.i_number_of_loops_checkpoint++;

		if (checkpoint.i_number_of_loops >= end){
			checkpoint.i_number_of_loops = start;

			if (settings.b_trainingFromFile){

				//Store training set
				getDataFromFile(settings.s_trainingSet, checkpoint.i_number_of_loops_checkpoint*settings.i_input, settings.i_number_of_training, settings.i_input, value);

			}

			if (settings.b_testingFromFile){
				getDataFromFile(settings.s_outputTrainingFile, checkpoint.i_number_of_loops_checkpoint*settings.i_output, settings.i_number_of_training, settings.i_output, results);
			}
		}
		//Get the mean_square_error when the number of loops reaches a user defined values
		if (checkpoint.i_number_of_loops_checkpoint%settings.i_number_before_growth_potential == 0){
			
			//Set the current mean_square_error as the previous error
			checkpoint.d_previous_mean_square_error = checkpoint.d_mean_square_error;

			//Retrieve the new mean_square_error
			checkpoint.d_mean_square_error = test.getMeanSquareError(testSetIn, testSetOut, testLength);
			
			//Set the current lowest
			if (checkpoint.d_mean_square_error < checkpoint.d_lowest_mean_square_error && checkpoint.d_mean_square_error != checkpoint.d_previous_mean_square_error){
				checkpoint.d_lowest_mean_square_error = checkpoint.d_mean_square_error;
				
				//Reset value since error was lowered
				checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
			}
			else{
				checkpoint.i_times_lowest_mean_square_error_to_large--;
				//If the below is true, something close to the limit has been reached, the network needs to change size 
				if (checkpoint.i_times_lowest_mean_square_error_to_large == 0){
					//Add new nodes to the network
					if (addToNetwork(test, settings,checkpoint, testSetIn, checkpoint.d_mean_square_error)){
						//Reset the number allowed
						//Since new ones may have been
						checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures + 10;
					}
					else{
						//Reset the number of times before a growth is attempted
						checkpoint.i_times_lowest_mean_square_error_to_large = settings.i_number_allowed_failures;
					}
				}
			}
			
		}

		//Loop until the error is smaller than the threshold
	} while (settings.d_threshold < checkpoint.d_mean_square_error && checkpoint.i_number_of_loops == start || checkpoint.i_number_of_loops != start);
}

//Returns true if a neuron was added
bool addToNetwork(CGraphicsNetwork &test,CSettings settings,SCheckpoint& checkpoint, double** testSet, double mean_square_error){

	//Get delta in success
	double success = test.getSuccessRate() - test.getPreviousSuccessRate();
	double averagedistance = abs(test.getPreviousAverageDistance() - test.getAverageDistance());
	double delta = abs(test.getAverageDelta());
	double mean_square_error_dif = mean_square_error - settings.d_threshold;
#ifdef FULL_SUCCESS
	double full_success = test.getFullSuccessRate() - test.getFullPreviousSuccessRate();
#endif
	//Get the number of test sets returning the exact same values
	int numberSame = numberFullSame(test,testSet, settings.i_number_of_training);

	//Add a new layer if the success is too low and the threshold has not been reached
	if (success <= settings.d_row_success_threshold && mean_square_error_dif > 0 && mean_square_error_dif >= checkpoint.d_row_distance_threshold && numberSame < settings.i_number_allowed_same){
		test.addLayer(-1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) * 5);
		//Increment the size of the need mean distance to get a new layer
		//And decrease the size of the needed mean distance to get a new neuron
		if (checkpoint.d_neuron_distance_threshold > 0 ){
			checkpoint.d_row_distance_threshold += settings.d_row_distance_threshold * .1;
			checkpoint.d_neuron_distance_threshold -= settings.d_neuron_distance_threshold * .1;
		}
		test.resetNetwork();
		return true;
	}
	else if (success <= settings.d_neuron_success_threshold && mean_square_error_dif > 0 && mean_square_error_dif >= checkpoint.d_neuron_distance_threshold){
		if (test.getNumLayers() == 2){
			test.addLayer(-1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) * 5);
			//Increment the size of the need mean distance to get a new layer
			//And decrease the size of the needed mean distance to get a new neuron
			if (checkpoint.d_neuron_distance_threshold > 0){
				checkpoint.d_row_distance_threshold += settings.d_row_distance_threshold * .1;
				checkpoint.d_neuron_distance_threshold -= settings.d_neuron_distance_threshold * .1;
			}
		}
		else{
			test.addNeuronToLayer(1, test.getNumLayers() - 2, 8);
			//Increment the size of the need mean distance to get a new neuron
			//And decrease the size of the needed mean distance to get a new row
			if (checkpoint.d_row_distance_threshold > 0){
				checkpoint.d_row_distance_threshold -= settings.d_row_distance_threshold * .1;
				checkpoint.d_neuron_distance_threshold += settings.d_neuron_distance_threshold * .1;
			}
		}
		test.resetNetwork();
		return true;
	}
	
	return false;


}

void testOutput(double* value_1, double* value_3, CGraphicsNetwork &test){
	vector<double> temp2;
	cout << "input";
	cout << endl;
	printArray(value_1, 2);
	test.feedForward(value_1);
	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;


	test.feedForward(value_3);
	temp2 = test.getOutput();

	cout << "input";
	cout << endl;
	printArray(value_3, 2);

	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;
}

void testOutput2(double** value, CGraphicsNetwork &test, int size){
	vector<double> temp2;

	for (int i = 0; i < size; i++){
		cout << "input";
		cout << endl;
		
			printArray(value[i], 6);
		
		test.feedForward(value[i]);
		
			temp2 = test.getOutput();
		
		cout << "output";
		cout << endl;
		printVectorOutput(temp2);
		cout << endl;
	}
}

//Output the network to a file
int writeToFile(CGraphicsNetwork &network, CSettings settings){
	static int file_number = 0;
	file_number++;
	std::ofstream outputfile;
	outputfile.open("networks/" + settings.s_network_name + std::to_string(file_number) + ".txt", ios::trunc);
	if (outputfile.is_open()){
		//Output the network
		outputfile << network << flush;
		outputfile.close();
	}
	else{
		cout << "Unable to write checkpoint to file." << endl;
		cout << "continue?";
	}

	return file_number;


}

bool loadFromFile(CGraphicsNetwork& network, string fileName){
	std::ifstream inputfile;

	inputfile.open(fileName, ios_base::beg);
	if (inputfile.is_open()){
		inputfile >> network;
		inputfile.close();
		return true;
	}
	else{
		cout << "Unable to read from file." << endl;
		cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}
		return false;
	}
}

CSettings loadSettings(string fileName){
	std::ifstream inputfile;
	inputfile.open(fileName, ios_base::beg);
	CSettings settings;
	if (inputfile.is_open()){

		inputfile >> settings;
		inputfile.close();
		return settings;
	}
	else{
		cout << "Unable to read from file." << endl;
		cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}

		return settings;
	}
}


#ifdef WINDOWS_COMPUTER
BOOL WINAPI ConsoleHandlerRoutine(DWORD dwCtrlType)
{
	if (dwCtrlType == CTRL_CLOSE_EVENT)
	{
		return TRUE;
	}

	return FALSE;
}


void initialize(){
	if (SetConsoleCtrlHandler(ConsoleHandlerRoutine, TRUE) == false){
		printf("Unable to attach Handler");
		cout << "continue? ";
		if (cin.get() == 'n'){
			exit(0);
		}
	}
}

#endif

//Retrieve the data from a file
//start: Where to start gathering characters from in the file
//numberOfRounds: The number of character sets which are needed to be retrieve
//numberResults: How many characters should be retrieved for a single round
//input : the storage container for the input
void getDataFromFile(string fileName, int start, int numberOfRounds, int numberResults, double** input){
	std::ifstream inputfile;

	inputfile.open(fileName);
	if (inputfile.is_open()){
		inputfile.seekg(start);
		int k = -1;
		int letterPosition = 0;
		for (int i = 0; i < (numberOfRounds)*numberResults; i++){
			if (i%numberResults == 0){
				k++;
				input[k] = new double[numberResults];
				letterPosition = 0;
			}

			input[k][letterPosition] = (int)inputfile.get();
#ifdef FIRST_TEST 
			if (input[k][letterPosition] == 48.0){
				input[k][letterPosition] = .1;
			}
			else if (input[k][letterPosition] == 49.0){
				input[k][letterPosition] = .9;
			}
#endif


			letterPosition++;



		}
		inputfile.close();
	}
	else{
		cout << "Unable to read from file." << endl;
		cout << "continue?";
		if (cin.get() == 'n'){

			exit(0);
		}
	}
}

void initialize_loops(int argc, char** argv){
	CGraphicsNetwork test;
	CSettings settings;
	if (argc > 1){
		settings = loadSettings(argv[1]);
	}

	int PROBLEMS = std::stoi(argv[2]);
	SCheckpoint checkpoint = SCheckpoint();
	if (settings.b_loadFromCheckpoint){
		//Load the checkpoint from a file
		LoadCheckpointFromFile(checkpoint,settings.s_checkpoint_file);
		//Load the information from the checkpoint
		test = CGraphicsNetwork();
		loadFromFile(test, checkpoint.s_network_file_name);
	}
	else if (settings.b_loadNetworkFromFile){//Load only the network from file
		test = CGraphicsNetwork();
		createNewCheckpoint(checkpoint,settings);
		loadFromFile(test, settings.s_loadNetworkFile);
	}
	else{//Start with a brand new network

		vector<int> temp = vector<int>();
		createNewCheckpoint(checkpoint,settings);
		temp.push_back(settings.i_input);
		temp.push_back(settings.i_output);
		vector<double> temp2 = vector<double>(settings.i_output);

		test = CGraphicsNetwork(temp, settings.d_beta, settings.d_alpha);
	}
	double **value;
	double **results;
	double **testIn;
	double **testOut;
	
	value = new double*[settings.i_number_of_training];
	results = new double*[settings.i_number_of_training];
	testIn = new double*[settings.i_number_of_training];
	testOut = new double*[settings.i_number_of_training];


	//Store training set
	getDataFromFile(settings.s_trainingSet, 0, settings.i_number_of_training, settings.i_input, value);
	getDataFromFile(settings.s_outputTrainingFile, 0, settings.i_number_of_training, settings.i_output, results);

	//Store training set
	getDataFromFile(settings.s_trainingSet, 0, settings.i_number_of_training, settings.i_input, testIn);
	getDataFromFile(settings.s_outputTrainingFile, 0, settings.i_number_of_training, settings.i_output, testOut);

	trainNetworkDelta(value, results, test, 0, settings.i_number_of_training, testIn, testOut, settings.i_number_of_training, settings, checkpoint);

	for (int i = 0; i < settings.i_number_of_training; i++){
		delete value[i];
		delete results[i];
	}
	
	
	/*if (!settings.b_trainingFromFile && !settings.b_trainingFromFile){
		int zero;
		int number2;
		value = new double*[PROBLEMS];
		results = new double*[PROBLEMS];
		for (int i = 0; i < PROBLEMS; i++){
			value[i] = new double[2];
			double number = (double)RandInt(0, PROBLEMS) + 1;
			value[i][0] = number;
			value[i][1] = number + 1;
			//number = (double)(1 / (number + number + 1));
			results[i] = new double[32];
			number2 = number + number + 1;
			zero = 1;
			for (int j = 31; j >= 0; j--){
				results[i][j] = (double)(((int)(number2 & zero)) != 0 ? .7 : .1);
				//Shift left by one
				zero = zero << 1;
			}
		}

		testIn = new double*[PROBLEMS];
		testOut = new double*[PROBLEMS];

		for (int i = 0; i < 100; i++){
			testIn[i] = new double[2];
			double number = (double)RandInt(0, PROBLEMS) + 1;
			testIn[i][0] = number;
			testIn[i][1] = number + 1;
			testOut[i] = new double[32];
			number2 = number + number + 1;
			zero = 1;
			for (int k = 31; k >= 0; k--){
				testOut[i][k] = (double)(((int)(number2 & zero)) != 0 ? .7 : .1);
				//Shift left by one
				zero = zero << 1;
			}
		}
		//trainNetwork2(value, results, test, 0, PROBLEMS, settings.i_loops);
		trainNetworkDelta(value, results, test, 0, PROBLEMS, testIn, testOut, 100, settings);
		for (int i = 0; i < PROBLEMS; i++){
			delete value[i];
			delete results[i];
		}
	}
	else{*/
		
	//}




	//Clean up memory
	delete value;
	delete results;
}

int main(int argc, char** argv){
#ifdef WINDOWS_COMPUTER
	initialize();
#endif
	initialize_loops(argc, argv);

	return 0;
}

