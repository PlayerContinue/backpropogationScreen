#include "hip/hip_runtime.h"


//---------------------------------------------------------------------------------------
//Author:David Greenberg
//Desc: Initizializing algorithm. Contains the main function and only the main function
//----------------------------------------------------------------------------------------

//#define PROBLEMS 5
#pragma once
#include <vector>
#include <stdlib.h>
#include <bitset>
#include <math.h>
#include <iostream>
#include <fstream>
#include <string>
#include "CSettings.h"
#include "CGraphicNetwork.cuh"
using namespace std;


void printVectorOutput(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}
	cout << value;
}

void printArray(double* arrayA, int size){
	for (int i = 0; i < size; i++){
		cout << arrayA[i];
		cout << endl;
	}
}

void trainNetwork(double* value_1, double* value_2, double* value_3, double* value_4, CGraphicsNetwork &test, int rounds){
	//Train the network on simple test values
	for (int i = 0; i < rounds; i++){
		if (i % 2 == 0){
			test.backprop(value_1, value_2);
		}
		else{
			test.backprop(value_3, value_4);
		}
	}

}

void trainNetwork2(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, int rounds){

	int count = 0;
	//Train the network on simple test values
	for (int i = start; i < rounds; i++){
		test.backprop(value[count], results[count]);
		count++;
		if (count >= end){
			count = start;
		}
	}

}

//Returns true if the current mean_square_error is less than the lowest_mean_square
template <typename T>
bool checkThreshold(T mean_square, T lowest_mean_square, T threshold){
	if (mean_square < threshold && threshold >= 0){
		return false;
	}
	else if (mean_square <= lowest_mean_square){
		return true;
	}
	else{
		return false;
	}
}

//Train the current network
//Check if the limit has been reached as the stopping point
//Use mean square error to check distance
void trainNetworkDelta(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, CSettings settings){
	int training_position = start;
	double mean_square_error = 0;
	double lowest_mean_square_error = (double) INT_MAX;
	do{
		
		test.backprop(value[training_position], results[training_position]);
		training_position++;

		if (training_position >= end){
			training_position = start;
			
			mean_square_error = test.getMeanSquareError(value, results, end - start);
			//Set the current lowest
			if (mean_square_error < lowest_mean_square_error){
				lowest_mean_square_error = mean_square_error;
			}
		}
		//Loop until the previously smallest mean is no longer the smallest
	} while (checkThreshold<double>(mean_square_error,lowest_mean_square_error + settings.d_fluctuate_square_mean,settings.d_threshold) && training_position==start || training_position != start);
}

void addToNetwork(CGraphicsNetwork &test,CSettings settings){

	//Get delta in success
	double success = test.getSuccessRate() - test.getPreviousSuccessRate();
	double averagedistance = test.getPreviousAverageDistance() - test.getAverageDistance();
	double delta = abs(test.getAverageDelta());
#ifdef FULL_SUCCESS
	double full_success = test.getFullSuccessRate() - test.getFullPreviousSuccessRate();
#endif
	//Add a new layer if the success is too low and the threshold has not been reached
	if (averagedistance<settings.d_row_distance_threshold && success <= settings.d_row_success_threshold){
		test.addLayer(test.getNumLayers() + 1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) / 5);
	}
	else if (averagedistance<settings.d_neuron_distance_threshold && success <= settings.d_neuron_success_threshold){
		if (test.getNumLayers() == 2){
			test.addLayer(test.getNumLayers() + 1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) / 5);
		}
		else{
			test.addNeuronToLayer(1, test.getNumLayers() - 2, 2);
		}
	}
	test.resetNetwork();

}

void testOutput(double* value_1, double* value_3, CGraphicsNetwork &test){
	vector<double> temp2;
	cout << "input";
	cout << endl;
	printArray(value_1, 2);
	test.feedForward(value_1);
	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;


	test.feedForward(value_3);
	temp2 = test.getOutput();

	cout << "input";
	cout << endl;
	printArray(value_3, 2);

	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;
}

void testOutput2(double** value, CGraphicsNetwork &test, int size){
	vector<double> temp2;

	for (int i = 0; i < size; i++){
		cout << "input";
		cout << endl;
		printArray(value[i], 2);
		test.feedForward(value[i]);
		temp2 = test.getOutput();
		cout << "output";
		cout << endl;
		printVectorOutput(temp2);
		cout << endl;
	}
}

//Output the network to a file
bool writeToFile(CGraphicsNetwork &network, int fileNumber){
	std::ofstream outputfile;
	char file_name[20];
	sprintf(file_name, "network%d.txt", fileNumber);
	outputfile.open(file_name, ios::trunc);
	if (outputfile.is_open()){
		//Output the network
		outputfile << network << flush;
		outputfile.close();
		return true;
	}
	else{
		cout << "Unable to write checkpoint to file." << endl;
		cout << "continue?";
		return false;
	}


}

bool loadFromFile(CGraphicsNetwork& network, string fileName){
	std::ifstream inputfile;

	inputfile.open(fileName, ios_base::beg);
	if (inputfile.is_open()){
		inputfile >> network;
		inputfile.close();
		return true;
	}
	else{
		cout << "Unable to read from file." << endl;
		cout << "continue?";
		return false;
	}
}

CSettings loadSettings(string fileName){
	std::ifstream inputfile;
	inputfile.open(fileName, ios_base::beg);
	CSettings settings;
	if (inputfile.is_open()){

		inputfile >> settings;
		inputfile.close();
		return settings;
	}
	else{
		cout << "Unable to read from file." << endl;
		cout << "continue?";
		return settings;
	}
}



int main(int argc, char** argv){
	
	CGraphicsNetwork test;
	CSettings settings;
	if (argc > 1){
		settings = loadSettings(argv[1]);
	}

	int PROBLEMS = std::stoi(argv[2]);

	if (settings.b_loadNetworkFromFile){
		test = CGraphicsNetwork();
		loadFromFile(test, settings.s_loadNetworkFile);
	}
	else{

		vector<int> temp = vector<int>();
		temp.push_back(2);
		temp.push_back(32);
		vector<double> temp2 = vector<double>(32);

		test = CGraphicsNetwork(temp, settings.d_beta, settings.d_alpha);
	}
	int zero;
	int number2;
	double **value = new double*[PROBLEMS];
	double **results = new double*[PROBLEMS];
	for (int i = 0; i < PROBLEMS; i++){
		value[i] = new double[2];
		double number = (double)i + 1;
		value[i][0] = number;
		value[i][1] = number + 1;
		//number = (double)(1 / (number + number + 1));
		results[i] = new double[32];
		number2 = number + number + 1;
		zero = 1;
		for (int j = 31; j >= 0; j--){
			results[i][j] = (double)(((int)(number2 & zero)) != 0 ? .7 : .1);
			//Shift left by one
			zero = zero << 1;
		}
	}

	for (int i = 0; i < 500; i++){
		try{
			if (i != 0){
				trainNetwork2(value, results, test, 0, PROBLEMS, settings.i_loops);
				trainNetworkDelta(value, results, test, 0, PROBLEMS,settings);
			}
			else{
				trainNetwork2(value, results, test, 0, PROBLEMS, settings.i_loops);
				trainNetworkDelta(value, results, test, 0, PROBLEMS, settings);
			}
		}
		catch (exception e){
			cout << i << endl;
		}

		//Test the output
		testOutput2(value, test, PROBLEMS);
		cout << " loop " << i << endl;
		writeToFile(test, i % 50);

		//Add new nodes to the network
		addToNetwork(test,settings);

	}

	testOutput2(value, test, PROBLEMS);

	for (int i = 0; i < PROBLEMS; i++){
		delete value[i];
		delete results[i];
	}

	//Clean up memory
	delete value;
	delete results;

	return 0;
}

