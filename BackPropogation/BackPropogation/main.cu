#include "hip/hip_runtime.h"


//---------------------------------------------------------------------------------------
//Author:David Greenberg
//Desc: Initizializing algorithm. Contains the main function and only the main function
//----------------------------------------------------------------------------------------

//#define PROBLEMS 5
#pragma once
#include <vector>
#include <stdlib.h>
#include <bitset>
#include <math.h>
#include <iostream>
#include <fstream>
#include <string>
#include "CGraphicNetwork.cuh"
using namespace std;


void printVectorOutput(vector<double> vectorA){
	int size = vectorA.size();
	int value = 0;//Store a int with 32 0
	int pos = 1;
	for (int i = vectorA.size() - 1; i >= 0; i--){

		//Either add one at the current position or add zero
		value = value | (vectorA[i] > .5 ? pos : 0);

		pos = pos << 1;
	}
	cout << value;
}

void printArray(double* arrayA, int size){
	for (int i = 0; i < size; i++){
		cout << arrayA[i];
		cout << endl;
	}
}

void trainNetwork(double* value_1, double* value_2, double* value_3, double* value_4, CGraphicsNetwork &test, int rounds){
	//Train the network on simple test values
	for (int i = 0; i < rounds; i++){
		if (i % 2 == 0){
			test.backprop(value_1, value_2);
		}
		else{
			test.backprop(value_3, value_4);
		}
	}

}

void trainNetwork2(double* value[], double* results[], CGraphicsNetwork &test, int start, int end, int rounds){

	int count = 0;
	//Train the network on simple test values
	for (int i = start; i < rounds; i++){
		test.backprop(value[count], results[count]);
		count++;
		if (count >= end){
			count = start;
		}
	}

}

void addToNetwork(CGraphicsNetwork &test){

	//Get delta in success
	double success = test.getSuccessRate() - test.getPreviousSuccessRate();
	double averagedistance = test.getPreviousAverageDistance() - test.getAverageDistance();
	double delta = abs(test.getAverageDelta());
	double distanceMeasure = 1;
#ifdef FULL_SUCCESS
	double full_success = test.getFullSuccessRate() - test.getFullPreviousSuccessRate();
#endif
	//Add a new layer if the success is too low and either the there are no hidden layers or the previous layer has too many nodes
	if (
#ifdef FULL_SUCCESS
		(success!=0 && success < .2 || success == 0 && full_success < .3) &&

#else
		success < .1 &&
#endif

		((-1 * distanceMeasure * .002) > averagedistance
		|| 0 < averagedistance && averagedistance < distanceMeasure * .00000000000000000000001))
	{
		test.addLayer(test.getNumLayers() + 1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) / 5);
	}
	else if (

#ifdef FULL_SUCCESS
		(success != 0 && success < .2 || success == 0 && full_success < .5) &&
#else

		success < .2 &&
#endif

		((-1 * distanceMeasure * .0002) > averagedistance
		|| 0 < averagedistance && averagedistance < (distanceMeasure * .00000003))){
		if (test.getNumLayers() == 2){
			test.addLayer(test.getNumLayers() + 1, test.getNumNeuronsInLayer(test.getNumLayers() - 1) / 5);
		}
		else{
			test.addNeuronToLayer(1, test.getNumLayers() - 2, 2);
		}
	}
	test.resetNetwork();

}

void testOutput(double* value_1, double* value_3, CGraphicsNetwork &test){
	vector<double> temp2;
	cout << "input";
	cout << endl;
	printArray(value_1, 2);
	test.feedForward(value_1);
	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;


	test.feedForward(value_3);
	temp2 = test.getOutput();

	cout << "input";
	cout << endl;
	printArray(value_3, 2);

	temp2 = test.getOutput();
	cout << "output";
	cout << endl;
	printVectorOutput(temp2);
	cout << endl;
}

void testOutput2(double** value, CGraphicsNetwork &test, int size){
	vector<double> temp2;

	for (int i = 0; i < size; i++){
		cout << "input";
		cout << endl;
		printArray(value[i], 2);
		test.feedForward(value[i]);
		temp2 = test.getOutput();
		cout << "output";
		cout << endl;
		printVectorOutput(temp2);
		cout << endl;
	}
}

//Output the network to a file
bool writeToFile(CGraphicsNetwork &network, int fileNumber){
	std::ofstream outputfile;
	char file_name[20];
	sprintf(file_name, "network%d.txt", fileNumber);
	outputfile.open(file_name, ios::trunc);
	if (outputfile.is_open()){
		//Output the network
		outputfile << network << flush;
		outputfile.close();
		return true;
	}
	else{
		cout << "Unable to write checkpoint to file." << endl;
		cout << "continue?";
		return false;
	}


}

bool loadFromFile(CGraphicsNetwork& network, string fileName){
	std::ifstream inputfile;

	inputfile.open(fileName, ios_base::beg);
	if (inputfile.is_open()){
		inputfile >> network;
		return true;
	}
	else{
		cout << "Unable to read from file." << endl;
		cout << "continue?";
		return false;
	}
}

int main(int argc, char** argv){
	int PROBLEMS = std::stoi(argv[1]);
	int loop = std::stoi(argv[2]);
	CGraphicsNetwork test;
	if (argc > 3){
		test = CGraphicsNetwork();
		loadFromFile(test, argv[3]);
	}
	else{

		vector<int> temp = vector<int>();
		temp.push_back(2);
		temp.push_back(32);
		vector<double> temp2 = vector<double>(32);
	
	test = CGraphicsNetwork(temp, 1, 2);
	}
	int zero;
	int number2;
	double **value = new double*[PROBLEMS];
	double **results = new double*[PROBLEMS];
	for (int i = 0; i < PROBLEMS; i++){
		value[i] = new double[2];
		double number = (double)i + 1;
		value[i][0] = number;
		value[i][1] = number + 1;
		//number = (double)(1 / (number + number + 1));
		results[i] = new double[32];
		number2 = number + number + 1;
		zero = 1;
		for (int j = 31; j >= 0; j--){
			results[i][j] = (double)(((int)(number2 & zero)) != 0 ? .7 : .1);
			//Shift left by one
			zero = zero << 1;
		}

	}

	for (int i = 0; i < 500; i++){
		try{
			if (i != 0){
				trainNetwork2(value, results, test, 0, PROBLEMS, loop);
			}
			else{
				trainNetwork2(value, results, test, 0, PROBLEMS, loop);
			}
		}
		catch (exception e){
			cout << i << endl;
		}

		//Test the output
		testOutput2(value, test, PROBLEMS);
		cout << " loop " << i << endl;;
		writeToFile(test, i);

		//Add new nodes to the network
		addToNetwork(test);

	}

	testOutput2(value, test, PROBLEMS);
	
	for (int i = 0; i < PROBLEMS; i++){
		delete value[i];
		delete results[i];
	}

	//Clean up memory
	delete value;
	delete results;

	return 0;
}

