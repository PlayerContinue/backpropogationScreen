#include "hip/hip_runtime.h"
#include "LongTermShortTermNetwork.cuh"
//#define TRAININGTEST
//#define TRAININGTEST2
//#define DELTA_TEST
//#define AVERAGE_TEST
//#define APPLY_DELTA_BIAS
//#define NVIDA_OUTPUT_TEST
//#define NVIDA_OUTPUT_TEST2
//#define DELTA_MAP_TEST
#define NUMBER_MEMORY_WEIGHTS 4
//*********************
//Training the Network
//*********************



void LongTermShortTermNetwork::InitializeLongShortTermMemory(){
	//Store all the values in the device
	//Will later add option for too little memory
	//Copy the information to the device
	this->UnrollNetwork(3);
	this->RealOutput = device_vector<weight_type>(this->settings.i_output);
	this->host_deltas = host_vector<weight_type>(this->GPUOutput_values.size() - this->numberNonWeights);
	this->device_deltas = device_vector<weight_type>(this->GPUOutput_values.size() - (this->settings.i_backprop_unrolled*this->numberNonWeights));
}

void LongTermShortTermNetwork::averageWeights(){

#ifdef AVERAGE_TEST
	testing::outputToFile<weight_type>(this->GPUOutput_values, "initialOutput", "tests/Testing6.txt");
#endif
	/*thrust::reduce_by_key(
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1/this->settings.i_backprop_unrolled
		),

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1 / this->settings.i_backprop_unrolled
		) + this->GPUOutput_values.size() - this->numberNonWeights,

		thrust::make_permutation_iterator(
		this->GPUOutput_values.begin() + this->numberNonWeights,
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		((this->numberOfNodes)* (_1%this->settings.i_backprop_unrolled)) + (_1 / this->settings.i_backprop_unrolled)
		)

		),

		thrust::make_discard_iterator(),

		this->GPUOutput_values.begin() + this->numberNonWeights
		);*/

#ifdef AVERAGE_TEST
	testing::outputToFile<weight_type>(this->GPUOutput_values, "outputbeforetransform", "tests/Testing6.txt");
#endif

	//Find the average from the sum
	/*thrust::transform(
		this->GPUOutput_values.begin() + this->numberNonWeights,
		this->GPUOutput_values.begin() + this->numberNonWeights + this->numberOfNodes,
		this->GPUOutput_values.begin() + this->numberNonWeights,
		_1 / this->settings.i_backprop_unrolled

		);*/
	int i = 0;
	thrust::copy(this->GPUOutput_values.end() - this->numberOfNodes, this->GPUOutput_values.end(), this->GPUOutput_values.begin() + this->numberNonWeights);//Replace the current input with the output from the last run
#ifdef AVERAGE_TEST
	testing::outputToFile<weight_type>(this->GPUOutput_values, "outputaftertransform", "tests/Testing6.txt");
#endif
	thrust::fill(this->GPUOutput_values.begin() + this->numberOfNodes + this->numberNonWeights, this->GPUOutput_values.end(), (weight_type)0);

#ifdef AVERAGE_TEST
	testing::outputToFile<weight_type>(this->GPUOutput_values, "outputAfterFill", "tests/Testing6.txt");
#endif

}

void LongTermShortTermNetwork::LongTermShortTermNetwork::LongShortTermMemoryTraining(weight_type** in, weight_type** out){
	//Reset the weights to the end of the weights
	this->averageWeights();

	//Get the number of weights in the output layer
	//This is needed because the output layer needs to be used only once, so we need to inform the system which weights to skip


	//Set the input values
	this->setInput(in);
	unsigned int number_nodes_to_beginning_of_layer = 0;
	unsigned int number_weights_in_layer = this->GPUWeights.size();
	for (int i = 1; i < this->settings.i_backprop_unrolled; i++){
	
		thrust::reduce_by_key(
			this->GPUMapTo.begin(),
			this->GPUMapTo.end(),

			//Multiply the weights x output
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.begin(),
			thrust::make_permutation_iterator(
			this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer,
			this->GPUMapFrom.begin()
			)
			)
			),
			functors::multiply<weight_type>()
			),
			thrust::make_discard_iterator(),
			this->GPUPreviousOutput_Values.begin()
			);
#ifdef NVIDA_OUTPUT_TEST2
		testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values, "PrevBias-3", "tests/prevbias3.txt");
		testing::outputToFile<weight_type>(thrust::make_permutation_iterator(
			this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer,
			this->GPUMapFrom.begin()
			),this->GPUMapTo.size(), "PrevBias-1", "tests/prevbias3.txt");
		cout << "2";
#endif

		if (i > 0){//Only increment it by the number of nodes when working from any layer which is not the initial layer
			//This lets the nodes use the previous layer as their input
			number_nodes_to_beginning_of_layer += this->numberOfNodes + this->numberNonWeights;
		}

		//Add the bias to the current value
		thrust::transform(this->GPUBias.begin(),
			this->GPUBias.end(),
			this->GPUPreviousOutput_Values.begin(),
			this->GPUPreviousOutput_Values.begin(),
			_1 + _2
			);


		//Transfer all values from the current to the next row
		thrust::transform(this->GPUPreviousOutput_Values.begin(),
			this->GPUPreviousOutput_Values.end(),
			this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->numberNonWeights, functors::sigmoid_functor<weight_type>());

#ifdef NVIDA_OUTPUT_TEST

		cout << "3";
#endif

	}

	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
	//Find the delta 
	this->FindBackPropDelta(out, 0);

}



//Find the delta gradiant for each of the "layers" of the network
void LongTermShortTermNetwork::FindBackPropDelta(weight_type** out, int current_layer){
#ifdef NVIDA_OUTPUT_TEST

	cout << "4";
#endif
	

	unsigned int number_weights_in_layer = this->GPUWeights.size();

	unsigned int number_nodes_to_end_of_layer = this->GPUOutput_values.size();

	
	unsigned int number_to_output_of_layer;

	unsigned int delta_next_start = this->device_deltas.size() - this->numberOfNodes;
	unsigned int delta_next_end = this->device_deltas.size();
	
	//Find the deltas of the output
	for (int i = this->settings.i_backprop_unrolled; i > 0; i--){
		//Store the output into a vector
		//Performed each round due to the output changing each round
		for (unsigned int j = 0; j < this->RealOutput.size(); j++){
			this->RealOutput[j] = out[i-1][j];
		}
		number_to_output_of_layer = this->settings.i_output + ((this->numberOfNodes + this->numberNonWeights) * (this->settings.i_backprop_unrolled - i));//Number nodes from end of list to end of the layer
		//Find the delta of the output for the current layer
		//output * (1-output) * (target - output)
		thrust::transform(
			this->RealOutput.begin(),
			this->RealOutput.end(),
			this->GPUOutput_values.end() - number_to_output_of_layer,
			this->device_deltas.begin() + delta_next_end - this->settings.i_output,
			functors::find_output_delta<weight_type>());

		number_nodes_to_end_of_layer -= this->settings.i_output;
#ifdef DELTA_TEST
		testing::outputToFile<weight_type>(this->GPUBias.begin(), 1, "test", "test11");
		testing::outputToFile<weight_type>(this->device_deltas, "PostOutput", "tests/testing.txt");
		//thrust::copy(thrust::make_counting_iterator(0), thrust::make_counting_iterator(0) + this->device_deltas.size(), this->device_deltas.begin());

		/*testing::outputToFile<weight_type>(
			thrust::make_permutation_iterator(
			this->device_deltas.begin() + end_of_succeeding_layer - number_from_delta_start - this->numberOfNodes,
			thrust::make_transform_iterator(
			this->GPUMapTo.begin(),
			_1 - this->numberNonWeights
			)
			), this->GPUWeights.size(), "Results2", "tests/testing.txt");


			testing::outputToFile<weight_type>(
			thrust::make_permutation_iterator(
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.begin(),

			thrust::make_permutation_iterator(
			this->device_deltas.begin() + end_of_succeeding_layer - number_from_delta_start - this->numberOfNodes,
			thrust::make_transform_iterator(
			this->GPUMapTo.begin(),
			_1 - this->numberNonWeights
			)
			)
			)
			),
			functors::multiply<weight_type>()
			),
			this->positionToSum.begin()

			), this->positionToSum.size(), "Results", "tests/testing.txt");*/
#endif
		if (i != this->settings.i_backprop_unrolled){//Only perform this action when we've gone past the output layer

			thrust::reduce_by_key(
				this->count.begin(),
				this->count.end(),
				thrust::make_permutation_iterator(
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->GPUWeights.begin(),

				thrust::make_permutation_iterator(
				this->device_deltas.begin() + delta_next_end,
				thrust::make_transform_iterator(
				this->GPUMapTo.begin(),
				_1 - this->numberNonWeights
				)
				)
				)
				),
				functors::multiply<weight_type>()
				),
				this->positionToSum.begin()

				),//Transform End
				thrust::make_discard_iterator(),
				this->GPUPreviousOutput_Values.begin()
				);

#ifdef DELTA_TEST
			testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values, "PreTransform", "tests/testing.txt");
#endif
			int k = 0;
			//Add the bias
			/*thrust::transform(
				this->GPUBias.begin(),
				this->GPUBias.end(),
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->device_deltas.begin() + end_of_succeeding_layer - this->numberNonWeights - this->numberOfNodes,
				this->GPUPreviousOutput_Values.begin()

				)
				),
				this->GPUPreviousOutput_Values.begin(),
				functors::add_bias<weight_type>(this->settings.i_backprop_unrolled == i)
				);*/
#ifdef DELTA_TEST
			testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values, "PostBias", "tests/testing.txt");
#endif
			//Find the new deltas
			thrust::transform(
				this->device_deltas.begin() + delta_next_start,
				this->device_deltas.begin() + delta_next_start + this->numberOfNodes,
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->GPUOutput_values.end() - number_to_output_of_layer + this->settings.i_output - this->numberOfNodes,
				this->GPUPreviousOutput_Values.begin()
				)), functors::find_non_output_delta<weight_type>()),
				this->device_deltas.begin() + delta_next_start,
				_1 + _2
				);
#ifdef DELTA_TEST
			testing::outputToFile<weight_type>(this->device_deltas, "Device","tests/testing.txt");
#endif
		}
		delta_next_end -= this->numberOfNodes;

		
		delta_next_start -= this->numberOfNodes;

		number_nodes_to_end_of_layer = (number_nodes_to_end_of_layer + this->settings.i_output) - this->numberOfNodes - this->numberNonWeights;
		


	}


}



//Apply the error
void LongTermShortTermNetwork::ApplyLongTermShortTermMemoryError(){
#ifdef NVIDA_OUTPUT_TEST

	cout << "5";
#endif
#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias-1", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUBias, "Bias-1", "tests/prevbias2.txt");


#endif

#ifdef DELTA_MAP_TEST 
	testing::outputToFile<int>(this->GPUMapFrom, "From", "tests/prevbias3.txt");
	testing::outputToFile<int>(this->GPUMapTo, "To", "tests/prevbias3.txt");
#endif
	this->ApplyErrorToBias();


	thrust::device_vector<weight_type> tempPrevBias = thrust::device_vector<weight_type>(this->GPUPreviousBias);
	thrust::device_vector<weight_type> tempBias = thrust::device_vector<weight_type>(this->GPUBias);
	thrust::device_vector<int> tempMapFrom = thrust::device_vector<int>(this->GPUMapFrom);
#ifdef TRAININGTEST2
	//testing::outputToFile<weight_type>(this->device_deltas, "Delta");
	//testing::outputToFile<weight_type>(this->GPUOutput_values, "Output");
#endif
	int length_between_adds = this->GPUWeights.size() + this->numberNonWeights;
	int number_delta_between_add = this->GPUMapTo.size();
#ifdef TRAININGTEST

	testing::outputToFile<weight_type>(
		thrust::make_permutation_iterator(
		thrust::make_permutation_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapTo.begin(),
		_1 - this->numberNonWeights
		),//End Transform Iterator (States to start with the 2nd layer)

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapTo.size()
		)//End Transform

		),//End Perm
		thrust::make_counting_iterator((int)0)
		)//End Tuple
		),//END zip
		functors::extend_value<int>(number_delta_between_add, 0, this->numberOfNodes, false)
		)
		)

		//Increase whenever the counter reaches the end
		, thrust::make_transform_iterator(//Weight 1 - 0, Weight 2-0,....
		thrust::make_counting_iterator((int)0),
		(this->GPUMapTo.size()*(_1 % (this->settings.i_backprop_unrolled - 1))) + (_1 / (this->settings.i_backprop_unrolled - 1))
		))
		, this->GPUMapTo.size()*(this->settings.i_backprop_unrolled - 1), "testing4", "tests/test4.txt"
		);



	testing::outputToFile<weight_type>(
		thrust::make_permutation_iterator(
		//Weight_permutation
		thrust::make_permutation_iterator(
		this->GPUOutput_values.begin(),
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapFrom.begin(),
		_1
		),//End Transform Iterator (States to start with the 2nd layer)
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapFrom.size()
		)
		),//End Perm
		thrust::make_counting_iterator((int)0)
		)
		),
		functors::extend_value<int>(this->GPUMapFrom.size(), 0, this->numberOfNodes + this->numberNonWeights, false)//Increase whenever the counter reaches the end
		)//End of transform iterator
		),//End Permutation Iterator


		thrust::make_transform_iterator(//Weight 1 - 0, Weight 2-0,....
		thrust::make_counting_iterator((int)0),
		(this->GPUMapTo.size()*(_1 % (this->settings.i_backprop_unrolled - 1))) + (_1 / (this->settings.i_backprop_unrolled - 1))
		)
		),
		this->GPUMapTo.size()*(this->settings.i_backprop_unrolled - 1),
		"weights",
		"tests/test7.txt"
		);

	testing::outputToFile<weight_type>(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		thrust::make_zip_iterator(
		thrust::make_tuple(

		thrust::make_permutation_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapTo.begin(),
		_1 - this->numberNonWeights
		),//End Transform Iterator (States to start with the 2nd layer)

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapTo.size()
		)//End Transform

		),//End Perm
		thrust::make_counting_iterator((int)0)
		)//End Tuple
		),//END zip
		functors::extend_value<int>(number_delta_between_add, 0, number_delta_between_add, false)
		)//End of transform iterator
		),

		//Weight_permutation
		thrust::make_permutation_iterator(
		this->GPUOutput_values.begin(),
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapFrom.begin(),
		_1
		),//End Transform Iterator (States to start with the 2nd layer)
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapFrom.size()
		)
		),//End Perm
		thrust::make_counting_iterator((int)0)
		)
		),
		functors::extend_value<int>(this->GPUMapFrom.size(), 0, this->numberOfNodes + this->numberNonWeights, false)//Increase whenever the counter reaches the end
		)//End of transform iterator
		)//End Permutation Iterator


		)
		),
		functors::find_previous_weight<weight_type>(this->settings.d_beta)
		),//End Transform Iterator
		thrust::make_transform_iterator(//Weight 1 - 0, Weight 2-0,....
		thrust::make_counting_iterator((int)0),
		(this->GPUMapTo.size()*(_1 % (this->settings.i_backprop_unrolled - 1))) + (_1 / (this->settings.i_backprop_unrolled - 1))
		)),
		this->GPUMapTo.size()*(this->settings.i_backprop_unrolled - 1),
		"test", "tests/test8.txt"
		);

#endif

	//Apply the alpha
	thrust::transform_if(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_constant_iterator(this->settings.d_alpha),
		this->GPUPreviousWeights.begin(),
		this->GPUWeights.begin()
		)
		),
		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_constant_iterator(this->settings.d_alpha),
		this->GPUPreviousWeights.end(),
		this->GPUWeights.end()
		)
		),
		this->GPUWeights.begin(),
		functors::multiply_add<weight_type>(),
		functors::compare_two<(unsigned int)1, weight_type>(5, 1)

		);

#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUBias, "Bias-3", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias-3", "tests/prevbias2.txt");
#endif
	
	//Find the previous weights
	thrust::reduce_by_key(
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		),
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		) + ((this->settings.i_backprop_unrolled - 1) *this->GPUOutput_values.size()),

		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		thrust::make_zip_iterator(
		thrust::make_tuple(

		thrust::make_permutation_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapTo.begin(),
		_1 - this->numberNonWeights
		),//End Transform Iterator (States to start with the 2nd layer)

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapTo.size()
		)//End Transform

		),//End Perm
		thrust::make_counting_iterator((int)0)
		)//End Tuple
		),//END zip
		functors::extend_value<int>(number_delta_between_add, 0, this->numberOfNodes, false)
		)//End of transform iterator
		),

		//Weight_permutation
		thrust::make_permutation_iterator(
		this->GPUOutput_values.begin(),
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapFrom.begin(),
		_1
		),//End Transform Iterator (States to start with the 2nd layer)
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapFrom.size()
		)
		),//End Perm
		thrust::make_counting_iterator((int)0)
		)
		),
		functors::extend_value<int>(this->GPUMapFrom.size(), 0, this->numberOfNodes + this->numberNonWeights, false)//Increase whenever the counter reaches the end
		)//End of transform iterator
		)//End Permutation Iterator


		)
		),
		functors::find_previous_weight<weight_type>(this->settings.d_beta)
		),//End Transform Iterator
		thrust::make_transform_iterator(//Weight 1 - 0, Weight 2-0,....
		thrust::make_counting_iterator((int)0),
		(this->GPUMapTo.size()*(_1 % (this->settings.i_backprop_unrolled - 1))) + (_1 / (this->settings.i_backprop_unrolled - 1))
		)

		),//End Permutation Iterator
		thrust::make_discard_iterator(),
		this->GPUPreviousWeights.begin()
		);

#ifdef DELTA_MAP_TEST 
	testing::outputToFile<int>(this->GPUMapFrom, "From", "tests/prevbias3.txt");
	testing::outputToFile<int>(this->GPUMapTo, "To", "tests/prevbias3.txt");
#endif

#ifdef APPLY_DELTA_BIAS

	testing::outputToFile<weight_type>(this->GPUBias, "Bias-2", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias-2", "tests/prevbias2.txt");
#endif
#ifdef TRAININGTEST2
	testing::outputToFile<weight_type>(this->device_deltas, "Delta", "tests/test5.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousWeights, "PrevGPUVal", "tests/test5.txt");
#endif

	thrust::transform_if(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUWeights.begin(),
		this->GPUPreviousWeights.begin()
		
		)
		),
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUWeights.end(),
		this->GPUPreviousWeights.end()
		
		)
		),
		this->GPUWeights.begin(),
		functors::add_and_store<weight_type>(this->settings.i_backprop_unrolled - 1),
		functors::compare_two<(unsigned int)0, weight_type>(5, 1));

	thrust::copy(tempPrevBias.begin(), tempPrevBias.end(), this->GPUPreviousBias.begin());
	thrust::copy(tempBias.begin(), tempBias.end(), this->GPUBias.begin());
	thrust::copy(tempMapFrom.begin(),tempMapFrom.end(),this->GPUMapFrom.begin());
#ifdef APPLY_DELTA_BIAS

	testing::outputToFile<weight_type>(this->GPUBias, "Bias-5", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias-5", "tests/prevbias2.txt");
#endif

	thrust::fill(this->device_deltas.begin(), this->device_deltas.end(), (weight_type)0);


}

void LongTermShortTermNetwork::ApplyErrorToBias(){
	//Apply the delta to the bias
#ifdef APPLY_DELTA_BIAS

	testing::outputToFile<weight_type>(this->GPUBias, "Bias1", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias1", "tests/prevbias2.txt");
#endif
	//Apply momentum to the bias

	if (this->settings.d_alpha != 0){
		//Apply the alpha to bias
		thrust::transform_if(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousBias.begin(),
			thrust::make_constant_iterator(this->settings.d_alpha),
			this->GPUBias.begin()
			)
			)
			,
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousBias.end(),
			thrust::make_constant_iterator(this->settings.d_alpha) + this->GPUPreviousBias.size(),
			this->GPUBias.end()
			)
			),
			this->GPUBias.begin(),
			functors::multiply_add<weight_type>(),
			functors::compare_two<(unsigned int)2, weight_type>(5, (weight_type)0)
			);
	}

#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUBias, "Bias2", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias2", "tests/prevbias2.txt");
#endif

	//Retrieve the new previous Bias
	//If I remove memory cells, remove permutation around device_delta so it doesn't skip
	thrust::reduce_by_key(
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		),
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		) + ((this->settings.i_backprop_unrolled - 1) * this->GPUBias.size()),

		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		functors::multiply_by_constant<weight_type>((weight_type)this->settings.d_beta)
		),

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(this->numberOfNodes*(_1 % (this->settings.i_backprop_unrolled - 1))) + (_1 / (this->settings.i_backprop_unrolled - 1))
		)
		),
		thrust::make_discard_iterator(),
		this->GPUPreviousBias.begin()
		);

#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUBias, "Bias3", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias3", "tests/prevbias2.txt");
#endif

	//Apply the error
	thrust::transform_if(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUBias.begin(),
		this->GPUPreviousBias.begin()
		)
		),
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUBias.end(),
		this->GPUPreviousBias.end()
		)
		),
		this->GPUBias.begin(),
		functors::add_and_store<weight_type>(this->settings.i_backprop_unrolled - 1),
		functors::compare_two<(unsigned int)0, weight_type>(5, (weight_type)0)
		);

#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUBias, "Bias4", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias4", "tests/prevbias2.txt");
#endif
}

//*********************
//Run The Network
//*********************


void LongTermShortTermNetwork::InitializeLongShortTermMemoryForRun(){
	//Form the delta objects
	this->CopyToDevice();
	this->moveBiasToGPU(false);//Don't create a previous_bias
	this->count_weights_in_layers();
}

device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in, run_type type){
	switch (type){
	case run_type::WITHOUT_MEMORY_CELLS:
		return this->runNetwork(in, 0);
	case run_type::WITH_MEMORY_CELLS:
		return this->runNetwork(in, this->GPUWeights.size() - this->numberOfWeightsInLayers[0]);
	default:
		return this->runNetwork(in, 0);
	}
}

thrust::device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in){
	return this->runNetwork(in,0);
}

thrust::device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in,int number_of_extra_weights){

	this->setInput(in);
	//Stores the numberofmblocks in a layer
	unsigned int numberMBlocks;
	//Number mBlocks in previous layer
	unsigned int previousnumberMBlocks = 0;
	unsigned int numberBlocksToLayer = 0;
	device_vector<weight_type> toReturn = device_vector<weight_type>(this->settings.i_output);

	int output_weight_size = ((this->mBlocksLayers[this->mBlocksLayers.size() - 2].size()));

	//unsigned int numberBias = 0;
	//Perform the transformation on each layer
	for (unsigned int i = 0; i < this->mBlocksLayers.size() - 1; i++){

		if (i != 0){
			previousnumberMBlocks += this->numberOfWeightsInLayers[i - 1] + number_of_extra_weights;
			numberBlocksToLayer += numberMBlocks;
		}
		numberMBlocks = this->mBlocksLayers[i].size();
		//Sum the values of the input/output/forget/potential_memory_cell_values nodes
		//The values in the GPU weights are in the order input, output, forget, memory cells
		//Subtracting this->mBlocksLayers[i].size() from the end will remove the memory cells from doing anything
		//Output to Previous

		
#ifdef TRAININGTEST
		testing::outputToFile(this->GPUMapTo, "Start", "tests/PrevOut1.txt");
		testing::outputToFile(this->GPUMapFrom, "Start", "tests/PrevOut1.txt");
		testing::outputToFile(this->GPUPreviousOutput_Values, "Start", "tests/PrevOut1.txt");
		testing::outputToFile(this->GPUOutput_values, "Start", "tests/PrevOut1.txt");
#endif
		thrust::reduce_by_key(
			this->GPUMapTo.begin() + previousnumberMBlocks + number_of_extra_weights,
			//Start at the beginning of the previous layer
			this->GPUMapTo.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i] + number_of_extra_weights, // End at the the number of nodes before the current layer + the number of nodes in the current layer
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.begin() + previousnumberMBlocks,//Start from the beginning of the layer
			thrust::make_permutation_iterator(//Permute the output values such that they start at the correct position
			this->GPUOutput_values.begin(),
			this->GPUMapFrom.begin() + previousnumberMBlocks
			)
			)
			),
			functors::multiply_or_return_zero<weight_type,1>() //Multiply the two together
			),
			thrust::make_discard_iterator(),
			this->GPUPreviousOutput_Values.begin()
			);
#ifdef TRAININGTEST
		testing::outputToFile(this->GPUPreviousOutput_Values, "GPUPreBias", "tests/PrevOut1.txt");
		testing::outputToFile(this->GPUOutput_values, "GPUPreBias", "tests/PrevOut1.txt");
#endif
		//Add Bias to the hidden layers
		thrust::transform(
			this->GPUBias.begin() + numberBlocksToLayer,
			this->GPUBias.begin() + numberBlocksToLayer + (numberMBlocks*4),
			this->GPUPreviousOutput_Values.begin(),
			this->GPUPreviousOutput_Values.begin(),
			_1 + _2
			);


#ifdef TRAININGTEST
		testing::outputToFile(this->GPUPreviousOutput_Values, "GPUPostBias", "tests/PrevOut1.txt");
		testing::outputToFile(this->GPUOutput_values, "GPUPostBias", "tests/PrevOut1.txt");
#endif
		//Create a input/output/forget/potential_memory_cell_values/memory_cell_value value
		//Essentially run the gate and get the output value
		thrust::for_each(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousOutput_Values.begin(), //input values
			this->GPUPreviousOutput_Values.begin() + numberMBlocks,//output values
			this->GPUPreviousOutput_Values.begin() + (2 * numberMBlocks),//forget values
			this->GPUPreviousOutput_Values.begin() + (3 * numberMBlocks),//potential_memory_cell_value
			this->GPUPreviousOutput_Values.begin() + (4 * numberMBlocks),
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 4),//Old Memory Cell Values
			this->GPUOutput_values.begin() + numberBlocksToLayer, //Old Input
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 2), //Old Forget
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 3) //Old Potential
			)),
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousOutput_Values.begin() + numberMBlocks, //input values
			this->GPUPreviousOutput_Values.begin() + (2 * numberMBlocks),//output values
			this->GPUPreviousOutput_Values.begin() + (3 * numberMBlocks),//forget values
			this->GPUPreviousOutput_Values.begin() + (4 * numberMBlocks),//potential_memory_cell_value
			this->GPUPreviousOutput_Values.begin() + (5 * numberMBlocks),
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 5),//Memory Cell Values
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 1), //Old Input
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 3), //Old Forget
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 4) //Old Potential
			)),
			functors::run_memory_block_functon<weight_type>());
#ifdef TRAININGTEST
		testing::outputToFile(this->GPUPreviousOutput_Values, "GPUMid", "tests/PrevOut1.txt");
#endif
		thrust::reduce_by_key(
			this->GPUMapTo.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i] + number_of_extra_weights,
			this->GPUMapTo.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i] + this->numberOfWeightsInLayers[i + 1] + number_of_extra_weights,
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i],
			thrust::make_permutation_iterator(
			this->GPUOutput_values.begin() ,
			this->GPUMapFrom.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i]
			)
			)
			),
			functors::multiply<weight_type>()
			),
			thrust::make_discard_iterator(),
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks*5)
			);

#ifdef TRAININGTEST
		testing::outputToFile(this->GPUPreviousOutput_Values, "GPUOut", "tests/PrevOut1.txt");
		testing::outputToFile(this->GPUBias, "GPUBiasOut", "tests/PrevOut1.txt");
#endif
		//Add the bias to the output


		thrust::transform(
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5),
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5) + this->settings.i_output,
			this->GPUBias.begin() + numberBlocksToLayer + (numberMBlocks * 5),
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5),
			_1 + _2
			);

#ifdef TRAININGTEST
		testing::outputToFile(this->GPUPreviousOutput_Values, "outpostbias", "tests/PrevOut1.txt");
		testing::outputToFile(this->GPUOutput_values, "outpostbias", "tests/PrevOut1.txt");
#endif

		thrust::transform(
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5),
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5) + this->settings.i_output,
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5),
			functors::sigmoid_functor<weight_type>()
			);
#ifdef TRAININGTEST
		testing::outputToFile(this->GPUPreviousOutput_Values, "Final", "tests/PrevOut1.txt");
		testing::outputToFile(this->GPUOutput_values, "Final", "tests/PrevOut1.txt");
#endif

		thrust::copy(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.begin() + (numberMBlocks * 5) + this->settings.i_output, this->GPUOutput_values.begin() + this->numberNonWeights + numberBlocksToLayer);
		thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
#ifdef TRAININGTEST
		testing::outputToFile(this->GPUPreviousOutput_Values, "GPUNew", "tests/PrevOut1.txt");
		testing::outputToFile(this->GPUOutput_values, "GPUNew", "tests/PrevOut1.txt");
#endif
	}

	


	

	thrust::copy(this->GPUOutput_values.begin() + this->numberNonWeights + numberBlocksToLayer + this->numberOfNodes - this->settings.i_output, this->GPUOutput_values.begin() + this->numberNonWeights + numberBlocksToLayer + this->numberOfNodes, toReturn.begin());

	return toReturn;
}
