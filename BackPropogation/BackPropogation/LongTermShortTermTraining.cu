#include "hip/hip_runtime.h"
#include "LongTermShortTermNetwork.cuh"
//#define TRAININGTEST
//#define TRAININGTEST2
//#define DELTA_TEST
//#define AVERAGE_TEST
//#define APPLY_DELTA_BIAS
//#define NVIDA_OUTPUT_TEST
//#define NVIDA_OUTPUT_TEST2
//#define DELTA_MAP_TEST
//*********************
//Training the Network
//*********************



void LongTermShortTermNetwork::InitializeLongShortTermMemory(){
	//Store all the values in the device
	//Will later add option for too little memory
	//Copy the information to the device
	this->UnrollNetwork(3);
	this->RealOutput = device_vector<weight_type>(this->settings.i_output);
	this->host_deltas = host_vector<weight_type>(this->GPUOutput_values.size() - this->numberNonWeights);
	this->device_deltas = device_vector<weight_type>(this->GPUOutput_values.size() - (this->settings.i_backprop_unrolled*this->numberNonWeights));
	this->training_previous_number_rows = this->settings.i_backprop_unrolled;
	this->count_weights_in_layers(true);
}

void LongTermShortTermNetwork::averageWeights(){

	thrust::copy(this->GPUOutput_values.begin() + ((this->numberOfNodes + this->numberNonWeights)* this->training_previous_number_rows - 2), this->GPUOutput_values.begin() + ((this->numberOfNodes + this->numberNonWeights)* this->training_previous_number_rows - 1), this->GPUOutput_values.begin());//Replace the current input with the output from the last run
	thrust::fill(this->GPUOutput_values.begin() + this->numberOfNodes + this->numberNonWeights, this->GPUOutput_values.end(), (weight_type)0);//Reset the rest of the output values


}


//Add the input
void LongTermShortTermNetwork::setInput(weight_type* in){
	//Place the input into the GPU values matrix
	for (int i = 0; i < this->settings.i_input; i++){
		this->GPUOutput_values[i] = in[i];
	}

}

//Add the input
void LongTermShortTermNetwork::setInput(weight_type** in){
	//Place the input into the GPU values matrix

	for (int j = 0; j < this->settings.i_backprop_unrolled; j++){
		for (int i = 0; i < this->numberNonWeights; i++){
			this->GPUOutput_values[i + (j*(this->numberNonWeights + this->numberOfNodes))] = in[j][i];
		}
	}

}

void LongTermShortTermNetwork::setInput(thrust::device_vector<weight_type> in){
	if (in.size() <= this->settings.i_input){
		thrust::copy(in.begin(), in.end(), this->GPUOutput_values.begin());
	}
	else{
		throw new exception("Input is too short");
	}
}

void LongTermShortTermNetwork::StartTraining(weight_type** in, weight_type** out){

	//Reset the weights to the end of the weights
	this->averageWeights();
	//Set the input values
	this->setInput(in);
	this->training_previous_number_rows = this->settings.i_backprop_unrolled;
	this->LongShortTermMemoryTraining(in, out);
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
	//Find the delta 
	this->FindBackPropDelta(out, 0);
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
	this->FindPreviousBias();
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
	this->FindPreviousWeights();
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
	thrust::fill(this->device_deltas.begin(), this->device_deltas.end(), (weight_type)0);
}

void LongTermShortTermNetwork::LongTermShortTermNetwork::LongShortTermMemoryTraining(weight_type** in, weight_type** out){

	int start = 1;

	//Get the number of weights in the output layer
	//This is needed because the output layer needs to be used only once, so we need to inform the system which weights to skip

	unsigned int number_nodes_to_internal_next_layer = 0;//Number nodes to the next "layer" in the current layer
	unsigned int number_weights_to_internal_next_layer = 0; // number weights to the next "layer" in the current layer

	//Number of nodes to the start of the current layer to which new numbers will be added
	unsigned int number_nodes_to_start_of_storage_layer = this->numberNonWeights + this->numberOfNodes + this->numberNonWeights;//Two number of non weights to get to the start of the next set of non input values

	//Number nodes to the beginning of the previous layer from which data will be gathered
	unsigned int number_nodes_to_beginning_of_layer = 0;

	unsigned int number_weights_in_layer = this->GPUWeights.size();
	for (int i = start; i < this->settings.i_backprop_unrolled; i++){
		number_nodes_to_internal_next_layer = 0;
		number_weights_to_internal_next_layer = 0;
		for (int j = 0; j < this->mBlocksLayers.size(); j++){
			thrust::reduce_by_key(
				this->GPUMapTo.begin() + number_weights_to_internal_next_layer,
				this->GPUMapTo.begin() + number_weights_to_internal_next_layer + this->numberOfWeightsInLayers[j],

				//Multiply the weights x output
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->GPUWeights.begin() + number_weights_to_internal_next_layer,
				thrust::make_permutation_iterator(
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer,
				this->GPUMapFrom.begin() + number_weights_to_internal_next_layer
				)
				)
				),
				functors::multiply<weight_type>()
				),
				thrust::make_discard_iterator(),
				this->GPUPreviousOutput_Values.begin()
				);

			//Redo the cell with the gate values
			/*thrust::for_each(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer,//Input
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->number_nodes_by_type[0][INPUT_CELL],//Output
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL],//Forget
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL],//Potential Memory Cell
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL] + this->number_nodes_by_type[0][POTENTIAL_MEMORY_CELL],//Old Memory Cell
				this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL] + this->number_nodes_by_type[0][POTENTIAL_MEMORY_CELL], // New Memory Cell
				this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] // New Output
				)

				),
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->number_nodes_by_type[0][INPUT_CELL],//Input
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL],//Output
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL],//Forget
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL] + this->number_nodes_by_type[0][POTENTIAL_MEMORY_CELL],//Potential Memory Cell
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL] + this->number_nodes_by_type[0][POTENTIAL_MEMORY_CELL] + this->number_nodes_by_type[0][MEMORY_CELL], //Old Memory Cell
				this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL] + this->number_nodes_by_type[0][POTENTIAL_MEMORY_CELL] + this->number_nodes_by_type[0][MEMORY_CELL], // New Memory Cell
				this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] // New Output

				)

				),
				functors::find_memory_cell_value<weight_type>()

				);*/


#ifdef NVIDA_OUTPUT_TEST2
				testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values, "prevout1" + std::to_string(j) + std::to_string(i), "tests/prevbias3.txt");
#endif

			//Add the bias to the current value
			thrust::transform(this->GPUBias.begin() + number_nodes_to_internal_next_layer,
				this->GPUBias.begin() + number_nodes_to_internal_next_layer + this->number_nodes_in_layer[j],
				this->GPUPreviousOutput_Values.begin(),
				this->GPUOutput_values.begin() + number_nodes_to_start_of_storage_layer + number_nodes_to_internal_next_layer,//Start + number of nodes to layer with searching values + number of nodes to current layer
				functors::sum_and_sigmoid<weight_type>()
				);

			number_nodes_to_internal_next_layer += this->number_nodes_in_layer[j];
			number_weights_to_internal_next_layer += this->numberOfWeightsInLayers[j];

#ifdef NVIDA_OUTPUT_TEST2
			testing::outputToFile<weight_type>(this->GPUOutput_values, "fullout2" + std::to_string(j) + std::to_string(i), "tests/prevbias3.txt");
			testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values, "prevout2" + std::to_string(j) + std::to_string(i), "tests/prevbias3.txt");
#endif

		}


#ifdef NVIDA_OUTPUT_TEST2
		testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values, "prevout", "tests/prevbias3.txt");
		testing::outputToFile<weight_type>(this->GPUWeights, "weights1", "tests/prevbias3.txt");
		testing::outputToFile<weight_type>(this->GPUBias, "bias1", "tests/prevbias3.txt");

#endif
		//Only increment it by the number of nodes when working from any layer which is not the initial layer
		//This lets the nodes use the previous layer as their input
		number_nodes_to_beginning_of_layer += this->numberOfNodes + this->numberNonWeights;
		number_nodes_to_start_of_storage_layer += this->numberNonWeights + this->numberOfNodes;


	}



}





//Find the delta gradiant for each of the "layers" of the network
void LongTermShortTermNetwork::FindBackPropDelta(weight_type** out, int current_layer){

	unsigned long delta_next_start = this->numberOfNodes * this->settings.i_backprop_unrolled - this->numberOfNodes;//this->device_deltas.size() - this->numberOfNodes;
	unsigned long delta_next_end = this->numberOfNodes * this->settings.i_backprop_unrolled; //this->device_deltas.size();
	unsigned long internal_delta_next_end = this->number_nodes_in_layer[this->number_nodes_in_layer.size() - 1];
	//Find the deltas of the output
	for (int i = this->settings.i_backprop_unrolled; i > 0; i--){
		//Store the output into a vector
		//Performed each round due to the output changing each round
		for (unsigned int j = 0; j < this->RealOutput.size(); j++){
			this->RealOutput[j] = out[i - 1][j];
		}

		//Find the delta of the output for the current layer
		//output * (1-output) * (target - output)
		thrust::transform(
			this->RealOutput.begin(),
			this->RealOutput.end(),
			this->GPUOutput_values.begin() + ((this->numberOfNodes + this->numberNonWeights) * i) - this->settings.i_output,
			this->device_deltas.begin() + delta_next_end - this->settings.i_output,
			functors::find_output_delta<weight_type>());




#ifdef DELTA_TEST
		this->device_deltas[0] = -1;
		testing::outputToFile<weight_type>(
			make_permutation_iterator(
			thrust::make_permutation_iterator(
			Unique_Iterator::make_return_zero_iterator(
			this->device_deltas.begin() + delta_next_start,
			this->device_deltas.end(),
			this->device_deltas.begin()
			),
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			thrust::make_transform_iterator(
			this->GPUMapTo.begin(),
			_1 - this->numberNonWeights
			),
			this->GPUMapFrom.begin()
			)
			),
			functors::add_when_less_than<long>(this->numberOfNodes, this->numberOfNodes + this->numberNonWeights)
			)
			),
			this->positionToSum.begin()
			),this->positionToSum.size() + 5, "delta_test","tests/delta_test.txt");

		testing::outputToFile<weight_type>(
			thrust::make_permutation_iterator(
			thrust::make_permutation_iterator(
			thrust::make_counting_iterator((int)0),
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			thrust::make_transform_iterator(
			this->GPUMapTo.begin(),
			_1 - this->numberNonWeights
			),
			this->GPUMapFrom.begin()
			)
			),
			functors::add_when_less_than<int>(this->numberOfNodes, this->numberOfNodes + this->numberNonWeights)
			)
			),
			this->positionToSum.begin()), this->count.size(), "outPos3", "tests/testing.txt");
		testing::outputToFile<weight_type>(
			thrust::make_permutation_iterator(
			thrust::make_permutation_iterator(
			this->device_deltas.begin() + delta_next_start,
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			thrust::make_transform_iterator(
			this->GPUMapTo.begin(),
			_1 - this->numberNonWeights
			),
			this->GPUMapFrom.begin()
			)
			),
			functors::add_when_less_than<int>(this->numberOfNodes, this->numberOfNodes + this->numberNonWeights)
			)
			),
			this->positionToSum.begin()), this->count.size(), "outPos", "tests/testing.txt");

		testing::outputToFile<weight_type>(
			thrust::make_permutation_iterator(
			this->GPUWeights.begin(),
			this->positionToSum.begin()

			), this->count.size(), "outPos4", "tests/testing.txt");
		testing::outputToFile<weight_type>(
			thrust::make_permutation_iterator(
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.begin(),
			thrust::make_permutation_iterator(
			this->device_deltas.begin() + delta_next_start,
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			thrust::make_transform_iterator(
			this->GPUMapTo.begin(),
			_1 - this->numberNonWeights
			),
			this->GPUMapFrom.begin()
			)
			),
			functors::add_when_less_than<int>(this->numberOfNodes, this->numberOfNodes + this->numberNonWeights)
			)
			)
			)
			),
			functors::multiply<weight_type>()
			),

			this->positionToSum.begin()

			), this->count.size(), "outPos2", "tests/testing.txt");

		testing::outputToFile<weight_type>(this->RealOutput, "output", "tests/testing.txt");

		testing::outputToFile<weight_type>(this->GPUOutput_values.begin() + ((this->numberOfNodes + this->numberNonWeights) * i) - this->settings.i_output, this->settings.i_output, "pred_out", "tests/testing.txt");
		testing::outputToFile<weight_type>(this->device_deltas.begin() + delta_next_end - this->settings.i_output,this->settings.i_output, "delta_output", "tests/testing.txt");
		testing::outputToFile<weight_type>(this->device_deltas, "PostOutput", "tests/testing.txt");

#endif
		thrust::reduce_by_key(
			this->count.begin(),
			this->count.end(),
			thrust::make_permutation_iterator(
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.begin(),
			thrust::make_permutation_iterator(
			Unique_Iterator::make_return_zero_iterator(
			this->device_deltas.begin() + delta_next_start,
			this->device_deltas.end(),
			this->device_deltas.begin()
			),
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			thrust::make_transform_iterator(
			this->GPUMapTo.begin(),
			_1 - this->numberNonWeights
			),
			this->GPUMapFrom.begin()
			)
			),
			functors::add_when_less_than<long>(this->numberOfNodes, this->numberOfNodes + this->numberNonWeights)
			)
			)
			)
			),
			functors::multiply<weight_type>()
			),

			this->positionToSum.begin()

			),//Transform End
			thrust::make_discard_iterator(),
			this->GPUPreviousOutput_Values.begin()
			);

#ifdef DELTA_TEST
		testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values, "Mid", "tests/testing.txt");
		testing::outputToFile<weight_type>(this->device_deltas, "DeviceMid", "tests/testing.txt");
#endif

		//Find the new deltas
		thrust::transform(
			this->device_deltas.begin() + delta_next_start,
			this->device_deltas.begin() + delta_next_start + this->numberOfNodes,
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUOutput_values.begin() + ((this->numberOfNodes + this->numberNonWeights) * (i)) - this->numberOfNodes,
			this->GPUPreviousOutput_Values.begin()
			)), functors::find_non_output_delta<weight_type>()),
			this->device_deltas.begin() + delta_next_start,
			_1 + _2
			);
#ifdef DELTA_TEST
		testing::outputToFile<weight_type>(this->device_deltas, "Device", "tests/testing.txt");
		testing::outputToFile<weight_type>(this->GPUOutput_values, "test", "tests/testing.txt");
		testing::outputToFile<weight_type>(this->GPUWeights, "test", "tests/testing.txt");
#endif
		delta_next_end -= this->numberOfNodes;


		delta_next_start -= this->numberOfNodes;


	}

}

void LongTermShortTermNetwork::FindPreviousBias(){
	int start_mem_cells = this->GPUBias.size() - this->settings.i_output - this->number_nodes_by_type[0][MEMORY_CELL];
	int end_mem_cells = this->GPUBias.size() - this->settings.i_output;
#ifdef APPLY_DELTA_BIAS

	testing::outputToFile<weight_type>(this->GPUBias, "Bias1", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias1", "tests/prevbias2.txt");
#endif
	//Apply momentum to the bias

	if (this->settings.d_alpha != 0){
		//Apply the alpha to bias
		thrust::transform_if(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousBias.begin(),
			thrust::make_constant_iterator(this->settings.d_alpha),
			this->GPUBias.begin(),
			thrust::make_counting_iterator(int(0))
			)
			)
			,
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousBias.begin(),
			thrust::make_constant_iterator(this->settings.d_alpha) + this->GPUPreviousBias.size(),
			this->GPUBias.begin(),
			thrust::make_counting_iterator(int(0))
			)
			) + this->GPUBias.size(),
			this->GPUBias.begin(),
			functors::multiply_add<weight_type>(),
			functors::compare_between<(unsigned int)3, int>(3, 0, start_mem_cells, end_mem_cells)// #bias-#output-#memory_cells < count < #bias - #output
			);
	}



#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUBias, "Bias2", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias2", "tests/prevbias2.txt");
#endif

	//Retrieve the new previous Bias
	//If I remove memory cells, remove permutation around device_delta so it doesn't skip
	thrust::reduce_by_key(
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		),
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		) + ((this->settings.i_backprop_unrolled - 1) * this->GPUBias.size()),

		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		functors::multiply_by_constant<weight_type>((weight_type)this->settings.d_beta)
		),

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(this->numberOfNodes*(_1 % (this->settings.i_backprop_unrolled - 1))) + (_1 / (this->settings.i_backprop_unrolled - 1))
		)
		),
		thrust::make_discard_iterator(),
		this->GPUPreviousTemp.begin()
		);

	thrust::transform(this->GPUPreviousBias.begin(), this->GPUPreviousBias.end(), this->GPUPreviousTemp.begin(), this->GPUPreviousBias.begin(), _1 + _2);

#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUBias, "Bias3", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias3", "tests/prevbias2.txt");
#endif
}

void LongTermShortTermNetwork::FindPreviousWeights(){



#ifdef TRAININGTEST2
	//thrust::sequence(this->GPUOutput_values.begin(), this->GPUOutput_values.end());
	//thrust::sequence(this->device_deltas.begin(), this->device_deltas.end());
	testing::outputToFile<weight_type>(this->device_deltas, "Delta", "tests/test5.txt");
	testing::outputToFile<weight_type>(this->GPUOutput_values, "Output", "tests/test5.txt");

#endif
	int length_between_adds = this->GPUWeights.size() + this->numberNonWeights;
	int number_delta_between_add = this->GPUMapTo.size();
#ifdef TRAININGTEST2
	testing::outputToFile<weight_type>(
		thrust::make_permutation_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapTo.begin(),
		_1 - this->numberNonWeights
		),//End Transform Iterator (States to start with the 2nd layer)

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapTo.size()
		)//End Transform

		),//End Perm
		thrust::make_counting_iterator((int)0)
		)//End Tuple
		),//END zip
		functors::extend_value<int>(number_delta_between_add, 0, this->numberOfNodes, false)
		)//End of transform iterator
		), (this->settings.i_backprop_unrolled - 1)  * this->GPUWeights.size(),
		"Intermediate1",

		"tests/test5.txt"
		);

	testing::outputToFile<weight_type>(
		//Weight_permutation
		thrust::make_permutation_iterator(
		this->GPUOutput_values.begin(),
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached
		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapFrom.begin(),
		_1
		),//End Transform Iterator (States to start with the 2nd layer)
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapFrom.size()
		)
		),//End Perm
		thrust::make_counting_iterator((int)0)
		)
		),
		functors::extend_value<int>(this->GPUMapFrom.size(), 0, this->numberOfNodes + this->numberNonWeights, false)//Increase whenever the counter reaches the end
		)//End of transform iterator
		),
		(this->settings.i_backprop_unrolled - 1)  * this->GPUWeights.size(),
		"Intermediate2",

		"tests/test5.txt"
		);

	testing::outputToFile<weight_type>(thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		thrust::make_zip_iterator(
		thrust::make_tuple(

		thrust::make_permutation_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapTo.begin(),
		_1 - this->numberNonWeights
		),//End Transform Iterator (States to start with the 2nd layer)

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapTo.size()
		)//End Transform

		),//End Perm
		thrust::make_counting_iterator((int)0)
		)//End Tuple
		),//END zip
		functors::extend_value<int>(number_delta_between_add, 0, this->numberOfNodes, false)
		)//End of transform iterator
		),

		//Weight_permutation
		thrust::make_permutation_iterator(
		this->GPUOutput_values.begin(),
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapFrom.begin(),
		_1
		),//End Transform Iterator (States to start with the 2nd layer)
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapFrom.size()
		)
		),//End Perm
		thrust::make_counting_iterator((int)0)
		)
		),
		functors::extend_value<int>(this->GPUMapFrom.size(), 0, this->numberOfNodes + this->numberNonWeights, false)//Increase whenever the counter reaches the end
		)//End of transform iterator
		)//End Permutation Iterator


		)
		),
		functors::find_previous_weight<weight_type>(this->settings.d_beta)
		),//End Transform Iterator
		thrust::make_transform_iterator(//Weight 1 - 0, Weight 2-0,....
		thrust::make_counting_iterator((int)0),
		(this->GPUMapTo.size()*(_1 % (this->settings.i_backprop_unrolled - 1))) + (_1 / (this->settings.i_backprop_unrolled - 1))
		)

		), (this->settings.i_backprop_unrolled - 1) * this->GPUWeights.size(),
		"Intermediate",

		"tests/test5.txt"

		);


#endif

	//Apply the alpha
	if (this->settings.d_alpha != 0){
		thrust::transform_if(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			thrust::make_constant_iterator(this->settings.d_alpha),
			this->GPUPreviousWeights.begin(),
			this->GPUWeights.begin()
			)
			),
			thrust::make_zip_iterator(
			thrust::make_tuple(
			thrust::make_constant_iterator(this->settings.d_alpha),
			this->GPUPreviousWeights.end(),
			this->GPUWeights.end()
			)
			),
			this->GPUWeights.begin(),
			functors::multiply_add<weight_type>(),
			functors::compare_two<(unsigned int)1, weight_type>(5, 1)
			);
	}

	//Find the previous weights
	thrust::reduce_by_key(
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		),
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		) + (this->settings.i_backprop_unrolled - 1)  * this->GPUWeights.size(),

		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		thrust::make_zip_iterator(
		thrust::make_tuple(

		thrust::make_permutation_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapTo.begin(),
		_1 - this->numberNonWeights
		),//End Transform Iterator (States to start with the 2nd layer)

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapTo.size()
		)//End Transform

		),//End Perm
		thrust::make_counting_iterator((int)0)
		)//End Tuple
		),//END zip
		functors::extend_value<int>(number_delta_between_add, 0, this->numberOfNodes, false)
		)//End of transform iterator
		),

		//Weight_permutation
		thrust::make_permutation_iterator(
		this->GPUOutput_values.begin(),
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached
		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapFrom.begin(),
		_1
		),//End Transform Iterator (States to start with the 2nd layer)
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapFrom.size()
		)
		),//End Perm
		thrust::make_counting_iterator((int)0)
		)
		),
		functors::extend_value<int>(this->GPUMapFrom.size(), 0, this->numberOfNodes + this->numberNonWeights, false)//Increase whenever the counter reaches the end
		)//End of transform iterator
		)//End Permutation Iterator


		)
		),
		functors::find_previous_weight<weight_type>(this->settings.d_beta)
		),//End Transform Iterator
		thrust::make_transform_iterator(//Weight 1 - 0, Weight 2-0,....
		thrust::make_counting_iterator((int)0),
		(this->GPUMapTo.size()*(_1 % (this->settings.i_backprop_unrolled - 1))) + (_1 / (this->settings.i_backprop_unrolled - 1))
		)

		),//End Permutation Iterator
		thrust::make_discard_iterator(),
		this->GPUPreviousTemp.begin()
		);

	thrust::transform(this->GPUPreviousWeights.begin(), this->GPUPreviousWeights.end(), this->GPUPreviousTemp.begin(), this->GPUPreviousWeights.begin(), _1 + _2);
	
	bool test = false;
	if (test == true){
		testing::outputToFile<weight_type>(this->GPUPreviousWeights, "Delta2", "tests/test5.txt");
	}

#ifdef TRAININGTEST2
	testing::outputToFile<weight_type>(this->device_deltas, "Delta2", "tests/test5.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousWeights, "PrevGPUVal", "tests/test5.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousTemp, "PreGPUTemp", "tests/test5.txt");
#endif

}

//Apply the error
void LongTermShortTermNetwork::ApplyLongTermShortTermMemoryError(){

#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias-1", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUBias, "Bias-1", "tests/prevbias2.txt");


#endif

#ifdef DELTA_MAP_TEST 
	testing::outputToFile<int>(this->GPUMapFrom, "From", "tests/prevbias3.txt");
	testing::outputToFile<int>(this->GPUMapTo, "To", "tests/prevbias3.txt");
#endif
	this->ApplyErrorToBias();


	thrust::transform_if(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUWeights.begin(),
		this->GPUPreviousWeights.begin()

		)
		),
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUWeights.end(),
		this->GPUPreviousWeights.end()

		)
		),
		this->GPUWeights.begin(),
		functors::add_and_store<weight_type>(this->settings.i_backprop_unrolled - 1),
		functors::compare_two<(unsigned int)0, weight_type>(5, 1));

#ifdef APPLY_DELTA_BIAS

	testing::outputToFile<weight_type>(this->GPUBias, "Bias-5", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias-5", "tests/prevbias2.txt");
#endif



}

void LongTermShortTermNetwork::ApplyErrorToBias(){
	int start_mem_cells = this->GPUBias.size() - this->settings.i_output - this->number_nodes_by_type[0][MEMORY_CELL];
	int end_mem_cells = this->GPUBias.size() - this->settings.i_output;
	//Apply the delta to the bias


	//Apply the error
	thrust::transform_if(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUBias.begin(),
		this->GPUPreviousBias.begin(),
		thrust::make_counting_iterator((int)0)
		)
		),
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUBias.begin(),
		this->GPUPreviousBias.begin(),
		thrust::make_counting_iterator((int)0)
		)
		) + this->GPUBias.size(),
		this->GPUBias.begin(),
		functors::add_and_store<weight_type>(this->settings.i_backprop_unrolled - 1),
		functors::compare_between<(unsigned int)2, int>(3, 0, start_mem_cells, end_mem_cells)
		);

#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUBias, "Bias4", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias4", "tests/prevbias2.txt");
#endif
}

//*********************
//Run The Network
//*********************


void LongTermShortTermNetwork::InitializeLongShortTermMemoryForRun(){
	//Form the delta objects
	this->CopyToDevice();
	this->moveBiasToGPU(false);//Don't create a previous_bias
	this->count_weights_in_layers();
}

device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in, run_type type){
	this->setInput(in);
	switch (type){
	case run_type::WITHOUT_MEMORY_CELLS:
		return this->runNetwork(in, 0, this->newSequence);
	case run_type::WITH_MEMORY_CELLS:
		//Pass the number of memory cells into the function so they can be skipped when finding the output
		return this->runNetwork(in, 1, this->newSequence);
	default:
		return this->runNetwork(in, 0, this->newSequence);
	}
}

device_vector<weight_type> LongTermShortTermNetwork::runNetwork(device_vector<weight_type> in, run_type type){
	this->setInput(in);//Set the input

	switch (type){
	case run_type::WITHOUT_MEMORY_CELLS:
		return this->runNetwork(0);
	case run_type::WITH_MEMORY_CELLS:
		//Pass the number of memory cells into the function so they can be skipped when finding the output
		return this->runNetwork(1);
	default:
		return this->runNetwork(0);
	}

}

thrust::device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in){
	return this->runNetwork(in, 0, this->newSequence);
}

thrust::device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in, int number_of_extra_weights, bool &newSequence){
	if (newSequence){
		//this->runNetwork(in, number_of_extra_weights);
		//newSequence = false;
	}

	return this->runNetwork(number_of_extra_weights);
}



thrust::device_vector<weight_type> LongTermShortTermNetwork::runNetwork(int number_of_extra_weights){


	//Stores the numberofmblocks in a layer
	unsigned int numberMBlocks;
	//Number mBlocks in previous layer
	unsigned int previousnumberMBlocks = 0;
	unsigned int numberBlocksToLayer = 0;
	device_vector<weight_type> toReturn = device_vector<weight_type>(this->settings.i_output);

	//unsigned int numberBias = 0;
	//Perform the transformation on each layer
	for (unsigned int i = 0; i < this->mBlocksLayers.size() - 1; i += 2){

		if (i != 0){
			previousnumberMBlocks += this->numberOfWeightsInLayers[i - 1] + this->numberOfWeightsInLayers[i - 2] + number_of_extra_weights;
			numberBlocksToLayer += numberMBlocks;
		}
		numberMBlocks = this->mBlocksLayers[i].size();



#ifdef TRAININGTEST

		if (number_of_extra_weights == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUPREV_START", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "GPU_START", "tests/PrevOut1.txt");
		}
		else if (this->settings.i_backprop_unrolled == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUPREV_START", "tests/PrevOut2.txt");
			testing::outputToFile(this->GPUOutput_values, "GPU_START", "tests/PrevOut2.txt");

		}

#endif

		//Sum the values of the input/output/forget/potential_memory_cell_values nodes
		//The values in the GPU weights are in the order input, output, forget, memory cells
		//Subtracting this->mBlocksLayers[i].size() from the end will remove the memory cells from doing anything
		//Output to Previous
		thrust::reduce_by_key(
			this->GPUMapTo.begin() + previousnumberMBlocks,
			//Start at the beginning of the previous layer
			this->GPUMapTo.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i], // End at the the number of nodes before the current layer + the number of nodes in the current layer
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.begin() + previousnumberMBlocks,//Start from the beginning of the layer
			thrust::make_permutation_iterator(//Permute the output values such that they start at the correct position
			this->GPUOutput_values.begin(),
			this->GPUMapFrom.begin() + previousnumberMBlocks
			)
			)
			),
			functors::multiply_or_return_zero<weight_type, 1>() //Multiply the two together
			),
			thrust::make_discard_iterator(),
			this->GPUPreviousOutput_Values.begin()
			);
#ifdef TRAININGTEST
		if (number_of_extra_weights == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUPreBias1", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUPreBias2", "tests/PrevOut1.txt");
			testing::outputToFile<weight_type>(this->GPUMapTo.begin() + previousnumberMBlocks, this->numberOfWeightsInLayers[i], "Map0", "tests/map1.txt");

		}
		else if (this->settings.i_backprop_unrolled == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUPreBias1", "tests/PrevOut2.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUPreBias2", "tests/PrevOut2.txt");
			testing::outputToFile<weight_type>(this->GPUMapTo.begin() + previousnumberMBlocks, this->numberOfWeightsInLayers[i], "Map0", "tests/map2.txt");

		}


#endif
		//Add Bias to the hidden layers
		thrust::transform(
			this->GPUBias.begin() + numberBlocksToLayer,
			this->GPUBias.begin() + numberBlocksToLayer + this->number_nodes_in_layer[i],
			this->GPUPreviousOutput_Values.begin(),
			this->GPUPreviousOutput_Values.begin(),
			_1 + _2
			);


#ifdef TRAININGTEST
		if (number_of_extra_weights == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUPostBias", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUPostBias", "tests/PrevOut1.txt");
			testing::outputToFile<weight_type>(this->GPUBias.begin() + numberBlocksToLayer, (numberMBlocks * 4), "Map1", "tests/map1.txt");
		}
		else if (this->settings.i_backprop_unrolled == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUPostBias", "tests/PrevOut2.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUPostBias", "tests/PrevOut2.txt");
			testing::outputToFile<weight_type>(this->GPUBias.begin() + numberBlocksToLayer, (numberMBlocks * 4), "Map1", "tests/map2.txt");
		}

#endif


		//Create a input/output/forget/potential_memory_cell_values/memory_cell_value value
		//Essentially run the gate and get the output value
		thrust::for_each(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousOutput_Values.begin(), //input values
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL],//output values
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL],//forget values
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL],//potential_memory_cell_value
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL] + this->number_nodes_by_type[0][POTENTIAL_MEMORY_CELL],
			this->GPUOutput_values.begin() + numberBlocksToLayer, //Old Input
			this->GPUOutput_values.begin() + numberBlocksToLayer + this->number_nodes_by_type[0][INPUT_CELL],//Old output
			this->GPUOutput_values.begin() + numberBlocksToLayer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL], //Old Forget
			this->GPUOutput_values.begin() + numberBlocksToLayer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL], //Old Potential
			this->GPUOutput_values.begin() + numberBlocksToLayer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL] + this->number_nodes_by_type[0][POTENTIAL_MEMORY_CELL]//Old Memory Cell Values
			)),
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL], //input values
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL],//output values
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL],//forget values
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL] + this->number_nodes_by_type[0][POTENTIAL_MEMORY_CELL],//potential_memory_cell_value
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL] + this->number_nodes_by_type[0][POTENTIAL_MEMORY_CELL] + +this->number_nodes_by_type[0][MEMORY_CELL],
			this->GPUOutput_values.begin() + numberBlocksToLayer + this->number_nodes_by_type[0][INPUT_CELL], //Old Input
			this->GPUOutput_values.begin() + numberBlocksToLayer + this->number_nodes_by_type[0][INPUT_CELL] + +this->number_nodes_by_type[0][OUTPUT_CELL],
			this->GPUOutput_values.begin() + numberBlocksToLayer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + +this->number_nodes_by_type[0][FORGET_CELL], //Old Forget
			this->GPUOutput_values.begin() + numberBlocksToLayer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL] + this->number_nodes_by_type[0][POTENTIAL_MEMORY_CELL], //Old Potential
			this->GPUOutput_values.begin() + numberBlocksToLayer + this->number_nodes_by_type[0][INPUT_CELL] + this->number_nodes_by_type[0][OUTPUT_CELL] + this->number_nodes_by_type[0][FORGET_CELL] + this->number_nodes_by_type[0][POTENTIAL_MEMORY_CELL] + this->number_nodes_by_type[0][MEMORY_CELL]//Old Memory Cell Values
			)),
			functors::run_memory_block_functon<weight_type>());

#ifdef TRAININGTEST
		if (number_of_extra_weights == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUMid1", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUMid1", "tests/PrevOut1.txt");
			testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values.begin() + numberMBlocks, numberMBlocks, "Map1", "tests/map1.txt");
		}
		else if (this->settings.i_backprop_unrolled == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUMid1", "tests/PrevOut2.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUMid1", "tests/PrevOut2.txt");
			testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values.begin() + numberMBlocks, numberMBlocks, "Map1", "tests/map2.txt");
		}

#endif


		//Find the output value
		if (number_of_extra_weights == 0){

			thrust::reduce_by_key(
				this->GPUMapTo.begin() + this->numberOfWeightsInLayers[i],
				this->GPUMapTo.begin() + this->numberOfWeightsInLayers[i] + this->numberOfWeightsInLayers[i + 1],
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->GPUWeights.begin() + this->numberOfWeightsInLayers[i],
				thrust::make_permutation_iterator(
				this->GPUPreviousOutput_Values.begin(),
				thrust::make_transform_iterator(
				this->GPUMapFrom.begin() + this->numberOfWeightsInLayers[i],
				_1 - this->numberNonWeights
				)
				)

				)
				),
				functors::multiply<weight_type>()
				),
				thrust::make_discard_iterator(),
				this->GPUPreviousOutput_Values.begin() + this->number_nodes_in_layer[i]
				);
		}
		else{
			thrust::reduce_by_key(
				this->GPUMapTo.begin() + this->numberOfWeightsInLayers[i],
				this->GPUMapTo.begin() + this->numberOfWeightsInLayers[i] + this->numberOfWeightsInLayers[i + 1],
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->GPUWeights.begin() + this->numberOfWeightsInLayers[i],
				thrust::make_permutation_iterator(
				this->GPUPreviousOutput_Values.begin(),
				thrust::make_transform_iterator(
				thrust::make_transform_iterator(
				this->GPUMapFrom.begin() + this->numberOfWeightsInLayers[i],
				_1 - this->numberNonWeights
				),
				functors::add_when_greater_than<int>(-(this->numberOfNodes + this->numberNonWeights), this->numberOfNodes)
				)

				)
				)
				),

				functors::multiply_or_return_zero<weight_type, 1>()
				),
				thrust::make_discard_iterator(),
				this->GPUPreviousOutput_Values.begin() + this->number_nodes_in_layer[i]
				);

		}

#ifdef TRAININGTEST

		if (number_of_extra_weights == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUOut", "tests/PrevOut1.txt");
			testing::outputToFile<weight_type>(this->GPUMapTo.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i], this->numberOfWeightsInLayers[i + 1], "Map2", "tests/map1.txt");
			testing::outputToFile<weight_type>(this->GPUMapFrom.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i], this->numberOfWeightsInLayers[i + 1], "From1", "tests/map1.txt");
		}
		else if (this->settings.i_backprop_unrolled == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUOut", "tests/PrevOut2.txt");
			testing::outputToFile<weight_type>(this->GPUMapTo.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i], this->numberOfWeightsInLayers[i + 1], "Map2", "tests/map2.txt");
			testing::outputToFile<weight_type>(this->GPUMapFrom.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i], this->numberOfWeightsInLayers[i + 1], "From1", "tests/map2.txt");

		}

#endif

		//Add the bias to the output
		thrust::transform(
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_in_layer[i],
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_in_layer[i] + this->number_nodes_in_layer[i + 1],
			this->GPUBias.begin() + this->number_nodes_in_layer[i],
			this->GPUPreviousOutput_Values.begin() + this->number_nodes_in_layer[i],
			functors::add_and_sigmoid<weight_type>()
			);


#ifdef TRAININGTEST

		if (number_of_extra_weights == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "Final", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "Final", "tests/PrevOut1.txt");
		}
		else if (this->settings.i_backprop_unrolled == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "Final", "tests/PrevOut2.txt");
			testing::outputToFile(this->GPUOutput_values, "Final", "tests/PrevOut2.txt");
		}


#endif
		thrust::copy(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.begin() + this->numberOfNodes, this->GPUOutput_values.begin() + this->numberNonWeights);
		thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);

#ifdef TRAININGTEST
		if (number_of_extra_weights == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUNew", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUNew", "tests/PrevOut1.txt");
		}
		else if (this->settings.i_backprop_unrolled == 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUNew", "tests/PrevOut2.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUNew", "tests/PrevOut2.txt");
		}

#endif
	}






	thrust::copy(this->GPUOutput_values.begin() + this->numberNonWeights + numberBlocksToLayer + this->numberOfNodes - this->settings.i_output, this->GPUOutput_values.begin() + this->numberNonWeights + numberBlocksToLayer + this->numberOfNodes, toReturn.begin());

	return toReturn;
}
