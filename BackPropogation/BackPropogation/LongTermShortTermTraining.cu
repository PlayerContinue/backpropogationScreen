#include "hip/hip_runtime.h"
#include "LongTermShortTermNetwork.cuh"
//#define TRAININGTEST
//#define TRAININGTEST2
//#define DELTA_TEST
//#define AVERAGE_TEST
//#define APPLY_DELTA_BIAS
//#define NVIDA_OUTPUT_TEST
//#define NVIDA_OUTPUT_TEST2
//#define DELTA_MAP_TEST
#define NUMBER_MEMORY_WEIGHTS 3
//*********************
//Training the Network
//*********************



void LongTermShortTermNetwork::InitializeLongShortTermMemory(){
	//Store all the values in the device
	//Will later add option for too little memory
	//Copy the information to the device
	this->UnrollNetwork(3);
	this->RealOutput = device_vector<weight_type>(this->settings.i_output);
	this->host_deltas = host_vector<weight_type>(this->GPUOutput_values.size() - this->numberNonWeights);
	this->device_deltas = device_vector<weight_type>(this->GPUOutput_values.size() - (this->settings.i_backprop_unrolled*this->numberNonWeights));
	this->training_previous_number_rows = this->settings.i_backprop_unrolled;
	this->count_weights_in_layers(true);
}

void LongTermShortTermNetwork::averageWeights(){

	thrust::copy(this->GPUOutput_values.begin() + ((this->numberOfNodes + this->numberNonWeights)* this->training_previous_number_rows-2), this->GPUOutput_values.begin() + ((this->numberOfNodes + this->numberNonWeights)* this->training_previous_number_rows-1), this->GPUOutput_values.begin());//Replace the current input with the output from the last run
	thrust::fill(this->GPUOutput_values.begin() + this->numberOfNodes + this->numberNonWeights, this->GPUOutput_values.end(), (weight_type)0);//Reset the rest of the output values


}


//Add the input
void LongTermShortTermNetwork::setInput(weight_type* in){
	//Place the input into the GPU values matrix
	for (int i = 0; i < this->settings.i_input; i++){
		this->GPUOutput_values[i] = in[i];
	}

}

//Add the input
void LongTermShortTermNetwork::setInput(weight_type** in){
	//Place the input into the GPU values matrix

	for (int j = 0; j < this->settings.i_backprop_unrolled; j++){
		for (int i = 0; i < this->numberNonWeights; i++){
			this->GPUOutput_values[i + (j*(this->numberNonWeights + this->numberOfNodes))] = in[j][i];
		}
	}

}

void LongTermShortTermNetwork::StartTraining(weight_type** in, weight_type** out){

	//Reset the weights to the end of the weights
	this->averageWeights();
	//Set the input values
	this->setInput(in);
	this->training_previous_number_rows = this->settings.i_backprop_unrolled;
	this->LongShortTermMemoryTraining(in, out);
	
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
	//Find the delta 
	this->FindBackPropDelta(out, 0);
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
	this->FindPreviousBias();
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
	this->FindPreviousWeights();
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
	thrust::fill(this->device_deltas.begin(), this->device_deltas.end(), (weight_type)0);
}

void LongTermShortTermNetwork::LongTermShortTermNetwork::LongShortTermMemoryTraining(weight_type** in, weight_type** out){
	
	int start = 1;

	//Get the number of weights in the output layer
	//This is needed because the output layer needs to be used only once, so we need to inform the system which weights to skip


	if (this->newSequence){
		//New sequence, so find the initial first layer
		start = 0;
		
	}
	
	unsigned int number_nodes_to_beginning_of_layer = 0;
	unsigned int number_weights_in_layer = this->GPUWeights.size();
	for (int i = start; i < this->settings.i_backprop_unrolled; i++){
	
		thrust::reduce_by_key(
			this->GPUMapTo.begin(),
			this->GPUMapTo.end(),

			//Multiply the weights x output
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.begin(),
			thrust::make_permutation_iterator(
			this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer,
			this->GPUMapFrom.begin()
			)
			)
			),
			functors::multiply<weight_type>()
			),
			thrust::make_discard_iterator(),
			this->GPUPreviousOutput_Values.begin()
			);

#ifdef NVIDA_OUTPUT_TEST2
		testing::outputToFile<weight_type>(this->GPUWeights, "weights", "tests/prevbias3.txt");
		testing::outputToFile<weight_type>(this->GPUBias, "weights", "tests/prevbias3.txt");
		testing::outputToFile<weight_type>(this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer,8, "GPU", "tests/prevbias3.txt");
		testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values, "PrevBias-3", "tests/prevbias3.txt");
		
#endif

		if (i > 0){//Only increment it by the number of nodes when working from any layer which is not the initial layer
			//This lets the nodes use the previous layer as their input
			number_nodes_to_beginning_of_layer += this->numberOfNodes + this->numberNonWeights;
		}

		//Add the bias to the current value
		thrust::transform(this->GPUBias.begin(),
			this->GPUBias.end(),
			this->GPUPreviousOutput_Values.begin(),
			this->GPUPreviousOutput_Values.begin(),
			_1 + _2
			);


		//Transfer all values from the current to the next row
		thrust::transform(this->GPUPreviousOutput_Values.begin(),
			this->GPUPreviousOutput_Values.end(),
			this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->numberNonWeights, functors::sigmoid_functor<weight_type>());

	}

	

}





//Find the delta gradiant for each of the "layers" of the network
void LongTermShortTermNetwork::FindBackPropDelta(weight_type** out, int current_layer){

	unsigned int number_weights_in_layer = this->GPUWeights.size();

	unsigned int number_nodes_to_end_of_layer = this->GPUOutput_values.size();

	


	unsigned int delta_next_start = this->numberOfNodes * this->settings.i_backprop_unrolled - this->numberOfNodes;//this->device_deltas.size() - this->numberOfNodes;
	unsigned int delta_next_end = this->numberOfNodes * this->settings.i_backprop_unrolled; //this->device_deltas.size();
	
	//Find the deltas of the output
	for (int i = this->settings.i_backprop_unrolled; i > 0; i--){
		//Store the output into a vector
		//Performed each round due to the output changing each round
		for (unsigned int j = 0; j < this->RealOutput.size(); j++){
			this->RealOutput[j] = out[i-1][j];
		}

		//Find the delta of the output for the current layer
		//output * (1-output) * (target - output)
		thrust::transform(
			this->RealOutput.begin(),
			this->RealOutput.end(),
			this->GPUOutput_values.begin() + ((this->numberOfNodes + this->numberNonWeights) * i) - this->settings.i_output,
			this->device_deltas.begin() + delta_next_end - this->settings.i_output,
			functors::find_output_delta<weight_type>());

		number_nodes_to_end_of_layer -= this->settings.i_output;
#ifdef DELTA_TEST
		testing::outputToFile<weight_type>(this->RealOutput, "output", "tests/testing.txt");
		testing::outputToFile<weight_type>(this->RealOutput, "output", "tests/testing.txt");
		testing::outputToFile<weight_type>(this->GPUOutput_values.begin() + ((this->numberOfNodes + this->numberNonWeights) * i) - this->settings.i_output, this->settings.i_output, "pred_out", "tests/testing.txt");
		testing::outputToFile<weight_type>(this->device_deltas.begin() + delta_next_end - this->settings.i_output,this->settings.i_output, "delta_output", "tests/testing.txt");
		testing::outputToFile<weight_type>(this->device_deltas, "PostOutput", "tests/testing.txt");
		
#endif
		if (i != this->settings.i_backprop_unrolled){//Only perform this action when we've gone past the output layer

			thrust::reduce_by_key(
				this->count.begin(),
				this->count.end(),
				thrust::make_permutation_iterator(
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->GPUWeights.begin(),

				thrust::make_permutation_iterator(
				this->device_deltas.begin() + delta_next_end,
				thrust::make_transform_iterator(
				this->GPUMapTo.begin(),
				_1 - this->numberNonWeights
				)
				)
				)
				),
				functors::multiply<weight_type>()
				),
				this->positionToSum.begin()

				),//Transform End
				thrust::make_discard_iterator(),
				this->GPUPreviousOutput_Values.begin()
				);

#ifdef DELTA_TEST
			testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values, "PreTransform", "tests/testing.txt");
#endif
			//Add the bias
			/*thrust::transform(
				this->GPUBias.begin(),
				this->GPUBias.end(),
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->device_deltas.begin() + end_of_succeeding_layer - this->numberNonWeights - this->numberOfNodes,
				this->GPUPreviousOutput_Values.begin()

				)
				),
				this->GPUPreviousOutput_Values.begin(),
				functors::add_bias<weight_type>(this->settings.i_backprop_unrolled == i)
				);*/
#ifdef DELTA_TEST
			testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values, "PostBias", "tests/testing.txt");
#endif
			//Find the new deltas
			thrust::transform(
				this->device_deltas.begin() + delta_next_start,
				this->device_deltas.begin() + delta_next_start + this->numberOfNodes,
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->GPUOutput_values.begin() + ((this->numberOfNodes + this->numberNonWeights) * (i)) - this->numberOfNodes,
				this->GPUPreviousOutput_Values.begin()
				)), functors::find_non_output_delta<weight_type>()),
				this->device_deltas.begin() + delta_next_start,
				_1 + _2
				);
#ifdef DELTA_TEST
			testing::outputToFile<weight_type>(this->device_deltas, "Device","tests/testing.txt");
#endif
		}
		delta_next_end -= this->numberOfNodes;

		
		delta_next_start -= this->numberOfNodes;

		number_nodes_to_end_of_layer = (number_nodes_to_end_of_layer + this->settings.i_output) - this->numberOfNodes - this->numberNonWeights;
		


	}

}

void LongTermShortTermNetwork::FindPreviousBias(){
	int start_mem_cells = this->GPUBias.size() - this->settings.i_output - this->mBlocksLayers[0].size();
	int end_mem_cells = this->GPUBias.size() - this->settings.i_output;
#ifdef APPLY_DELTA_BIAS

	testing::outputToFile<weight_type>(this->GPUBias, "Bias1", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias1", "tests/prevbias2.txt");
#endif
	//Apply momentum to the bias

	if (this->settings.d_alpha != 0){
		//Apply the alpha to bias
		thrust::transform_if(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousBias.begin(),
			thrust::make_constant_iterator(this->settings.d_alpha),
			this->GPUBias.begin(),
			thrust::make_counting_iterator(int(0))
			)
			)
			,
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousBias.begin(),
			thrust::make_constant_iterator(this->settings.d_alpha) + this->GPUPreviousBias.size(),
			this->GPUBias.begin(),
			thrust::make_counting_iterator(int(0))
			)
			) + this->GPUBias.size(),
			this->GPUBias.begin(),
			functors::multiply_add<weight_type>(),
			functors::compare_between<(unsigned int)3, int>(3, 0, start_mem_cells, end_mem_cells)// #bias-#output-#memory_cells < count < #bias - #output
			);
	}



#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUBias, "Bias2", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias2", "tests/prevbias2.txt");
#endif

	//Retrieve the new previous Bias
	//If I remove memory cells, remove permutation around device_delta so it doesn't skip
	thrust::reduce_by_key(
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		),
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		) + ((this->settings.i_backprop_unrolled - 1) * this->GPUBias.size()),

		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		functors::multiply_by_constant<weight_type>((weight_type)this->settings.d_beta)
		),

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(this->numberOfNodes*(_1 % (this->settings.i_backprop_unrolled - 1))) + (_1 / (this->settings.i_backprop_unrolled - 1))
		)
		),
		thrust::make_discard_iterator(),
		this->GPUPreviousTemp.begin()
		);

	thrust::transform(this->GPUPreviousBias.begin(),this->GPUPreviousBias.end(),this->GPUPreviousTemp.begin(),this->GPUPreviousBias.begin(), _1 + _2);

#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUBias, "Bias3", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias3", "tests/prevbias2.txt");
#endif
}

void LongTermShortTermNetwork::FindPreviousWeights(){

	

#ifdef TRAININGTEST2
	//thrust::sequence(this->GPUOutput_values.begin(), this->GPUOutput_values.end());
	//thrust::sequence(this->device_deltas.begin(), this->device_deltas.end());
	testing::outputToFile<weight_type>(this->device_deltas, "Delta", "tests/test5.txt");
	testing::outputToFile<weight_type>(this->GPUOutput_values, "Output", "tests/test5.txt");
	
#endif
	int length_between_adds = this->GPUWeights.size() + this->numberNonWeights;
	int number_delta_between_add = this->GPUMapTo.size();
#ifdef TRAININGTEST2
	testing::outputToFile<weight_type>(
		thrust::make_permutation_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapTo.begin(),
		_1 - this->numberNonWeights
		),//End Transform Iterator (States to start with the 2nd layer)

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapTo.size()
		)//End Transform

		),//End Perm
		thrust::make_counting_iterator((int)0)
		)//End Tuple
		),//END zip
		functors::extend_value<int>(number_delta_between_add, 0, this->numberOfNodes, false)
		)//End of transform iterator
		), (this->settings.i_backprop_unrolled - 1)  * this->GPUWeights.size(),
		"Intermediate1",

		"tests/test5.txt"
		);

	testing::outputToFile<weight_type>(thrust::make_permutation_iterator(
		this->GPUOutput_values.begin(),
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapFrom.begin(),
		_1
		),//End Transform Iterator (States to start with the 2nd layer)
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapFrom.size()
		)
		),//End Perm
		thrust::make_counting_iterator((int)0)
		)
		),
		functors::extend_value<int>(this->GPUMapFrom.size(), 0, this->numberOfNodes + this->numberNonWeights, false)//Increase whenever the counter reaches the end
		)//End of transform iterator
		), (this->settings.i_backprop_unrolled - 1)  * this->GPUWeights.size(),
		"Intermediate2",

		"tests/test5.txt"
		);

	testing::outputToFile<weight_type>(thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		thrust::make_zip_iterator(
		thrust::make_tuple(

		thrust::make_permutation_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapTo.begin(),
		_1 - this->numberNonWeights
		),//End Transform Iterator (States to start with the 2nd layer)

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapTo.size()
		)//End Transform

		),//End Perm
		thrust::make_counting_iterator((int)0)
		)//End Tuple
		),//END zip
		functors::extend_value<int>(number_delta_between_add, 0, this->numberOfNodes, false)
		)//End of transform iterator
		),

		//Weight_permutation
		thrust::make_permutation_iterator(
		this->GPUOutput_values.begin(),
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapFrom.begin(),
		_1
		),//End Transform Iterator (States to start with the 2nd layer)
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapFrom.size()
		)
		),//End Perm
		thrust::make_counting_iterator((int)0)
		)
		),
		functors::extend_value<int>(this->GPUMapFrom.size(), 0, this->numberOfNodes + this->numberNonWeights, false)//Increase whenever the counter reaches the end
		)//End of transform iterator
		)//End Permutation Iterator


		)
		),
		functors::find_previous_weight<weight_type>(this->settings.d_beta)
		),//End Transform Iterator
		thrust::make_transform_iterator(//Weight 1 - 0, Weight 2-0,....
		thrust::make_counting_iterator((int)0),
		(this->GPUMapTo.size()*(_1 % (this->settings.i_backprop_unrolled - 1))) + (_1 / (this->settings.i_backprop_unrolled - 1))
		)

		), (this->settings.i_backprop_unrolled - 1) * this->GPUWeights.size(),
		"Intermediate",

		"tests/test5.txt"
		
		);
	

#endif

	//Apply the alpha
	thrust::transform_if(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_constant_iterator(this->settings.d_alpha),
		this->GPUPreviousWeights.begin(),
		this->GPUWeights.begin()
		)
		),
		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_constant_iterator(this->settings.d_alpha),
		this->GPUPreviousWeights.end(),
		this->GPUWeights.end()
		)
		),
		this->GPUWeights.begin(),
		functors::multiply_add<weight_type>(),
		functors::compare_two<(unsigned int)1, weight_type>(5, 1)
		);

	//Find the previous weights
	thrust::reduce_by_key(
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		),
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		(_1 / (this->settings.i_backprop_unrolled - 1))
		) + (this->settings.i_backprop_unrolled - 1)  * this->GPUWeights.size(),

		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		thrust::make_zip_iterator(
		thrust::make_tuple(

		thrust::make_permutation_iterator(
		this->device_deltas.begin() + this->numberOfNodes,
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapTo.begin(),
		_1 - this->numberNonWeights
		),//End Transform Iterator (States to start with the 2nd layer)

		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapTo.size()
		)//End Transform

		),//End Perm
		thrust::make_counting_iterator((int)0)
		)//End Tuple
		),//END zip
		functors::extend_value<int>(number_delta_between_add, 0, this->numberOfNodes, false)
		)//End of transform iterator
		),

		//Weight_permutation
		thrust::make_permutation_iterator(
		this->GPUOutput_values.begin(),
		thrust::make_transform_iterator(//Add the number of nodes when the end of the mapto is reached

		thrust::make_zip_iterator(
		thrust::make_tuple(
		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		this->GPUMapFrom.begin(),
		_1
		),//End Transform Iterator (States to start with the 2nd layer)
		thrust::make_transform_iterator(
		thrust::make_counting_iterator((int)0),
		_1%this->GPUMapFrom.size()
		)
		),//End Perm
		thrust::make_counting_iterator((int)0)
		)
		),
		functors::extend_value<int>(this->GPUMapFrom.size(), 0, this->numberOfNodes + this->numberNonWeights, false)//Increase whenever the counter reaches the end
		)//End of transform iterator
		)//End Permutation Iterator


		)
		),
		functors::find_previous_weight<weight_type>(this->settings.d_beta)
		),//End Transform Iterator
		thrust::make_transform_iterator(//Weight 1 - 0, Weight 2-0,....
		thrust::make_counting_iterator((int)0),
		(this->GPUMapTo.size()*(_1 % (this->settings.i_backprop_unrolled - 1))) + (_1 / (this->settings.i_backprop_unrolled - 1))
		)

		),//End Permutation Iterator
		thrust::make_discard_iterator(),
		this->GPUPreviousTemp.begin()
		);

	thrust::transform(this->GPUPreviousWeights.begin(),this->GPUPreviousWeights.end(),this->GPUPreviousTemp.begin(),this->GPUPreviousWeights.begin(), _1 + _2);


#ifdef TRAININGTEST2
	testing::outputToFile<weight_type>(this->device_deltas, "Delta2", "tests/test5.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousWeights, "PrevGPUVal", "tests/test5.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousTemp, "PreGPUTemp", "tests/test5.txt");
#endif

}

//Apply the error
void LongTermShortTermNetwork::ApplyLongTermShortTermMemoryError(){

#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias-1", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUBias, "Bias-1", "tests/prevbias2.txt");


#endif

#ifdef DELTA_MAP_TEST 
	testing::outputToFile<int>(this->GPUMapFrom, "From", "tests/prevbias3.txt");
	testing::outputToFile<int>(this->GPUMapTo, "To", "tests/prevbias3.txt");
#endif
	this->ApplyErrorToBias();


	thrust::transform_if(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUWeights.begin(),
		this->GPUPreviousWeights.begin()
		
		)
		),
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUWeights.end(),
		this->GPUPreviousWeights.end()
		
		)
		),
		this->GPUWeights.begin(),
		functors::add_and_store<weight_type>(this->settings.i_backprop_unrolled-1),
		functors::compare_two<(unsigned int)0, weight_type>(5, 1));

#ifdef APPLY_DELTA_BIAS

	testing::outputToFile<weight_type>(this->GPUBias, "Bias-5", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias-5", "tests/prevbias2.txt");
#endif



}

void LongTermShortTermNetwork::ApplyErrorToBias(){
	int start_mem_cells = this->GPUBias.size() - this->settings.i_output - this->mBlocksLayers[0].size();
	int end_mem_cells = this->GPUBias.size() - this->settings.i_output;
	//Apply the delta to the bias


	//Apply the error
	thrust::transform_if(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUBias.begin(),
		this->GPUPreviousBias.begin(),
		thrust::make_counting_iterator((int)0)
		)
		),
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUBias.begin(),
		this->GPUPreviousBias.begin(),
		thrust::make_counting_iterator((int)0)
		)
		) + this->GPUBias.size(),
		this->GPUBias.begin(),
		functors::add_and_store<weight_type>(this->settings.i_backprop_unrolled - 1),
		functors::compare_between<(unsigned int)2, int>(3, 0, start_mem_cells, end_mem_cells)
		);

#ifdef APPLY_DELTA_BIAS
	testing::outputToFile<weight_type>(this->GPUBias, "Bias4", "tests/prevbias2.txt");
	testing::outputToFile<weight_type>(this->GPUPreviousBias, "PrevBias4", "tests/prevbias2.txt");
#endif
}

//*********************
//Run The Network
//*********************


void LongTermShortTermNetwork::InitializeLongShortTermMemoryForRun(){
	//Form the delta objects
	this->CopyToDevice();
	this->moveBiasToGPU(false);//Don't create a previous_bias
	this->count_weights_in_layers();
}

device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in, run_type type){
	switch (type){
	case run_type::WITHOUT_MEMORY_CELLS:
		return this->runNetwork(in, 0, this->newSequence);
	case run_type::WITH_MEMORY_CELLS:
		//Pass the number of memory cells into the function so they can be skipped when finding the output
		return this->runNetwork(in, 1, this->newSequence);
	default:
		return this->runNetwork(in, 0, this->newSequence);
	}
}

thrust::device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in){
	return this->runNetwork(in,0,this->newSequence);
}

thrust::device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in, int number_of_extra_weights, bool &newSequence){
	if (newSequence){
		this->runNetwork(in, number_of_extra_weights);
		newSequence = false;
	}

	return this->runNetwork(in, number_of_extra_weights);
}

thrust::device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in,int number_of_extra_weights){

	this->setInput(in);
	//Stores the numberofmblocks in a layer
	unsigned int numberMBlocks;
	//Number mBlocks in previous layer
	unsigned int previousnumberMBlocks = 0;
	unsigned int numberBlocksToLayer = 0;
	device_vector<weight_type> toReturn = device_vector<weight_type>(this->settings.i_output);

	//unsigned int numberBias = 0;
	//Perform the transformation on each layer
	for (unsigned int i = 0; i < this->mBlocksLayers.size() - 1; i+=2){
		
		if (i != 0){
			previousnumberMBlocks += this->numberOfWeightsInLayers[i - 1] + this->numberOfWeightsInLayers[i-2] + number_of_extra_weights;
			numberBlocksToLayer += numberMBlocks;
		}
		numberMBlocks = this->mBlocksLayers[i].size();
		

		
#ifdef TRAININGTEST

		if (number_of_extra_weights != 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUPREV_START", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "GPU_START", "tests/PrevOut1.txt");
		}
		
#endif
		
		//Sum the values of the input/output/forget/potential_memory_cell_values nodes
		//The values in the GPU weights are in the order input, output, forget, memory cells
		//Subtracting this->mBlocksLayers[i].size() from the end will remove the memory cells from doing anything
		//Output to Previous
		thrust::reduce_by_key(
			this->GPUMapTo.begin() + previousnumberMBlocks,
			//Start at the beginning of the previous layer
			this->GPUMapTo.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i], // End at the the number of nodes before the current layer + the number of nodes in the current layer
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.begin() + previousnumberMBlocks,//Start from the beginning of the layer
			thrust::make_permutation_iterator(//Permute the output values such that they start at the correct position
			this->GPUOutput_values.begin(),
			this->GPUMapFrom.begin() + previousnumberMBlocks
			)
			)
			),
			functors::multiply_or_return_zero<weight_type,1>() //Multiply the two together
			),
			thrust::make_discard_iterator(),
			this->GPUPreviousOutput_Values.begin()
			);
#ifdef TRAININGTEST
		if (number_of_extra_weights != 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUPreBias1", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUPreBias2", "tests/PrevOut1.txt");
			testing::outputToFile<weight_type>(this->GPUMapTo.begin() + previousnumberMBlocks, this->numberOfWeightsInLayers[i], "Map0", "tests/map1.txt");
			
		}
		
#endif
		//Add Bias to the hidden layers
		thrust::transform(
			this->GPUBias.begin() + numberBlocksToLayer,
			this->GPUBias.begin() + numberBlocksToLayer + (numberMBlocks*4),
			this->GPUPreviousOutput_Values.begin(),
			this->GPUPreviousOutput_Values.begin(),
			_1 + _2
			);


#ifdef TRAININGTEST
		if (number_of_extra_weights != 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUPostBias", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUPostBias", "tests/PrevOut1.txt");
			testing::outputToFile<weight_type>(this->GPUBias.begin() + numberBlocksToLayer, (numberMBlocks * 4), "Map1", "tests/map1.txt");
		}
		
#endif
		//Create a input/output/forget/potential_memory_cell_values/memory_cell_value value
		//Essentially run the gate and get the output value
		thrust::for_each(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousOutput_Values.begin(), //input values
			this->GPUPreviousOutput_Values.begin() + numberMBlocks,//output values
			this->GPUPreviousOutput_Values.begin() + (2 * numberMBlocks),//forget values
			this->GPUPreviousOutput_Values.begin() + (3 * numberMBlocks),//potential_memory_cell_value
			this->GPUPreviousOutput_Values.begin() + (4 * numberMBlocks),
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 4),//Old Memory Cell Values
			this->GPUOutput_values.begin() + numberBlocksToLayer, //Old Input
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 2), //Old Forget
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 3), //Old Potential
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 1)
			)),
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousOutput_Values.begin() + numberMBlocks, //input values
			this->GPUPreviousOutput_Values.begin() + (2 * numberMBlocks),//output values
			this->GPUPreviousOutput_Values.begin() + (3 * numberMBlocks),//forget values
			this->GPUPreviousOutput_Values.begin() + (4 * numberMBlocks),//potential_memory_cell_value
			this->GPUPreviousOutput_Values.begin() + (5 * numberMBlocks),
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 5),//Memory Cell Values
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 1), //Old Input
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 3), //Old Forget
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 4), //Old Potential
			this->GPUOutput_values.begin() + numberBlocksToLayer + (numberMBlocks * 2)
			)),
			functors::run_memory_block_functon<weight_type>());
		
#ifdef TRAININGTEST
		if (number_of_extra_weights != 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUMid1", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUMid1", "tests/PrevOut1.txt");
			testing::outputToFile<weight_type>(this->GPUPreviousOutput_Values.begin() + numberMBlocks, numberMBlocks, "Map1", "tests/map1.txt");
		}
		
#endif
			thrust::reduce_by_key(
				this->GPUMapTo.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i],
				this->GPUMapTo.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i] + this->numberOfWeightsInLayers[i + 1],
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				this->GPUWeights.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i],
				thrust::make_permutation_iterator(
				this->GPUOutput_values.begin(),
				thrust::make_transform_iterator(
				this->GPUMapFrom.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i],
				_1
				)
				)
				)
				),
				functors::multiply<weight_type>()
				),
				thrust::make_discard_iterator(),
				this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5)
				);
		

#ifdef TRAININGTEST
		
		if (number_of_extra_weights != 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUOut", "tests/PrevOut1.txt");
			testing::outputToFile<weight_type>(this->GPUMapTo.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i], this->numberOfWeightsInLayers[i + 1], "Map2", "tests/map1.txt");
			testing::outputToFile<weight_type>(this->GPUMapFrom.begin() + previousnumberMBlocks + this->numberOfWeightsInLayers[i], this->numberOfWeightsInLayers[i + 1], "From1", "tests/map1.txt");
		}
		
#endif
		
		//Add the bias to the output
		thrust::transform(
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5),
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5) + this->settings.i_output,
			this->GPUBias.begin() + numberBlocksToLayer + (numberMBlocks * 5),
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5),
			_1 + _2
			);
		
#ifdef TRAININGTEST
		if (number_of_extra_weights != 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "outpostbias", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "outpostbias", "tests/PrevOut1.txt");
		}
		
#endif
		
		thrust::transform(
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5),
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5) + this->settings.i_output,
			this->GPUPreviousOutput_Values.begin() + numberBlocksToLayer + (numberMBlocks * 5),
			functors::sigmoid_functor<weight_type>()
			);
		
#ifdef TRAININGTEST
		
		if (number_of_extra_weights != 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "Final", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "Final", "tests/PrevOut1.txt");
		}
		

#endif
		
		thrust::copy(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.begin() + (numberMBlocks * 5) + this->settings.i_output, this->GPUOutput_values.begin() + this->numberNonWeights + numberBlocksToLayer);
		
		thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
		
#ifdef TRAININGTEST
		if (number_of_extra_weights != 0){
			testing::outputToFile(this->GPUPreviousOutput_Values, "GPUNew", "tests/PrevOut1.txt");
			testing::outputToFile(this->GPUOutput_values, "GPUNew", "tests/PrevOut1.txt");
		}
		
#endif
	}

	


	

	thrust::copy(this->GPUOutput_values.begin() + this->numberNonWeights + numberBlocksToLayer + this->numberOfNodes - this->settings.i_output, this->GPUOutput_values.begin() + this->numberNonWeights + numberBlocksToLayer + this->numberOfNodes, toReturn.begin());

	return toReturn;
}
