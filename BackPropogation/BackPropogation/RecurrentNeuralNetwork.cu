#include "hip/hip_runtime.h"
#include "RecurrentNeuralNetwork.cuh"

RecurrentNeuralNetwork::RecurrentNeuralNetwork(){
	this->settings = CSettings();
	RecurrentNeuralNetwork(this->settings);
}

RecurrentNeuralNetwork::RecurrentNeuralNetwork(CSettings& settings){
	this->settings = settings;
	this->initialize_network();
}

void RecurrentNeuralNetwork::initialize_network(){
	this->weights = host_vector<weight_type>();
	this->mapTo = host_vector<int>();
	this->mapFrom = host_vector<int>();
	this->output_values = host_vector<weight_type>();
	positionOfLastWeightToNode = vector<long>();
	//The initial size of the network will consist of only the input and output layers
	//This is an approximation of a feedforward network
	//Add the total number of nodes in the input layer as the first set of nodes
	//This set will change when running the system
	for (int i = 0; i < this->settings.i_input; i++){
		//Set the first output_values as the input
		this->output_values.push_back(0);
	}

	//Create a container for current value of each node
	for (int i = 0; i < this->settings.i_output; i++){
		this->output_values.push_back(0);
	}
	this->numberNonWeights = this->settings.i_input + this->settings.i_output;

	//Seed the random
	srand((unsigned)(time(NULL)));

	//This set will also change over time and contains all the output values
	for (int i = 0; i < this->settings.i_output; i++){
		if (i > 0){
			this->positionOfLastWeightToNode.push_back(this->positionOfLastWeightToNode[i - 1]);
		}else{
			this->positionOfLastWeightToNode.push_back(-1);
		}
		
		//Map a weight from each of the input to each of the outputs
		for (int j = 0; j < this->settings.i_input; j++){
			//Each one starts with a random value as the output weight
			this->weights.push_back(RandomClamped());
			this->mapFrom.push_back(j);
			this->mapTo.push_back(i);
			//Increment the position of the weights
			this->positionOfLastWeightToNode[i]++;
		}
	}
	this->numberOfNodes = this->settings.i_output;
	this->input_weights = this->weights.size();
}


//***************************
//Modify Structure Of Neuron
//***************************
int RecurrentNeuralNetwork::decideNodeToAttachTo(){
	vector<int> notFullyConnected = vector<int>();
	//Find how many nodes are not fully connected
	for (int k = this->settings.i_output; k<this->numberOfNodes; k++){
		if (this->positionOfLastWeightToNode[k] - (k > this->settings.i_output ? this->positionOfLastWeightToNode[k-1] : -1) < (this->numberOfNodes + this->settings.i_input - this->settings.i_output)){
			notFullyConnected.push_back(k);
		}
	}

	if (notFullyConnected.size() > 0){
		//Return a random number in the set of not fully connected nodes
		//It's the number contained in the positionOfLastWeightToNode which is this->settings.i_input less than its actual position
		return notFullyConnected[RandInt(0, notFullyConnected.size()-1)];
	}
	else{
		//All nodes are fully connected and no new weights can be added
		return -1;
	}
}

int RecurrentNeuralNetwork::decideNodeToAttachFrom(int attachTo){
	vector<int> notConnectedTo = vector<int>();
	bool containsValue = false;
	int start = (this->settings.i_output != attachTo ? this->positionOfLastWeightToNode[attachTo-1] : 0);
	int end = (this->settings.i_output != attachTo ? this->positionOfLastWeightToNode[attachTo] : this->positionOfLastWeightToNode[attachTo]);
	for (unsigned int k = this->numberNonWeights; k < this->output_values.size(); k++){
		for (int i = start; i<=end; i++){
			if (this->mapFrom[i] == k){
				//The value is already contained in the system
				containsValue = true;
				break;
			}
		}

		if (!containsValue){
			notConnectedTo.push_back(k);
		}
		containsValue = false;


	}


	
	if (notConnectedTo.size() != 0){
		//It's the number contained in the positionOfLastWeightToNode which is this->settings.i_input less than its actual position
		return notConnectedTo[RandInt(0, notConnectedTo.size() - 1)];
	}
	else{
		return -1;
	}

}





weight_type RecurrentNeuralNetwork::getNewWeight(){
	return RandomClamped();
}



void RecurrentNeuralNetwork::addWeight(int numberWeightsToAdd){
	int decideTo;//Where the input should go to
	int decideFrom;//Where the input should come from
	int positionToAdd;//The node to add in positionOfLastWeightToNode
	host_vector<weight_type>::iterator itWeights;
	host_vector<int>::iterator itMapFrom;
	host_vector<int>::iterator itMapTo;
	for (int i = 0; i < numberWeightsToAdd; i++){
		//A new weight should be added between existing neurons
		//Add one weight from any current node to the new nodes
		decideTo = this->decideNodeToAttachTo();
		if (decideTo != -1){//At least one node isn't completely connected to every other node
			decideFrom = this->decideNodeToAttachFrom(decideTo);
			if (decideFrom != -1){//The node which was chosen has at least one node not connected to it
				positionToAdd = (this->positionOfLastWeightToNode[decideTo]) + 1;
				decideTo += this->settings.i_input;
				itWeights = this->weights.begin() + positionToAdd;
				itMapFrom = this->mapFrom.begin() + positionToAdd;
				itMapTo = this->mapTo.begin() + positionToAdd;

				//Add the new weight into the position
				this->weights.insert(itWeights, this->getNewWeight());
				this->mapTo.insert(itMapTo, decideTo);
				this->mapFrom.insert(itMapFrom, decideFrom);

				//Increment the position for any following the current node
				for (unsigned int j = decideTo - this->settings.i_input; j < this->positionOfLastWeightToNode.size(); j++){
					this->positionOfLastWeightToNode[j] += 1;
				}
				this->input_weights++;
			}
		}

	}

	//Add the new weight positions to the output weight list
	for (int i = 0; i < this->settings.i_output; i++){
		this->positionOfLastWeightToNode[i] += numberWeightsToAdd;
	}
}

void RecurrentNeuralNetwork::addNeuron(int numberNeuronsToAdd){
	int addNewNeuron = 0;//Count the number of neurons to add. Multiple insertions at one time are easier than a single insertion

	//Add the new nodes defined by the numberofNodesToAdd
	for (int i = 0; i < numberNeuronsToAdd; i++){
		if (this->numberOfNodes == this->settings.i_output){
			//There are currently only the input/output nodes
			//In order to not need to delete any weights (which would cost quite a bit of time, we add in X new nodes, where X is the number of nodes in the output
			for (int j = 0; j < this->settings.i_output; j++){
				//Add the new node
				this->output_values.push_back(0);
				
				
				for (int k = (this->settings.i_input*j); k < ((this->settings.i_input*j)) + this->settings.i_input; k++){
					//We need to move all those inputs/outputs weights to the single new nodes
					//Move all weights between the current input/output pair to be input -> new node -> output
					this->mapTo[k] = this->output_values.size() - 1;
				}
				this->positionOfLastWeightToNode.push_back(this->positionOfLastWeightToNode[j]);
				
			}
			//Add new weights from the new nodes to the output nodes
			for (int j = this->settings.i_input; j < this->numberNonWeights; j++){
				for (unsigned int k = this->numberNonWeights; k < this->output_values.size(); k++){
					//Create a new weight from the current node to the weight
					//Create a new weight
					this->weights.push_back(RandomClamped());
					//Set a new pointer from the one new node to each of the output nodes
					this->mapFrom.push_back(k);
					//Map the new weights to the output
					this->mapTo.push_back(j);
				}
				this->positionOfLastWeightToNode[j - this->settings.i_input] = this->weights.size() - 1;
			}
			this->numberOfNodes += this->settings.i_output;

		}
		else if (true){
			//A new neuron is added
			addNewNeuron++;
		}



	}

	if (addNewNeuron > 0){
		host_vector<weight_type>::iterator it = this->weights.begin() + this->input_weights;
		host_vector<int>::iterator itInt = this->mapFrom.begin() + this->input_weights;
		host_vector<int>::iterator itInt2 = this->mapTo.begin() + this->input_weights;
		int output_size = this->output_values.size();
		int total_nodes_weights_before_output_added = 0;//Count the number of weights which are added before the to output nodes are found
		//Insert any new Neurons which were chosen to be created
		//Create connection from input to new node
		for (int i = addNewNeuron-1; i > -1; i--){
			//Add the new neuron
			this->output_values.push_back(0);
			for (int j = this->settings.i_input - 1; j > -1; j--){
				//Insert the connections to the input
				it = this->weights.insert(it, getNewWeight());
				itInt = this->mapFrom.insert(itInt, j);
				itInt2 = this->mapTo.insert(itInt2,  output_size + i);
				this->input_weights++;//Increase input_weights end position
				total_nodes_weights_before_output_added++;//Increment the number of weights added
			}

			this->positionOfLastWeightToNode.push_back(this->input_weights - 1);
			
		}


		it = this->weights.begin() + this->input_weights + this->numberOfNodes - this->settings.i_output;
		itInt = this->mapFrom.begin() + this->input_weights + this->numberOfNodes - this->settings.i_output;
		itInt2 = this->mapTo.begin() + this->input_weights + this->numberOfNodes - this->settings.i_output;
		//Create connection from new node to output node
		for (int i = 0; i < this->settings.i_output; i++){
			for (int j = addNewNeuron - 1; j > -1; j--){
				//Insert a connection to the output
				it = this->weights.insert(it, getNewWeight());
				itInt = this->mapFrom.insert(itInt, j + this->numberOfNodes + this->settings.i_input);
				itInt2 = this->mapTo.insert(itInt2, i + this->settings.i_input);
			}
			it += addNewNeuron + numberOfNodes - this->settings.i_output;
			itInt += addNewNeuron + numberOfNodes - this->settings.i_output;
			itInt2 += addNewNeuron + numberOfNodes - this->settings.i_output;
		}
		this->numberOfNodes += addNewNeuron;
		this->positionOfLastWeightToNode[0] = this->input_weights + (this->numberOfNodes - this->settings.i_output-1);
		//Increment the stored position of the last weight
		for (int i = 1; i < this->settings.i_output; i++){
			this->positionOfLastWeightToNode[i] = this->positionOfLastWeightToNode[i-1] + this->numberOfNodes - this->settings.i_output;
		}
	}


}



//*********************
//Run The Network
//*********************
//Multiply two values
template <typename T>
struct multiply : public thrust::unary_function < T, T > {

	//Overload the function operator
	template <typename Tuple>
	__host__ __device__
		T operator()(Tuple &x) const{
		return (thrust::get<0>(x) * thrust::get<1>(x));
	}

};

//Perform a sigmoid function
template <typename T>
struct sigmoid_functor : public thrust::unary_function < T, T > {
	sigmoid_functor(){};

	__host__ __device__
		T operator()(const T &x) const{
		T z = thrust::exp(((T) -1) * x);
		return (T)1 / ((T)1 + z);
	}

};

thrust::device_vector<weight_type> RecurrentNeuralNetwork::runNetwork(weight_type* in){
	//Sum all the input values
	device_vector<weight_type> GPUOutput_values = this->output_values;//Copy the output_nodes
	device_vector<weight_type> GPUPreviousOutput_Values = this->output_values;
	device_vector<int> GPUMapFrom = this->mapFrom;//Copy the map from
	device_vector<int> GPUMapTo = this->mapTo; //Copy the mapTo
	device_vector<weight_type> GPUWeights = this->weights;

	//Copy the input into the GPU memory
	for (int i = 0; i < this->settings.i_input; i++){
		GPUOutput_values[i] = (weight_type)in[i];
	}
	this->sumNetworkValues(GPUOutput_values,//Copy the output_nodes
		GPUPreviousOutput_Values,
		GPUMapFrom,//Copy the map from
		GPUMapTo, //Copy the mapTo
		GPUWeights, 1);

	//Free the used memory
	clear_vector::free(GPUMapFrom);
	clear_vector::free(GPUMapTo);
	clear_vector::free(GPUWeights);
	//Return either of these two as the output
	if (this->settings.i_recurrent_flip_flop % 2 == 0){
		clear_vector::free(GPUOutput_values);
		return GPUPreviousOutput_Values;
	}
	else{
		clear_vector::free(GPUPreviousOutput_Values);
		return GPUOutput_values;
	}

}
//*********************
//Training the Network
//*********************
template <typename T>
struct find_error : public thrust::unary_function < T, T > {

	//Overload the function operator
	template <typename Tuple>
	__host__ __device__
		T operator()(Tuple &x) const{
		return thrust::pow((thrust::get<0>(x) - thrust::get<1>(x)),(T)2);
	}

};


void RecurrentNeuralNetwork::LongShortTermMemoryTraining(device_vector<weight_type> in, weight_type* out){

}

void RecurrentNeuralNetwork::InitializeRealTimeRecurrentTraining(){
	//Store all the values in the device
	//Will later add option for too little memory
	this->host_deltas = host_vector<weight_type>(this->weights.size());
	//Copy the information to the device
	this->CopyToDevice();
	this->total_error = 0;
}

void RecurrentNeuralNetwork::RealTimeRecurrentLearningTraining(weight_type* in, weight_type* out){
	this->total_error = this->RealTimeRecurrentLearningTraining(in, out, this->total_error, this->GPUMapTo, this->GPUMapFrom, this->GPUWeights, this->GPUOutput_values, this->GPUPreviousOutput_Values,
		this->device_deltas);
}

//Incomplete
//Need more info about how it works
weight_type RecurrentNeuralNetwork::RealTimeRecurrentLearningTraining(
	weight_type* in, 
	weight_type* out, 
	weight_type total_error,
	thrust::device_vector<int> &GPUMapTo,
	thrust::device_vector<int> &GPUMapFrom, 
	thrust::device_vector<weight_type> &GPUWeights, 
	thrust::device_vector<weight_type> &GPUOutput_values, 
	thrust::device_vector<weight_type> &GPUPreviousOutput_Values,
	thrust::device_vector<weight_type> &GPU_Deltas){
	
	device_vector<weight_type> output = device_vector<weight_type>(this->settings.i_output);
	
	//Copy the desired output into GPU memory
	for (int i = 0; i < this->settings.i_output; i++){
		output[i] = (weight_type)out[i];
	}

	//Copy the input into the GPU memory
	for (int i = 0; i < this->settings.i_input; i++){
		GPUOutput_values[i] = (weight_type)in[i];
	}

	
	//The GPU Sum is now set
	this->sumNetworkValues(GPUOutput_values,//Copy the output_nodes
		GPUOutput_values,
		GPUMapFrom,//Copy the map from
		GPUMapTo, //Copy the mapTo
		GPUWeights, 1);
	
	//Get the sum of the error
	weight_type current_total = thrust::reduce(thrust::make_transform_iterator(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		GPUOutput_values.begin() + this->settings.i_input,
		output.begin())),
		find_error<weight_type>()
		),
		thrust::make_transform_iterator(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		GPUOutput_values.begin() + this->settings.i_input + this->settings.i_output,
		output.end())),
		find_error<weight_type>()
		)
		);	

	thrust::fill(GPUPreviousOutput_Values.begin(), GPUPreviousOutput_Values.end(), (weight_type) 0);
	thrust::transform(GPUOutput_values.begin(), GPUOutput_values.begin() + this->settings.i_output, output.begin(), GPUPreviousOutput_Values.begin(), _2/_1);

	//Add the new change to the deltas to the current delta
	thrust::transform(GPUWeights.begin(), GPUWeights.end(),
		GPU_Deltas.begin(),GPU_Deltas.begin(),
		((((weight_type)(this->settings.d_alpha)) * current_total) / _1));
	weight_type temp = thrust::reduce(GPU_Deltas.begin(), GPU_Deltas.end());
	return current_total;

}

void RecurrentNeuralNetwork::RealTimeRecurrentLearningApplyError(){

	thrust::transform(this->GPUWeights.begin(), this->GPUWeights.end(), this->device_deltas.begin(), this->GPUWeights.begin(), _1 + _2);

}

//*********************
//Hessian Free
//*********************

void RecurrentNeuralNetwork::InitializeHessianFreeOptimizationTraining(){

	//Store all the values in the device
	//Will later add option for too little memory
	this->host_deltas = host_vector<weight_type>(this->GPUOutput_values.size());
	//Copy the information to the device
	this->CopyToDevice();
	//Fill the intial previous output as 0
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
}
void RecurrentNeuralNetwork::HessianFreeOptimizationTraining(weight_type* in, weight_type* out){

	device_vector<weight_type> output = device_vector<weight_type>(this->settings.i_output);
	device_vector<weight_type> GPUOutput_Values_Copy(this->GPUOutput_values);//Contains a copy of the current GPU weights
	//Copy the desired output into GPU memory
	for (int i = 0; i < this->settings.i_output; i++){
		output[i] = (weight_type)out[i];
	}

	//Copy the input into the GPU memory
	for (int i = 0; i < this->settings.i_input; i++){
		this->GPUOutput_values[i] = (weight_type)in[i];
	}
	


	//The GPU Sum is now set
	this->sumNetworkValues(this->GPUOutput_values, 
		GPUOutput_Values_Copy,//Copy the output_nodes
		this->GPUMapFrom,//Copy the map from
		this->GPUMapTo, //Copy the mapTo
		this->GPUWeights, 1);

	thrust::transform(this->GPUOutput_values.begin() + this->settings.i_input, GPUOutput_values.end(), this->GPUPreviousOutput_Values.begin() + this->settings.i_input, this->device_deltas.begin(), (_1 - _2) / (weight_type)this->settings.d_alpha);

	//Get the next iteration of values
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
	//Difference between target and current value
	//Store the current round of values into the GPUPreviousOutput_Values
	//Essentailly delta f(x_n)
	thrust::transform(this->GPUOutput_values.begin() + this->settings.i_input, this->GPUOutput_values.begin() + this->settings.i_input + this->settings.i_output, output.begin(), this->GPUPreviousOutput_Values.begin(), _2 - _1);

	clear_vector::free(output);
	clear_vector::free(output);
}
void RecurrentNeuralNetwork::HessianFreeOptimizationApplyError(){
	//Apply the found delta to all of the values 
	//Essentially, add the delta
	thrust::transform(this->GPUWeights.begin(), this->GPUWeights.end(), 
		thrust::make_permutation_iterator(this->device_deltas.begin(),this->GPUMapFrom.begin()), this->GPUWeights.begin(), _1 + _2);
}

//*********************
//Perform Functionality
//*********************

void RecurrentNeuralNetwork::sumNetworkValues(device_vector<weight_type> &GPUOutput_values,//Copy the output_nodes
	device_vector<weight_type> &GPUPreviousOutput_Values,
	device_vector<int> &GPUMapFrom,//Copy the map from
	device_vector<int> &GPUMapTo, //Copy the mapTo
	device_vector<weight_type> &GPUWeights, int number_of_rounds){

	for (int i = 0; i < number_of_rounds; i++){
		if (i % 2 != 0){//Store the results in the previous Output
			//Reduce the input into the sum for each neuron
			thrust::reduce_by_key(
				GPUMapTo.begin(),
				GPUMapTo.end(),
				//Transform by multiplying the weight by the previous output
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				GPUWeights.begin(),
				make_permutation_iterator(
				GPUOutput_values.begin(),
				GPUMapFrom.begin()
				))
				),
				multiply<weight_type>()),
				thrust::make_discard_iterator(),
				GPUPreviousOutput_Values.begin()
				);
			//Transform the output using the sigmoid function
			thrust::transform(GPUPreviousOutput_Values.begin(), GPUPreviousOutput_Values.end(), GPUPreviousOutput_Values.begin(), sigmoid_functor<weight_type>());
		}
		else{//Store in current output
			//Reduce the input into the sum for each neuron
			thrust::reduce_by_key(
				GPUMapTo.begin(),
				GPUMapTo.end(),
				//Transform by multiplying the weight by the previous output
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(GPUPreviousOutput_Values.begin(),
				make_permutation_iterator(
				GPUOutput_values.begin(),
				GPUMapFrom.begin()
				))
				),
				multiply<weight_type>()),
				thrust::make_discard_iterator(),
				GPUOutput_values.begin()
				);

			//Transform the output using the sigmoid function
			thrust::transform(GPUOutput_values.begin(), GPUOutput_values.end(), GPUOutput_values.begin(), sigmoid_functor<weight_type>());
		}
	}

}

void RecurrentNeuralNetwork::ResetSequence(){
	thrust::fill(this->GPUOutput_values.begin(),this->GPUOutput_values.end(), 0);
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), 0);
}


void RecurrentNeuralNetwork::CopyToHost(){
	//Copy the device memory to local
	thrust::copy(this->GPUMapFrom.begin(), this->GPUMapFrom.end(), this->mapFrom.begin());
	thrust::copy(this->GPUMapTo.begin(), this->GPUMapTo.end(), this->mapTo.begin());
	thrust::copy(this->GPUWeights.begin(), this->GPUWeights.end(), this->weights.begin());
	thrust::copy(this->device_deltas.begin(), this->device_deltas.end(), this->host_deltas.begin());
	thrust::copy(this->device_deltas.begin(), this->device_deltas.end(), this->host_deltas.begin());
	thrust::copy(this->GPUOutput_values.begin(), this->GPUOutput_values.end(), this->output_values.begin());
}

void RecurrentNeuralNetwork::CopyToDevice(){
	this->device_deltas = this->host_deltas;
	this->GPUMapTo = this->mapTo;
	this->GPUMapFrom = this->mapFrom;
	this->GPUOutput_values = this->output_values;
	this->GPUPreviousOutput_Values = this->output_values;
	this->GPUWeights = this->weights;
}

void  RecurrentNeuralNetwork::cleanNetwork(){
	this->CopyToHost();
	//Free the used memory
	clear_vector::free(this->GPUMapFrom);
	clear_vector::free(this->GPUMapTo);
	clear_vector::free(this->GPUWeights);
	clear_vector::free(this->device_deltas);
	clear_vector::free(this->GPUOutput_values);
	clear_vector::free(this->GPUPreviousOutput_Values);
}

void  RecurrentNeuralNetwork::emptyGPUMemory(){

}
//*********************
//Misc
//*********************
ostream& RecurrentNeuralNetwork::OutputNetwork(ostream &os){
	return os;
}

void RecurrentNeuralNetwork::VisualizeNetwork(){
	cout.precision(20);
	std::cout << "Weight" << "\t" << "In" << "\t" << "Out" << endl;
	for (unsigned int i = 0; i < this->weights.size(); i++){
		std::cout << i << ") " << this->weights[i] << "\t" << this->mapFrom[i] << "\t" << this->mapTo[i] << endl;
	}
	std::cout << endl;
	cout << "Neuron Values" << endl;
	
	for (unsigned int i = this->settings.i_input; i < this->output_values.size(); i++){
		std::cout << i << ") " << this->output_values[i] << endl;
	}

	std::cout << endl;

	std::cout << endl;
	cout << "deltas" << endl;
	for (unsigned int i = 0; i < this->host_deltas.size(); i++){
		std::cout << i << ") " << this->host_deltas[i] << endl;
	}



}