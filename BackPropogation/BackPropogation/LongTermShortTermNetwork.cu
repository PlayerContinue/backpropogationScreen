#include "hip/hip_runtime.h"
#include "LongTermShortTermNetwork.cuh"

LongTermShortTermNetwork::LongTermShortTermNetwork(){
	this->settings = CSettings();
	LongTermShortTermNetwork(this->settings);
}

LongTermShortTermNetwork::LongTermShortTermNetwork(CSettings& settings){
	this->settings = settings;
	this->initialize_network();
}

//***************************
//Initialize Network
//***************************

void LongTermShortTermNetwork::initialize_network(){
	this->weights = host_vector<weight_type>();
	this->mapTo = host_vector<int>();
	this->mapFrom = host_vector<int>();
	this->bias = host_vector<weight_type>();

	//Initialize the position
	this->weight_position = vector<thrust::host_vector<weight_type>::iterator>();
	this->mapTo_position = vector<thrust::host_vector<int>::iterator>();
	this->mapFrom_position = vector<thrust::host_vector<int>::iterator>();

	positionOfLastWeightToNode = vector<long>();

	for (int i = 0; i < this->settings.i_input; i++){
		this->bias.push_back(0);
	}

	

	//Initially only create on memory block connected to all input/output
	this->InitialcreateMemoryBlock(1);
	for (int i = 0; i < this->settings.i_output; i++){
		this->addNewPositionInList();
		this->addNewNeuron(this->bias.size() - this->settings.i_input, this->bias.size() - this->settings.i_input, 1, this->last_output_cell_pos, this->bias.size() - this->settings.i_input);
		this->bias.push_back(0);
	}

	this->createMemoryBlock(1);

}


//***************************
//Modify Structure Of Neuron
//***************************
int LongTermShortTermNetwork::decideNodeToAttachTo(){
	vector<int> notFullyConnected = vector<int>();
	//Find how many nodes are not fully connected
	for (int k = this->settings.i_output; k<this->numberOfNodes; k++){
		if (this->positionOfLastWeightToNode[k] - (k > this->settings.i_output ? this->positionOfLastWeightToNode[k - 1] : -1) < (this->numberOfNodes + this->settings.i_input - this->settings.i_output)){
			notFullyConnected.push_back(k);
		}
	}

	if (notFullyConnected.size() > 0){
		//Return a random number in the set of not fully connected nodes
		//It's the number contained in the positionOfLastWeightToNode which is this->settings.i_input less than its actual position
		return notFullyConnected[RandInt(0, notFullyConnected.size() - 1)];
	}
	else{
		//All nodes are fully connected and no new weights can be added
		return -1;
	}
}

int LongTermShortTermNetwork::decideNodeToAttachFrom(int attachTo){
	vector<int> notConnectedTo = vector<int>();
	bool containsValue = false;
	int start = (this->settings.i_output != attachTo ? this->positionOfLastWeightToNode[attachTo - 1] : 0);
	int end = (this->settings.i_output != attachTo ? this->positionOfLastWeightToNode[attachTo] : this->positionOfLastWeightToNode[attachTo]);
	for (int k = this->numberNonWeights; k < this->bias.size(); k++){
		for (int i = start; i <= end; i++){
			if (this->mapFrom[i] == k){
				//The value is already contained in the system
				containsValue = true;
				break;
			}
		}

		if (!containsValue){
			notConnectedTo.push_back(k);
		}
		containsValue = false;


	}



	if (notConnectedTo.size() != 0){
		//It's the number contained in the positionOfLastWeightToNode which is this->settings.i_input less than its actual position
		return notConnectedTo[RandInt(0, notConnectedTo.size() - 1)];
	}
	else{
		return -1;
	}

}





weight_type LongTermShortTermNetwork::getNewWeight(){
	return RandomClamped();
}



void LongTermShortTermNetwork::addWeight(int numberWeightsToAdd){
}

void LongTermShortTermNetwork::addNeuron(int numberNeuronsToAdd){
	
}

void LongTermShortTermNetwork::addNewNeuron(int store,int position, weight_type weight, int mapFrom, int mapTo){
	this->weight_position[store] = this->weights.insert(this->weight_position[position], weight) + 1;
	//Push back the node the values is coming from
	this->mapFrom_position[store] = this->mapFrom.insert(this->mapFrom_position[position], mapFrom) + 1;
	this->mapTo_position[store] = this->mapTo.insert(this->mapTo_position[position], mapTo) + 1;
}

void LongTermShortTermNetwork::addNewPositionInList(){
	this->mapTo_position.push_back(this->mapTo.begin() + this->settings.i_input - 1);
	this->mapFrom_position.push_back(this->mapFrom.begin() + this->settings.i_input - 1);
	this->weight_position.push_back(this->weights.begin() + this->settings.i_input - 1);
}

void LongTermShortTermNetwork::createMemoryBlock(int numberMemoryCells){
	//Create the input block
	//Create the input block
	//Create the input Lock

	this->addNewPositionInList();
	//Add the weight for the inputs
	for (int i = 0; i < this->settings.i_input; i++){
		this->addNewNeuron(this->last_input_cell_pos, this->last_input_cell_pos-1, this->getNewWeight(), i, this->last_input_cell_pos);
	}
	
	this->bias.push_back(this->getNewWeight());

	this->weight_position[this->last_input_cell_pos]++;
	//Push back the node the values is coming from
	this->mapFrom_position[this->last_input_cell_pos]++;
	this->mapTo_position[this->last_input_cell_pos]++;

	this->last_input_cell_pos++;


	//Create the memory cell
	for (int i = 0; i < numberMemoryCells; i++){
		this->addNewPositionInList();
		//Add connection to input
		this->addNewNeuron(this->last_memory_cell_pos, this->last_memory_cell_pos - 1, 1, this->last_input_cell_pos - 1, this->last_memory_cell_pos);
		//Add conection to itself
		this->addNewNeuron(this->last_memory_cell_pos,this->last_memory_cell_pos-1, 1, this->last_input_cell_pos - 1, this->last_memory_cell_pos);

		
		this->bias.push_back(this->getNewWeight());
		
		this->last_memory_cell_pos++;
	}

	this->addNewPositionInList();
	//Add connections from memory cells to node
	for (int j = 0; j < numberMemoryCells; j++){
		this->addNewNeuron(this->last_output_cell_pos, this->last_output_cell_pos - 1, 1, this->last_memory_cell_pos - j, this->last_memory_cell_pos);
	}

	//Add connections from input to the output nodes
	for (int i = this->settings.i_input - 1; i < this->settings.i_input + this->settings.i_output - 1; i++){
		this->addNewNeuron(this->last_output_cell_pos, this->last_output_cell_pos - 1, this->getNewWeight(), i, this->last_output_cell_pos - 1);
	}
	
	//Add conections to output
	for (int i = 0; i < this->settings.i_output; i++){
		this->addNewNeuron(this->bias.size() - 1 - i - this->settings.i_input, this->bias.size() - 1 - this->settings.i_input - i, 1, this->weight_position.size() + this->settings.i_input, this->bias.size() - this->settings.i_input - 1 - i);
	}

	this->last_output_cell_pos++;
	this->bias.push_back(this->getNewWeight());

	

	//Increment number of memory blocks
	this->numberOfNodes++;
}

void LongTermShortTermNetwork::InitialcreateMemoryBlock(int numberMemoryCells){
	


}



//*********************
//Run The Network
//*********************
//Multiply two values
template <typename T>
struct multiply : public thrust::unary_function < T, T > {

	//Overload the function operator
	template <typename Tuple>
	__host__ __device__
		T operator()(Tuple &x) const{
		return (thrust::get<0>(x) * thrust::get<1>(x));
	}

};

//Perform a sigmoid function
template <typename T>
struct sigmoid_functor : public thrust::unary_function < T, T > {
	sigmoid_functor(){};

	__host__ __device__
		T operator()(const T &x) const{
		T z = thrust::exp(((T)-1) * x);
		return (T)1 / ((T)1 + z);
	}

};

thrust::device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in){
	//Sum all the input values
	device_vector<weight_type> GPUOutput_values = this->bias;//Copy the output_nodes
	device_vector<weight_type> GPUPreviousOutput_Values = this->bias;
	device_vector<int> GPUMapFrom = this->mapFrom;//Copy the map from
	device_vector<int> GPUMapTo = this->mapTo; //Copy the mapTo
	device_vector<weight_type> GPUWeights = this->weights;

	//Copy the input into the GPU memory
	for (int i = 0; i < this->settings.i_input; i++){
		GPUOutput_values[i] = (weight_type)in[i];
	}
	this->sumNetworkValues(GPUOutput_values,//Copy the output_nodes
		GPUPreviousOutput_Values,
		GPUMapFrom,//Copy the map from
		GPUMapTo, //Copy the mapTo
		GPUWeights, 1);

	//Free the used memory
	clear_vector::free(GPUMapFrom);
	clear_vector::free(GPUMapTo);
	clear_vector::free(GPUWeights);
	//Return either of these two as the output
	if (this->settings.i_recurrent_flip_flop % 2 == 0){
		clear_vector::free(GPUOutput_values);
		return GPUPreviousOutput_Values;
	}
	else{
		clear_vector::free(GPUPreviousOutput_Values);
		return GPUOutput_values;
	}

}
//*********************
//Training the Network
//*********************
template <typename T>
struct find_error : public thrust::unary_function < T, T > {

	//Overload the function operator
	template <typename Tuple>
	__host__ __device__
		T operator()(Tuple &x) const{
		return thrust::pow((thrust::get<0>(x) -thrust::get<1>(x)), (T)2);
	}

};


//*********************
//Hessian Free
//*********************

void LongTermShortTermNetwork::InitializeLongShortTermMemory(){

	//Store all the values in the device
	//Will later add option for too little memory
	this->host_deltas = host_vector<weight_type>(this->GPUOutput_values.size());
	//Copy the information to the device
	this->CopyToDevice();
	//Fill the intial previous output as 0
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
}
void LongTermShortTermNetwork::LongTermShortTermNetwork::LongShortTermMemoryTraining(weight_type* in, weight_type* out){

	
}
void LongTermShortTermNetwork::ApplyLongTermShortTermMemoryError(){
	
}

//*********************
//Perform Functionality
//*********************

void LongTermShortTermNetwork::sumNetworkValues(device_vector<weight_type> &GPUOutput_values,//Copy the output_nodes
	device_vector<weight_type> &GPUPreviousOutput_Values,
	device_vector<int> &GPUMapFrom,//Copy the map from
	device_vector<int> &GPUMapTo, //Copy the mapTo
	device_vector<weight_type> &GPUWeights, int number_of_rounds){

	for (int i = 0; i < number_of_rounds; i++){
		if (i % 2 != 0){//Store the results in the previous Output
			//Reduce the input into the sum for each neuron
			thrust::reduce_by_key(
				GPUMapTo.begin(),
				GPUMapTo.end(),
				//Transform by multiplying the weight by the previous output
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(
				GPUWeights.begin(),
				make_permutation_iterator(
				GPUOutput_values.begin(),
				GPUMapFrom.begin()
				))
				),
				multiply<weight_type>()),
				thrust::make_discard_iterator(),
				GPUPreviousOutput_Values.begin()
				);
			//Transform the output using the sigmoid function
			thrust::transform(GPUPreviousOutput_Values.begin(), GPUPreviousOutput_Values.end(), GPUPreviousOutput_Values.begin(), sigmoid_functor<weight_type>());
		}
		else{//Store in current output
			//Reduce the input into the sum for each neuron
			thrust::reduce_by_key(
				GPUMapTo.begin(),
				GPUMapTo.end(),
				//Transform by multiplying the weight by the previous output
				thrust::make_transform_iterator(
				thrust::make_zip_iterator(
				thrust::make_tuple(GPUPreviousOutput_Values.begin(),
				make_permutation_iterator(
				GPUOutput_values.begin(),
				GPUMapFrom.begin()
				))
				),
				multiply<weight_type>()),
				thrust::make_discard_iterator(),
				GPUOutput_values.begin()
				);

			//Transform the output using the sigmoid function
			thrust::transform(GPUOutput_values.begin(), GPUOutput_values.end(), GPUOutput_values.begin(), sigmoid_functor<weight_type>());
		}
	}

}

void LongTermShortTermNetwork::ResetSequence(){
	thrust::fill(this->GPUOutput_values.begin(), this->GPUOutput_values.end(), 0);
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), 0);
}


void LongTermShortTermNetwork::CopyToHost(){
	//Copy the device memory to local
	thrust::copy(this->GPUMapFrom.begin(), this->GPUMapFrom.end(), this->mapFrom.begin());
	thrust::copy(this->GPUMapTo.begin(), this->GPUMapTo.end(), this->mapTo.begin());
	thrust::copy(this->GPUWeights.begin(), this->GPUWeights.end(), this->weights.begin());
	thrust::copy(this->device_deltas.begin(), this->device_deltas.end(), this->host_deltas.begin());
	thrust::copy(this->device_deltas.begin(), this->device_deltas.end(), this->host_deltas.begin());
	thrust::copy(this->GPUOutput_values.begin(), this->GPUOutput_values.end(), this->bias.begin());
}

void LongTermShortTermNetwork::CopyToDevice(){
	this->device_deltas = this->host_deltas;
	this->GPUMapTo = this->mapTo;
	this->GPUMapFrom = this->mapFrom;
	this->GPUOutput_values = this->bias;
	this->GPUPreviousOutput_Values = this->bias;
	this->GPUWeights = this->weights;
}

void  LongTermShortTermNetwork::cleanNetwork(){
	this->CopyToHost();
	//Free the used memory
	clear_vector::free(this->GPUMapFrom);
	clear_vector::free(this->GPUMapTo);
	clear_vector::free(this->GPUWeights);
	clear_vector::free(this->device_deltas);
	clear_vector::free(this->GPUOutput_values);
	clear_vector::free(this->GPUPreviousOutput_Values);
}
//*********************
//Misc
//*********************
void LongTermShortTermNetwork::VisualizeNetwork(){
	cout.precision(20);
	std::cout << "Weight" << "\t" << "In" << "\t" << "Out" << endl;
	for (int i = 0; i < this->weights.size(); i++){
		std::cout << i << ") " << this->weights[i] << "\t" << this->mapFrom[i] << "\t" << this->mapTo[i] << endl;
	}
	std::cout << endl;
	cout << "Neuron Values" << endl;

	for (int i = this->settings.i_input; i < this->bias.size(); i++){
		std::cout << i << ") " << this->bias[i] << endl;
	}

	std::cout << endl;

	std::cout << endl;
	cout << "deltas" << endl;
	for (int i = 0; i < this->host_deltas.size(); i++){
		std::cout << i << ") " << this->host_deltas[i] << endl;
	}



}