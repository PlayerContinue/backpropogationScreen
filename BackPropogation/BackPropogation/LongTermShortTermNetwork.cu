#include "hip/hip_runtime.h"
#include "LongTermShortTermNetwork.cuh"
#define NUMBER_NODES_IN_MEMORY_CELL 5

LongTermShortTermNetwork::LongTermShortTermNetwork(){
	this->settings = CSettings();
	LongTermShortTermNetwork(this->settings);
}

LongTermShortTermNetwork::LongTermShortTermNetwork(CSettings& settings){
	this->settings = settings;
	this->initialize_network();
}

//***************************
//Initialize Network
//***************************

void LongTermShortTermNetwork::initialize_network(){
	this->weights = host_vector<weight_type>();
	this->mapTo = host_vector<int>();
	this->mapFrom = host_vector<int>();
	this->bias = host_vector<weight_type>();

	//Initialize the layers
	this->mBlocksLayers = vector<vector<Memory_Block>>();


	positionOfLastWeightToNode = vector<long>();
	this->numberNonWeights = this->settings.i_input;

	this->createMemoryBlock(2);

}


//***************************
//Modify Structure Of Neuron
//***************************
int LongTermShortTermNetwork::decideNodeToAttachTo(){
	vector<int> notFullyConnected = vector<int>();
	int numberMBlocks = this->mBlocksLayers[0].size();
	//Find how many nodes are not fully connected
	for (int i = 0; i < numberMBlocks; i++){
		//A node is considered not fully connected if it is not connected to at least one other memory block
		if (this->mBlocksLayers[0][i].input_weights.size() < this->numberNonWeights + numberMBlocks){
			notFullyConnected.push_back(i);
		}

	}

	if (notFullyConnected.size() > 0){
		//Return a random number in the set of not fully connected nodes
		//It's the number contained in the positionOfLastWeightToNode which is this->settings.i_input less than its actual position
		return notFullyConnected[RandInt(0, notFullyConnected.size() - 1)];
	}
	else{
		//All nodes are fully connected and no new weights can be added
		return -1;
	}
}

int LongTermShortTermNetwork::decideNodeToAttachFrom(int attachTo){
	vector<int> notConnectedTo = vector<int>();
	bool containsValue = false;
	int start = (this->settings.i_output != attachTo ? this->positionOfLastWeightToNode[attachTo - 1] : 0);
	int end = (this->settings.i_output != attachTo ? this->positionOfLastWeightToNode[attachTo] : this->positionOfLastWeightToNode[attachTo]);
	for (int k = this->numberNonWeights; k < this->bias.size() - this->settings.i_output; k++){
		for (int i = start; i <= end; i++){
			if (this->mapFrom[i] == k){
				//The value is already contained in the system
				containsValue = true;
				break;
			}
		}

		if (!containsValue){
			notConnectedTo.push_back(k);
		}
		containsValue = false;


	}



	if (notConnectedTo.size() != 0){
		//It's the number contained in the positionOfLastWeightToNode which is this->settings.i_input less than its actual position
		return notConnectedTo[RandInt(0, notConnectedTo.size() - 1)];
	}
	else{
		return -1;
	}

}





weight_type LongTermShortTermNetwork::getNewWeight(){
	srand(time(NULL));
	return RandomClamped();
}



void LongTermShortTermNetwork::addWeight(int numberWeightsToAdd){
	int decideTo = this->decideNodeToAttachTo();
	if (decideTo != -1){
		this->mBlocksLayers[0][decideTo].addNewConnection(this->numberNonWeights - 1, this->mBlocksLayers[0].size() + this->numberNonWeights);
	}
	else{

	}

}

void LongTermShortTermNetwork::addNeuron(int numberNeuronsToAdd){
	this->createMemoryBlock(numberNeuronsToAdd);
}


void LongTermShortTermNetwork::createMemoryBlock(int numberMemoryCells){
	if (this->mBlocksLayers.size() == 0){
		this->mBlocksLayers.push_back(vector<Memory_Block>());//Add one hidden layer
		this->mBlocksLayers.push_back(vector<Memory_Block>());//Add one output layer
		for (unsigned int i = 0; i < this->settings.i_output; i++){
			this->mBlocksLayers[1].push_back(Memory_Block(numberMemoryCells + this->numberNonWeights, numberMemoryCells));
		}
	}

	for (int i = 0; i < numberMemoryCells; i++){
		this->mBlocksLayers[0].push_back(Memory_Block(this->settings.i_input));
	}

}

void LongTermShortTermNetwork::InitialcreateMemoryBlock(int numberMemoryCells){
	if (this->mBlocksLayers.size() == 0){
		this->mBlocksLayers.push_back(vector<Memory_Block>());
	}
	this->mBlocksLayers[0].push_back(Memory_Block(this->settings.i_input));
}



//*********************
//Run The Network
//*********************
//Multiply two values
template <typename T>
struct multiply : public thrust::unary_function <T,T> {

	//Overload the function operator
	template <typename Tuple>
	__host__ __device__
		T operator()(Tuple x) const{
		return (thrust::get<0>(x) * thrust::get<1>(x));
	}

};
template <typename T>
struct run_memory_block_functon : public::unary_function < T, T > {


	template <typename Tuple>
	__host__ __device__
	void operator()(Tuple &x){//Received Tuple is in the form input, output, forget, potential memory cell, memory cell value
		thrust::get<3>(x) = thrust::get<0>(x)*thrust::get<3>(x);//Multiply the input by the potential_memory_value
		thrust::get<2>(x) = thrust::get<2>(x) * thrust::get<4>(x); //Multiply the forget * the old memory cell value
		thrust::get<4>(x) = thrust::get<2>(x) + thrust::get<3>(x) + thrust::get<4>(x); //Sum the forget,input, and old cell value to get the new vaue the new potential memory cell value
		thrust::get<1>(x) = thrust::get<4>(x) * thrust::get<1>(x); //Multiply the new memory_cell value by the new output value 
	}

};

//Perform Sigmoid Operation of a Tuple
template <typename T>
struct sigmoid_tuple_functor : public thrust::unary_function <T, T> {

	//Overload the function operator
	template <typename Tuple>
	__host__ __device__
	T operator()(Tuple x) const{
		T z = (T)(thrust::get<0>(x)*thrust::get<1>(x));
		z = thrust::exp(((T)-1) * z);
		return (T)1 / ((T)1 + z);
	}

};



//Perform a sigmoid function
template <typename T>
struct sigmoid_functor : public thrust::unary_function <T,T> {
	sigmoid_functor(){};

	__host__ __device__
		T operator()(const T &x) const{
		T z = thrust::exp(((T)-1) * x);
		return (T)1 / ((T)1 + z);
	}

};

void LongTermShortTermNetwork::setInput(weight_type* in){
	//Place the input into the GPU values matrix
	for (int i = 0; i < this->settings.i_input; i++){
		this->GPUOutput_values[i] = in[i];
		this->GPUPreviousOutput_Values[i] = in[i];
	}

}

thrust::device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in){

	this->setInput(in);
	//Stores the numberofmblocks in a layer
	unsigned int numberMBlocks;
	unsigned int previousnumberMBlocks = 0;
	//Perform the transformation on each layer
	for (unsigned int i = 0; i < this->mBlocksLayers.size(); i++){
		
		if (i != 0){
			previousnumberMBlocks = numberMBlocks;
		}
		numberMBlocks = this->mBlocksLayers[i].size();
		//Sum the values of the input/output/forget/potential_memory_cell_values nodes
		//The values in the GPU weights are in the order input, output, forget, memory cells
		//Subtracting this->mBlocksLayers[i].size() from the end will remove the memory cells from doing anything
		//Output to Previous
		thrust::reduce_by_key(this->GPUMapTo.begin(), this->GPUMapTo.end(), make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.begin() + this->numberNonWeights + previousnumberMBlocks, // We don't want to multiply the actual input/out values, so we skip them
			make_permutation_iterator( // Create an iterator which maps the values coming from to those going to
			this->GPUOutput_values.begin(),
			this->GPUMapFrom.begin())
			)
			),
			sigmoid_tuple_functor<weight_type>()), //Multiply the two values then run them through a sigmoid function
			thrust::make_discard_iterator(), // Discard the retrieved order, the order should be constant
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + previousnumberMBlocks//Store in the previous in order to not overwrite the saved values
			);

		//Create a input/output/forget/potential_memory_cell_values/memory_cell_value value
		//Essentially run the gate and get the output value
		thrust::for_each(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + previousnumberMBlocks, //input values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + numberMBlocks + previousnumberMBlocks,//output values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (2 * numberMBlocks) + previousnumberMBlocks,//forget values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (3 * numberMBlocks) + previousnumberMBlocks,//potential_memory_cell_value
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (4 * numberMBlocks) + previousnumberMBlocks
			)),
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + numberMBlocks + previousnumberMBlocks, //input values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (2 * numberMBlocks) + previousnumberMBlocks,//output values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (3 * numberMBlocks) + previousnumberMBlocks,//forget values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (4 * numberMBlocks) + previousnumberMBlocks,//potential_memory_cell_value
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (5 * numberMBlocks) + previousnumberMBlocks
			)),
			run_memory_block_functon<weight_type>());
	}



	return device_vector<weight_type>();
}
//*********************
//Training the Network
//*********************
template <typename T>
struct find_error : public thrust::unary_function < T, T > {

	//Overload the function operator
	template <typename Tuple>
	__host__ __device__
		T operator()(Tuple &x) const{
		return thrust::pow((thrust::get<0>(x) -thrust::get<1>(x)), (T)2);
	}

};


void LongTermShortTermNetwork::InitializeLongShortTermMemory(){
	//Store all the values in the device
	//Will later add option for too little memory
	//Copy the information to the device
	this->UnrollNetwork(3);

	//Form the delta objects
	this->host_deltas = host_vector<weight_type>(this->GPUOutput_values.size());
	this->device_deltas = device_vector<weight_type>(this->GPUOutput_values.size());
	this->VisualizeNetwork();
}
void LongTermShortTermNetwork::LongTermShortTermNetwork::LongShortTermMemoryTraining(weight_type* in, weight_type* out){
	
	//Set the input values
	this->setInput(in);
	//Special functionality is required for the first layer
	thrust::reduce_by_key
		(
		thrust::make_transo
		
		);

	for (int i = 1; i < this->settings.i_backprop_unrolled - 1; i++){

	}

	//Special functionality is required for the output layer

}


void LongTermShortTermNetwork::ApplyLongTermShortTermMemoryError(){

}

//*********************
//Perform Functionality
//*********************

void LongTermShortTermNetwork::UnrollNetwork(int numLayers){
	vector<vector<Memory_Block>> Unrolled_Layers = vector<vector<Memory_Block>>();//Storage of the memory blocks as new layers
	//Add room for the intial input values
	this->GPUOutput_values.resize(this->numberNonWeights);
	
	for (unsigned int i = 0; i < this->mBlocksLayers.size() - 1; i++){
		this->loadUnrolledToDevice(0, i);
	}

	//Unroll the output layer only once
	this->loadUnrolledToDevice(1, this->mBlocksLayers.size() - 1);
	//Create an empty array for the current values in the network
	this->ResetSequence();
}

void LongTermShortTermNetwork::ResetSequence(){
	thrust::fill(this->GPUOutput_values.begin(), this->GPUOutput_values.end(), 0);
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), 0);
}

template <typename T>
void copyValuesToHost(int start, device_vector<T> &GPU_Vector, host_vector<T> &local_host_Vector){
	//Copy the values into the network
	thrust::copy(GPU_Vector.begin() + start, GPU_Vector.begin() + local_host_Vector.size() + start, local_host_Vector.begin());
}



void LongTermShortTermNetwork::CopyToHost(){
	//Copy the device memory to local
	this->output_bias.resize(this->GPUOutput_values.size());
	this->bias.resize(this->GPUPreviousOutput_Values.size());
	//Copy the output
	thrust::copy(this->GPUOutput_values.begin(), this->GPUOutput_values.end(), this->output_bias.begin());
	//Copy the secondary output
	thrust::copy(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), this->bias.begin());
	int start = 0;
	for (int j = 0; j < this->mBlocksLayers.size(); j++){
		//Copy back the input
		for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
			copyValuesToHost<weight_type>(start, this->GPUWeights, this->mBlocksLayers[j][i].input_weights);
			start += this->mBlocksLayers[j][i].input_weights.size();
		}
		//Copy back to output
		for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
			copyValuesToHost<weight_type>(start, this->GPUWeights, this->mBlocksLayers[j][i].output_weights);
			start += this->mBlocksLayers[j][i].output_weights.size();
		}

		//Copy back to forget
		for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
			copyValuesToHost<weight_type>(start, this->GPUWeights, this->mBlocksLayers[j][i].forget_weights);
			start += this->mBlocksLayers[j][i].forget_weights.size();
		}

		//Copy back to potential memory cell
		for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
			copyValuesToHost<weight_type>(start, this->GPUWeights, this->mBlocksLayers[j][i].potential_memory_cell_value);
			start += this->mBlocksLayers[j][i].potential_memory_cell_value.size();
		}

		//Get the new memory_cell_weight
		for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
			this->mBlocksLayers[j][i].memory_cell_weights = this->GPUOutput_values[this->GPUOutput_values.size() - i - this->mBlocksLayers[j].size()];
		}
	}
}

template <typename T>
void LongTermShortTermNetwork::copyNodesToDevice(device_vector<T> &GPU_Vector, device_vector<int> &fromPosition, host_vector<T> &local_host_Vector, host_vector<int> host_from_vector){
	int GPU_VecSize = GPU_Vector.size();
	int GPUFromPos = fromPosition.size();
	GPU_Vector.resize(GPU_VecSize + local_host_Vector.size());
	fromPosition.resize(GPU_VecSize + local_host_Vector.size());
	//Copy the values into the network
	thrust::copy(local_host_Vector.begin(), local_host_Vector.end(), GPU_Vector.begin() + GPU_VecSize);
	thrust::copy(host_from_vector.begin(), host_from_vector.end(), fromPosition.begin() + GPUFromPos);
}

//Copies only the input and not the device
template <typename T>
void LongTermShortTermNetwork::specialCopyToNodes(int start_output, int number_output, device_vector<T> &GPUWeightVector, device_vector<int> &toPosition, device_vector<int> &fromPosition, host_vector<T> &local_weights, host_vector<int> map){
	int GPU_VecSize = GPUWeightVector.size();
	int GPUFromPos = fromPosition.size();
	
	//We need to store the number a special copy of a map, such that it has input from both the input of the sequence and the output of the previous layer
	GPUWeightVector.resize(GPU_VecSize + local_weights.size());
	fromPosition.resize(GPU_VecSize + map.size());
	thrust::copy(map.begin(), map.end(),fromPosition.begin() + GPUFromPos);
	thrust::copy(local_weights.begin(), local_weights.end(), GPUWeightVector.begin() + GPU_VecSize);
	GPU_VecSize = GPU_VecSize + local_weights.size();
	
	for (unsigned int i = start_output; i <start_output + number_output; i++){
		fromPosition.push_back(i);
		GPUWeightVector.push_back(1);
	}

	toPosition.resize(fromPosition.size());
	thrust::fill(toPosition.end() - (local_weights.size() + number_output), toPosition.end(), this->GPUOutput_values.size());

}

void LongTermShortTermNetwork::loadUnrolledToDevice(int type_of_row,unsigned int j){
	//We need to keep track of the end of the number of inputs in order to add in a connection to the outputs for the next level
	unsigned int start_output_position = 0;
	unsigned int number_output_to_add = 0;
	
	if (type_of_row == 0){//Is not an output row
		number_output_to_add = this->mBlocksLayers[j].size();
		for (unsigned int i = 0; i < this->mBlocksLayers[j].size(); i++){
			start_output_position = this->mBlocksLayers[j][i].input_weights.size();
		}



		//Increment it by the input numbers
		start_output_position += this->settings.i_input - 1;
	}
	//Set all the input values
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		specialCopyToNodes<weight_type>(start_output_position, number_output_to_add, this->GPUWeights, this->GPUMapTo, this->GPUMapFrom, this->mBlocksLayers[j][i].input_weights, this->mBlocksLayers[j][i].mapFrom);
		
		this->GPUOutput_values.push_back(0);
	}


	//Set all the outputs
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		specialCopyToNodes<weight_type>(start_output_position, number_output_to_add, this->GPUWeights, this->GPUMapTo, this->GPUMapFrom, this->mBlocksLayers[j][i].output_weights, this->mBlocksLayers[j][i].mapFrom);
		this->GPUOutput_values.push_back(0);
	}

	//Set all the forget nodes
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		specialCopyToNodes<weight_type>(start_output_position, number_output_to_add, this->GPUWeights, this->GPUMapTo, this->GPUMapFrom, this->mBlocksLayers[j][i].forget_weights, this->mBlocksLayers[j][i].mapFrom);
		this->GPUOutput_values.push_back(0);
	}

	//Set all the potential_output nodes
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		specialCopyToNodes<weight_type>(start_output_position, number_output_to_add, this->GPUWeights, this->GPUMapTo, this->GPUMapFrom, this->mBlocksLayers[j][i].potential_memory_cell_value, this->mBlocksLayers[j][i].mapFrom);
		this->GPUOutput_values.push_back(0);
	}

	this->GPUPreviousOutput_Values.resize(this->GPUOutput_values.size());
	//Set the values of the Memory Cells
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		this->GPUOutput_values.push_back(this->mBlocksLayers[j][i].memory_cell_weights);
		this->GPUPreviousOutput_Values.push_back(this->mBlocksLayers[j][i].memory_cell_weights);
	}
}


void LongTermShortTermNetwork::loadLayerToDevice(unsigned int j){
	//Set all the input values
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		copyNodesToDevice<weight_type>(this->GPUWeights, this->GPUMapFrom, this->mBlocksLayers[j][i].input_weights, this->mBlocksLayers[j][i].mapFrom);
		this->GPUMapTo.resize(this->GPUMapTo.size() + this->mBlocksLayers[j][i].input_weights.size());
		thrust::fill(this->GPUMapTo.end() - this->mBlocksLayers[j][i].input_weights.size(), this->GPUMapTo.end(), this->GPUOutput_values.size());
		this->GPUOutput_values.push_back(0);
	}


	//Set all the outputs
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		copyNodesToDevice<weight_type>(this->GPUWeights, this->GPUMapFrom, this->mBlocksLayers[j][i].output_weights, this->mBlocksLayers[j][i].mapFrom);
		this->GPUMapTo.resize(this->GPUMapTo.size() + this->mBlocksLayers[j][i].output_weights.size());
		thrust::fill(this->GPUMapTo.end() - this->mBlocksLayers[j][i].output_weights.size(), this->GPUMapTo.end(), this->GPUOutput_values.size());
		this->GPUOutput_values.push_back(0);
	}

	//Set all the forget nodes
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		copyNodesToDevice<weight_type>(this->GPUWeights, this->GPUMapFrom, this->mBlocksLayers[j][i].forget_weights, this->mBlocksLayers[j][i].mapFrom);
		this->GPUMapTo.resize(this->GPUMapTo.size() + this->mBlocksLayers[j][i].forget_weights.size());
		thrust::fill(this->GPUMapTo.end() - this->mBlocksLayers[j][i].forget_weights.size(), this->GPUMapTo.end(), this->GPUOutput_values.size());
		this->GPUOutput_values.push_back(0);
	}

	//Set all the potential_output nodes
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		copyNodesToDevice<weight_type>(this->GPUWeights, this->GPUMapFrom, this->mBlocksLayers[j][i].potential_memory_cell_value, this->mBlocksLayers[j][i].mapFrom);
		this->GPUMapTo.resize(this->GPUMapTo.size() + this->mBlocksLayers[j][i].potential_memory_cell_value.size());
		thrust::fill(this->GPUMapTo.end() - this->mBlocksLayers[j][i].potential_memory_cell_value.size(), this->GPUMapTo.end(), this->GPUOutput_values.size());
		this->GPUOutput_values.push_back(0);
	}

	this->GPUPreviousOutput_Values.resize(this->GPUOutput_values.size());
	//Set the values of the Memory Cells
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		this->GPUOutput_values.push_back(this->mBlocksLayers[j][i].memory_cell_weights);
		this->GPUPreviousOutput_Values.push_back(this->mBlocksLayers[j][i].memory_cell_weights);
	}
}

void LongTermShortTermNetwork::CopyToDevice(){
	this->device_deltas = device_vector<weight_type>();
	this->GPUMapTo = device_vector<int>();
	this->GPUMapFrom = device_vector<int>();
	this->GPUOutput_values = device_vector<weight_type>();
	this->GPUPreviousOutput_Values = device_vector<weight_type>();
	this->GPUWeights = device_vector<weight_type>();

	//Set the input values to 0
	this->GPUOutput_values.resize(this->numberNonWeights);
	for (unsigned int j = 0; j < this->mBlocksLayers.size(); j++){
		this->loadLayerToDevice(j);
	}

}

void  LongTermShortTermNetwork::cleanNetwork(){
	this->CopyToHost();
	//Free the used memory
	clear_vector::free(this->GPUMapFrom);
	clear_vector::free(this->GPUMapTo);
	clear_vector::free(this->GPUWeights);
	clear_vector::free(this->device_deltas);
	clear_vector::free(this->GPUOutput_values);
	clear_vector::free(this->GPUPreviousOutput_Values);
}

void LongTermShortTermNetwork::emptyGPUMemory(){
	clear_vector::free(this->GPUMapFrom);
	clear_vector::free(this->GPUMapTo);
	clear_vector::free(this->GPUWeights);
	clear_vector::free(this->device_deltas);
	clear_vector::free(this->GPUOutput_values);
	clear_vector::free(this->GPUPreviousOutput_Values);
}
//*********************
//Misc
//*********************
void LongTermShortTermNetwork::VisualizeNetwork(){
	cout << *this;
}

ostream& LongTermShortTermNetwork::OutputNetwork(ostream& os){
	os << *this;
	return os;
}