#include "hip/hip_runtime.h"
#include "LongTermShortTermNetwork.cuh"
#define NUMBER_NODES_IN_MEMORY_CELL 5
#define TEST_DEBUG

//#define _DEBUG_WEIGHTS
LongTermShortTermNetwork::LongTermShortTermNetwork(){
	this->settings = CSettings();
	LongTermShortTermNetwork(this->settings);
}

LongTermShortTermNetwork::LongTermShortTermNetwork(CSettings& settings){
	this->settings = settings;
	this->initialize_network();
}

//***************************
//Initialize Network
//***************************

void LongTermShortTermNetwork::initialize_network(){
	this->weights = host_vector<weight_type>();
	this->mapTo = host_vector<int>();
	this->mapFrom = host_vector<int>();
	this->bias = host_vector<weight_type>();

	//Initialize the layers
	this->mBlocksLayers = vector<vector<Memory_Block>>();


	positionOfLastWeightToNode = vector<long>();
	this->numberNonWeights = this->settings.i_input;
	srand(time(NULL));
	this->createMemoryBlock(2);

}


//***************************
//Modify Structure Of Neuron
//***************************
int LongTermShortTermNetwork::decideNodeToAttachTo(){
	vector<int> notFullyConnected = vector<int>();
	int numberMBlocks = this->mBlocksLayers[0].size();
	//Find how many nodes are not fully connected
	for (int i = 0; i < numberMBlocks; i++){
		//A node is considered not fully connected if it is not connected to at least one other memory block
		if (this->mBlocksLayers[0][i].input_weights.size() < this->numberNonWeights + numberMBlocks){
			notFullyConnected.push_back(i);
		}

	}

	if (notFullyConnected.size() > 0){
		//Return a random number in the set of not fully connected nodes
		//It's the number contained in the positionOfLastWeightToNode which is this->settings.i_input less than its actual position
		return notFullyConnected[RandInt(0, notFullyConnected.size() - 1)];
	}
	else{
		//All nodes are fully connected and no new weights can be added
		return -1;
	}
}

int LongTermShortTermNetwork::decideNodeToAttachFrom(int attachTo){
	vector<int> notConnectedTo = vector<int>();
	bool containsValue = false;
	int start = (this->settings.i_output != attachTo ? this->positionOfLastWeightToNode[attachTo - 1] : 0);
	int end = (this->settings.i_output != attachTo ? this->positionOfLastWeightToNode[attachTo] : this->positionOfLastWeightToNode[attachTo]);
	for (int k = this->numberNonWeights; k < this->bias.size() - this->settings.i_output; k++){
		for (int i = start; i <= end; i++){
			if (this->mapFrom[i] == k){
				//The value is already contained in the system
				containsValue = true;
				break;
			}
		}

		if (!containsValue){
			notConnectedTo.push_back(k);
		}
		containsValue = false;


	}



	if (notConnectedTo.size() != 0){
		//It's the number contained in the positionOfLastWeightToNode which is this->settings.i_input less than its actual position
		return notConnectedTo[RandInt(0, notConnectedTo.size() - 1)];
	}
	else{
		return -1;
	}

}





weight_type LongTermShortTermNetwork::getNewWeight(){
	srand(time(NULL));
	return RandomClamped();
}



void LongTermShortTermNetwork::addWeight(int numberWeightsToAdd){
	int decideTo = this->decideNodeToAttachTo();
	if (decideTo != -1){
		this->mBlocksLayers[0][decideTo].addNewConnection(this->numberNonWeights - 1, this->mBlocksLayers[0].size() + this->numberNonWeights);
	}
	else{

	}

}

void LongTermShortTermNetwork::addNeuron(int numberNeuronsToAdd){
	this->createMemoryBlock(numberNeuronsToAdd);
}


void LongTermShortTermNetwork::createMemoryBlock(int numberMemoryCells){
	if (this->mBlocksLayers.size() == 0){
		this->mBlocksLayers.push_back(vector<Memory_Block>());//Add one hidden layer
		this->mBlocksLayers.push_back(vector<Memory_Block>());//Add one output layer
		for (unsigned int i = 0; i < this->settings.i_output; i++){
			this->mBlocksLayers[1].push_back(Memory_Block(numberMemoryCells + this->numberNonWeights, numberMemoryCells,Memory_Block::OUTPUT));
		}
	}

	for (int i = 0; i < numberMemoryCells; i++){
		this->mBlocksLayers[0].push_back(Memory_Block(this->settings.i_input));
	}

}

void LongTermShortTermNetwork::InitialcreateMemoryBlock(int numberMemoryCells){
	if (this->mBlocksLayers.size() == 0){
		this->mBlocksLayers.push_back(vector<Memory_Block>());
	}
	this->mBlocksLayers[0].push_back(Memory_Block(this->settings.i_input));
}



//*********************
//Run The Network
//*********************
//Multiply two values
template <typename T>
struct multiply : public thrust::unary_function < T, T > {

	//Overload the function operator
	template <typename Tuple>
	__host__ __device__
		T operator()(Tuple x) const{
		return ((T)thrust::get<0>(x) * (T)thrust::get<1>(x));
	}

};

template <typename T>
struct subtract_tuple : public thrust::unary_function < T, T > {

	//Overload the function operator
	template <typename Tuple>
	__host__ __device__
		T operator()(const Tuple &x){
		return (thrust::get<0>(x) -thrust::get<1>(x));
	}
};

template <typename T>
struct run_memory_block_functon : public::unary_function < T, T > {


	template <typename Tuple>
	__host__ __device__
		void operator()(Tuple &x){//Received Tuple is in the form input, output, forget, potential memory cell, memory cell value
		weight_type memory_value = sigmoid_function(thrust::get<0>(x) * thrust::get<3>(x));//Multiply the input by the potential_memory_value

		weight_type forget = (weight_type)thrust::get<2>(x);

		thrust::get<2>(x) = sigmoid_function((weight_type)thrust::get<2>(x) * (weight_type)thrust::get<4>(x)); //Multiply the forget * the old memory cell value
		thrust::get<4>(x) = thrust::get<4>(x) + memory_value + forget; //Sum the forget,input, and old cell value to get the new value the new potential memory cell value
		thrust::get<1>(x) = sigmoid_function((weight_type)thrust::get<4>(x) * (weight_type)thrust::get<1>(x)); //Multiply the new memory_cell value by the new output value 

	}

	__host__ __device__
	weight_type sigmoid_function(weight_type value){
		return (weight_type)1 / ((weight_type)1 + thrust::exp(((weight_type)-1 * value)));
	}

};

//Perform Sigmoid Operation of a Tuple
template <typename T>
struct sigmoid_tuple_functor : public thrust::unary_function < T, T > {

	//Overload the function operator
	template <typename Tuple>

	__host__ __device__
		T operator()(Tuple x) const{
		T z = (T)((T)thrust::get<0>(x)*(T)thrust::get<1>(x));
		z = thrust::exp(((T)-1) * z);
		return (T)1 / ((T)1 + z);
	}

};




//Perform a sigmoid function
template <typename T>
struct sigmoid_functor : public thrust::unary_function < T, T > {
	sigmoid_functor(){};

	__host__ __device__
		T operator()(const T &x) const{
		T z = thrust::exp(((T)-1) * x);
		return (T)1 / ((T)1 + z);
	}

};

void LongTermShortTermNetwork::setInput(weight_type* in){
	//Place the input into the GPU values matrix
	for (int i = 0; i < this->settings.i_input; i++){
		this->GPUOutput_values[i] = in[i];
		this->GPUPreviousOutput_Values[i] = in[i];
	}

}

thrust::device_vector<weight_type> LongTermShortTermNetwork::runNetwork(weight_type* in){

	this->setInput(in);
	//Stores the numberofmblocks in a layer
	unsigned int numberMBlocks;
	unsigned int previousnumberMBlocks = 0;
	//Perform the transformation on each layer
	for (unsigned int i = 0; i < this->mBlocksLayers.size(); i++){

		if (i != 0){
			previousnumberMBlocks = numberMBlocks;
		}
		numberMBlocks = this->mBlocksLayers[i].size();
		//Sum the values of the input/output/forget/potential_memory_cell_values nodes
		//The values in the GPU weights are in the order input, output, forget, memory cells
		//Subtracting this->mBlocksLayers[i].size() from the end will remove the memory cells from doing anything
		//Output to Previous
		thrust::reduce_by_key(this->GPUMapTo.begin(), this->GPUMapTo.end(), thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.begin() + this->numberNonWeights + previousnumberMBlocks, // We don't want to multiply the actual input/out values, so we skip them
			make_permutation_iterator( // Create an iterator which maps the values coming from to those going to
			this->GPUOutput_values.begin(),
			this->GPUMapFrom.begin())
			)
			),
			multiply<weight_type>()), //Multiply the two values then run them through a sigmoid function
			thrust::make_discard_iterator(), // Discard the retrieved order, the order should be constant
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + previousnumberMBlocks//Store in the previous in order to not overwrite the saved values
			);

		thrust::transform(this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + previousnumberMBlocks,
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + previousnumberMBlocks + (5 * numberMBlocks),
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + previousnumberMBlocks,
			sigmoid_functor<weight_type>());

		//Create a input/output/forget/potential_memory_cell_values/memory_cell_value value
		//Essentially run the gate and get the output value
		thrust::for_each(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + previousnumberMBlocks + this->settings.i_input, //input values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + numberMBlocks + previousnumberMBlocks + this->settings.i_input,//output values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (2 * numberMBlocks) + previousnumberMBlocks + this->settings.i_input,//forget values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (3 * numberMBlocks) + previousnumberMBlocks + this->settings.i_input,//potential_memory_cell_value
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (4 * numberMBlocks) + previousnumberMBlocks + this->settings.i_input
			)),
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + numberMBlocks + previousnumberMBlocks + this->settings.i_input, //input values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (2 * numberMBlocks) + previousnumberMBlocks + this->settings.i_input,//output values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (3 * numberMBlocks) + previousnumberMBlocks + this->settings.i_input,//forget values
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (4 * numberMBlocks) + previousnumberMBlocks + this->settings.i_input,//potential_memory_cell_value
			this->GPUPreviousOutput_Values.begin() + this->numberNonWeights + (5 * numberMBlocks) + previousnumberMBlocks + this->settings.i_input
			)),
			run_memory_block_functon<weight_type>());
		thrust::copy(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), this->GPUOutput_values.begin());
		
	}

	device_vector<weight_type> toReturn = device_vector<weight_type>(this->settings.i_output);
	//Transform the output of the current
	thrust::transform(this->GPUOutput_values.end() - 
		(5 * this->mBlocksLayers[this->mBlocksLayers.size() - 1].size()) + this->settings.i_output,
		this->GPUOutput_values.end() - 
		(5 * this->mBlocksLayers[this->mBlocksLayers.size() - 1].size())+ this->settings.i_output + this->settings.i_output,
		toReturn.begin(), _1 * (weight_type)1);

	return toReturn;
}
//*********************
//Training the Network
//*********************
template <typename T>
struct find_error : public thrust::unary_function < T, T > {

	//Overload the function operator
	template <typename Tuple>
	__host__ __device__
		T operator()(Tuple &x) const{
		return thrust::pow((thrust::get<0>(x) -thrust::get<1>(x)), (T)2);
	}

};

//sum a value
template <typename T>
struct add_constant_value : public thrust::unary_function < T, T > {
	const T c;
	const unsigned int input;
	add_constant_value() : c(0), input(0){};

	add_constant_value(T _c, unsigned int _input) :c(_c), input(_input){};


	__host__ __device__
		T operator()(const T &x)const{

		if (x >= input){//The value is not an input
			return ((T)x + (T)c);
		}
		else{//The value is an input
			return x;
		}
	}


};

struct compare_plus : public thrust::unary_function < int, int > {
	int input;
	int numberIncrease;//Number to increase the number by 
	compare_plus(int max_input, int numberIncrease){
		this->input = max_input;
		this->numberIncrease = numberIncrease;
	}


	__host__ __device__
		int operator()(int &x) const{
		if (x < this->input){//Returns this directly, as it is an input
			return x;
		}
		else{
			return (x + numberIncrease);
		}
	}
};

template <typename T>
struct add_one_when_equal_to : public thrust::unary_function < T, T > {
	const T equal_to;
	const T divide_by;
	add_one_when_equal_to(T _divide_by, T _equal_to) :equal_to(_equal_to), divide_by(_divide_by){}
	__host__ __device__
		T operator()(const T &x){
		if (x >= equal_to){
			return  x;
		}
		return (x / divide_by);
	}

};

//Function is _add_to, _greater_than_this
template <typename T>
struct add_when_greater_than : public thrust::unary_function < T, T > {
	const T greater_than_this;
	const T add_to;
	add_when_greater_than(T _add_to, T _greater_than_this) :greater_than_this(_greater_than_this), add_to(_add_to){}
	__host__ __device__
		T operator()(const T &x){
		if (x >= _equal_to){
			return (x + add_to);
		}
		else{
			return x;
		}
	}

};

//Apply the error from the delta and the weight
template <typename T>
struct apply_error: public thrust::binary_function<T,T,T>{
	const T alpha;
	const T beta;
	const T divide;
	apply_error(T _alpha, T _beta, T _divide) : alpha(_alpha), beta(_beta),divide(_divide){

	}

	//w = weight, d = delta, beta * (d/(number summed) + (w + (w*alpha)
	__host__ __device__
		T operator()(const T &d, const T &w)const{
		return (beta * (d / divide)) + (w + (w*alpha));
	}
};

template <typename T>
struct find_non_output_delta : public thrust::unary_function < T, T > {


	find_non_output_delta(){};

	template <typename Tuple>
	__host__ __device__
		T operator()(Tuple &t){
		return (T)thrust::get<0>(t) * ((T)1 - (T)thrust::get<0>(t)) * (T)thrust::get<1>(t);

	}

};

void LongTermShortTermNetwork::InitializeLongShortTermMemoryForRun(){
	//Form the delta objects
	this->CopyToDevice();
}

void LongTermShortTermNetwork::InitializeLongShortTermMemory(){
	//Store all the values in the device
	//Will later add option for too little memory
	//Copy the information to the device
	this->UnrollNetwork(3);
	this->host_deltas = host_vector<weight_type>(this->GPUOutput_values.size());
	this->device_deltas = device_vector<weight_type>(this->GPUOutput_values.size());
	this->RealOutput = device_vector<weight_type>(this->settings.i_output);
}

template <typename T>
void testCopy(thrust::device_vector<T> vector,int start,int end){
	thrust::copy(vector.begin() + start, vector.begin() + end, std::ostream_iterator<T>(std::cout, "\n"));
}

void LongTermShortTermNetwork::LongTermShortTermNetwork::LongShortTermMemoryTraining(weight_type* in, weight_type* out){
	//Get the number of weights in the output layer
	//This is needed because the output layer needs to be used only once, so we need to inform the system which weights to skip
	
	//Set the input values
	this->setInput(in);
	unsigned int number_weights_to_beginining_of_layer = 0;
	unsigned int number_nodes_to_beginning_of_layer = 0;
	unsigned int number_weights_in_layer = this->GPUWeights.size();
	for (int i = 0; i < this->settings.i_backprop_unrolled; i++){
		thrust::reduce_by_key(
			this->GPUMapTo.begin(),
			this->GPUMapTo.end(),
			
			//Multiply the weights x output
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			
			this->GPUWeights.begin(),
			
			thrust::make_permutation_iterator(
			this->GPUOutput_values.begin(),
			thrust::make_transform_iterator(
			this->GPUMapFrom.begin(), add_constant_value<int>(number_nodes_to_beginning_of_layer, this->settings.i_input))
			)
			)
			),
			multiply<weight_type>()
			),
			thrust::make_discard_iterator(),
			this->GPUPreviousOutput_Values.begin()
			);

		if (i > 0){//Only increment it by the number of nodes when working from any layer which is not the initial layer
			//This lets the nodes use the previous layer as their input
			number_weights_to_beginining_of_layer += number_weights_in_layer;
			number_nodes_to_beginning_of_layer += this->numberOfNodes;
		}
		
		

		if (true){
			//Transfer all values from the current to the next row
			thrust::transform(this->GPUPreviousOutput_Values.begin(),
				this->GPUPreviousOutput_Values.end(),
				this->GPUOutput_values.begin() + number_nodes_to_beginning_of_layer + this->numberNonWeights, sigmoid_functor<weight_type>());
		}

		
	}

}

//Find the delta gradiant for each of the "layers" of the network
void LongTermShortTermNetwork::FindBackPropDelta(weight_type* out){
	//Retrieve the length of the output
	unsigned int numberCellsInLayers = this->mBlocksLayers[this->mBlocksLayers.size() - 1].size();
	unsigned int lengthOfOutput = (this->mBlocksLayers[this->mBlocksLayers.size() - 1].size() * 4) + this->getNumberMemoryCells(this->mBlocksLayers.size() - 1);
	unsigned int numberInLayers = this->mBlocksLayers[this->mBlocksLayers.size() - 1].size();
	unsigned int numberOfWeightsOfInputType = getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 1, INPUT_CELL);
	unsigned int numberOfWeightsOfOutputType = getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 1, OUTPUT_CELL);
	unsigned int numberOfWeightsOfForgetType = getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 1, FORGET_CELL);
	unsigned int numberOfWeightsOfPotentialMemoryCellType = getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 1, POTENTIAL_MEMORY_CELL);
	unsigned int numberOfWeightsOfMemoryCellType = getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 1, MEMORY_CELL);
	unsigned int numberOfWeightsInLayer = numberOfWeightsOfInputType + numberOfWeightsOfOutputType + numberOfWeightsOfForgetType + numberOfWeightsOfPotentialMemoryCellType + numberOfWeightsOfMemoryCellType;
	unsigned int numberNodesOfSingleType = this->mBlocksLayers[this->mBlocksLayers.size() - 1].size();//Number of non memory cells in a node



	//Find the output delta
	//Start from the begining + the number of input nodes
	thrust::transform(this->RealOutput.begin(), this->RealOutput.end(), 
		this->GPUOutput_values.end() - lengthOfOutput + numberNodesOfSingleType, 
		this->device_deltas.end() - lengthOfOutput + numberNodesOfSingleType,
		_2 * (((weight_type)1) - _2) * (_1 - _2));//Output * (1- output) * (target-output)
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), 0);
	
	thrust::fill(this->device_deltas.begin(), this->device_deltas.end(), 1);

	//n,k used to form the sequence
	//placed here to make the function easier to read
	unsigned int n = numberCellsInLayers;
	unsigned int k = numberOfWeightsOfOutputType;
	//Backpropogate to the input of the output memory cells

	//Multiply the output weights by their deltas
	//deltas.end() - totalLengthOfTheOutput + #input nodes, deltas.end() - totalLengthOfTheOutput + #input nodes + #output nodes
	//The output layer has a special feature wherin the number of weights to each output is equal, and the only different value is the last one for each node
	// as such a formula can be made to place each one next to each other
	thrust::reduce_by_key(
		thrust::make_transform_iterator(thrust::make_counting_iterator((int)0),
		add_one_when_equal_to<int>((int)(numberOfWeightsOfOutputType / numberCellsInLayers), numberOfWeightsOfOutputType - numberCellsInLayers)),
		thrust::make_transform_iterator(thrust::make_counting_iterator((int)0),
		add_one_when_equal_to<int>((int)(numberOfWeightsOfOutputType / numberCellsInLayers), numberOfWeightsOfOutputType - numberCellsInLayers)) + numberOfWeightsOfOutputType,

		thrust::make_permutation_iterator(
		thrust::make_transform_iterator(
		thrust::make_zip_iterator(
		thrust::make_tuple(
		this->GPUWeights.end() - numberOfWeightsInLayer + numberOfWeightsOfInputType,//Beginning of the output of the output layer memory cells
		
		thrust::make_permutation_iterator(
		this->device_deltas.begin(), //Beginning of the deltas of the output
		this->GPUMapTo.end() - numberOfWeightsInLayer + numberOfWeightsOfInputType
		)
		)
		),
		multiply<weight_type>()
		),
		thrust::make_transform_iterator(thrust::make_counting_iterator((int)0), (((_1%n)*k) + (_1/n)))),
		thrust::make_discard_iterator(),
		this->GPUPreviousOutput_Values.begin()
		);





	//Multiply the memory cells by their memory to get the needed values
	thrust::transform(this->GPUPreviousOutput_Values.begin(),
		this->GPUPreviousOutput_Values.begin() + this->mBlocksLayers[this->mBlocksLayers.size() - 2].size() + (numberOfWeightsOfMemoryCellType / numberCellsInLayers),
		this->GPUOutput_values.end() - (numberOfWeightsOfMemoryCellType / 3),
		this->device_deltas.end() - (numberOfWeightsOfMemoryCellType / 3),
		_2*((weight_type)1 - _2)*_1);

	//Copy the weights from the memory cell to the input/forget/potential, since all connections to the memory cell are always weight one
	// and 1 * n =n

	thrust::transform(
		this->GPUPreviousOutput_Values.begin() + this->mBlocksLayers[this->mBlocksLayers.size() - 2].size(),
		this->GPUPreviousOutput_Values.begin() + this->mBlocksLayers[this->mBlocksLayers.size() - 2].size() + (numberOfWeightsOfMemoryCellType / 3),
		this->GPUOutput_values.end() - numberOfWeightsInLayer,
		this->device_deltas.end() - lengthOfOutput,
		_2*((weight_type)1 - _2)*_1
		);


	//Forget Nodes
	thrust::transform(
		this->GPUPreviousOutput_Values.begin() + this->mBlocksLayers[this->mBlocksLayers.size() - 2].size(),
		this->GPUPreviousOutput_Values.begin() + this->mBlocksLayers[this->mBlocksLayers.size() - 2].size() + (numberOfWeightsOfMemoryCellType / 3),
		this->GPUOutput_values.end() - numberOfWeightsInLayer + numberOfWeightsOfInputType + numberOfWeightsOfOutputType,
		this->device_deltas.end() - lengthOfOutput + (2 * numberCellsInLayers),
		_2*((weight_type)1 - _2)*_1
		);

	//Potential Nodes
	thrust::transform(
		this->GPUPreviousOutput_Values.begin() + this->mBlocksLayers[this->mBlocksLayers.size() - 2].size(),
		this->GPUPreviousOutput_Values.begin() + this->mBlocksLayers[this->mBlocksLayers.size() - 2].size() + (numberOfWeightsOfMemoryCellType / 3),
		this->GPUOutput_values.end() - numberOfWeightsInLayer + numberOfWeightsOfInputType + numberOfWeightsOfOutputType + numberOfWeightsOfForgetType,
		this->device_deltas.end() - lengthOfOutput + (3 * numberCellsInLayers),
		_2*((weight_type)1 - _2)*_1
		);



	//Lengths of weights in the next layer. I.e. output layer if second layer from top
	unsigned int numberOfWeightsOfInputTypeInNextLayer = numberOfWeightsOfInputType;
	unsigned int numberOfWeightsOfOutputTypeInNextLayer = numberOfWeightsOfOutputType;
	unsigned int numberOfWeightsOfForgetTypeInNextLayer = numberOfWeightsOfForgetType;
	unsigned int numberOfWeightsOfPotentialMemoryCellTypeInNextLayer = numberOfWeightsOfPotentialMemoryCellType;
	unsigned int numberOfWeightsOfMemoryCellTypeInNextLayer = numberOfWeightsOfMemoryCellType;
	unsigned int numberOfWeightsInNextLayer = numberOfWeightsInLayer;
	unsigned int numberCellsInNextLayer = numberCellsInLayers;


	//Rempty the delta * weight holder
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), 0);

	numberCellsInLayers = this->mBlocksLayers[this->mBlocksLayers.size() - 2].size();
	unsigned int length_to_previous_output = lengthOfOutput;
	unsigned int length_to_previous_weight = numberOfWeightsInNextLayer;
	unsigned int delta_start = lengthOfOutput;
	unsigned int end_of_count = numberOfWeightsInNextLayer;
	//Find the delta from the gradiant of each other layer in the unrolled network
	for (int i = this->settings.i_backprop_unrolled; i > 0; i--){
		
		//Lengths of weights in the next layer. I.e. output layer if second layer from top
		
		thrust::reduce_by_key(
			this->count.end() - length_to_previous_weight,
			this->count.end() - length_to_previous_weight + end_of_count,//Sum over start of layer to end of layer

			thrust::make_permutation_iterator(
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			//Permute the Output_values such that each one occurs with it's particular weight
			thrust::make_permutation_iterator(
			this->GPUOutput_values.end(),
			this->GPUMapFrom.end() - length_to_previous_weight - this->numberOfNodes),

			//Weight x Delta
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.end() - length_to_previous_weight,//End - number of weights in next layer + the number of values which have no weight
			thrust::make_permutation_iterator(
			this->device_deltas.end() - delta_start, //Start from the beginning of the previous layer
			thrust::make_transform_iterator(
			thrust::make_counting_iterator((int)0), _1 / numberCellsInLayers)))
			),
			multiply<weight_type>()
			)//Permute the deltas such that it matches the weights
			)),
			find_non_output_delta<weight_type>()),
			this->positionToSum.end() - length_to_previous_weight),
			thrust::make_discard_iterator(),
			this->GPUPreviousOutput_Values.begin()
			);



#ifdef _DEBUG_WEIGHTS		
		thrust::copy(thrust::make_permutation_iterator(
			this->device_deltas.end() - delta_start, //Start from the beginning of the previous layer
			thrust::make_transform_iterator(
			thrust::make_counting_iterator((int)0), _1 / numberCellsInLayers)), 
			
			thrust::make_permutation_iterator(
			this->device_deltas.end() - delta_start, //Start from the beginning of the previous layer
			thrust::make_transform_iterator(
			thrust::make_counting_iterator((int)0), _1 / numberCellsInLayers)) + end_of_count, std::ostream_iterator<weight_type>(std::cout, "\n"));
		
		std::cout << "______________________";

		//Weight x Delta
		thrust::copy(thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.end() - length_to_previous_weight,//End - number of weights in next layer + the number of values which have no weight
			thrust::make_permutation_iterator(
			this->device_deltas.end() - delta_start, //Start from the beginning of the previous layer
			thrust::make_transform_iterator(
			thrust::make_counting_iterator((int)0), _1 / numberCellsInLayers)))
			),
			multiply<weight_type>()
			), thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.end() - length_to_previous_weight,//End - number of weights in next layer + the number of values which have no weight
			thrust::make_permutation_iterator(
			this->device_deltas.end() - delta_start, //Start from the beginning of the previous layer
			thrust::make_transform_iterator(
			thrust::make_counting_iterator((int)0), _1 / numberCellsInLayers)))
			),
			multiply<weight_type>()
			) + end_of_count, std::ostream_iterator<weight_type>(std::cout, "\n"));

		std::cout << "______________________";

		//Weight x Delta
		thrust::copy(
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			//Permute the Output_values such that each one occurs with it's particular weight
			thrust::make_permutation_iterator(
			this->GPUOutput_values.end() - length_to_previous_weight + this->numberOfNodes,
			this->GPUMapFrom.end() - length_to_previous_weight + this->numberOfNodes),

			//Weight x Delta
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.end() - length_to_previous_weight,//End - number of weights in next layer + the number of values which have no weight
			thrust::make_permutation_iterator(
			this->device_deltas.end() - delta_start, //Start from the beginning of the previous layer
			thrust::make_transform_iterator(
			thrust::make_counting_iterator((int)0), _1 / numberCellsInLayers)))
			),
			multiply<weight_type>()
			)//Permute the deltas such that it matches the weights
			)),
			find_non_output_delta<weight_type>()),
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			//Permute the Output_values such that each one occurs with it's particular weight
			thrust::make_permutation_iterator(
			this->GPUOutput_values.end() - length_to_previous_weight + this->numberOfNodes,
			this->GPUMapFrom.end() - length_to_previous_weight + this->numberOfNodes),

			//Weight x Delta
			thrust::make_transform_iterator(
			thrust::make_zip_iterator(
			thrust::make_tuple(
			this->GPUWeights.end() - length_to_previous_weight,//End - number of weights in next layer + the number of values which have no weight
			thrust::make_permutation_iterator(
			this->device_deltas.end() - delta_start, //Start from the beginning of the previous layer
			thrust::make_transform_iterator(
			thrust::make_counting_iterator((int)0), _1 / numberCellsInLayers)))
			),
			multiply<weight_type>()
			)//Permute the deltas such that it matches the weights
			)),
			find_non_output_delta<weight_type>()) + end_of_count, std::ostream_iterator<weight_type>(std::cout, "\n"));

#endif

		//Increase the position of the weights
		length_to_previous_output += this->numberOfNodes;
		length_to_previous_weight += numberOfWeightsInLayer;
		delta_start += this->numberOfNodes;
		end_of_count = this->numberOfNodes;
		thrust::copy(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.begin() + this->numberOfNodes, this->device_deltas.end() - delta_start);

	}


}

//Apply the error
void LongTermShortTermNetwork::ApplyLongTermShortTermMemoryError(){
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), (weight_type)0);
	unsigned int lengthOfOutput = (this->mBlocksLayers[this->mBlocksLayers.size() - 1].size() * 4) + this->getNumberMemoryCells(this->mBlocksLayers.size() - 1);


	unsigned int numberOfWeightsInLayer = getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 1, MEMORY_CELL) + getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 1, POTENTIAL_MEMORY_CELL) + getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 1, FORGET_CELL) + getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 1, OUTPUT_CELL) + getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 1, INPUT_CELL);
	unsigned int numberOfWeightsInCurrentLayer = getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 2, MEMORY_CELL) + getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 2, POTENTIAL_MEMORY_CELL) + getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 2, FORGET_CELL) + getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 2, OUTPUT_CELL) + getNumberTypeWeightsInLayer(this->mBlocksLayers.size() - 2, INPUT_CELL);
	unsigned int numberMemoryCellsOutput = this->getNumberMemoryCells(this->mBlocksLayers.size() - 1);
	unsigned int numberMemoryCellsLayers = this->getNumberMemoryCells(this->mBlocksLayers.size() - 2);
	//Average the deltas
	thrust::reduce_by_key(
		thrust::make_transform_iterator(thrust::make_counting_iterator((int)0),
		_1 / this->settings.i_backprop_unrolled
		),
		thrust::make_transform_iterator(thrust::make_counting_iterator((int)0),
		_1 / this->settings.i_backprop_unrolled
		) + this->device_deltas.size() - lengthOfOutput,

		make_permutation_iterator(this->device_deltas.begin() + this->numberNonWeights,
		thrust::make_transform_iterator(thrust::make_counting_iterator((int)0),
		((_1%this->settings.i_backprop_unrolled) * this->numberOfNodes) + (_1/this->settings.i_backprop_unrolled))
		),

		thrust::make_discard_iterator(),
		this->GPUPreviousOutput_Values.begin()
		);

	//Subtract the deltas from the weights from each non-output nodes
	thrust::transform(
		this->GPUWeights.begin() + this->numberNonWeights,
		this->GPUWeights.begin() + this->numberNonWeights + numberOfWeightsInCurrentLayer - numberMemoryCellsLayers,
		make_permutation_iterator(
		this->GPUPreviousOutput_Values.begin(), 
		this->GPUMapTo.begin() + this->numberNonWeights
		),
		this->GPUWeights.begin() + this->numberNonWeights,
		//Beta * (average of delta) + (weights + (weights * alpha))
		apply_error<weight_type>((weight_type)this->settings.d_alpha, (weight_type)this->settings.d_beta, (weight_type)this->settings.i_backprop_unrolled)
		);


	//Subtract the deltas from the weights of the output
	thrust::transform(
		this->GPUWeights.end() - numberOfWeightsInLayer ,
		this->GPUWeights.end() - numberMemoryCellsOutput,
		make_permutation_iterator(this->device_deltas.begin(),
		this->GPUMapTo.end() - numberOfWeightsInLayer),
		this->GPUWeights.end() - numberOfWeightsInLayer, 
		apply_error<weight_type>((weight_type)this->settings.d_alpha, (weight_type) this->settings.d_beta, (weight_type)1)
		);




}

//*********************
//Perform Functionality
//*********************

void LongTermShortTermNetwork::UnrollNetwork(int numLayers){
	vector<vector<Memory_Block>> Unrolled_Layers = vector<vector<Memory_Block>>();//Storage of the memory blocks as new layers
	this->numberOfNodes = 0;
	//Add room for the intial input values
	this->GPUOutput_values.resize(this->numberNonWeights);
	for (unsigned int i = 0; i < this->mBlocksLayers.size() - 1; i++){
		this->loadUnrolledToDevice(2, i);
	}

	//Unroll the output layer only once
	//The output layer will contain only n node (n is the number of output) and will merely sum all input passed into it
	//This makes performing analysis far easier than using a extra layer of memory cells
	this->loadUnrolledToDevice(2, this->mBlocksLayers.size()-1);
	//Expand the output container
	
	this->GPUPreviousOutput_Values.resize(this->GPUOutput_values.size() - this->numberNonWeights);


	int GPUOutput_values_size = this->GPUOutput_values.size();

	//Resize the network to contain locations for the other layer
	this->GPUOutput_values.resize(this->GPUOutput_values.size() + ((this->settings.i_backprop_unrolled - 1)*(this->GPUOutput_values.size() - this->numberNonWeights)));
	
	this->getSumPermutation();



	//Create an empty array for the current values in the network
	this->ResetSequence();
}

void LongTermShortTermNetwork::ResetSequence(){
	thrust::fill(this->GPUOutput_values.begin(), this->GPUOutput_values.end(), 0);
	thrust::fill(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), 0);
}

template <typename T>
void copyValuesToHost(int start, device_vector<T> &GPU_Vector, host_vector<T> &local_host_Vector){
	//Copy the values into the network
	thrust::copy(GPU_Vector.begin() + start, GPU_Vector.begin() + local_host_Vector.size() + start, local_host_Vector.begin());
}



void LongTermShortTermNetwork::CopyToHost(){
	//Copy the device memory to local
	this->output_bias.resize(this->GPUOutput_values.size());
	this->bias.resize(this->GPUPreviousOutput_Values.size());
	//Copy the output
	thrust::copy(this->GPUOutput_values.begin(), this->GPUOutput_values.end(), this->output_bias.begin());
	//Copy the secondary output
	thrust::copy(this->GPUPreviousOutput_Values.begin(), this->GPUPreviousOutput_Values.end(), this->bias.begin());
	int start = this->settings.i_input;
	for (int j = 0; j < this->mBlocksLayers.size(); j++){
		//Copy back the input
		for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
			copyValuesToHost<weight_type>(start, this->GPUWeights, this->mBlocksLayers[j][i].input_weights);
			start += this->mBlocksLayers[j][i].input_weights.size();
		}
		//Copy back to output
		for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
			copyValuesToHost<weight_type>(start, this->GPUWeights, this->mBlocksLayers[j][i].output_weights);
			start += this->mBlocksLayers[j][i].output_weights.size();
		}

		//Copy back to forget
		for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
			copyValuesToHost<weight_type>(start, this->GPUWeights, this->mBlocksLayers[j][i].forget_weights);
			start += this->mBlocksLayers[j][i].forget_weights.size();
		}

		//Copy back to potential memory cell
		for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
			copyValuesToHost<weight_type>(start, this->GPUWeights, this->mBlocksLayers[j][i].potential_memory_cell_value);
			start += this->mBlocksLayers[j][i].potential_memory_cell_value.size();
		}

		//Get the new memory_cell_weight
		for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
			if (this->mBlocksLayers[j][i].memory_cell_weights.size() > 0){
			this->mBlocksLayers[j][i].memory_cell_weights[0] = 0;
			}
		}
	}
}

template <typename T>
void LongTermShortTermNetwork::copyNodesToDevice(device_vector<T> &GPU_Vector, device_vector<int> &fromPosition, host_vector<T> &local_host_Vector, host_vector<int> host_from_vector){
	int GPU_VecSize = GPU_Vector.size();
	int GPUFromPos = fromPosition.size();
	GPU_Vector.resize(GPU_VecSize + local_host_Vector.size());
	fromPosition.resize(GPUFromPos + local_host_Vector.size());
	//Copy the values into the network
	thrust::copy(local_host_Vector.begin(), local_host_Vector.end(), GPU_Vector.begin() + GPU_VecSize);
	thrust::copy(host_from_vector.begin(), host_from_vector.end(), fromPosition.begin() + GPUFromPos);
}

//Copies only the input and not the device
template <typename T>
void LongTermShortTermNetwork::specialCopyToNodes(int start_output, int number_output, device_vector<T> &GPUWeightVector, device_vector<int> &toPosition, device_vector<int> &fromPosition, host_vector<T> &local_weights, host_vector<int> map){
	int GPU_VecSize = GPUWeightVector.size();
	int GPUFromPos = fromPosition.size();

	//We need to store the number a special copy of a map, such that it has input from both the input of the sequence and the output of the previous layer
	GPUWeightVector.resize(GPU_VecSize + local_weights.size());
	fromPosition.resize(GPUFromPos + map.size());
	thrust::copy(map.begin(), map.end(), fromPosition.begin() + GPUFromPos);
	thrust::copy(local_weights.begin(), local_weights.end(), GPUWeightVector.begin() + GPU_VecSize);
	GPU_VecSize = GPU_VecSize + local_weights.size();

	for (unsigned int i = start_output; i < start_output + number_output; i++){
		fromPosition.push_back(i);
		GPUWeightVector.push_back(1);
	}

	toPosition.resize(fromPosition.size());
	thrust::fill(toPosition.end() - (local_weights.size() + number_output), toPosition.end(), this->GPUOutput_values.size());

}

void LongTermShortTermNetwork::loadUnrolledToDevice(int type_of_row, unsigned int j){
	//We need to keep track of the end of the number of inputs in order to add in a connection to the outputs for the next level
	unsigned int start_output_position = 0;
	unsigned int number_output_to_add = 0;
	unsigned int* input_nodes = new unsigned int[this->mBlocksLayers[j].size() * 3];
	host_vector<int> memory_cell_from = host_vector<int>(4);
	host_vector<weight_type> memory_cell_weights = host_vector<weight_type>(4);
	memory_cell_weights[0] = 1;//From the input
	memory_cell_weights[1] = 1;//From the potential
	memory_cell_weights[2] = 1;//From the forget
	memory_cell_weights[3] = 1;//From itself

	if (type_of_row == 0){//Is not an output row
		number_output_to_add = this->mBlocksLayers[j].size();
		for (unsigned int i = 0; i < this->mBlocksLayers[j].size(); i++){
			start_output_position = this->mBlocksLayers[j][i].input_weights.size();
		}



		//Increment it by the input numbers
		start_output_position += this->settings.i_input - 1;
	}

	//Set all the input values
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		if (this->mBlocksLayers[j][i].getTypeOfMemoryBlock() == Memory_Block::LAYER){
			specialCopyToNodes<weight_type>(start_output_position, number_output_to_add, this->GPUWeights, this->GPUMapTo, this->GPUMapFrom, this->mBlocksLayers[j][i].input_weights, this->mBlocksLayers[j][i].mapFrom);
			input_nodes[i] = this->GPUOutput_values.size();//Get the position of an input node
			if (type_of_row == 2){
				this->numberOfNodes++;
			}
			this->GPUOutput_values.push_back(0);
		}
	}


	//Set all the outputs
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		if (this->mBlocksLayers[j][i].getTypeOfMemoryBlock() == Memory_Block::LAYER){
			specialCopyToNodes<weight_type>(start_output_position, number_output_to_add, this->GPUWeights, this->GPUMapTo, this->GPUMapFrom, this->mBlocksLayers[j][i].output_weights, this->mBlocksLayers[j][i].mapFrom);
			//Add a connection to the memory cell
			this->weights.push_back(1);//Push back a 1 for the multiplication value
			input_nodes[i + this->mBlocksLayers[j].size()] = this->GPUOutput_values.size() + (this->mBlocksLayers[j].size() * 2) + this->mBlocksLayers[j].size();//Store the position of the memory cell for the forget node
			this->GPUMapFrom.push_back(input_nodes[i + this->mBlocksLayers[j].size()]);//Push back connection to the memory cell
			this->GPUMapTo.push_back(this->GPUOutput_values.size());
			this->GPUWeights.push_back((weight_type)1);
			if (type_of_row == 2){
				this->numberOfNodes++;
			}
			this->GPUOutput_values.push_back(0);
		}
	}

	//Set all the forget nodes
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		if (this->mBlocksLayers[j][i].getTypeOfMemoryBlock() == Memory_Block::LAYER){
			if (type_of_row == 2){
				this->numberOfNodes++;
			}

			specialCopyToNodes<weight_type>(start_output_position, number_output_to_add, this->GPUWeights, this->GPUMapTo, this->GPUMapFrom, this->mBlocksLayers[j][i].forget_weights, this->mBlocksLayers[j][i].mapFrom);
			this->weights.push_back(1);//Push back a 1 for the multiplication value
			this->GPUMapFrom.push_back(input_nodes[i + this->mBlocksLayers[j].size()]);//Push back connection to the memory cell
			this->GPUMapTo.push_back(this->GPUOutput_values.size());
			this->GPUWeights.push_back((weight_type)1);
			input_nodes[i + this->mBlocksLayers[j].size()] = this->GPUOutput_values.size();//Set position of the forget blocks
			this->GPUOutput_values.push_back(0);
		}
	}

	//Set all the potential_output nodes
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		
		input_nodes[i + (this->mBlocksLayers[j].size() * 2)] = this->GPUOutput_values.size();//Store the position of the 
		
		if (type_of_row == 2){
			this->numberOfNodes++;
		}

		specialCopyToNodes<weight_type>(start_output_position, number_output_to_add, this->GPUWeights, this->GPUMapTo, this->GPUMapFrom, this->mBlocksLayers[j][i].potential_memory_cell_value, this->mBlocksLayers[j][i].mapFrom);
		
		this->GPUOutput_values.push_back(0);
	}

	//Set the values of the Memory Cells
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		if (this->mBlocksLayers[j][i].getTypeOfMemoryBlock() == Memory_Block::LAYER){
			memory_cell_from[0] = input_nodes[i];//Get input in
			memory_cell_from[1] = input_nodes[i + this->mBlocksLayers[j].size()];//The potential input
			memory_cell_from[2] = input_nodes[i + (this->mBlocksLayers[j].size() * 2)];
			memory_cell_from[3] = this->GPUOutput_values.size();//itself

			if (type_of_row == 2){
				this->numberOfNodes++;
			}

			specialCopyToNodes<weight_type>(start_output_position, number_output_to_add, this->GPUWeights, this->GPUMapTo, this->GPUMapFrom, memory_cell_weights, memory_cell_from);
			this->GPUOutput_values.push_back(this->mBlocksLayers[j][i].memory_cell_weights[0]);
			this->GPUPreviousOutput_Values.push_back(this->mBlocksLayers[j][i].memory_cell_weights[0]);
		}
	}

	free(input_nodes);
}


void LongTermShortTermNetwork::loadLayerToDevice(unsigned int j){

	//Add a place for the input
	if (j == 0){
		this->GPUOutput_values.push_back(0);
		this->GPUPreviousOutput_Values.push_back(0);
	}

	//Set all the input values
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		copyNodesToDevice<weight_type>(this->GPUWeights, this->GPUMapFrom, this->mBlocksLayers[j][i].input_weights, this->mBlocksLayers[j][i].mapFrom);
		this->GPUMapTo.resize(this->GPUMapTo.size() + this->mBlocksLayers[j][i].input_weights.size());
		thrust::fill(this->GPUMapTo.end() - this->mBlocksLayers[j][i].input_weights.size(), this->GPUMapTo.end(), this->GPUOutput_values.size());
		this->GPUOutput_values.push_back(0);
	}


	//Set all the outputs
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		copyNodesToDevice<weight_type>(this->GPUWeights, this->GPUMapFrom, this->mBlocksLayers[j][i].output_weights, this->mBlocksLayers[j][i].mapFrom);
		this->GPUMapTo.resize(this->GPUMapTo.size() + this->mBlocksLayers[j][i].output_weights.size());
		thrust::fill(this->GPUMapTo.end() - this->mBlocksLayers[j][i].output_weights.size(), this->GPUMapTo.end(), this->GPUOutput_values.size());
		this->GPUOutput_values.push_back(0);
	}

	//Set all the forget nodes
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		copyNodesToDevice<weight_type>(this->GPUWeights, this->GPUMapFrom, this->mBlocksLayers[j][i].forget_weights, this->mBlocksLayers[j][i].mapFrom);
		this->GPUMapTo.resize(this->GPUMapTo.size() + this->mBlocksLayers[j][i].forget_weights.size());
		thrust::fill(this->GPUMapTo.end() - this->mBlocksLayers[j][i].forget_weights.size(), this->GPUMapTo.end(), this->GPUOutput_values.size());
		this->GPUOutput_values.push_back(0);
	}

	//Set all the potential_output nodes
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		copyNodesToDevice<weight_type>(this->GPUWeights, this->GPUMapFrom, this->mBlocksLayers[j][i].potential_memory_cell_value, this->mBlocksLayers[j][i].mapFrom);
		this->GPUMapTo.resize(this->GPUMapTo.size() + this->mBlocksLayers[j][i].potential_memory_cell_value.size());
		thrust::fill(this->GPUMapTo.end() - this->mBlocksLayers[j][i].potential_memory_cell_value.size(), this->GPUMapTo.end(), this->GPUOutput_values.size());
		
		this->GPUOutput_values.push_back(0);
	}

	this->GPUPreviousOutput_Values.resize(this->GPUOutput_values.size());
	//Set the values of the Memory Cells
	for (int i = 0; i < this->mBlocksLayers[j].size(); i++){
		if (this->mBlocksLayers[j][i].getTypeOfMemoryBlock() == Memory_Block::LAYER){
			this->GPUOutput_values.push_back(this->mBlocksLayers[j][i].memory_cell_weights[0]);
			this->GPUPreviousOutput_Values.push_back(this->mBlocksLayers[j][i].memory_cell_weights[0]);
		}
	}
}

void LongTermShortTermNetwork::CopyToDevice(){
	this->device_deltas = device_vector<weight_type>();
	this->GPUMapTo = device_vector<int>();
	this->GPUMapFrom = device_vector<int>();
	this->GPUOutput_values = device_vector<weight_type>();
	this->GPUPreviousOutput_Values = device_vector<weight_type>();
	this->GPUWeights = device_vector<weight_type>();

	//Set the input values to 0
	this->GPUOutput_values.resize(this->numberNonWeights);
	for (unsigned int j = 0; j < this->mBlocksLayers.size(); j++){
		this->loadLayerToDevice(j);
	}

}

unsigned int LongTermShortTermNetwork::getNumberMemoryCells(unsigned int layer){
	if (layer >= this->mBlocksLayers.size()){
		throw new exception("Layer does not exist");
	}
	unsigned int memory_cell_count = 0;
	for (int i = 0; i < this->mBlocksLayers[layer].size(); i++){
		memory_cell_count += this->mBlocksLayers[layer][i].number_memory_cells;
	}
	return memory_cell_count;

}

unsigned int LongTermShortTermNetwork::getNumberWeightsInLayer(unsigned int layer){
	if (layer >= this->mBlocksLayers.size()){
		throw new exception("Layer does not exist");
	}

	unsigned int weights_count = 0;
	for (int i = 0; i < this->mBlocksLayers[layer].size(); i++){
		weights_count += this->mBlocksLayers[layer][i].input_weights.size();
		weights_count += this->mBlocksLayers[layer][i].forget_weights.size();
		weights_count += this->mBlocksLayers[layer][i].output_weights.size();
		weights_count += (this->mBlocksLayers[layer][i].number_memory_cells * 2);//2 is because there is a weight between the memory cell, the forget node, and the output node
		weights_count += (this->mBlocksLayers[layer][i]).number_memory_cells * 3; //The number of weights from the input,potential, and the forget node
	}
	return weights_count;
}

//Returns number of weights going to the type of node in the layer
unsigned int LongTermShortTermNetwork::getNumberTypeWeightsInLayer(unsigned int layer, cell_type cell){
	if (layer >= this->mBlocksLayers.size()){
		throw new exception("Layer does not exist");
	}
	unsigned int number_types = 0;

	for (int i = 0; i < this->mBlocksLayers[layer].size(); i++){
		switch (cell){
		case MEMORY_CELL:
			number_types += (this->mBlocksLayers[layer][i].number_memory_cells) * 3;
			break;
		case POTENTIAL_MEMORY_CELL:
			number_types += (this->mBlocksLayers[layer][i].potential_memory_cell_value.size());
			break;
		case FORGET_CELL:
			number_types += (this->mBlocksLayers[layer][i].forget_weights.size()) + this->mBlocksLayers[layer][i].number_memory_cells;
			break;
		case INPUT_CELL:
			number_types += this->mBlocksLayers[layer][i].input_weights.size();
			break;
		case OUTPUT_CELL:
			number_types += this->mBlocksLayers[layer][i].output_weights.size() + this->mBlocksLayers[layer][i].number_memory_cells;//+1 for the number of memory cell connections
			break;
		}

	}



	return number_types;

}
void LongTermShortTermNetwork::getSumPermutation(){
	//Create a permutation list containing a list of object
	this->positionToSum = thrust::device_vector<int>();
	this->count = thrust::device_vector<int>();
	unsigned int weights[5];
	unsigned int start = 0;
	unsigned int counter = 0;
	unsigned int length = 0;
#ifdef  _DEBUG
	vector<int> temp = vector<int>();
#endif
	for (int k = 0; k < this->mBlocksLayers.size(); k++){//For Each Layer
		weights[0] = getNumberTypeWeightsInLayer(k, INPUT_CELL);
		weights[1] = getNumberTypeWeightsInLayer(k, OUTPUT_CELL);
		weights[2] = getNumberTypeWeightsInLayer(k, FORGET_CELL);
		weights[3] = getNumberTypeWeightsInLayer(k, POTENTIAL_MEMORY_CELL);
		weights[4] = getNumberTypeWeightsInLayer(k, MEMORY_CELL);
		if (k == this->mBlocksLayers.size() - 1){
			length = (weights[0] + weights[1] + weights[2] + weights[3] + weights[4]);
			start = this->GPUMapFrom.size() - (weights[0] + weights[1] + weights[2] + weights[3] + weights[4]);
		}
		else{
			length = this->numberOfNodes;
		}

		
			for (int i = start; i < start + length ; i++){
				for (int j = i; j < start + length ; j++){
					if (this->GPUMapFrom[i] == this->GPUMapFrom[j]){
						this->positionToSum.push_back(j - start);
						this->count.push_back(counter);
#ifdef _DEBUG
						temp.push_back(j - start);
#endif
						
					}
				}
				
				counter++;
			}
			start += length;
		}
		
	

}

void  LongTermShortTermNetwork::cleanNetwork(){
	this->CopyToHost();
	//Free the used memory
	this->emptyGPUMemory();
}

void LongTermShortTermNetwork::emptyGPUMemory(){
	clear_vector::free(this->GPUMapFrom);
	clear_vector::free(this->GPUMapTo);
	clear_vector::free(this->GPUWeights);
	clear_vector::free(this->device_deltas);
	clear_vector::free(this->GPUOutput_values);
	clear_vector::free(this->GPUPreviousOutput_Values);
	clear_vector::free(this->positionToSum);
	clear_vector::free(this->count);
}
//*********************
//Misc
//*********************
void LongTermShortTermNetwork::VisualizeNetwork(){
	cout << *this;
}

ostream& LongTermShortTermNetwork::OutputNetwork(ostream& os){
	os << *this;
	return os;
}