#include "hip/hip_runtime.h"
#include "ReccurentLoops.cuh"

//*****************************
//Constructor
//*****************************
ReccurentLoops::ReccurentLoops()
{

}

ReccurentLoops::ReccurentLoops(CSettings settings){
	this->settings = settings;
	this->InitializeNetwork();
	this->checkpoint = CRecurrentCheckpoint();
	this->mainNetwork = new RecurrentNeuralNetwork(settings);

}

ReccurentLoops::ReccurentLoops(CSettings settings, int type){
	this->settings = settings;

	switch (type){
	case ReccurentLoops::RealTimeTraining:
		//Train the network using real time recurrent
		//this->train_network_RealTimeRecurrentTraining();
		break;
	case ReccurentLoops::HessianFreeOptimization:
		this->mainNetwork = new RecurrentNeuralNetwork(settings);
		break;
	case ReccurentLoops::LongTermShortTerm:
		this->mainNetwork = new LongTermShortTermNetwork(settings);
		break;
	}

	
	this->InitializeNetwork();
	this->checkpoint = CRecurrentCheckpoint();
}

ReccurentLoops::ReccurentLoops(CSettings settings, CRecurrentCheckpoint checkpoint) :ReccurentLoops(settings){
	this->checkpoint = checkpoint;

}

//*****************************
//Intialzie the Internal Requirements For Training
//*****************************

void ReccurentLoops::InitializeNetwork(){
	this->input = new weight_type*[this->settings.i_number_of_training];
	this->output = new weight_type*[this->settings.i_number_of_training];
}


//*****************************
//Reload a Network From a File
//*****************************
bool ReccurentLoops::loadNetworkFromFile(){
	
	return true;
}

//*****************************
//Convert Input Types to Required Type
//*****************************
template <typename T>
weight_type* ReccurentLoops::convert_array(T* in){
	weight_type* temp = new weight_type[this->settings.i_input];
	for (int i = 0; i < settings.i_input; i++){
		temp[i] = (weight_type)in[i];
	}
	return temp;
}

vector<RETURN_WEIGHT_TYPE> ReccurentLoops::runNetwork(int* in){

	return this->runNetwork(this->convert_array<int>(in));

}

vector<RETURN_WEIGHT_TYPE> ReccurentLoops::runNetwork(weight_type* in){
	//this->mainNetwork->InitializeRun();
	device_vector<weight_type> temp_device = this->mainNetwork->runNetwork(in);
	vector<RETURN_WEIGHT_TYPE> to_return = vector <RETURN_WEIGHT_TYPE>(temp_device.size());

	for (unsigned int i = 0; i < temp_device.size(); i++){
		to_return[i] = temp_device[i];
	}
	clear_vector::free(temp_device);

	return to_return;
}
//*****************************
//Get Data From the users file
//*****************************
bool ReccurentLoops::load_training_data_from_file(){
	for (int i = 0; i < this->settings.i_number_of_training; i++){
		this->input[i] = this->createTestInputOutput(this->settings.i_input,0);
	}
	for (int i = 0; i < this->settings.i_number_of_training; i++){
		this->output[i] = this->createTestInputOutput(this->settings.i_output, 1);
	}
	
	return true;
}

//**********************
//Training
//**********************
void ReccurentLoops::startTraining(int type){
	//Load the data from a file
	if (!load_training_data_from_file()){
		throw exception("Unable to read from file.");
	}
	
	
	switch (type){
	case ReccurentLoops::RealTimeTraining:
		//Train the network using real time recurrent
		this->train_network_RealTimeRecurrentTraining();
		break;
	case ReccurentLoops::HessianFreeOptimization:
		this->train_network_HessianFreeOptimizationTraining();
		break;
	}
}

#ifdef _DEBUG 
void ReccurentLoops::testTraining(){
	try{
		this->load_training_data_from_file();
		this->mainNetwork->InitializeTraining();
		for (int i = 0; i < this->settings.i_loops; i++){
			this->mainNetwork->StartTraining(this->input[this->checkpoint.i_number_of_loops_checkpoint], this->output[this->checkpoint.i_number_of_loops_checkpoint]);
			
			if (i%this->settings.i_number_allowed_same == 0){
				this->createCheckpoint();
			}

			this->mainNetwork->ApplyError();
			if (i%this->settings.i_number_allowed_same == 0){
				this->createCheckpoint();
			}
			
			//Apply the error
			
		
			if (i%this->settings.i_number_in_sequence == 0 && i!=0){//Reset the sequence once the sequence has finished
				
				this->mainNetwork->ResetSequence();
			}
			this->checkpoint.i_number_of_loops_checkpoint += 1;
			
		}
		try{
			this->createCheckpoint();
			/*this->mainNetwork->ResetSequence();
			for (int i = 0; i < this->settings.i_number_in_sequence; i++){
				cout << i << ") " << endl;
				thrust::device_vector<weight_type> temp = this->mainNetwork->runNetwork(this->input[i]);
				testing::outputToFile<weight_type>(temp, "results", "tests/results.txt");
			}*/
			this->createCheckpoint("RunResultsInMemory");
			this->mainNetwork->cleanNetwork();
			this->mainNetwork->InitializeRun();
			this->mainNetwork->ResetSequence();
			this->createCheckpoint("RunStart");
			for (int i = 0; i < this->settings.i_number_in_sequence; i++){
				if (i == 0){
					testing::outputArrayToFile<weight_type>(this->input[i], this->settings.i_input, "tests/results2.txt");
					testing::outputArrayToFile<weight_type>(this->output[i], this->settings.i_output, "tests/results2.txt");
				}
				testing::outputArrayToFile<weight_type>(this->input[i], this->settings.i_input, "tests/results2.txt");
				testing::outputArrayToFile<weight_type>(this->output[i], this->settings.i_output, "tests/results2.txt");
				std::vector<weight_type> temp2 = this->runNetwork(this->input[i]);
				testing::outputVectorToFile<weight_type>(temp2, "results", "tests/results2.txt");
			}
			this->createCheckpoint("RunResultsFromHost");
			this->mainNetwork->emptyGPUMemory();
		}
		catch (exception e){
			cout << e.what();
			cin.sync();
			cin.get();
			//this->mainNetwork->emptyGPUMemory();
		}

		
	}
	catch (exception e){//Edit to write the problems to file later
		cout << e.what();
		cin.sync();
		cin.get();
	}
}
#endif

bool ReccurentLoops::train_network_HessianFreeOptimizationTraining(){
	this->mainNetwork->InitializeTraining();
	do{
		this->mainNetwork->StartTraining(this->input[this->checkpoint.i_number_of_loops_checkpoint], this->output[this->checkpoint.i_number_of_loops_checkpoint]);
		if (this->checkpoint.i_number_of_loops % this->settings.i_loops == 0){
			this->mainNetwork->VisualizeNetwork();
			this->mainNetwork->ApplyError();//Apply the error gained from the last steps
			this->mainNetwork->CopyToHost();
			this->mainNetwork->VisualizeNetwork();
			this->createCheckpoint();
			this->mainNetwork->ResetSequence();
		}

		this->checkpoint.i_number_of_loops_checkpoint++;
		this->checkpoint.i_number_of_loops++;
	} while (checkpoint.i_number_of_loops_checkpoint < this->settings.i_number_of_training);
	this->mainNetwork->cleanNetwork();
	return true;
}

bool ReccurentLoops::train_network_RealTimeRecurrentTraining(){
	return true;
}

//*********************
//DEBUG FUNCTIONS
//*********************

//Creates a test input/output
weight_type* ReccurentLoops::createTestInputOutput(int numberOfInput, int input_output){
	static int position = 0;
	weight_type* temp = new weight_type[numberOfInput];
	weight_type count = .1;
	for (int i = position; i < position + numberOfInput; i++){

		if (input_output == 0){
			temp[i - position] = (weight_type)(i%this->settings.i_number_in_sequence) + 1;
		}
		else{
			temp[i - position] = (weight_type)(.1*(i%this->settings.i_number_in_sequence)) + .1 + count;
		}
		count += .1;
	}
	position += numberOfInput;
	return  temp;
}

//Create a checkpoint with the network name
//Default function
void ReccurentLoops::createCheckpoint(){
	this->createCheckpoint(this->settings.s_network_name);
}

//Create a Checkpoint with any name
void ReccurentLoops::createCheckpoint(string file_name){
	static int count = 0 ;

		std::ofstream outputfile;
		outputfile.open("recurrent_networks/" + file_name + std::to_string(count) + ".txt", ios::trunc);
		if (outputfile.is_open()){
			outputfile << *this << flush;
			outputfile << endl;
			outputfile.close();
		}
		else{
			std::cout << "Unable to write checkpoint to file." << endl;
			std::cout << "continue?";
		}
		
		count++;
		

	

}