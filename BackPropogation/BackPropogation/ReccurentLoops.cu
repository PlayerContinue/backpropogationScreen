#include "hip/hip_runtime.h"
#include "ReccurentLoops.cuh"
//#define TESTING
#ifndef SEQUENCE_DELIMITER
#define SEQUENCE_DELIMITER 4
#endif
//*****************************
//Constructor
//*****************************
ReccurentLoops::ReccurentLoops()
{

}

ReccurentLoops::ReccurentLoops(CSettings settings){
	this->settings = settings;
	this->InitializeNetwork();
	this->checkpoint = CRecurrentCheckpoint();
	this->mainNetwork = new LongTermShortTermNetwork(settings, true);
	this->loadCheckpoint();
}

ReccurentLoops::ReccurentLoops(CSettings settings, int type){
	this->settings = settings;

	switch (type){
	case ReccurentLoops::RealTimeTraining:
		//Train the network using real time recurrent
		//this->train_network_RealTimeRecurrentTraining();
		break;
	case ReccurentLoops::HessianFreeOptimization:
		this->mainNetwork = new RecurrentNeuralNetwork(settings);
		break;
	case ReccurentLoops::LongTermShortTerm:
		this->mainNetwork = new LongTermShortTermNetwork(settings);
		break;
	}

	
	this->InitializeNetwork();
	this->checkpoint = CRecurrentCheckpoint(settings);
}

ReccurentLoops::ReccurentLoops(CSettings settings, CRecurrentCheckpoint checkpoint){
	this->settings = settings;
	this->checkpoint = checkpoint;
	this->mainNetwork = new LongTermShortTermNetwork(settings,true);
	this->InitializeNetwork();
}

//*****************************
//Intialzie the Internal Requirements For Training
//*****************************

void ReccurentLoops::InitializeNetwork(){
	this->input = new weight_type*[this->settings.i_number_of_training];
	this->output = new weight_type*[this->settings.i_number_of_training];
	this->inputfile = new std::fstream();
	this->outputfile = new std::fstream();
	this->inputfile->open(this->settings.s_trainingSet);
	this->outputfile->open(this->settings.s_outputTrainingFile);
	this->LoadTrainingSet();
}



//*****************************
//Reload a Network From a File
//*****************************
bool ReccurentLoops::loadNetworkFromFile(){
	
	return true;
}

void ReccurentLoops::loadCheckpoint(){
	std::ifstream is;
	is.open(this->settings.s_checkpoint_file);
	is >> this->checkpoint;
	
	this->mainNetwork->LoadNetwork(is);
}

//*****************************
//Convert Input Types to Required Type
//*****************************
template <typename T>
weight_type* ReccurentLoops::convert_array(T* in){
	weight_type* temp = new weight_type[this->settings.i_input];
	for (int i = 0; i < settings.i_input; i++){
		temp[i] = (weight_type)in[i];
	}
	return temp;
}

vector<RETURN_WEIGHT_TYPE> ReccurentLoops::runNetwork(int* in){

	return this->runNetwork(this->convert_array<int>(in));

}

vector<RETURN_WEIGHT_TYPE> ReccurentLoops::runNetwork(weight_type* in){
	device_vector<weight_type> temp_device;
	if (this->checkpoint.b_still_running){
		temp_device = this->mainNetwork->runNetwork(in, NetworkBase::run_type::WITH_MEMORY_CELLS);
	}
	else{
		temp_device = this->mainNetwork->runNetwork(in,NetworkBase::run_type::WITHOUT_MEMORY_CELLS);
	}
	
	vector<RETURN_WEIGHT_TYPE> to_return = vector <RETURN_WEIGHT_TYPE>(temp_device.size());

	for (unsigned int i = 0; i < temp_device.size(); i++){
		to_return[i] = temp_device[i];
	}
	clear_vector::free(temp_device);

	return to_return;
}
//*****************************
//Get Data From the users file
//*****************************
bool ReccurentLoops::load_training_data_from_file(){
#ifdef TESTING 
	for (int i = 0; i < this->settings.i_number_of_training; i++){
		this->input[i] = this->createTestInputOutput(this->settings.i_input,0);

		testing::outputArrayToFile(this->input[i], this->settings.i_input, "tests/inout.txt");

	}
#endif
#ifdef TESTING 
	for (int i = 0; i < this->settings.i_number_of_training; i++){
		this->output[i] = this->createTestInputOutput(this->settings.i_output, 1);

		testing::outputArrayToFile(this->output[i], this->settings.i_output, "tests/inout.txt");

	}
#endif
	return true;
}

//file - the file to read from
//length_of_results - maximum length of the array
//storage - array to contain the results
//sequence_length - [0] = length of the current sequence, [1] - if the sequence is longer than storage returns 0, else returns 1
//type - the type of data which should be retrieved
void ReccurentLoops::loadFromFile(std::fstream &file, int length_of_results, double** storage, int sequence_length[2], data_type type){

	//reset sequence length
	sequence_length[0] = 0;

	char individual_delimiter = (char)37;
	char group_delimiter = (char)30;
	char sequence_delimiter = (char)SEQUENCE_DELIMITER;
	char current_char = '1';
	string current_string = "";
	int start = file.tellg();
	if (file.is_open()){
		int letterPosition = 0;
		for (int i = 0; i < this->settings.i_number_of_training;i++){
			//Reset Everything
			storage[i] = new weight_type[length_of_results];
			current_char = '1';
			current_string = "";
			letterPosition = 0;

			//While not at the end of a group, retrieve the current dataset
			while (current_char != group_delimiter && !file.eof() && current_char!=sequence_delimiter){
				//Get the current char
				current_char = file.get();

				if (current_char == individual_delimiter){//Reached the end of the current set
					if (type == OUTPUT){
						storage[i][letterPosition] = stod(current_string);
					}
					else if(type==INPUT){
						storage[i][letterPosition] = (weight_type)current_string.at(0);
					}
					letterPosition++;
					current_string = "";
				}
				else{//String has not ended and is still correct
					current_string += current_char;
				}
			}
			//Increment Length Of String
			sequence_length[0]++;
			if (current_char == sequence_delimiter){
				i++;//Increment i and store a special sequence which is used to represent a new sequence
				storage[i] = new weight_type[length_of_results];
				for (int j = 0; j < length_of_results; j++){
					storage[i][j] = SEQUENCE_DELIMITER;
				}
			}
		}
		//Get current location in file
		int currentPosition = file.tellg();
		if (file.eof()){
			sequence_length[1] = -1;//File has ended
		}
		else if (current_char == sequence_delimiter){
			sequence_length[1] = 0;//The sequence has ended
		}
		else{
			sequence_length[1] = 1;//There is more to the sequence
		}
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){
			exit(0);
		}
	}
}

void ReccurentLoops::LoadTrainingSet(){
	this->training_input = new weight_type*[this->settings.i_number_of_training];
	this->training_output = new weight_type*[this->settings.i_number_of_training];
	int training_length[2];
	std::fstream stream;
	if (this->settings.b_testingFromFile){//A training file has been included and should be read from for the training set
		stream.open(this->settings.s_testSet);
		this->loadFromFile(stream, this->settings.i_number_of_training, this->training_input, training_length,INPUT);
		stream.close();
		stream.open(this->settings.s_outputTestSet);
		this->loadFromFile(stream, this->settings.i_number_of_training, this->training_output, training_length, OUTPUT);
		stream.close();
		this->number_in_training_sequence = training_length[0];
	}
	else{//A training file has not been included, get a random set from the input file
		stream.open(this->settings.s_trainingSet);
		this->loadFromFile(stream, this->settings.i_number_of_training, this->training_input, training_length, INPUT);
		stream.close();
		stream.open(this->settings.s_outputTrainingFile);
		this->loadFromFile(stream, this->settings.i_number_of_training, this->training_output, training_length, OUTPUT);
		stream.close();
		this->number_in_training_sequence = training_length[0];
	}
}

//**********************
//Training
//**********************
void ReccurentLoops::startTraining(int type){
	//Load the data from a file
	//if (!load_training_data_from_file()){
		//throw exception("Unable to read from file.");
	//}
	this->inputfile->open(this->settings.s_outputTrainingFile);
	this->outputfile->open(this->settings.s_outputTestSet);
	if (this->inputfile->is_open() && this->outputfile->is_open()){
		switch (type){
		case ReccurentLoops::RealTimeTraining:
			//Train the network using real time recurrent
			this->train_network_RealTimeRecurrentTraining();
			break;
		case ReccurentLoops::HessianFreeOptimization:
			this->train_network_HessianFreeOptimizationTraining();
			break;
		case ReccurentLoops::LongTermShortTerm:
			this->testTraining();
			break;
		}
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){
			exit(0);
		}
	}
}


void ReccurentLoops::testTraining(){
	weight_type** trainingInput = new weight_type*[this->settings.i_backprop_unrolled];
	weight_type** trainingOutput = new weight_type*[this->settings.i_backprop_unrolled];
	int length[2];
	bool sequence_end = false;//Tell if the sequence ends
	
	try{
		this->load_training_data_from_file();
		if (!this->checkpoint.b_still_running){
			this->mainNetwork->InitializeTraining();
		}
		this->checkpoint.b_still_running = true;
		this->createCheckpoint("Initial Checkpoint");
		while (length[1] != -1){

			this->loadFromFile(*(this->outputfile), this->settings.i_output, this->output, length, OUTPUT);
			this->loadFromFile(*(this->inputfile), this->settings.i_input, this->input, length, INPUT);
			//testing::outputArrayToFile(this->input, length[0], this->settings.i_input, "tests/input_output.txt");
			//testing::outputArrayToFile(this->output, length[0], this->settings.i_output, "tests/input_output.txt");
			for (int i = 0; i < this->settings.i_number_of_training; i += this->settings.i_backprop_unrolled){

				for (int j = i, k = 0; k < this->settings.i_backprop_unrolled; j++, k++){
					if (!sequence_end && (this->input[j][0] != SEQUENCE_DELIMITER || this->output[j][0] != SEQUENCE_DELIMITER)){//If both are a sequence_delimiter, then the sequence has ended
						trainingInput[k] = this->input[j];
						trainingOutput[k] = this->output[j];
					}
					else{
						sequence_end = true;
						if (k < this->settings.i_backprop_unrolled){
							trainingInput[k] = this->input[0];
							trainingOutput[k] = this->output[0];
						}
						//Reached the end of the sequence, load the next sequence
						//Normally load more from the file
						//break;
					}
				}
				this->mainNetwork->StartTraining(trainingInput, trainingOutput);

				if (this->checkpoint.i_number_of_loops_checkpoint%this->settings.i_number_allowed_same == 0){
					this->createCheckpoint();
				}
				//Apply the error
				this->mainNetwork->ApplyError();
				if (this->checkpoint.i_number_of_loops_checkpoint%this->settings.i_number_allowed_same == 0){
					this->createCheckpoint();
				}




				
				this->checkpoint.i_number_of_loops_checkpoint += 1;


				if (sequence_end){
					//The sequence has ended, so we need to reset the sequence
					this->mainNetwork->ResetSequence();
					sequence_end = false;
				}

			}
			if (length[1] == 0){//Reset the sequence once the sequence has finished

				this->mainNetwork->ResetSequence();
			}
			//Load more data from the file
			
			
		}
		//No longer running loops
		
		try{
			this->createCheckpoint();
			this->mainNetwork->ResetSequence();
			for (int i = 0; i < this->settings.i_number_of_training; i++){
				if (i == 0){
					testing::outputArrayToFile<weight_type>(this->training_input[i], this->settings.i_input, "tests/results.txt");
					testing::outputArrayToFile<weight_type>(this->training_output[i], this->settings.i_output, "tests/results.txt");
				}
				if (this->training_input[i][0] != SEQUENCE_DELIMITER || this->training_output[i][0] != SEQUENCE_DELIMITER){
					testing::outputArrayToFile<weight_type>(this->training_input[i], this->settings.i_input, "tests/results.txt");
					testing::outputArrayToFile<weight_type>(this->training_output[i], this->settings.i_output, "tests/results.txt");
					thrust::device_vector<weight_type> temp = this->mainNetwork->runNetwork(this->training_input[i]);
					testing::outputToFile<weight_type>(temp, "results", "tests/results.txt");
				}
				else{
					this->mainNetwork->ResetSequence();
				}
			}
			this->checkpoint.b_still_running = false;
			this->createCheckpoint("RunResultsInMemory");
			this->mainNetwork->cleanNetwork();
			this->mainNetwork->InitializeRun();
			this->mainNetwork->ResetSequence();
			this->createCheckpoint("RunStart");
			for (int i = 0; i < this->settings.i_number_of_training; i++){
				if (i == 0){
					testing::outputArrayToFile<weight_type>(this->training_input[i], this->settings.i_input, "tests/results2.txt");
					testing::outputArrayToFile<weight_type>(this->training_output[i], this->settings.i_output, "tests/results2.txt");
				}
				if (this->training_input[i][0] != SEQUENCE_DELIMITER || this->training_output[i][0] != SEQUENCE_DELIMITER){
					testing::outputArrayToFile<weight_type>(this->training_input[i], this->settings.i_input, "tests/results2.txt");
					testing::outputArrayToFile<weight_type>(this->training_output[i], this->settings.i_output, "tests/results2.txt");
					std::vector<weight_type> temp2 = this->runNetwork(this->training_input[i]);
					testing::outputVectorToFile<weight_type>(temp2, "results", "tests/results2.txt");
				}
				else{
					this->mainNetwork->ResetSequence();
				}
			}
			this->createCheckpoint("RunResultsFromHost");
			this->mainNetwork->emptyGPUMemory();
		}
		catch (exception e){
			cout << e.what();
			cin.sync();
			cin.get();
			//this->mainNetwork->emptyGPUMemory();
		}

		
	}

	catch (exception e){//Edit to write the problems to file later
		cout << e.what();
		cin.sync();
		cin.get();
	}
}


bool ReccurentLoops::train_network_HessianFreeOptimizationTraining(){
	this->mainNetwork->InitializeTraining();
	do{
		this->mainNetwork->StartTraining(this->input[this->checkpoint.i_number_of_loops_checkpoint], this->output[this->checkpoint.i_number_of_loops_checkpoint]);
		if (this->checkpoint.i_number_of_loops % this->settings.i_loops == 0){
			this->mainNetwork->VisualizeNetwork();
			this->mainNetwork->ApplyError();//Apply the error gained from the last steps
			this->mainNetwork->CopyToHost();
			this->mainNetwork->VisualizeNetwork();
			this->createCheckpoint();
			this->mainNetwork->ResetSequence();
		}

		this->checkpoint.i_number_of_loops_checkpoint++;
		this->checkpoint.i_number_of_loops++;
	} while (checkpoint.i_number_of_loops_checkpoint < this->settings.i_number_of_training);
	this->mainNetwork->cleanNetwork();
	return true;
}

bool ReccurentLoops::train_network_RealTimeRecurrentTraining(){
	return true;
}

//*********************
//DEBUG FUNCTIONS
//*********************

//Creates a test input/output
weight_type* ReccurentLoops::createTestInputOutput(int numberOfInput, int input_output){
	static int position = 0;
	static int previous = input_output;
	if (input_output != previous){
		position = 0;
		previous = input_output;
	}
	if (input_output == 0 && position >= this->settings.i_number_in_sequence){
		position = 0;
	}
	else if (position >= this->settings.i_number_in_sequence){
		position = 0;
	}
	weight_type* temp = new weight_type[numberOfInput];
	weight_type count = .1;
	for (int i = 0; i < numberOfInput; i++){

		if (input_output == 0){
			temp[i] = (weight_type)(position);
		}
		else{
			temp[i] = (weight_type)(.01*(position));
		}
		count += .1;
		position++;
	}
	
	return  temp;
}

void ReccurentLoops::cleanLoops(){
	for (int i = 0; i < this->settings.i_number_of_training; i++){
		std::free(this->input[i]);
		std::free(this->output[i]);
		std::free(this->training_input[i]);
		std::free(this->training_output[i]);
	}
	std::free(this->input);
	std::free(this->output);
	std::free(this->training_input);
	std::free(this->training_output);
}

//Create a checkpoint with the network name
//Default function
void ReccurentLoops::createCheckpoint(){
	this->createCheckpoint(this->settings.s_network_name);
}

//Create a Checkpoint with any name
void ReccurentLoops::createCheckpoint(string file_name){
	static int count = 0 ;

		std::ofstream outputfile;
		outputfile.open("recurrent_networks/" + file_name + std::to_string(count) + ".txt", ios::trunc);
		if (outputfile.is_open()){
			outputfile << *this << flush;
			outputfile << endl;
			outputfile.close();
		}
		else{
			std::cout << "Unable to write checkpoint to file." << endl;
			std::cout << "continue?";
		}
		
		count++;
		

	

}