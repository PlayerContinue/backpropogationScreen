#include "hip/hip_runtime.h"
#include "ReccurentLoops.cuh"
//#define TESTING
#ifndef SEQUENCE_DELIMITER
#define SEQUENCE_DELIMITER 4
#endif
//*****************************
//Constructor
//*****************************
ReccurentLoops::ReccurentLoops()
{

}

ReccurentLoops::ReccurentLoops(CSettings settings){
	this->settings = settings;
	this->InitializeNetwork();
	this->checkpoint = CRecurrentCheckpoint();
	this->mainNetwork = new LongTermShortTermNetwork(settings, true);
	this->loadCheckpoint();
}

ReccurentLoops::ReccurentLoops(CSettings settings, int type){
	this->settings = settings;

	switch (type){
	case ReccurentLoops::RealTimeTraining:
		//Train the network using real time recurrent
		//this->train_network_RealTimeRecurrentTraining();
		break;
	case ReccurentLoops::HessianFreeOptimization:
		this->mainNetwork = new RecurrentNeuralNetwork(settings);
		break;
	case ReccurentLoops::LongTermShortTerm:
		this->mainNetwork = new LongTermShortTermNetwork(settings);
		break;
	}


	this->InitializeNetwork();
	this->checkpoint = CRecurrentCheckpoint(settings);
}

ReccurentLoops::ReccurentLoops(CSettings settings, CRecurrentCheckpoint checkpoint){
	this->settings = settings;
	this->checkpoint = checkpoint;
	this->mainNetwork = new LongTermShortTermNetwork(settings, true);
	this->InitializeNetwork();
}

//*****************************
//Intialzie the Internal Requirements For Training
//*****************************

void ReccurentLoops::InitializeNetwork(){
	this->input = new weight_type*[this->settings.i_number_of_training];
	this->output = new weight_type*[this->settings.i_number_of_training];
	this->mean_square_error_results_new = host_vector<weight_type>(this->settings.i_output + 1);
	this->mean_square_error_results_old = host_vector<weight_type>(this->settings.i_output + 1);
	this->inputfile = new std::fstream();
	this->outputfile = new std::fstream();
	this->inputfile->open(this->settings.s_trainingSet);
	this->outputfile->open(this->settings.s_outputTrainingFile);
	this->LoadTrainingSet();
}



//*****************************
//Reload a Network From a File
//*****************************
bool ReccurentLoops::loadNetworkFromFile(){

	return true;
}

void ReccurentLoops::loadCheckpoint(){
	std::ifstream is;
	is.open(this->settings.s_checkpoint_file);
	is >> this->checkpoint;

	this->mainNetwork->LoadNetwork(is);
}

//*****************************
//Convert Input Types to Required Type
//*****************************
template <typename T>
weight_type* ReccurentLoops::convert_array(T* in){
	weight_type* temp = new weight_type[this->settings.i_input];
	for (int i = 0; i < settings.i_input; i++){
		temp[i] = (weight_type)in[i];
	}
	return temp;
}

vector<RETURN_WEIGHT_TYPE> ReccurentLoops::runNetwork(int* in){

	return this->runNetwork(this->convert_array<int>(in));

}

vector<RETURN_WEIGHT_TYPE> ReccurentLoops::runNetwork(weight_type* in){

	device_vector<weight_type> temp_device = this->runTrainingNetwork(in);
	vector<RETURN_WEIGHT_TYPE> to_return = vector<RETURN_WEIGHT_TYPE>(temp_device.size());


	for (unsigned int i = 0; i < temp_device.size(); i++){
		to_return[i] = temp_device[i];
	}
	clear_vector::free(temp_device);

	return to_return;
}

//Run the network during training
device_vector<weight_type> ReccurentLoops::runTrainingNetwork(weight_type* in){
	device_vector<weight_type> temp_device;
	if (this->checkpoint.b_still_running){
		temp_device = this->mainNetwork->runNetwork(in, NetworkBase::run_type::WITH_MEMORY_CELLS);
	}
	else{
		temp_device = this->mainNetwork->runNetwork(in, NetworkBase::run_type::WITHOUT_MEMORY_CELLS);
	}

	return temp_device;

}
//*****************************
//Get Data From the users file
//*****************************
bool ReccurentLoops::load_training_data_from_file(){
#ifdef TESTING 
	for (int i = 0; i < this->settings.i_number_of_training; i++){
		this->input[i] = this->createTestInputOutput(this->settings.i_input,0);

		testing::outputArrayToFile(this->input[i], this->settings.i_input, "tests/inout.txt");

	}
#endif
#ifdef TESTING 
	for (int i = 0; i < this->settings.i_number_of_training; i++){
		this->output[i] = this->createTestInputOutput(this->settings.i_output, 1);

		testing::outputArrayToFile(this->output[i], this->settings.i_output, "tests/inout.txt");

	}
#endif
	return true;
}

//file - the file to read from
//length_of_results - maximum length of the array
//storage - array to contain the results
//sequence_length - [0] = length of the current sequence, [1] - if the sequence is longer than storage returns 0, else returns 1
//type - the type of data which should be retrieved
void ReccurentLoops::loadFromFile(std::fstream &file, int length_of_results, double** storage, int sequence_length[2], data_type type, bool first_run){
	this->loadFromFile(file, length_of_results, storage, sequence_length,this->settings.i_number_of_training, type,first_run);
}

void ReccurentLoops::loadFromFile(std::fstream &file, int length_of_results, double** storage, int sequence_length[2], int length, data_type type,bool first_run){

	//reset sequence length
	sequence_length[0] = 0;

	char individual_delimiter = (char)37;
	char group_delimiter = (char)30;
	char sequence_delimiter = (char)SEQUENCE_DELIMITER;
	char current_char = '1';
	string current_string = "";
	int start = file.tellg();
	if (file.is_open()){
		int letterPosition = 0;
		for (int i = 0; i < length && !file.eof(); i++){
			//Reset Everything
			if (first_run){
				storage[i] = new weight_type[length_of_results];
			}
			current_char = '1';
			current_string = "";
			letterPosition = 0;

			//While not at the end of a group, retrieve the current dataset
			while (current_char != group_delimiter && !file.eof() && current_char != sequence_delimiter){
				//Get the current char
				current_char = file.get();

				if (current_char == individual_delimiter){//Reached the end of the current set
					if (type == OUTPUT){
						storage[i][letterPosition] = stod(current_string);
					}
					else if (type == INPUT){
						storage[i][letterPosition] = (weight_type)stod(current_string);
					}
					letterPosition++;
					current_string = "";
				}
				else{//String has not ended and is still correct
					current_string += current_char;
				}
			}
			//Increment Length Of String
			sequence_length[0]++;
			if (current_char == sequence_delimiter || file.eof()){
				for (int j = 0; j < length_of_results; j++){
					storage[i][j] = SEQUENCE_DELIMITER;
				}
			}

			
		}
		//Get current location in file
		int currentPosition = file.tellg();
		if (file.eof()){
			if (sequence_length[0] < length){
				if (first_run){
					//storage[sequence_length[0]] = new weight_type[length_of_results];
				}
				for (int j = 0; j < length_of_results; j++){
					//storage[sequence_length[0]][j] = SEQUENCE_DELIMITER;
				}
			}
			sequence_length[1] = -1;//File has ended
		}
		else if (current_char == sequence_delimiter){
			sequence_length[1] = 0;//The sequence has ended
		}
		else{
			sequence_length[1] = 1;//There is more to the sequence
		}
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){
			exit(0);
		}
	}
}

void ReccurentLoops::LoadTrainingSet(){
	this->training_input = new weight_type*[this->settings.i_number_of_training];
	this->training_output = new weight_type*[this->settings.i_number_of_training];
	int training_length[2];
	std::fstream stream;
	if (this->settings.b_testingFromFile){//A training file has been included and should be read from for the training set
		stream.open(this->settings.s_testSet);
		this->loadFromFile(stream, this->settings.i_input, this->training_input, training_length,this->settings.i_number_of_testing_items, INPUT,true);
		stream.close();
		stream.open(this->settings.s_outputTestSet);
		this->loadFromFile(stream, this->settings.i_output, this->training_output, training_length, this->settings.i_number_of_testing_items, OUTPUT, true);
		stream.close();
		this->number_in_training_sequence = training_length[0];
	}
	else{//A training file has not been included, get a random set from the input file
		stream.open(this->settings.s_trainingSet);
		this->loadFromFile(stream, this->settings.i_input, this->training_input, training_length, this->settings.i_number_of_testing_items, INPUT, true);
		stream.close();
		stream.open(this->settings.s_outputTrainingFile);
		this->loadFromFile(stream, this->settings.i_output, this->training_output, training_length, this->settings.i_number_of_testing_items, OUTPUT, true);
		stream.close();
		this->number_in_training_sequence = training_length[0];
	}
}

//**********************
//Training
//**********************
void ReccurentLoops::startTraining(int type){
	//Load the data from a file
	//if (!load_training_data_from_file()){
	//throw exception("Unable to read from file.");
	//}
	this->inputfile->open(this->settings.s_outputTrainingFile);
	this->outputfile->open(this->settings.s_outputTestSet);
	if (this->inputfile->is_open() && this->outputfile->is_open()){
		switch (type){
		case ReccurentLoops::RealTimeTraining:
			//Train the network using real time recurrent
			this->train_network_RealTimeRecurrentTraining();
			break;
		case ReccurentLoops::HessianFreeOptimization:
			this->train_network_HessianFreeOptimizationTraining();
			break;
		case ReccurentLoops::LongTermShortTerm:
			this->testTraining();
			break;
		}
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){
			exit(0);
		}
	}
}


void ReccurentLoops::testTraining(){
	weight_type** trainingInput = new weight_type*[this->settings.i_backprop_unrolled];
	weight_type** trainingOutput = new weight_type*[this->settings.i_backprop_unrolled+1];
	int length[2];
	bool sequence_end = false;//Tell if the sequence ends
	int count_sequences = 0;
	int k = 0;
	bool first_run = true;
	int length_of_sequence = 0;
	this->mean_square_error_results_new[0] = this->settings.d_threshold + 1;
	try{
		if (!this->checkpoint.b_still_running){
			this->mainNetwork->InitializeTraining();
		}
		this->checkpoint.b_still_running = true;
		this->createCheckpoint("Initial Checkpoint");
		//Get the mean Square error
		this->getMeanSquareError();
		testing::outputToFile<weight_type>(this->mean_square_error_results_new, "new", "tests/meansquare.txt");
		this->mainNetwork->ResetSequence();
		while (length[1] != -1 && this->mean_square_error_results_new[0] > this->settings.d_threshold){

			this->loadFromFile(*(this->outputfile), this->settings.i_output, this->output, length, OUTPUT, first_run);
			this->loadFromFile(*(this->inputfile), this->settings.i_input, this->input, length, INPUT, first_run);
			first_run = false;
			
			for (int i = 0; i < length[0] && this->mean_square_error_results_new[0] > this->settings.d_threshold;){
				for (; k < this->settings.i_backprop_unrolled; k++){
					if (!sequence_end && i < length[0] && (this->input[i][0] != SEQUENCE_DELIMITER || this->output[i][0] != SEQUENCE_DELIMITER)){//If both are a sequence_delimiter, then the sequence has ended
						trainingInput[k] = this->input[i];
						if (k == 0){
							trainingOutput[k] = this->output[i];
						}
						trainingOutput[k+1] = this->output[i];
						i++;//Increment i here because the next sequence follows
					}
					else{
						//Since the sequence ended, but we have not reached the end of the backdrop, we need to add an extra layer
						trainingInput[k] = this->input[i - 1];
						trainingOutput[k+1] = this->output[i-1];
						k++;
						if (!sequence_end){
							i++;//Skip passed the end of the sequence
						}
						
						sequence_end = true;
						
						break;
					}
				}
				//Set the i_backpropunrolled of the mainNetworks settings so it only applys the information on the current sequence length
				//Allows for multilength sequences
				this->mainNetwork->seti_backprop_unrolled(k);
				if (k > 0){
					length_of_sequence += k;
					//Run the sequence to find the results
					
					this->mainNetwork->StartTraining(trainingInput, trainingOutput);
					
					this->checkpoint.i_number_of_loops_checkpoint += 1;
					
					if (this->checkpoint.i_number_of_loops_checkpoint>=this->settings.i_number_allowed_same){
						this->createCheckpoint();
					}

					
				}


				if (sequence_end){
					if (length_of_sequence > 0){
						this->mainNetwork->seti_backprop_unrolled(length_of_sequence);
						//Apply the error at the end of the sequence
						this->mainNetwork->ApplyError();
						if (this->checkpoint.i_number_of_loops_checkpoint>=this->settings.i_number_allowed_same){
							this->createCheckpoint("AfterError");
							this->checkpoint.i_number_of_loops_checkpoint = 0;
						}
						length_of_sequence = 0;
					}
					//The sequence has ended, so we need to reset the sequence
					this->mainNetwork->ResetSequence();
					if (count_sequences >= this->settings.i_loops){
						//Copy the previous set of error to the new set of errors
						std::copy(this->mean_square_error_results_new.begin(), this->mean_square_error_results_new.end(), this->mean_square_error_results_old.begin());
						
						//Get the mean Square error
						this->getMeanSquareError();
						weight_type old = this->mean_square_error_results_old[0];
						weight_type new_val = this->mean_square_error_results_new[0];
						testing::outputToFile<weight_type>(this->mean_square_error_results_new, "new", "tests/meansquare.txt");
						this->mainNetwork->ResetSequence();
						count_sequences = 0;
					}
					count_sequences++;
					sequence_end = false;
					//A new sequence, start from the beginning
					k = 0;
				}
				else{
					//The input of the initial is the beginning of it.
					trainingInput[0] = trainingInput[k-1];
					trainingOutput[0] = trainingOutput[k];
					//Continue the sequence, we need to keep the previous input
					k = 1;
				}

			}
			if (length[1] == 0){//Reset the sequence once the sequence has finished
				this->mainNetwork->ResetSequence();

			}
			//Load more data from the file


		}
		//No longer running loops

		try{
			//Copy the previous set of error to the new set of errors
			std::copy(this->mean_square_error_results_new.begin(), this->mean_square_error_results_new.end(), this->mean_square_error_results_old.begin());
			//Get the mean Square error
			this->getMeanSquareError();
			testing::outputToFile<weight_type>(this->mean_square_error_results_new, "new", "tests/meansquare.txt");
			this->mainNetwork->ResetSequence();
			this->createCheckpoint();
			this->mainNetwork->ResetSequence();
			for (int i = 0; i < this->number_in_training_sequence; i++){
				if (i == 0){
					testing::outputArrayToFile<weight_type>(this->training_input[i], this->settings.i_input, "tests/results.txt");
					testing::outputArrayToFile<weight_type>(this->training_output[i], this->settings.i_output, "tests/results.txt");
				}
				if (this->training_input[i][0] != SEQUENCE_DELIMITER || this->training_output[i][0] != SEQUENCE_DELIMITER){
					testing::outputArrayToFile<weight_type>(this->training_input[i], this->settings.i_input, "tests/results.txt");
					testing::outputArrayToFile<weight_type>(this->training_output[i], this->settings.i_output, "tests/results.txt");
					thrust::device_vector<weight_type> temp = this->mainNetwork->runNetwork(this->training_input[i], NetworkBase::run_type::WITH_MEMORY_CELLS);
					testing::outputToFile<weight_type>(temp, "results", "tests/results.txt");
				}
				else{
					this->mainNetwork->ResetSequence();
				}
			}
			this->checkpoint.b_still_running = false;
			this->createCheckpoint("RunResultsInMemory");
			this->mainNetwork->cleanNetwork();
			this->mainNetwork->InitializeRun();
			this->mainNetwork->ResetSequence();
			this->createCheckpoint("RunStart");
			for (int i = 0; i < this->number_in_training_sequence; i++){
				if (i == 0){
					testing::outputArrayToFile<weight_type>(this->training_input[i], this->settings.i_input, "tests/results2.txt");
					testing::outputArrayToFile<weight_type>(this->training_output[i], this->settings.i_output, "tests/results2.txt");
				}
				if (this->training_input[i][0] != SEQUENCE_DELIMITER || this->training_output[i][0] != SEQUENCE_DELIMITER){
					testing::outputArrayToFile<weight_type>(this->training_input[i], this->settings.i_input, "tests/results2.txt");
					testing::outputArrayToFile<weight_type>(this->training_output[i], this->settings.i_output, "tests/results2.txt");
					std::vector<weight_type> temp2 = this->runNetwork(this->training_input[i]);
					testing::outputVectorToFile<weight_type>(temp2, "results", "tests/results2.txt");
				}
				else{
					this->mainNetwork->ResetSequence();
				}
			}
			this->createCheckpoint("RunResultsFromHost");
			this->mainNetwork->emptyGPUMemory();
		}
		catch (exception e){
			cout << "error" << endl;
			cout << e.what();
			cin.sync();
			cin.get();
			//this->mainNetwork->emptyGPUMemory();
		}


	}

	catch (exception e){//Edit to write the problems to file later
		cout << "error" << endl;
		cout << e.what();
		cin.sync();
		cin.get();
	}
}

void ReccurentLoops::getMeanSquareError(){
	thrust::device_vector<weight_type> vec;
	thrust::device_vector<weight_type> real_output = thrust::device_vector<weight_type>(this->settings.i_output);
	for (int i = 0; i < this->number_in_training_sequence; i++){
		if (this->training_input[i][0] == SEQUENCE_DELIMITER && this->training_output[i][0] == SEQUENCE_DELIMITER){
			
			this->mainNetwork->ResetSequence();
			
		}
		else{
			vec = this->runTrainingNetwork(this->training_input[i]);
			for (int j = 0; j < this->settings.i_output; j++){
				real_output[j] = this->training_output[i][j];
			}

			if (i != 0){
				value_testing::getMeanSquareErrorSum(vec.begin(), vec.end(), real_output.begin(), real_output.end(), this->mean_square_error_results_new);
			}
			else{
				value_testing::getMeanSquareError(vec.begin(), vec.end(), real_output.begin(), real_output.end(), this->mean_square_error_results_new);
			}
		}
	}

	//Divide the summed value
	for (int i = 0; i < this->mean_square_error_results_new.size(); i++){
		this->mean_square_error_results_new[i] /= this->number_in_training_sequence;
	}
}

bool ReccurentLoops::train_network_HessianFreeOptimizationTraining(){
	this->mainNetwork->InitializeTraining();
	do{
		this->mainNetwork->StartTraining(this->input[this->checkpoint.i_number_of_loops_checkpoint], this->output[this->checkpoint.i_number_of_loops_checkpoint]);
		if (this->checkpoint.i_number_of_loops % this->settings.i_loops == 0){
			this->mainNetwork->VisualizeNetwork();
			this->mainNetwork->ApplyError();//Apply the error gained from the last steps
			this->mainNetwork->CopyToHost();
			this->mainNetwork->VisualizeNetwork();
			this->createCheckpoint();
			this->mainNetwork->ResetSequence();
		}

		this->checkpoint.i_number_of_loops_checkpoint++;
		this->checkpoint.i_number_of_loops++;
	} while (checkpoint.i_number_of_loops_checkpoint < this->settings.i_number_of_training);
	this->mainNetwork->cleanNetwork();
	return true;
}

bool ReccurentLoops::train_network_RealTimeRecurrentTraining(){
	return true;
}

//*********************
//DEBUG FUNCTIONS
//*********************

//Creates a test input/output
weight_type* ReccurentLoops::createTestInputOutput(int numberOfInput, int input_output){
	static int position = 0;
	static int previous = input_output;
	if (input_output != previous){
		position = 0;
		previous = input_output;
	}
	if (input_output == 0 && position >= this->settings.i_number_in_sequence){
		position = 0;
	}
	else if (position >= this->settings.i_number_in_sequence){
		position = 0;
	}
	weight_type* temp = new weight_type[numberOfInput];
	weight_type count = .1;
	for (int i = 0; i < numberOfInput; i++){

		if (input_output == 0){
			temp[i] = (weight_type)(position);
		}
		else{
			temp[i] = (weight_type)(.01*(position));
		}
		count += .1;
		position++;
	}

	return  temp;
}

void ReccurentLoops::cleanLoops(){
	for (int i = 0; i < this->settings.i_number_of_training; i++){
		std::free(this->input[i]);
		std::free(this->output[i]);
		std::free(this->training_input[i]);
		std::free(this->training_output[i]);
	}
	std::free(this->input);
	std::free(this->output);
	std::free(this->training_input);
	std::free(this->training_output);
}

//Create a checkpoint with the network name
//Default function
void ReccurentLoops::createCheckpoint(){
	this->createCheckpoint(this->settings.s_network_name);
}

//Create a Checkpoint with any name
void ReccurentLoops::createCheckpoint(string file_name){
	static int count = 0;

	std::ofstream outputfile;
	outputfile.open("recurrent_networks/" + file_name + std::to_string(count) + ".txt", ios::trunc);
	if (outputfile.is_open()){
		outputfile << *this << flush;
		outputfile << endl;
		outputfile.close();
	}
	else{
		std::cout << "Unable to write checkpoint to file." << endl;
		std::cout << "continue?";
	}

	count++;




}