#include "hip/hip_runtime.h"
#include "ReccurentLoops.cuh"

//*****************************
//Constructor
//*****************************
ReccurentLoops::ReccurentLoops()
{

}

ReccurentLoops::ReccurentLoops(CSettings settings){
	this->settings = settings;
	this->InitializeNetwork();
	this->checkpoint = CRecurrentCheckpoint();
	this->mainNetwork = new RecurrentNeuralNetwork(settings);

}

ReccurentLoops::ReccurentLoops(CSettings settings, int type){
	this->settings = settings;

	switch (type){
	case ReccurentLoops::RealTimeTraining:
		//Train the network using real time recurrent
		//this->train_network_RealTimeRecurrentTraining();
		break;
	case ReccurentLoops::HessianFreeOptimization:
		this->mainNetwork = new RecurrentNeuralNetwork(settings);
		break;
	case ReccurentLoops::LongTermShortTerm:
		this->mainNetwork = new LongTermShortTermNetwork(settings);
		break;
	}

	
	this->InitializeNetwork();
	this->checkpoint = CRecurrentCheckpoint();
}

ReccurentLoops::ReccurentLoops(CSettings settings, CRecurrentCheckpoint checkpoint) :ReccurentLoops(settings){
	this->checkpoint = checkpoint;

}

//*****************************
//Intialzie the Internal Requirements For Training
//*****************************

void ReccurentLoops::InitializeNetwork(){
	this->input = new weight_type*[this->settings.i_number_of_training];
	this->output = new weight_type*[this->settings.i_number_of_training];
}


//*****************************
//Reload a Network From a File
//*****************************
bool ReccurentLoops::loadNetworkFromFile(){
	
	return true;
}

//*****************************
//Convert Input Types to Required Type
//*****************************
template <typename T>
weight_type* ReccurentLoops::convert_array(T* in){
	weight_type* temp = new weight_type[this->settings.i_input];
	for (int i = 0; i < settings.i_input; i++){
		temp[i] = (weight_type)in[i];
	}
	return temp;
}

vector<RETURN_WEIGHT_TYPE> ReccurentLoops::runNetwork(double* in){

	return this->runNetwork(this->convert_array<double>(in));

}


vector<RETURN_WEIGHT_TYPE> ReccurentLoops::runNetwork(weight_type* in){
	this->mainNetwork->InitializeRun();
#ifdef _DEBUG
	this->createCheckpoint();
#endif
	device_vector<weight_type> temp_device = this->mainNetwork->runNetwork(in);
#ifdef _DEBUG
	this->createCheckpoint();
#endif
	vector<RETURN_WEIGHT_TYPE> to_return = vector <RETURN_WEIGHT_TYPE>(temp_device.size());

	for (unsigned int i = 0; i < temp_device.size(); i++){
		to_return[i] = temp_device[i];
	}
	this->mainNetwork->emptyGPUMemory();
	clear_vector::free(temp_device);

	return to_return;
}
//*****************************
//Get Data From the users file
//*****************************
bool ReccurentLoops::load_training_data_from_file(){
	for (int i = 0; i < this->settings.i_number_of_training; i++){
		this->input[i] = this->createTestInputOutput(this->settings.i_input,0);
		this->output[i] = this->createTestInputOutput(this->settings.i_output,1);
	}
	return true;
}

//**********************
//Training
//**********************
void ReccurentLoops::startTraining(int type){
	//Load the data from a file
	if (!load_training_data_from_file()){
		throw exception("Unable to read from file.");
	}
	
	
	switch (type){
	case ReccurentLoops::RealTimeTraining:
		//Train the network using real time recurrent
		this->train_network_RealTimeRecurrentTraining();
		break;
	case ReccurentLoops::HessianFreeOptimization:
		this->train_network_HessianFreeOptimizationTraining();
		break;
	}
}

#ifdef _DEBUG 
void ReccurentLoops::testTraining(){
	try{
		this->load_training_data_from_file();
		this->mainNetwork->InitializeTraining();
		for (int i = 0; i < this->settings.i_loops; i++){
			this->mainNetwork->StartTraining(this->input[this->checkpoint.i_number_of_loops_checkpoint], this->output[this->checkpoint.i_number_of_loops_checkpoint]);
			//Apply the error
			//this->mainNetwork->ApplyError();
			if (i%this->settings.i_number_allowed_same == 0){
				this->createCheckpoint();
			}
			this->checkpoint.i_number_of_loops_checkpoint += 1;
			
			if (i%this->settings.i_number_in_sequence == 0){//Reset the sequence once the sequence has finished
				this->mainNetwork->ResetSequence();
			}
			
		}
		try{
			this->createCheckpoint();
			this->mainNetwork->cleanNetwork();
			//this->runNetwork(this->input[0]);

		}
		catch (exception e){
			this->mainNetwork->emptyGPUMemory();
		}

		
	}
	catch (exception e){//Edit to write the problems to file later
		cout << e.what();
		cin.sync();
		cin.get();
	}
}
#endif

bool ReccurentLoops::train_network_HessianFreeOptimizationTraining(){
	//this->mainNetwork->addNeuron(1);
	//this->mainNetwork->VisualizeNetwork();
	//this->mainNetwork->addNeuron(2);
	//this->mainNetwork->VisualizeNetwork();

	//this->mainNetwork->addWeight(5);
	//this->mainNetwork->VisualizeNetwork();
	this->mainNetwork->InitializeTraining();
	do{
		this->mainNetwork->StartTraining(this->input[this->checkpoint.i_number_of_loops_checkpoint], this->output[this->checkpoint.i_number_of_loops_checkpoint]);
		if (this->checkpoint.i_number_of_loops % this->settings.i_loops == 0){
			this->mainNetwork->VisualizeNetwork();
			this->mainNetwork->ApplyError();//Apply the error gained from the last steps
			this->mainNetwork->CopyToHost();
			this->mainNetwork->VisualizeNetwork();
			this->createCheckpoint();
			this->mainNetwork->ResetSequence();
		}

		this->checkpoint.i_number_of_loops_checkpoint++;
		this->checkpoint.i_number_of_loops++;
	} while (checkpoint.i_number_of_loops_checkpoint < this->settings.i_number_of_training);
	this->mainNetwork->cleanNetwork();
	return true;
}

bool ReccurentLoops::train_network_RealTimeRecurrentTraining(){
	return true;
}

//*********************
//DEBUG FUNCTIONS
//*********************

//Creates a test input/output
weight_type* ReccurentLoops::createTestInputOutput(int numberOfInput, int input_output){
	static int position = 0;
	weight_type* temp = new weight_type[numberOfInput];
	for (int i = position; i < position + numberOfInput; i++){
		if (input_output == 0){
			temp[i - position] = (weight_type)i;
		}
		else{
			temp[i - position] = (weight_type).1;
		}
	}
	position += numberOfInput;
	return  temp;
}

void ReccurentLoops::createCheckpoint(){
	static int count = 0 ;

		std::ofstream outputfile;
		outputfile.open("networks/" + settings.s_network_name + std::to_string(count) + ".txt", ios::trunc);
		if (outputfile.is_open()){
			for (int i = 0; i < this->settings.i_input; i++){
				//cout << i << ") " << this->input[0][i] << endl;
			}


			for (int i = 0; i < this->settings.i_input; i++){
				//cout << i << ") " << this->output[0][i] << endl;
			}
			//Output the network
			outputfile << *this << flush;
			outputfile << endl;
			cout << *this << endl;
			//vector<weight_type> vect = this->runNetwork(this->input[0]);

			for (int i = 0; i < this->settings.i_input; i++){
				//cout << i << ") " << vect[i] << endl;
			}
			outputfile.close();
		}
		else{
			std::cout << "Unable to write checkpoint to file." << endl;
			std::cout << "continue?";
		}
		
		count++;
		

	

}