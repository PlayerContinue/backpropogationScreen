#include "hip/hip_runtime.h"
#include "ReccurentLoops.cuh"
//#define TESTING
//*****************************
//Constructor
//*****************************
ReccurentLoops::ReccurentLoops()
{

}

ReccurentLoops::ReccurentLoops(CSettings settings){
	this->settings = settings;
	this->InitializeNetwork();
	this->checkpoint = CRecurrentCheckpoint();
	this->mainNetwork = new LongTermShortTermNetwork(settings,true);
	this->loadCheckpoint();
}

ReccurentLoops::ReccurentLoops(CSettings settings, int type){
	this->settings = settings;

	switch (type){
	case ReccurentLoops::RealTimeTraining:
		//Train the network using real time recurrent
		//this->train_network_RealTimeRecurrentTraining();
		break;
	case ReccurentLoops::HessianFreeOptimization:
		this->mainNetwork = new RecurrentNeuralNetwork(settings);
		break;
	case ReccurentLoops::LongTermShortTerm:
		this->mainNetwork = new LongTermShortTermNetwork(settings);
		break;
	}

	
	this->InitializeNetwork();
	this->checkpoint = CRecurrentCheckpoint(settings);
}

ReccurentLoops::ReccurentLoops(CSettings settings, CRecurrentCheckpoint checkpoint){
	this->settings = settings;
	this->checkpoint = checkpoint;
	this->mainNetwork = new LongTermShortTermNetwork(settings,true);
	this->InitializeNetwork();

}

//*****************************
//Intialzie the Internal Requirements For Training
//*****************************

void ReccurentLoops::InitializeNetwork(){
	this->input = new weight_type*[this->settings.i_number_of_training];
	this->output = new weight_type*[this->settings.i_number_of_training];
	this->inputfile = new std::fstream();
	this->outputfile = new std::fstream();
	this->inputfile->open(this->settings.s_trainingSet);
	this->outputfile->open(this->settings.s_outputTrainingFile);
}


//*****************************
//Reload a Network From a File
//*****************************
bool ReccurentLoops::loadNetworkFromFile(){
	
	return true;
}

void ReccurentLoops::loadCheckpoint(){
	std::ifstream is;
	is.open(this->settings.s_checkpoint_file);
	is >> this->checkpoint;
	this->mainNetwork->LoadNetwork(is);
}

//*****************************
//Convert Input Types to Required Type
//*****************************
template <typename T>
weight_type* ReccurentLoops::convert_array(T* in){
	weight_type* temp = new weight_type[this->settings.i_input];
	for (int i = 0; i < settings.i_input; i++){
		temp[i] = (weight_type)in[i];
	}
	return temp;
}

vector<RETURN_WEIGHT_TYPE> ReccurentLoops::runNetwork(int* in){

	return this->runNetwork(this->convert_array<int>(in));

}

vector<RETURN_WEIGHT_TYPE> ReccurentLoops::runNetwork(weight_type* in){
	//this->mainNetwork->InitializeRun();
	device_vector<weight_type> temp_device = this->mainNetwork->runNetwork(in);
	vector<RETURN_WEIGHT_TYPE> to_return = vector <RETURN_WEIGHT_TYPE>(temp_device.size());

	for (unsigned int i = 0; i < temp_device.size(); i++){
		to_return[i] = temp_device[i];
	}
	clear_vector::free(temp_device);

	return to_return;
}
//*****************************
//Get Data From the users file
//*****************************
bool ReccurentLoops::load_training_data_from_file(){
#ifdef TESTING 
	for (int i = 0; i < this->settings.i_number_of_training; i++){
		this->input[i] = this->createTestInputOutput(this->settings.i_input,0);

		testing::outputArrayToFile(this->input[i], this->settings.i_input, "tests/inout.txt");

	}
#endif
#ifdef TESTING 
	for (int i = 0; i < this->settings.i_number_of_training; i++){
		this->output[i] = this->createTestInputOutput(this->settings.i_output, 1);

		testing::outputArrayToFile(this->output[i], this->settings.i_output, "tests/inout.txt");

	}
#endif
	return true;
}

void ReccurentLoops::loadFromFile(std::fstream &file, int length_of_results, double** storage, int* sequence_length, int type){

	//reset sequence length
	sequence_length[0] = 0;

	char individual_delimiter = (char)37;
	char group_delimiter = (char)30;
	char sequence_delimiter = (char)4;
	char current_char = '1';
	string current_string = "";
	int start = file.tellg();
	if (file.is_open()){
		int letterPosition = 0;
		for (int i = 0; i < this->settings.i_number_of_training && current_char != sequence_delimiter;i++){
			//Reset Everything
			storage[i] = new double[length_of_results];
			current_char = '1';
			current_string = "";
			letterPosition = 0;

			//While not at the end of a group, retrieve the current dataset
			while (current_char != group_delimiter && !file.eof() && current_char!=sequence_delimiter){
				//Get the current char
				current_char = file.get();

				if (current_char == individual_delimiter){//Reached the end of the current set
					if (type == 0){
						storage[i][letterPosition] = stod(current_string);
					}
					else{
						storage[i][letterPosition] = (double)current_string.at(0);
					}
					letterPosition++;
					current_string = "";
				}
				else{//String has not ended and is still correct
					current_string += current_char;
				}
			}
			//Increment Length Of String
			sequence_length[0]++;
		}
		//Get current location in file
		int currentPosition = file.tellg();
		if (file.eof()){
			sequence_length[1] = -1;//File has ended
		}
		else if (current_char == sequence_delimiter){
			sequence_length[1] = 0;//The sequence has ended
		}
		else{
			sequence_length[1] = 1;//There is more to the sequence
		}
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){
			exit(0);
		}
	}
}

//**********************
//Training
//**********************
void ReccurentLoops::startTraining(int type){
	//Load the data from a file
	//if (!load_training_data_from_file()){
		//throw exception("Unable to read from file.");
	//}
	this->inputfile->open(this->settings.s_outputTrainingFile);
	this->outputfile->open(this->settings.s_outputTestSet);
	if (this->inputfile->is_open() && this->outputfile->is_open()){
		switch (type){
		case ReccurentLoops::RealTimeTraining:
			//Train the network using real time recurrent
			this->train_network_RealTimeRecurrentTraining();
			break;
		case ReccurentLoops::HessianFreeOptimization:
			this->train_network_HessianFreeOptimizationTraining();
			break;
		case ReccurentLoops::LongTermShortTerm:
			this->testTraining();
			break;
		}
	}
	else{
		std::cout << "Unable to read from file." << endl;
		std::cout << "continue?";
		if (cin.get() == 'n'){
			exit(0);
		}
	}
}

#ifdef _DEBUG 
void ReccurentLoops::testTraining(){
	weight_type** trainingInput = new weight_type*[this->settings.i_backprop_unrolled];
	weight_type** trainingOutput = new weight_type*[this->settings.i_backprop_unrolled];
	int length[2];
	
	this->loadFromFile(*(this->outputfile), this->settings.i_output, this->output, length,0);
	this->loadFromFile(*(this->inputfile), this->settings.i_input, this->input, length,1);
	try{
		this->load_training_data_from_file();
		if (!this->checkpoint.b_still_running){
			this->mainNetwork->InitializeTraining();
		}
		this->checkpoint.b_still_running = true;
		while (length[1] != -1){
			
			for (int i = 0; i < length[0]; i += this->settings.i_backprop_unrolled){

				for (int j = i, k = 0; k < this->settings.i_backprop_unrolled; j++, k++){
					if (j < length[0]){
						trainingInput[k] = this->input[j];
						trainingOutput[k] = this->output[j];
					}
					else{
						if (k < this->settings.i_backprop_unrolled){
							trainingInput[k] = this->input[0];
							trainingOutput[k] = this->output[0];
						}
						//Reached the end of the sequence, load the next sequence
						//Normally load more from the file
						//break;
					}
				}


				this->mainNetwork->StartTraining(trainingInput, trainingOutput);

				if (this->checkpoint.i_number_of_loops_checkpoint%this->settings.i_number_allowed_same == 0){
					this->createCheckpoint();
				}
				//Apply the error
				this->mainNetwork->ApplyError();
				if (this->checkpoint.i_number_of_loops_checkpoint%this->settings.i_number_allowed_same == 0){
					this->createCheckpoint();
				}





				
				this->checkpoint.i_number_of_loops_checkpoint += 1;

			}
			if (length[1] == 0){//Reset the sequence once the sequence has finished

				this->mainNetwork->ResetSequence();
			}
			//Load more data from the file
			
			this->loadFromFile(*(this->outputfile), this->settings.i_output, this->output, length,0);
			this->loadFromFile(*(this->inputfile), this->settings.i_input, this->input, length,1);
			
		}
		//No longer running loops
		this->checkpoint.b_still_running = false;
		try{
			this->createCheckpoint();
			/*this->mainNetwork->ResetSequence();
			for (int i = 0; i < this->settings.i_number_in_sequence; i++){
				cout << i << ") " << endl;
				thrust::device_vector<weight_type> temp = this->mainNetwork->runNetwork(this->input[i]);
				testing::outputToFile<weight_type>(temp, "results", "tests/results.txt");
			}*/
			this->createCheckpoint("RunResultsInMemory");
			this->mainNetwork->cleanNetwork();
			this->mainNetwork->InitializeRun();
			this->mainNetwork->ResetSequence();
			this->createCheckpoint("RunStart");
			for (int i = 0; i < this->settings.i_number_in_sequence; i++){
				if (i == 0){
					testing::outputArrayToFile<weight_type>(this->input[i], this->settings.i_input, "tests/results2.txt");
					testing::outputArrayToFile<weight_type>(this->output[i], this->settings.i_output, "tests/results2.txt");
				}
				testing::outputArrayToFile<weight_type>(this->input[i], this->settings.i_input, "tests/results2.txt");
				testing::outputArrayToFile<weight_type>(this->output[i], this->settings.i_output, "tests/results2.txt");
				std::vector<weight_type> temp2 = this->runNetwork(this->input[i]);
				testing::outputVectorToFile<weight_type>(temp2, "results", "tests/results2.txt");
			}
			this->createCheckpoint("RunResultsFromHost");
			this->mainNetwork->emptyGPUMemory();
		}
		catch (exception e){
			cout << e.what();
			cin.sync();
			cin.get();
			//this->mainNetwork->emptyGPUMemory();
		}

		
	}

	catch (exception e){//Edit to write the problems to file later
		cout << e.what();
		cin.sync();
		cin.get();
	}
}
#endif

bool ReccurentLoops::train_network_HessianFreeOptimizationTraining(){
	this->mainNetwork->InitializeTraining();
	do{
		this->mainNetwork->StartTraining(this->input[this->checkpoint.i_number_of_loops_checkpoint], this->output[this->checkpoint.i_number_of_loops_checkpoint]);
		if (this->checkpoint.i_number_of_loops % this->settings.i_loops == 0){
			this->mainNetwork->VisualizeNetwork();
			this->mainNetwork->ApplyError();//Apply the error gained from the last steps
			this->mainNetwork->CopyToHost();
			this->mainNetwork->VisualizeNetwork();
			this->createCheckpoint();
			this->mainNetwork->ResetSequence();
		}

		this->checkpoint.i_number_of_loops_checkpoint++;
		this->checkpoint.i_number_of_loops++;
	} while (checkpoint.i_number_of_loops_checkpoint < this->settings.i_number_of_training);
	this->mainNetwork->cleanNetwork();
	return true;
}

bool ReccurentLoops::train_network_RealTimeRecurrentTraining(){
	return true;
}

//*********************
//DEBUG FUNCTIONS
//*********************

//Creates a test input/output
weight_type* ReccurentLoops::createTestInputOutput(int numberOfInput, int input_output){
	static int position = 0;
	static int previous = input_output;
	if (input_output != previous){
		position = 0;
		previous = input_output;
	}
	if (input_output == 0 && position >= this->settings.i_number_in_sequence){
		position = 0;
	}
	else if (position >= this->settings.i_number_in_sequence){
		position = 0;
	}
	weight_type* temp = new weight_type[numberOfInput];
	weight_type count = .1;
	for (int i = 0; i < numberOfInput; i++){

		if (input_output == 0){
			temp[i] = (weight_type)(position);
		}
		else{
			temp[i] = (weight_type)(.01*(position));
		}
		count += .1;
		position++;
	}
	
	return  temp;
}

//Create a checkpoint with the network name
//Default function
void ReccurentLoops::createCheckpoint(){
	this->createCheckpoint(this->settings.s_network_name);
}

//Create a Checkpoint with any name
void ReccurentLoops::createCheckpoint(string file_name){
	static int count = 0 ;

		std::ofstream outputfile;
		outputfile.open("recurrent_networks/" + file_name + std::to_string(count) + ".txt", ios::trunc);
		if (outputfile.is_open()){
			outputfile << *this << flush;
			outputfile << endl;
			outputfile.close();
		}
		else{
			std::cout << "Unable to write checkpoint to file." << endl;
			std::cout << "continue?";
		}
		
		count++;
		

	

}