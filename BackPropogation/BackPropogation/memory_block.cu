#include "hip/hip_runtime.h"
#include "memory_block.cuh"

Memory_Block::Memory_Block(){

}

Memory_Block::Memory_Block(unsigned int start, unsigned int numberInput){
	//Initialize the values
	this->input_weights = host_vector<weight_type>();
	this->output_weights = host_vector<weight_type>();
	this->forget_weights = host_vector<weight_type>();
	this->potential_memory_cell_value = host_vector<weight_type>();
	this->memory_cell_weights = this->getNewWeight();
	this->mapFrom = host_vector<int>();

	//Make the input weights
	for (int i = 0; i < numberInput; i++){
		this->input_weights.push_back(this->getNewWeight());
		this->output_weights.push_back(this->getNewWeight());
		this->forget_weights.push_back(this->getNewWeight());
		this->potential_memory_cell_value.push_back(this->getNewWeight());
		this->mapFrom.push_back(i + start);
	}
}

Memory_Block::Memory_Block(unsigned int input) :Memory_Block(0, input){
	
}

void Memory_Block::addNewConnection(int min, int max){
	
}

weight_type Memory_Block::getNewWeight(){
	return RandomClamped();
}
