#include "hip/hip_runtime.h"
#include "memory_block.cuh"

Memory_Block::Memory_Block(){

}

Memory_Block::Memory_Block(unsigned int input) :Memory_Block(0, input){

}

Memory_Block::Memory_Block(unsigned int start, unsigned int numberInput):Memory_Block(start,numberInput,LAYER){
	
}

Memory_Block::Memory_Block(unsigned int start, unsigned int numberInput, memory_block_type type){
	//Initialize the values
	this->input_weights = host_vector<weight_type>();
	this->output_weights = host_vector<weight_type>();
	this->forget_weights = host_vector<weight_type>();
	this->potential_memory_cell_value = host_vector<weight_type>();
	this->memory_cell_weights = host_vector<weight_type>();
	this->bias = host_vector<weight_type>();
	this->number_weights = 0;
	this->type = type;//Set the type of memory block this is
	if (type == LAYER){//Output layer does not require this, as it is only a set of input	
		this->memory_cell_weights.push_back(this->getNewWeight());
	}
	this->number_memory_cells = 1;
	this->mapFrom = host_vector<int>();
	//Add weights which connect from the input nodes to the output nodes
	for (int i = 0; i < numberInput; i++){

		
		if (type==LAYER){//Only add these if the current node is in a layer which is not an output
			this->input_weights.push_back(this->getNewWeight());
			this->output_weights.push_back(this->getNewWeight());
			this->forget_weights.push_back(this->getNewWeight());
			
		}
		//If the layer is an output, it needs both a map from where the ouput is
		//and a cell for containing the output
		this->potential_memory_cell_value.push_back(this->getNewWeight());
		this->mapFrom.push_back(i + start);
		if (type == LAYER){
			number_weights += 4;//Increment the number of weights in the list
		}
		else if (type == OUTPUT){
			number_weights += 1;
		}
	}

	//Create Biases for each node
	//The biases are currently randomly chosen, but may change on future iterations
	//4 is the number of non-memory-cell nodes, memory cells have a bias of 0
	if (type == LAYER){
		for (int i = 0; i < 4; i++){
			this->bias.push_back(this->getNewWeight());
		}
	}
	else if (type == OUTPUT){
		this->bias.push_back(this->getNewWeight());
	}

}



void Memory_Block::addNewConnection(int min, int max){
	
}

weight_type Memory_Block::getNewWeight(){
	return RandomClamped();
}

//Return the type of node it is
Memory_Block::memory_block_type Memory_Block::getTypeOfMemoryBlock(){
	return this->type;
}