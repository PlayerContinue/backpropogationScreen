#include "hip/hip_runtime.h"
#include "memory_block.cuh"

Memory_Block::Memory_Block(){

}

Memory_Block::Memory_Block(unsigned int input) :Memory_Block(0, input){

}

Memory_Block::Memory_Block(unsigned int start, unsigned int numberInput):Memory_Block(start,numberInput,LAYER){
	
}

Memory_Block::Memory_Block(unsigned int start, unsigned int numberInput, memory_block_type type){
	//Initialize the values
	this->input_weights = host_vector<weight_type>();
	this->output_weights = host_vector<weight_type>();
	this->forget_weights = host_vector<weight_type>();
	this->potential_memory_cell_value = host_vector<weight_type>();
	this->memory_cell_weights = host_vector<weight_type>();
	if (type == LAYER){//Output layer does not require this, as it is only a set of input
		
		this->memory_cell_weights.push_back(this->getNewWeight());
	}
	this->number_memory_cells = 1;
	this->mapFrom = host_vector<int>();
	//Make the input weights
	for (int i = 0; i < numberInput; i++){

		
		if (type==LAYER){//Only add these if the current node is in a layer which is not an output
			this->input_weights.push_back(this->getNewWeight());
			this->output_weights.push_back(this->getNewWeight());
			this->forget_weights.push_back(this->getNewWeight());
			
		}
		//If the layer is an output, it needs both a map from where the ouput is
		//and a cell for containing the output
		this->potential_memory_cell_value.push_back(this->getNewWeight());
		this->mapFrom.push_back(i + start);
	}
}



void Memory_Block::addNewConnection(int min, int max){
	
}

weight_type Memory_Block::getNewWeight(){
	return RandomClamped();
}


Memory_Block::memory_block_type Memory_Block::getTypeOfMemoryBlock(){
	if (this->memory_cell_weights.size() > 0){
		return LAYER;
	}
	else{
		return OUTPUT;
	}
}