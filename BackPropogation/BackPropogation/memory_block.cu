#include "hip/hip_runtime.h"
#include "memory_block.cuh"

Memory_Block::Memory_Block(){

}

Memory_Block::Memory_Block(unsigned int input) :Memory_Block(0, input){

}

Memory_Block::Memory_Block(unsigned int start, unsigned int numberInput):Memory_Block(start,numberInput,LAYER){
	
}

Memory_Block::Memory_Block(unsigned int start, unsigned int numberInput, memory_block_type type){
	//Initialize the values
	this->input_weights = host_vector<weight_type>();
	this->output_weights = host_vector<weight_type>();
	this->forget_weights = host_vector<weight_type>();
	this->potential_memory_cell_value = host_vector<weight_type>();
	this->memory_cell_weights = host_vector<weight_type>();
	this->bias = host_vector<weight_type>();
	this->number_weights = 0;
	this->number_inputs = numberInput;
	this->type = type;//Set the type of memory block this is
	if (type == LAYER){//Output layer does not require this, as it is only a set of input	
		this->memory_cell_weights.push_back(this->getNewWeight());
		this->number_memory_cells = 1;
	}
	else{
		this->number_memory_cells = 0;
	}
	this->mapFrom = host_vector<int>();
	
	setInitialWeights(start, numberInput, type);
	createStorage();
}

Memory_Block::Memory_Block(unsigned int start, unsigned int numberInput, unsigned int extra_at_start, memory_block_type type){

	//Initialize the values
	this->input_weights = host_vector<weight_type>();
	this->output_weights = host_vector<weight_type>();
	this->forget_weights = host_vector<weight_type>();
	this->potential_memory_cell_value = host_vector<weight_type>();
	this->memory_cell_weights = host_vector<weight_type>();
	this->bias = host_vector<weight_type>();
	this->number_weights = 0;
	this->number_inputs = numberInput;
	this->type = type;//Set the type of memory block this is
	
	if (type == LAYER){//Output layer does not require this, as it is only a set of input	
		this->memory_cell_weights.push_back(this->getNewWeight());
		this->number_memory_cells = 1;
	}
	else{
		this->number_memory_cells = 0;
	}

	this->mapFrom = host_vector<int>();
	//setInitialWeights(0, extra_at_start,type);
	setInitialWeights(start, numberInput, type);
	
	createStorage();
}

void Memory_Block::createStorage(){
	this->weight_lists = vector<thrust::host_vector<weight_type>>(5);
	this->weight_lists[cell_type::INPUT_CELL] = this->input_weights;
	this->weight_lists[cell_type::OUTPUT_CELL] = this->output_weights;
	this->weight_lists[cell_type::MEMORY_CELL] = this->memory_cell_weights;
	this->weight_lists[cell_type::FORGET_CELL] = this->forget_weights;
	this->weight_lists[cell_type::POTENTIAL_MEMORY_CELL] = this->potential_memory_cell_value;
}

void Memory_Block::setInitialWeights(int start, int numberInput, memory_block_type type){
	//Add weights which connect from the input nodes to the output nodes
	for (int i = 0; i < numberInput; i++){
		if (type == LAYER){//Only add these if the current node is in a layer which is not an output
			this->input_weights.push_back(this->getNewWeight());
			this->output_weights.push_back(this->getNewWeight());
			this->forget_weights.push_back(this->getNewWeight());

		}
		//If the layer is an output, it needs both a map from where the ouput is
		//and a cell for containing the output
		this->potential_memory_cell_value.push_back(this->getNewWeight());
		this->mapFrom.push_back(i + start);
		if (type == LAYER){
			number_weights += 4;//Increment the number of weights in the list
		}
		else if (type == OUTPUT){
			number_weights += 1;
		}
	}

	//Create Biases for each node
	//The biases are currently randomly chosen, but may change on future iterations
	//4 is the number of non-memory-cell nodes, memory cells have a bias of 0
	if (type == LAYER){
		for (int i = 0; i < 4; i++){
			this->bias.push_back(this->getNewWeight());
		}
	}
	else if (type == OUTPUT){
		this->bias.push_back(this->getNewWeight());
	}
}

weight_type Memory_Block::getBias(cell_type type){
	switch (type){
	case INPUT_CELL:
		return this->bias[0];
	case OUTPUT_CELL:
		return this->bias[1];
	case FORGET_CELL:
		return this->bias[2];
	case POTENTIAL_MEMORY_CELL:
		return this->bias[3];
	case MEMORY_CELL:
		return 0;
	default:
		return -1;
	}
}



void Memory_Block::addNewConnection(int pos){
	this->mapFrom.push_back(pos);
	
	this->potential_memory_cell_value.push_back(this->getNewWeight());
	this->weight_lists[POTENTIAL_MEMORY_CELL].push_back(this->potential_memory_cell_value[this->potential_memory_cell_value.size() - 1]);
	if (this->type != OUTPUT){
		this->input_weights.push_back(this->getNewWeight());
		this->output_weights.push_back(this->getNewWeight());
		this->forget_weights.push_back(this->getNewWeight());
		this->weight_lists[INPUT_CELL].push_back(this->input_weights[this->input_weights.size() - 1]);
		this->weight_lists[OUTPUT_CELL].push_back(this->output_weights[this->output_weights.size() - 1]);
		this->weight_lists[FORGET_CELL].push_back(this->forget_weights[this->forget_weights.size() - 1]);
		this->number_weights += 4;
	}
	else{
		this->number_weights += 1;
	}
	
}
void Memory_Block::addNewConnection(int min, int max){
	bool mappedFrom = false;
	for (int i = min; i < max; i++){
		mappedFrom = false;
		for (int j = 0; j < this->mapFrom.size(); j++){
			if (this->mapFrom[j] == i){
				mappedFrom = true;
			}
		}

		if (!mappedFrom){
			addNewConnection(i);
			break;
		}
	}
}

bool Memory_Block::removeConnection(int toRemove){
	

	if (toRemove < this->potential_memory_cell_value.size()){
		if (this->type == LAYER){
			for (unsigned int start = INPUT_CELL; start <= MEMORY_CELL; start++){
				if (start != POTENTIAL_MEMORY_CELL){
					this->weight_lists[start].erase(this->weight_lists[start].begin() + toRemove);
				}
			}
			this->input_weights.erase(this->input_weights.begin() + toRemove);
			this->output_weights.erase(this->output_weights.begin() + toRemove);
			this->forget_weights.erase(this->forget_weights.begin() + toRemove);
		}
		this->potential_memory_cell_value.erase(this->potential_memory_cell_value.begin() + toRemove);
		this->weight_lists[POTENTIAL_MEMORY_CELL].erase(this->weight_lists[POTENTIAL_MEMORY_CELL].begin() + toRemove);
		this->mapFrom.erase(this->mapFrom.begin() + toRemove);
		this->number_weights--;
		return true;
	}
	else{
		return false;
	}
}

void Memory_Block::incrementFromPosition(int add){
	this->incrementFromPosition(add, 0);
}
void Memory_Block::incrementFromPosition(int add, int add_from){
	thrust::transform(this->mapFrom.begin() + add_from, this->mapFrom.end(), this->mapFrom.begin() + add_from, _1 + add);
}

weight_type Memory_Block::getNewWeight(){
	return RandomClamped();
}

//Return the type of node it is
Memory_Block::memory_block_type Memory_Block::getTypeOfMemoryBlock(){
	return this->type;
}