#include "hip/hip_runtime.h"
#include "CGraphicNetwork.cuh"


CGraphicsNetwork::CGraphicsNetwork()
{
}

CGraphicsNetwork::CGraphicsNetwork(CSettings* settings){
	this->settings = settings;
}

CGraphicsNetwork::CGraphicsNetwork(vector<int> &sizes)
{
	//Set the number of inputs
	this->I_input = sizes.at(0);

	//Set the number of outputs
	this->I_output = sizes.back();

	//Get the number of layers
	this->v_num_layers = sizes.size();

	//Set the number of layers
	this->v_layers.resize(this->v_num_layers);

	//Create a temporary location for new neuron
	SNeuron tempNeuron;

	//Seed the random
	srand((unsigned)(time(NULL)));

	//Assign the number of Neuron Layers

	//Create the intial layer
	this->v_layers[0] = SNeuronLayer(sizes.at(0), 0);
	this->total_num_nodes += sizes.at(0);
	//Create every other layer
	for (int i = 1; i < this->v_num_layers; i++){//Travel through layers

		
		this->v_layers[i] = SNeuronLayer(sizes.at(i), sizes.at(i - 1));
		//Increase the count on the total number of nodes
		this->total_num_nodes+=sizes.at(i);

		
	}
}

CGraphicsNetwork::CGraphicsNetwork(vector<int> &sizes, double beta, double alpha) :CGraphicsNetwork(sizes){
	this->beta = beta;
	this->alpha = alpha;
}

CGraphicsNetwork::CGraphicsNetwork(vector<int> &sizes, CSettings* settings) : CGraphicsNetwork(sizes, settings->d_beta, settings->d_alpha){
	this->settings = settings;

	this->setSettings(settings);
}


//Needs Testing
//TODO Use up a tiny bit of memory to create a pointer to the different objects which are used multiple times


void CGraphicsNetwork::feedForward(double *in){

	//Store the sumation from the previous layer


	//Store the input in the input layer
	//Allows future calculations to be performed easier
	for (int i = 0; i < this->v_layers[0].number_per_layer; i++){
		this->v_layers[0].output[i] = in[i];
	}
#ifdef TRIAL5
	//Define a holder for the output
	thrust::host_vector<double> output;
	int failure;
#endif

	//Perform the following actions on each hidden layer
	for (int i = 1; i < this->v_num_layers; i++){
#ifdef TRIAL5
		double sum;
		output.resize(this->v_layers[i].number_per_layer);

		//For each neuron in the current layer
		//take the output of the previous layer
		//and perform the calculation on it
		for (int j = 0; j < this->v_layers[i].number_per_layer; j++){
			if (!checkNeuronRemoved(this->v_layers[i].neurons[j])){//The current node has not been removed, use it
				sum = 0.0;//Reset the sum
				//For input from each neuron in the preceding layer
				for (int k = 0; k < this->v_layers[i - 1].number_per_layer; k++){
					if (!checkNeuronRemoved(this->v_layers[i - 1].neurons[k])){//The neuron in the previous layer has not been removed, add it
						//Add the output from the nodes from the previous layer times the weights for that neuron on the current layer
						sum += this->v_layers[i - 1].output[k]*this->v_layers[i].neurons[j].weights[k];
						cout << this->v_layers[i - 1].output[k] * this->v_layers[i].neurons[j].weights[k] << endl;
					}
				}

				//Apply the bias
				sum += this->v_layers[i].neurons[j].bias;
				cout << sum << endl;
				cout << "___________________" << endl;
				//Apply the sigmoid function
				output[j] = CGraphicsNetwork::sigmoid(sum);
				//Possibly Temporary
				//States if the neuron has been activated
				if (isNeuronActivated(this->v_layers[i].neurons[j])){
					this->v_layers[i].neurons[j].activated += 1;
				}
			}
		}

#endif

			//Get the feedforward value
			feedForwardGPU(this->v_layers[i], this->v_layers[i - 1]);
		

#ifdef TRIAL5
		failure = 0;
		//Test the output
		for (int m = 0; m < this->v_layers[i].output.size(); m++){
			if (this->v_layers[i].output[m] != output[m]){
				cout.precision(20);
				cout << this->v_layers[i].output[m] << " | " << output[m] << endl;
				failure++;
			}

		}
		cout << failure << endl;
#endif

	}



}





void CGraphicsNetwork::backprop(double *in, double *tgt){


	//Host_Vector containing the current target
	thrust::host_vector<double> target_vector;

	//Set the size of the target vector
	target_vector = thrust::host_vector<double>(this->I_output);

	//Perform the feedforward algorithm to retrieve the output of 
	//each node in the network
	this->feedForward(in);



	//Check if results were successful
	updateSuccess(tgt);

	//Copy the target values into the target vector for use in processing by the GPU
	for (int i = 0; i < this->I_output; i++){
		target_vector[i] = tgt[i];
	}

#if defined(TRIAL2) || defined(TRIAL1)|| defined(TRIAL3) || defined(TRIAL4)
	double sum;
	int count_fail = 0;
	thrust::host_vector<double> weights;
	//Stores the current neuron
	SNeuron *currentNeuron;
	thrust::host_vector<double> delta = thrust::host_vector<double>(this->v_layers[this->v_num_layers - 1].delta.size());
#endif

#ifdef TRIAL1

	//Find Delta for the output Layer
	//The required change to have the correct answer
	for (int i = 0; i < this->v_layers[this->v_num_layers - 1].number_per_layer; i++){

		//Store a pointer to the variable
		currentNeuron = &(this->v_layers[this->v_num_layers - 1].neurons[i]);
		delta[i] = this->v_layers[this->v_num_layers - 1].output[i] * (1 - this->v_layers[this->v_num_layers - 1].output[i]) * (tgt[i] - this->v_layers[this->v_num_layers - 1].output[i]);


	}//Removed due to better implementation
#endif

	//Find the delta of the output layer
	findOutputDelta(this->v_layers[this->v_num_layers - 1].getOutput(), target_vector, this->v_layers[this->v_num_layers - 1]);

#ifdef TRIAL1

	for (int i = 0; i < delta.size(); i++){
		if (delta[i] != target_vector[i]){
			count_fail++;
		}
	}
	cout << count_fail << endl;
#endif

	//this->v_layers[this->v_num_layers - 1].delta = target_vector;

	//Find the delta for the hidden layers
	for (int layerPosition = this->v_num_layers - 2; layerPosition > 0; layerPosition--){


#ifdef TRIAL2
		count_fail = 0;
		delta.resize(this->v_layers[layerPosition].neurons.size());
		for (int j = 0; j < this->v_layers[layerPosition].number_per_layer; j++){
			sum = 0.0;

			//Find the delta for the current neuron
			for (int k = 0; k < this->v_layers[layerPosition + 1].number_per_layer; k++){

				//Delta * each weight of the neuron
				sum += this->v_layers[layerPosition + 1].delta[k] *
					this->v_layers[layerPosition + 1].neurons[k].weights[j];
				cout.precision(20);
				cout << this->v_layers[layerPosition + 1].delta[k] *
					this->v_layers[layerPosition + 1].neurons[k].weights[j] << endl;

			}
			cout << "___________________" << endl;
			cout << sum << endl;
			delta[j] = this->v_layers[layerPosition].output[j] * (1 - this->v_layers[layerPosition].output[j]) * sum;
		}
		cout << "_____________________" << endl;
#endif

		//Retrieve the changed delta and store in the output vector
		findHiddenDelta(this->v_layers[layerPosition + 1], this->v_layers[layerPosition]);

#ifdef TRIAL2

		for (int i = 0; i < delta.size(); i++){
			if (delta[i] != this->v_layers[layerPosition].delta[i]){
				std::cout.precision(20);
				cout << delta[i] << " | " << this->v_layers[layerPosition].delta[i] << endl;
				count_fail++;
			}
		}
		cout << count_fail << endl;

#endif
	}

	//Find Delta for the hidden layers
	//The change needed to recieve the correct answer
	//All Layers except input and output
	/*for (int layerPosition = this->v_num_layers - 2; layerPosition > 0; layerPosition--){

	}*/

	//Apply the momentum
	//Does nothing if alpha = 0;
	if (this->alpha != 0){
		for (int layerPos = 1; layerPos < this->v_num_layers; layerPos++){

#ifdef TRIAL3
			weights.clear();
			for (int neuronPos = 0; neuronPos < this->v_layers[layerPos].number_per_layer; neuronPos++){


				//Apply the alpha to each weight
				for (int weightPos = 0; weightPos < this->v_layers[layerPos].neurons[0].weights.size(); weightPos++){
					weights.push_back(this->v_layers[layerPos].neurons[neuronPos].weights[weightPos] + (this->alpha * this->v_layers[layerPos].neurons[neuronPos].previousWeight[weightPos]));
				}

				//Add the bias
				weights.push_back(this->v_layers[layerPos].neurons[neuronPos].bias + (this->alpha * this->v_layers[layerPos].neurons[neuronPos].previousBias));
			}

#endif

			applyMomentum(this->v_layers[layerPos], this->alpha);

#ifdef TRIAL3 
			int position2 = 0;
			int wrong_count = 0;
			for (int neuronPos = 0; neuronPos < this->v_layers[layerPos].number_per_layer; neuronPos++){
				for (int weightPos = 0; weightPos < this->v_layers[layerPos - 1].number_per_layer; weightPos++){
					if (this->v_layers[layerPos].neurons[neuronPos].weights[weightPos] != weights[position2]){
						cout << this->v_layers[layerPos].neurons[neuronPos].weights[weightPos] << " | " << weights[position2] << endl;
						wrong_count++;
					}
					position2++;

				}

				if (this->v_layers[layerPos].neurons[neuronPos].bias != weights[position2]){
					cout << this->v_layers[layerPos].neurons[neuronPos].bias << " | " << weights[position2] << endl;
					wrong_count++;
				}
				position2++;

			}
			cout << wrong_count << endl;
#endif
		}

	}

#ifdef TRIAL4

	thrust::host_vector<double> previousWeights;
	double previousWeights3;

	//Apply the correction
	for (int layerNum = 1; layerNum < this->v_num_layers; layerNum++){
		weights.clear();
		previousWeights.clear();

		for (int neuronPos = 0; neuronPos < this->v_layers[layerNum].number_per_layer; neuronPos++){
			if (!checkNeuronRemoved(this->v_layers[layerNum].neurons[neuronPos]) && !checkNeuronLocked(this->v_layers[layerNum].neurons[neuronPos])){//Check if Neuron is temp removed

				for (int weightPos = 0; weightPos < this->v_layers[layerNum - 1].number_per_layer; weightPos++){

					//Check if the weight should be updated based on previous layer neuron availability
					if (!checkNeuronRemoved(this->v_layers[layerNum - 1].neurons[weightPos])){

						//BETA * delta * output
						previousWeights3 = this->beta * this->v_layers[layerNum].delta[neuronPos] *
							this->v_layers[layerNum - 1].output[weightPos];
						previousWeights.push_back(previousWeights3);

						weights.push_back(this->v_layers[layerNum].neurons[neuronPos].weights[weightPos] +
							previousWeights3);
					}
				}

				previousWeights3 = this->beta * this->v_layers[layerNum].delta[neuronPos];
				previousWeights.push_back(this->beta * this->v_layers[layerNum].delta[neuronPos]);

				weights.push_back(this->v_layers[layerNum].neurons[neuronPos].bias +
					previousWeights3);
			}


		}


		applyCorrection(this->v_layers[layerNum], this->v_layers[layerNum - 1].getOutput(1, 1), this->beta);
		int position = 0;
		count_fail = 0;
		for (int j = 0; j < this->v_layers[layerNum].number_per_layer; j++){
			for (int i = 0; i < this->v_layers[layerNum].neurons[j].previousWeight.size(); i++){
				if (this->v_layers[layerNum].neurons[j].previousWeight[i] != previousWeights[position]
					|| this->v_layers[layerNum].neurons[j].weights[i] != weights[position]
					){
					cout.precision(10);
					cout << " " << this->v_layers[layerNum].neurons[j].previousWeight[i] << " | " << previousWeights[position] << endl
						<< this->v_layers[layerNum].neurons[j].weights[i]
						<< " | " << weights[position] << endl << layerNum << "," << j << "," << i << endl;
					count_fail++;
				}

				position++;
			}

			if (this->v_layers[layerNum].neurons[j].bias != weights[position] || this->v_layers[layerNum].neurons[j].previousBias != previousWeights[position]){
				count_fail++;
			}
			position++;
		}

		cout << count_fail << endl;

	}
#else
	for (int layerNum = 1; layerNum < this->v_num_layers; layerNum++){
		applyCorrection(this->v_layers[layerNum], this->v_layers[layerNum - 1].getOutput(1, 1), this->beta);
	}

#endif
	//Free the target vector memory
	vector_free::free(target_vector);

}

//**********************************************
//Add and Remove Layers and Neurons
//**********************************************

//Add a new neuron which causes will not activate until after
// it is taught at least once
//By keeping the neuron non active, the neural network should be able to better 
//update the values
void CGraphicsNetwork::addNeuronToLayer(int layerPositionStart, int layerPositionEnd, int numToAdd){
	int layerPosition = 1;
	int minNeurons = INT_MAX;
	//Can't add neurons to non-hidden layers
	//Changing the number of inputs would change the value to greatly
	//as would changing the number of outputs
	//Special version later maybe
	if (layerPositionStart < 1 || layerPositionStart >= (int) this->v_layers.size() - 1){
		//Change the position to the layer below the output
		layerPositionStart = this->v_layers.size() - 1;
	}

	//Add the new Neurons to the one which can get the most use out of them
	//TODO figure out a good algorithm
	for (int i = layerPositionStart; i <= layerPositionEnd; i++){
		if (this->v_layers[i].number_per_layer < minNeurons || minNeurons == 0){
			minNeurons = this->v_layers[i].number_per_layer;
			layerPosition = i;
		}
	}

	this->v_layers[layerPosition].resizeNetwork(this->v_layers[layerPosition].number_per_layer + numToAdd);
	//Seed the random
	srand((unsigned)(time(NULL)));
	//Add the new Neuron
	for (int i = minNeurons; i < minNeurons + numToAdd; i++){
		//Add the weights
		//TODO find a better algorithm for deciding the weight
		for (int k = 0; k < this->v_layers[layerPosition - 1].number_per_layer; k++){//Number of neurons in next layer used as number of outgoing outputs
			//this->v_layers[layerPosition].neurons[i].weights.push_back(this->v_layers[layerPosition].neurons[i - minNeurons].weights[k] / 2);//Add a random weight between 0 and 1
			//this->v_layers[layerPosition].neurons[i - minNeurons].weights[k] = this->v_layers[layerPosition].neurons[i - minNeurons].weights[k] / 2;//Set the weight to half so that it takes the results
			this->v_layers[layerPosition].neurons[i].weights.push_back(RandomClamped());
			this->v_layers[layerPosition].neurons[i].previousWeight.push_back(0);//Set previous weight to 0
		}

		//Add the bias (Random Number between 0 and 1)
		this->v_layers[layerPosition].neurons[i].bias = RandomClamped();


		//Set the initial previousbias to 0
		this->v_layers[layerPosition].neurons[i].previousBias = 0;


	}

	//Add a new weight for the new node on the next level
	this->v_layers[layerPosition + 1].addNewWeights(numToAdd);
	//Increase the count on the total number of nodes
	this->total_num_nodes += numToAdd;
}

//Create a new layer with no effect on the current output of the network
//By utilizing a no change new layer, the system can learn new values while 
//leaving the previous layer unchanged
void CGraphicsNetwork::addLayer(int position, int neuronPerLayer){

	//Create iterator for insertion
	vector<SNeuronLayer>::iterator it;
	bool replaceOutput = false;
	//Add a new layer below the output layer
	//Used to deal with negative values and overly large values
	if (position < 0 ){
		//Change the position to the output layer position
		position = this->v_layers.size() - 1;
	}
	//Adding a new output layer
	//Requires the number of neurons to equal the output
	else if (position > (int) this->v_layers.size()){
		neuronPerLayer = this->I_output;
		position = this->v_layers.size();
		replaceOutput = true;
	}



	//Add a new layer at the given position
	it = this->v_layers.begin() + position;

	//Insert the new layer
	if (!replaceOutput){
		this->v_layers.insert(it, SNeuronLayer(neuronPerLayer, this->v_layers[position - 1].number_per_layer, this->settings));
	}else{
		this->v_layers.insert(it, SNeuronLayer(neuronPerLayer));
		this->v_layers.back().settings = this->settings;
	}




	//Randomly create a bias for each of the neurons
	for (int j = 0; j < neuronPerLayer; j++){//Travel through neurons

		//Add the weights
		if (position > 0 && replaceOutput){
			this->v_layers[position].neurons.push_back(SNeuron());
			for (int k = 0; k < this->v_layers[position - 1].number_per_layer; k++){//Number of neurons in next layer used as number of outgoing outputs
					//By only taking in one value, the current node becomes a copy of that node
					if (k == j){
						this->v_layers.back().neurons[j].weights.push_back(1);//Take only one output, output the same value as the one passed in
						this->v_layers.back().neurons[j].previousWeight.push_back(0);//Set previous weight to 0
					}
					else{
						this->v_layers.back().neurons[j].weights.push_back(0);//Add a random weight between 0 and 1
						this->v_layers.back().neurons[j].previousWeight.push_back(0);//Set previous weight to 0
					}
				}
			}

		
		if (replaceOutput){
			//Set the bias to zero to have no effect on output
			this->v_layers.back().neurons[j].bias = 0;
			this->v_layers.back().neurons[j].previousBias = 0;
		}else{
			//Remove uneeded weights or add new weights
			this->v_layers[position + 1].keepXWeights(neuronPerLayer);
		}
	}

	//Increase the count on the total number of nodes
	this->total_num_nodes += neuronPerLayer;
	this->v_num_layers += 1;
}

void CGraphicsNetwork::reloadNetwork(){
	//Reload the network from a file
	std::ifstream outputfile;
	outputfile.open("backups/removedNodes.txt", ios_base::beg);
	if (outputfile.is_open()){
		//Output the network
		outputfile >> *this;
		outputfile.close();
	}
	else{
		cout << "Unable reload from file." << endl;
		cout << "continue?";
		if (cin.get() == 'n'){
			exit(0);
		}
	}
}

//Remove a neuron from the current layer
void CGraphicsNetwork::removeNeuron(int layerPosition, int neuronPosition){

	if (layerPosition >= this->v_num_layers || layerPosition < 0){//Layer doesn't exist
		throw 20;//Out of bound layer
	}
	else if (neuronPosition >= this->v_layers[layerPosition].number_per_layer || neuronPosition < 0){
		throw 21; //Out of bound Neuron
	}
	else{
		//Write the network to a file in case the previous version was better
		//Allows keeping of replacing the old network
		std::ofstream outputfile;
		outputfile.open("backups/removedNodes.txt", ios::trunc);
		if (outputfile.is_open()){
			//Output the network
			outputfile << *this << flush;
			outputfile.close();
		}
		else{
			cout << "Unable to write backup containing removed nodes to file." << endl;
			cout << "continue?";
			if (cin.get() == 'n'){
				exit(0);
			}
		}

		//Remove the weights from the next layer
		this->v_layers[layerPosition+1].removeWeightsAtY(neuronPosition);

		this->total_num_nodes -= 1;

		//Remove the neuron from the array
		this->v_layers[layerPosition].neurons.erase(this->v_layers[layerPosition].neurons.begin() + neuronPosition);
		
		//Remove the node from the count
		this->v_layers[layerPosition].number_per_layer -= 1;

		//Shorten the output/input such that the nuerons information is ignored
		this->v_layers[layerPosition].delta.resize(this->v_layers[layerPosition].number_per_layer);
		this->v_layers[layerPosition].output.resize(this->v_layers[layerPosition].number_per_layer);
		this->v_layers[layerPosition].locked_nodes.resize(this->v_layers[layerPosition].number_per_layer);

	}

}

void CGraphicsNetwork::removeLayer(int layerPosition){

}

//***********************************************
//Overload Operators
//***********************************************
//Overload the output operator
ostream& operator<<(ostream& os, const CGraphicsNetwork& network){
	//Set the precision
	os.precision(60);
	//Print the number of layers
	os << network.v_num_layers << endl;
	//Print the total number of nodes
	os << network.total_num_nodes << endl;
	//Print number of inputs and outputs
	os << network.I_input << " " << network.I_output << endl;
	//Print the alpha and beta
	os << network.alpha << " " << network.beta << endl;
	//seperator
	os << "/" << endl;

	for (int i = 0; i < network.v_num_layers; i++){
		//Print the layers
		os << network.v_layers[i] << "/" << endl;
	}

	return os;
}

istream& operator>>(istream& is, CGraphicsNetwork& network){
	char next;
	//Retrieve the number of layer
	is >> network.v_num_layers;
	//Set the number of layers
	network.v_layers = vector<SNeuronLayer>(network.v_num_layers);
	//Retrieve the total number of nodes
	is >> network.total_num_nodes;
	//Retrieve the number of inputs
	is >> network.I_input;
	//Retrieve the number of outputs
	is >> network.I_output;
	//Retrieve the alpha
	is >> network.alpha;
	//Retrieve the beta
	is >> network.beta;
	//Skip the seperator
	is >> next;
	if (next != '/'){
		throw new exception("Text file not formatted correctly");
	}
	//Add the network
	for (int i = 0; i < network.v_num_layers; i++){
		is >> network.v_layers[i];
		is >> next;
		if (next != '/'){//Check if the file is formatted correctly
			throw new exception("Text file not formatted correctly");
		}
	}

	
	
	//Add function to check if network was created correctly


	return is;
}


